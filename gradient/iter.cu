#include <thrust/iterator/functional_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/shared_for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <cstdlib>
#include <ctime>
#define D 500
#define N 10
__constant__ float weights[D];

class funcOp
{
public:
  __host__ __device__ long operator() (long index) const
  {
    return index/D;
  }
};
class funcOp2
{
public:
  __host__ __device__ long operator() (long index) const
  {
    return index/D + (index%D)*N;
  }
};
class funcOp3
{
public:
  __host__ __device__ long operator() (long index) const
  {
    return index/N + 1;
  }
};
class squareOp
{
public:
  __host__ __device__ float operator() (float index) const
  {
    return index*index;
  }
};
class printFunctor
{
public:
  __host__ __device__ void operator() (long index) const
  {
    printf("PF - %ld\n",index);
  }
};

struct floatsD
{
float data[D];
__host__ __device__ float& operator[] (int index)
{
  return data[index];
}
};

struct dotProductFunctor
{
  __device__ float operator() (floatsD &vector)
  {
    float temp;
    for(int i = 0; i<D;i++)
    {
      temp+=vector[i]*weights[i];
    }
    return temp;
  }
};
struct gradSub
{
  float operator() (float &weight, float&gradient)
  {
    return weight - 0.001*gradient/N;
  }
};
int main()
{
 srand (time(NULL));
  float hostRandomArray[D*N];
  float hostYval[N];
  printf("Begin\n");
  for(int i = 0; i<D*N;i++)
  {
    hostRandomArray[i]=static_cast <float> (rand()) / static_cast <float> (RAND_MAX/5);
    // printf("%f\n",hostRandomArray[i]);
  }
  thrust::host_vector<float> host_gradient(D),host_weights(D);
  for(int i = 0; i<D;i++)
  {
    host_weights[i]=-0.1*(static_cast <float> (rand()) / static_cast <float> (RAND_MAX));
    // printf("%f\n",host_weights[i]);
  }
  for(int i = 0; i<N;i++)
  {
    hostYval[i]=5.0*(static_cast <float> (rand()) / static_cast <float> (RAND_MAX));
    // printf("%f\n",host_weights[i]);
  }
  thrust::device_vector<floatsD>Xinput(N);
  thrust::device_vector<float>Y_actual(N);
  thrust::device_vector<float>Xtemp(D*N);
  thrust::device_vector<float>Xtemp2(D*N);
  thrust::device_vector<float> y_pred(N),error(N);
  thrust::device_vector<int> emptyVector(D*N);
  thrust::device_vector<float> gradient(D);
  thrust::device_vector<float>errSquare(N);
  hipMemcpy(Xinput.data().get(),hostRandomArray,sizeof(float)*D*N,hipMemcpyHostToDevice);
  hipMemcpy(Xtemp.data().get(),hostRandomArray,sizeof(float)*D*N,hipMemcpyHostToDevice);
  hipMemcpy(Y_actual.data().get(),hostYval,sizeof(float)*N,hipMemcpyHostToDevice);
  int count = 0;
  while(count<10000)
  {
  hipMemcpyToSymbol(HIP_SYMBOL(weights),host_weights.data(),sizeof(float)*D);
  thrust::transform(Xinput.begin(),Xinput.end(),y_pred.begin(),dotProductFunctor());
  // for(int i = 0; i< N;i++)
  // {
  //   printf("%f\n",((float)Xtemp[i]));
  // }

  thrust::transform(y_pred.begin(),y_pred.end(),Y_actual.begin(),error.begin(),thrust::minus<float>());
  thrust::transform(error.begin(),error.end(),errSquare.begin(),squareOp());
  float errorVal = thrust::reduce(errSquare.begin(),errSquare.end());
  printf("@ %d Error = %f\n",count,errorVal);


  // auto xtb =   thrust::make_permutation_iterator(Xtemp2.begin(),thrust::functional_iterator<funcOp2>(funcOp2()));
  // auto erb = thrust::make_permutation_iterator(error.begin(),thrust::functional_iterator<funcOp>(funcOp()));
  //
  // thrust::transform(Xtemp.begin(),Xtemp.end(),erb,xtb, thrust::multiplies<float>());
  // thrust::transform(thrust::functional_iterator<funcOp3>(funcOp3()),thrust::functional_iterator<funcOp3>(funcOp3(),D*N),emptyVector.begin(),thrust::identity<long>());
  // // printf("Keys\n");
  // // for(int i = 0; i<3;i++)
  // // {
  // //   printf("%d\n",(int)emptyVector[i]);
  // // }
  // auto new_end = thrust::reduce_by_key(emptyVector.begin(),emptyVector.end(),
  //                                      Xtemp2.begin(),y_pred.begin(),gradient.begin());

  // printf("Size of gradient = %d\n",new_end.second-gradient.begin());

  // host_gradient = gradient;
  // printf("Gradient\n");
  // for(int i = 0; i<D;i++)
  // {
  //   printf("%f\n",(float)host_gradient[i]);
  // }
  // printf("Error\n");
  // for(int i = 0; i<N;i++)
  // {
  //   printf("%f\n",(float)error[i]);
  // }
  // printf("X values\n");
  // for(int i = 0; i<N;i++)
  // {
  //   for(int j = 0; j<D;j++)
  //   {
  //     printf("%f ",(float)Xtemp2[i*D + j]);
  //   }
  //   printf("|---------|");
  //   for(int j = 0; j<D;j++)
  //   {
  //     printf("%f ",(float)Xtemp[i*D+j]);
  //   }
  //   printf("\n");
  // }

  // thrust::transform(thrust::host,host_weights.begin(),host_weights.end(),host_gradient.begin(),host_weights.begin(),gradSub());
  // printf("alkdsf");
  count++;
  }
  return 0;
}
