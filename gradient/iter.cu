#include <thrust/iterator/functional_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/shared_for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <cstdlib>
#include <ctime>
#define D 32
#define N 10000
__constant__ float weights[D];
struct GenRand
{
    __host__ __device__ float operator() (float idx)
    {
        thrust::default_random_engine randEng;
        thrust::uniform_real_distribution<float> uniDist;
        randEng.discard(idx);
        return uniDist(randEng)*5;
    }
};
class funcOp
{
public:
  __host__ __device__ long operator() (long index) const
  {
    return index/D;
  }
};
class funcOp2
{
public:
  __host__ __device__ long operator() (long index) const
  {
    return index/D + (index%D)*N;
  }
};
class funcOp3
{
public:
  __host__ __device__ long operator() (long index) const
  {
    return index/N;
  }
};
struct floatsD
{
float data[D];
__host__ __device__ float& operator[] (int index)
{
  return data[index];
}
};

struct dotProductFunctor
{
  __device__ float operator() (floatsD &vector)
  {
    float temp;
    for(int i = 0; i<D;i++)
    {
      temp+=vector[i]*weights[i];
    }
    return temp;
  }
};
struct gradSub
{
  float operator() (float &weight, float&gradient)
  {
    return weight - 0.00001*gradient/N;
  }
};
int main()
{
  srand (static_cast <unsigned> (time(0)));
  float hostRandomArray[D*N];
  for(int i = 0; i<D*N;i++)
  {
    hostRandomArray[i]=static_cast <float> (rand()) / static_cast <float> (RAND_MAX/5);
  }
  thrust::host_vector<float> host_gradient(D),host_weights(D);
  for(int i = 0; i<D;i++)
  {
    host_weights[i]=0.1*(static_cast <float> (rand()) / static_cast <float> (RAND_MAX));
  }
  thrust::device_vector<floatsD>Xinput(N);
  thrust::device_vector<float>Y_actual(N);
  thrust::device_vector<float>Xtemp(D*N);
  thrust::device_vector<float>Xtemp2(D*N);
  thrust::device_vector<float> y_pred(N),error(N);
  thrust::device_vector<float> gradient(D);
  hipMemcpy(Xinput.data().get(),hostRandomArray,sizeof(float)*D*N,hipMemcpyHostToDevice);
  hipMemcpy(Xtemp.data().get(),Xinput.data().get(),sizeof(float)*D*N,hipMemcpyDeviceToDevice);
  thrust::transform(thrust::make_counting_iterator(0),thrust::make_counting_iterator(N),Y_actual.begin(),GenRand());
  int count = 0;
  while(count<1000)
  {
  hipMemcpyToSymbol(HIP_SYMBOL(weights),host_weights.data(),sizeof(float)*D);
  thrust::transform(Xinput.begin(),Xinput.end(),y_pred.begin(),dotProductFunctor());
  // for(int i = 0; i< 100;i++)
  // {
  //   printf("%f\n",((floatD)Xinput[i])[0]);
  // }
  for(int i = 0; i< 10;i++)
  {
    printf("%f-%f\n",(float)Y_actual[i],(float)y_pred[i]);
  }
  printf("-----\n");
  thrust::transform(y_pred.begin(),y_pred.end(),Y_actual.begin(),error.begin(),thrust::minus<float>());
  auto xtb =   thrust::make_permutation_iterator(Xtemp2.begin(),thrust::functional_iterator<funcOp2>(funcOp2()));
  auto erb = thrust::make_permutation_iterator(error.begin(),thrust::functional_iterator<funcOp>(funcOp()));

  thrust::transform(Xtemp.begin(),Xtemp.end(),erb,xtb, thrust::multiplies<float>());
  auto new_end = thrust::reduce_by_key(thrust::functional_iterator<funcOp3>(funcOp3()),
                                       thrust::functional_iterator<funcOp3>(funcOp3(),D*N),
                                       Xtemp2.begin(),Xtemp.begin(),gradient.begin());
  // printf("Size of gradient = %d\n",new_end.second-gradient.begin());
  host_gradient = gradient;
  thrust::transform(thrust::host,host_weights.begin(),host_weights.end(),host_gradient.begin(),host_weights.begin(),gradSub());
  // printf("alkdsf");
  count++;
  }
  return 0;
}
