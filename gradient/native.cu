
#include <hip/hip_runtime.h>
#include <fstream>
__constant__ float c_weights[10000];
int main(int argc, char **argv)
{
  std::ifstream values;
  values.open("/dev/shm/values.txt");
  int D,N;
  int niter = atoi(argv[1]);
  float learn = atof(argv[2]);
  float *xvalues,*y_actual,*real_weights,*weights;
  values>>D>>N;
  xvalues = new float [D*N];
  for(int i = 0 ; i<N;i++)
  {
    for (int j = 0; j<D;j++)
    {
      values>>xvalues[j*N + i];
    }
  }
  y_actual = new float[N];
  for(int i = 0; i<N;i++)
  {
    values>>y_actual[i];
  }
  real_weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>real_weights[i];
  }
  weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>weights[i];
  }
  printf("Done Reading Data\n");
  float * d_xval, *d_yval,* d_weights;
  hipMalloc((void**)&d_xval,sizeof(float)*D*N);
  hipMalloc((void**)&d_weights,sizeof(float)*D);
  hipMalloc((void**)&d_yval,sizeof(float)*N);
  hipMemcpy(d_xval,xvalues,sizeof(float)*D*N,hipMemcpyHostToDevice);
  hipMemcpy(d_yval,(void *)y_actual,sizeof(float)*N,hipMemcpyHostToDevice);
  hipMemcpy(d_weights,(void *)weights,sizeof(float)*D,hipMemcpyHostToDevice);
  int count = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  while(count<niter)
  {
    hipMemcpyToSymbol(HIP_SYMBOL(c_weights),d_weights,sizeof(float)*D,hipMemcpyDeviceToDevice);
  }


}
