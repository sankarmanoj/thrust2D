#include "hip/hip_runtime.h"
#include <fstream>
#include "native_kernel.h"
int main(int argc, char **argv)
{
  std::ifstream values;
  values.open("/dev/shm/values.txt");
  int D,N;
  int niter = atoi(argv[1]);
  float learn = atof(argv[2]);
  float *xvalues,*y_actual,*real_weights,*weights;
  values>>D>>N;
  xvalues = new float [D*N];
  for(int i = 0 ; i<N;i++)
  {
    for (int j = 0; j<D;j++)
    {
      values>>xvalues[j*N + i];
    }
  }
  y_actual = new float[N];
  for(int i = 0; i<N;i++)
  {
    values>>y_actual[i];
  }
  real_weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>real_weights[i];
  }
  weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>weights[i];
  }
  printf("Done Reading Data\n");
  float * d_xval, *d_yval,* d_weights, *d_error,*h_error;
  hipMalloc((void**)&d_xval,sizeof(float)*D*N);
  hipMalloc((void**)&d_weights,sizeof(float)*D);
  hipMalloc((void**)&d_yval,sizeof(float)*N);
  hipMalloc((void**)&d_error,sizeof(float)*N);
  hipMemcpy(d_xval,xvalues,sizeof(float)*D*N,hipMemcpyHostToDevice);
  hipMemcpy(d_yval,(void *)y_actual,sizeof(float)*N,hipMemcpyHostToDevice);
  hipMemcpy(d_weights,(void *)weights,sizeof(float)*D,hipMemcpyHostToDevice);
  int count = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  printf("Loop Start\n");
  while(count<niter)
  {
    hipMemcpyToSymbol(HIP_SYMBOL(c_weights),d_weights,sizeof(float)*D,0,hipMemcpyDeviceToDevice);
    getError<<<iDivUp(N,1024),1024>>>(N,D,d_xval,d_yval,d_error);
    updateWeight<<<iDivUp(D,1024),1024>>>(N,D,learn,d_xval,d_error,d_weights);
    count++;
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float time_in_ms;
  hipEventElapsedTime(&time_in_ms,start,stop);
  h_error = new float[N];
  hipMemcpy(h_error,d_error,sizeof(float)*N,hipMemcpyDeviceToHost);
  for(int i = 0; i<100;i++)
  {
    printf("%f ",h_error[i]);
    if(i%10==0)
      printf("\n");
  }
  printf("Compute Time = %f\n",time_in_ms);
  // sdiff = sqrt(sdiff/D);
  // printf("Final Error = %f\n",sdiff);

}
