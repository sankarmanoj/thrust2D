#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/constant_memory.h>
#include <thrust/shared_algorithms.h>
#include "descent-struct.h"
#include <fstream>

int main(int argc, char **argv)
{
  std::ifstream values;
  values.open("/dev/shm/values.txt");
  int D,N;
  int niter = atoi(argv[1]);
  float learn = atof(argv[2]);
  float *xvalues,*y_actual,*real_weights,*weights;
  values>>D>>N;
  xvalues = new float [D*N];
  for(int i = 0 ; i<N;i++)
  {
    for (int j = 0; j<D;j++)
    {
      values>>xvalues[j*N + i];
    }
  }
  y_actual = new float[N];
  for(int i = 0; i<N;i++)
  {
    values>>y_actual[i];
  }
  real_weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>real_weights[i];
  }
  weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>weights[i];
  }
  // printf("Done Reading Data\n");
  thrust::device_vector<float> d_Xvalues(xvalues,xvalues+D*N);
  thrust::device_vector<float> d_Yactual(y_actual,y_actual+N);
  thrust::device_vector<float> d_Ypredict(N);
  thrust::device_vector<float> d_error(N);
  thrust::device_vector<float> d_weights(D);
  d_weights.assign(weights, weights + D);
  thrust::device_vector<float> d_gradient(D);
  thrust::host_vector<floatD> h_XD(N);
  thrust::host_vector<float> h_gradient(D);
  thrust::host_vector<float> h_error(N);
  for(int i = 0; i<N;i++)
  {
    h_XD[i].data = d_Xvalues.data().get() + i;
    h_XD[i].N=N;
  }
  thrust::device_vector<floatD> d_XD;
  d_XD = h_XD;
  float threshold_error = 1.0f;
  float error;
  do
  {
    thrust::constant_vector<float> ca_weights(d_weights.begin(),d_weights.end());
    thrust::transform(thrust::cuda::shared,d_XD.begin(),d_XD.end(),d_Yactual.begin(),d_error.begin(),dotProductFunctor<thrust::constant_vector<float>>(D,ca_weights));
    error = (thrust::transform_reduce(thrust::cuda::shared,d_error.begin(),d_error.end(),squareOp()))/N;
    for(int i = 0; i<D;i++)
    {
      h_gradient[i]=thrust::transform_reduce(thrust::cuda::shared,d_Xvalues.begin()+i*N,d_Xvalues.begin()+(i+1)*N,d_error.begin(),thrust::multiplies<float>())/N;

    }
    d_gradient = h_gradient;

    thrust::transform(thrust::cuda::shared, d_weights.begin(),d_weights.end(),d_gradient.begin(),d_weights.begin(),update_weights(learn));
  }
  while(error > threshold_error);
  h_error = d_error;
  // for(int i = 0; i<100;i++)
  // {
  //   printf("%f ",h_error[i]);
  //   if(i%10==0)
  //     printf("\n");
  // }
  // printf("Compute Time = %f\n",time_in_ms);
  // sdiff = sqrt(sdiff/D);
  // printf("Final Error = %f\n",sdiff);
  // delete xvalues;
  // delete real_weights;
  // delete y_actual;
  // delete weights;
  return 0;
}
