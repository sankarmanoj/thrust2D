#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/constant_memory.h>
#include <thrust/shared_for_each.h>
#include "descent-struct.h"
#include <fstream>

int main(int argc, char **argv)
{
  std::ifstream values;
  values.open("/dev/shm/values.txt");
  int D,N;
  int niter = atoi(argv[1]);
  float learn = atof(argv[2]);
  float *xvalues,*y_actual,*real_weights,*weights;
  values>>D>>N;
  xvalues = new float [D*N];
  for(int i = 0 ; i<N;i++)
  {
    for (int j = 0; j<D;j++)
    {
      values>>xvalues[j*N + i];
    }
  }
  y_actual = new float[N];
  for(int i = 0; i<N;i++)
  {
    values>>y_actual[i];
  }
  real_weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>real_weights[i];
  }
  weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>weights[i];
  }
  // printf("Done Reading Data\n");
  thrust::device_vector<float> d_Xvalues(xvalues,xvalues+D*N);
  thrust::device_vector<float> d_Yactual(y_actual,y_actual+N);
  thrust::device_vector<float> d_Ypredict(N);
  thrust::device_vector<float> d_error(N);
  thrust::host_vector<floatD> h_XD(N);
  thrust::host_vector<float> h_gradient(D);
  thrust::host_vector<float> h_error(N);
  for(int i = 0; i<N;i++)
  {
    h_XD[i].data = d_Xvalues.data().get() + i;
    h_XD[i].N=N;
  }
  thrust::device_vector<floatD> d_XD;
  d_XD = h_XD;
  int count = 0;
  while(count<niter)
  {
    float* ca_weights = thrust::get_constant_memory_pointer(weights,weights+D,hipMemoryTypeHost);
    // float *ca_weights = d_weights.data().get();
    thrust::transform(thrust::cuda::shared,d_XD.begin(),d_XD.end(),d_Ypredict.begin(),dotProductFunctor(D,ca_weights));
    thrust::transform(thrust::cuda::shared,d_Ypredict.begin(),d_Ypredict.end(),d_Yactual.begin(),d_error.begin(),thrust::minus<float>());
    // for (size_t i = 0; i < 10; i++)
    // {
    //   printf("%f\n",(float) d_Ypredict[i]);
    // }
    printf("%d Error = %.9f\n",count,(float)thrust::transform_reduce(thrust::cuda::shared,d_error.begin(),d_error.end(),squareOp(),0,thrust::plus<float>())/N);
    for(int i = 0; i<30;i++)
    {
      h_gradient[i]=thrust::transform_reduce(thrust::cuda::shared,d_Xvalues.begin()+i*N,d_Xvalues.begin()+(i+1)*N,d_error.begin(),thrust::multiplies<float>())/N;
      // h_gradient[i]=thrust::reduce(thrust::cuda::shared,d_Ypredict.begin(),d_Ypredict.end())/N;
      // printf("%f\n",h_gradient[i]);
    }
    for(int i = 0; i<D;i++)
    {
      weights[i] = weights[i] - learn*h_gradient[i];
    }
    count++;
  }
  h_error = d_error;
  for(int i = 0; i<100;i++)
  {
    printf("%f ",h_error[i]);
    if(i%10==0)
      printf("\n");
  }
  // printf("Compute Time = %f\n",time_in_ms);
  // sdiff = sqrt(sdiff/D);
  // printf("Final Error = %f\n",sdiff);
  // delete xvalues;
  // delete real_weights;
  // delete y_actual;
  // delete weights;
  return 0;
}
