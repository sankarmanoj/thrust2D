#include <thrust/device_vector.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/sequence.h>
#include <thrust/for_each.h>
#include <thrust/random.h>
#include <thrust/reduce.h>

#define D 20
#define N 10000
__constant__ float weights[D];
struct indexfunctor1
{
  __device__ void operator() (int &i)
  {
    i= i/D;
  }
};
struct indexfunctor2
{
  __device__ void operator() (int &i)
  {
    i= (i%D)*N + i/D;
  }
};
struct GenRand
{
    __device__ float operator() (float idx)
    {
        thrust::default_random_engine randEng;
        thrust::uniform_real_distribution<float> uniDist;
        randEng.discard(idx);
        return uniDist(randEng)*5;
    }
};
struct dotFunctor
{
    __device__ float operator() (float &input,int &index)
    {
      return input*weights[index%D];
    }
};
struct dotFunctor2
{
    __device__ float operator() (float &input,float &index)
    {
      float returnval = input*index;
      // printf("%f*%f=%f\n",input,index,returnval);
      return returnval;
    }
};

int main()
{

  thrust::device_vector<int> map(D*N);
  thrust::device_vector<int> tap(N*D);
  thrust::sequence(map.begin(),map.end());
  thrust::for_each(map.begin(),map.end(),indexfunctor1());
  thrust::for_each(tap.begin(),tap.end(),indexfunctor2());
  thrust::device_vector<float>Xinput(D*N);
  thrust::device_vector<float>Y_actual(N);
  float host_weights[] = {2,3,4,5,2,2,3,4,5,2,2,3,4,5,2,2,3,4,5,2,2,3,4,5,2,2,3,4,5,2,2,3,4,5,2,2,3,4,5,2,2,3,4,5,2,2,3,4,5,2,2,3,4,5,2};
  hipMemcpyToSymbol(HIP_SYMBOL(weights),host_weights,sizeof(float)*D);
  thrust::transform(thrust::make_counting_iterator(0),thrust::make_counting_iterator(D*N),Xinput.begin(),GenRand());
  thrust::transform(thrust::make_counting_iterator(0),thrust::make_counting_iterator(N),Y_actual.begin(),GenRand());

  thrust::device_vector<float>Xtemp(D*N);
  thrust::transform(Xinput.begin(),Xinput.end(),thrust::make_counting_iterator(0),Xtemp.begin(),dotFunctor());
  // for(int i = 0; i<30;i++)
  // {
  //  printf("%f*%f=%f\n",(float)Xinput[i],host_weights[i%D],(float)Xtemp[i]);
  // }
  thrust::device_vector<int> map_out(N);
  thrust::device_vector<float> y_pred(N),error(N);
  thrust::pair<thrust::detail::normal_iterator<thrust::device_ptr<int> >,thrust::detail::normal_iterator<thrust::device_ptr<float> > > new_end;
  new_end = thrust::reduce_by_key(map.begin(),map.end(),Xtemp.begin(),map_out.begin(),y_pred.begin());
  // for(int i = 0; i<N;i++)
  // {
  //  printf("%f=%f___%f-%f-%f-%f-%f\n",(float) y_pred[i],(float) Xtemp[i*D]+(float) Xtemp[i*D+1]+(float) Xtemp[i*D+2]+(float) Xtemp[i*D+3]+(float) Xtemp[i*D+4],
  //                                               (float) Xtemp[i*D],(float) Xtemp[i*D+1],(float) Xtemp[i*D+2],(float) Xtemp[i*D+3],(float) Xtemp[i*D+4]);
  // }
  thrust::transform(y_pred.begin(),y_pred.end(),Y_actual.begin(),error.begin(),thrust::minus<float>());
  // for(int i = 0; i<N;i++)
  // {
  //   printf("%f-%f=%f\n",(float)y_pred[i],(float)Y_actual[i],(float)error[i]);
  // }
  // thrust::transform(thrust::make_permutation_iterator(Xinput.begin(),tap.begin()),
  //                   thrust::make_permutation_iterator(Xinput.end(),tap.end()),
  //                   thrust::make_permutation_iterator(error.begin(),map.begin()),Xtemp.begin(),dotFunctor2());
  // for(int i = 0; i<D;i++)
  // {
  //   for(int j = 0;j<N;j++)
  //     {
  //         printf("%f*%f=%f\n",(float)Xinput[i*N + j],(float)error[j],(float)Xtemp[i*N + j]);
  //     }
  // }

  return 0;
}
