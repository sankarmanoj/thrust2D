#include "hip/hip_runtime.h"
#include <thrust/iterator/functional_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/shared_for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include "descent-struct.h"
#include <fstream>

int main()
{
  std::ifstream values;
  values.open("values.txt");
  int D,N;
  float *xvalues,*y_actual,*real_weights,*weights;
  values>>D>>N;
  printf("D=%d N=%d\n",D,N);
  xvalues = new float [D*N];
  for(int i = 0 ; i<N;i++)
  {
    for (int j = 0; j<D;j++)
    {
      values>>xvalues[j*N + i];
    }
  }
  y_actual = new float[N];
  for(int i = 0; i<N;i++)
  {
    values>>y_actual[i];
  }
  real_weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>real_weights[i];
  }
  weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>weights[i];
  }
  thrust::device_vector<float> d_Xvalues(xvalues,xvalues+D*N);
  thrust::device_vector<float> d_Yactual(y_actual,y_actual+N);
  thrust::device_vector<float> d_Ypredict(N);
  thrust::device_vector<float> d_error(N);
  thrust::device_vector<float> d_weights(weights,weights+D);
  thrust::host_vector<floatD> h_XD(N);
  thrust::host_vector<float> h_gradient(D);
  for(int i = 0; i<N;i++)
  {
    h_XD[i].data = d_Xvalues.data().get() + i;
    h_XD[i].N=N;
  }
  thrust::device_vector<floatD> d_XD;
  d_XD = h_XD;
  int count = 0;
  while(count<10)
  {
    hipMemcpyToSymbol(HIP_SYMBOL(c_weights),weights,sizeof(float)*D);
    thrust::transform(d_XD.begin(),d_XD.end(),d_Ypredict.begin(),dotProductFunctor(D));
    thrust::transform(d_Ypredict.begin(),d_Ypredict.end(),d_Yactual.begin(),d_error.begin(),thrust::minus<float>());
    printf("%d Error = %f\n",count,sqrt(thrust::transform_reduce(d_error.begin(),d_error.end(),squareOp(),0,thrust::plus<float>())/N));
    for(int i = 0; i<D;i++)
    {
      thrust::transform(d_Xvalues.begin()+i*N,d_Xvalues.begin()+(i+1)*N,d_error.begin(),d_Ypredict.begin(),thrust::multiplies<float>());
      h_gradient[i]=thrust::reduce(d_Ypredict.begin(),d_Ypredict.end())/N;
    }
    for(int i = 0; i<D;i++)
    {
      weights[i] = weights[i] -0.001*h_gradient[i];
    }
    count++;
  }
  float sdiff=0.0;
  for (int i=0; i<D; i++)
  {
    sdiff += (weights[i]-real_weights[i])*(weights[i]-real_weights[i]);
  }
  sdiff = sqrt(sdiff/D);
  printf("Final Error = %f\n",sdiff);
}
