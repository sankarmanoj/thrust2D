#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/constant_memory.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/reduce.h>
#include "descent_struct.h"
#include <fstream>

int main(int argc, char **argv)
{
  std::ifstream values;
  values.open("/dev/shm/values.txt");
  int D,N;
  int niter = atoi(argv[1]);
  float learn = atof(argv[2]);
  float *xvalues,*y_actual,*real_weights,*weights;
  values>>D>>N;
  xvalues = new float [D*N];
  for(int i = 0 ; i<N;i++)
  {
    for (int j = 0; j<D;j++)
    {
      values>>xvalues[j*N + i];
    }
  }
  y_actual = new float[N];
  for(int i = 0; i<N;i++)
  {
    values>>y_actual[i];
  }
  real_weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>real_weights[i];
  }
  weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>weights[i];
  }
  // printf("Done Reading Data\n");
  float threshold_error = 1;
  thrust::device_vector<float> d_Xvalues(xvalues,xvalues+D*N);
  thrust::device_vector<float> d_Yactual(y_actual,y_actual+N);
  thrust::device_vector<float> d_Ypredict(N);
  thrust::device_vector<float> d_error(N);
  thrust::device_vector<float> d_weights(D);
  d_weights.assign(weights, weights + D);
  thrust::device_vector<float> d_gradient(D);
  thrust::host_vector<floatD> h_XD(N);
  thrust::host_vector<float> h_gradient(D);
  thrust::host_vector<float> h_error(N);
  for(int i = 0; i<N;i++)
  {
    h_XD[i].data = d_Xvalues.data().get() + i;
    h_XD[i].N=N;
  }
  thrust::device_vector<floatD> d_XD;
  d_XD = h_XD;
  float error ;
  do
  {
    float *ca_weights = d_weights.data().get();
    thrust::transform(d_XD.begin(),d_XD.end(),d_Yactual.begin(),d_error.begin(),dotProductFunctor<float *>(D,ca_weights));
    error = thrust::transform_reduce(d_error.begin(),d_error.end(),squareOp(),0,thrust::plus<float>())/N;
    printf("Error = %f\n",error);
    for(int i = 0; i<D;i++)
    {
      thrust::transform(d_Xvalues.begin()+i*N,d_Xvalues.begin()+(i+1)*N,d_error.begin(),d_Ypredict.begin(),thrust::multiplies<float>());
      h_gradient[i]=thrust::reduce(d_Ypredict.begin(),d_Ypredict.end())/N;
    }
    d_gradient = h_gradient;
    thrust::transform(d_weights.begin(),d_weights.end(),d_gradient.begin(),d_weights.begin(),update_weights(learn));
  }
  while(error>=threshold_error);


  h_error = d_error;
  // printf("Compute Time = %f\n",time_in_ms);
  // sdiff = sqrt(sdiff/D);
  // printf("Final Error = %f\n",sdiff);
  // delete xvalues;
  // delete real_weights;
  // delete y_actual;
  // delete weights;
  return 0;
}
