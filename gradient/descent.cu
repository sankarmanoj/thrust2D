#include "hip/hip_runtime.h"
#include <thrust/iterator/functional_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/shared_for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include "descent-struct.h"
#include <fstream>

int main(int argc, char **argv)
{
  std::ifstream values;
  values.open("/dev/shm/values.txt");
  int D,N;
  int niter = atoi(argv[1]);
  float learn = atof(argv[2]);
  float *xvalues,*y_actual,*real_weights,*weights;
  values>>D>>N;
  xvalues = new float [D*N];
  for(int i = 0 ; i<N;i++)
  {
    for (int j = 0; j<D;j++)
    {
      values>>xvalues[j*N + i];
    }
  }
  y_actual = new float[N];
  for(int i = 0; i<N;i++)
  {
    values>>y_actual[i];
  }
  real_weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>real_weights[i];
  }
  weights = new float[D];
  for(int i = 0; i<D;i++)
  {
    values>>weights[i];
  }
  printf("Done Reading Data\n");
  thrust::device_vector<float> d_Xvalues(xvalues,xvalues+D*N);
  thrust::device_vector<float> d_Yactual(y_actual,y_actual+N);
  thrust::device_vector<float> d_Ypredict(N);
  thrust::device_vector<float> d_error(N);
  thrust::device_vector<float> d_weights(weights,weights+D);
  thrust::host_vector<floatD> h_XD(N);
  thrust::host_vector<float> h_gradient(D);
  for(int i = 0; i<N;i++)
  {
    h_XD[i].data = d_Xvalues.data().get() + i;
    h_XD[i].N=N;
  }
  thrust::device_vector<floatD> d_XD;
  d_XD = h_XD;
  int count = 0;
  while(count<niter)
  {
    hipMemcpyToSymbol(HIP_SYMBOL(c_weights),weights,sizeof(float)*D);
    thrust::transform(d_XD.begin(),d_XD.end(),d_Ypredict.begin(),dotProductFunctor(D));
    thrust::transform(d_Ypredict.begin(),d_Ypredict.end(),d_Yactual.begin(),d_error.begin(),thrust::minus<float>());
    // for (size_t i = 0; i < 10; i++)
    // {
    //   printf("%f\n",(float) d_Ypredict[i]);
    // }
    // printf("%d Error = %.9f\n",count,(float)thrust::transform_reduce(d_error.begin(),d_error.end(),squareOp(),0,thrust::plus<float>())/N);
    for(int i = 0; i<D;i++)
    {
      thrust::transform(d_Xvalues.begin()+i*N,d_Xvalues.begin()+(i+1)*N,d_error.begin(),d_Ypredict.begin(),thrust::multiplies<float>());
      h_gradient[i]=thrust::reduce(d_Ypredict.begin(),d_Ypredict.end())/N;
      // printf("%f\n",h_gradient[i]);
    }
    for(int i = 0; i<D;i++)
    {
      weights[i] = weights[i] - learn*h_gradient[i];
    }
    count++;
  }
  // float sdiff=0.0;
  for (int i=0; i<D; i++)
  {
    printf("%f %f\n", weights[i],real_weights[i]);
    // sdiff += (weights[i]-real_weights[i])*(weights[i]-real_weights[i]);
  }
  // sdiff = sqrt(sdiff/D);
  // printf("Final Error = %f\n",sdiff);
}
