#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>

#ifdef RD_WG_SIZE_0_0
        #define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
        #define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
        #define BLOCK_SIZE RD_WG_SIZE
#else
        #define BLOCK_SIZE 16
#endif

#define STR_SIZE 256

/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5

#define AMBIENT_TEMP 80

/* chip parameters	*/
float t_chip = 0.0005;
float chip_height = 0.016;
float chip_width = 0.016;
/* ambient temperature, assuming no package at all	*/
float amb_temp = 80.0;

void run(int argc, char** argv);

/* define timer macros */
#define pin_stats_reset()   startCycle()
#define pin_stats_pause(cycles)   stopCycle(cycles)
#define pin_stats_dump(cycles)    printf("timer: %Lu\n", cycles)

void fatal(char *s)
{
	fprintf(stderr, "error: %s\n", s);

}

void writeoutput(thrust::host_vector<float> vect, int grid_rows, int grid_cols, char *file){
	int i,j, index=0;
	FILE *fp;
	char str[STR_SIZE];

	if( (fp = fopen(file, "w" )) == 0 )
        printf( "The file was not opened\n" );


	for (i=0; i < grid_rows; i++)
	 for (j=0; j < grid_cols; j++)
	 {

		sprintf(str, "%d\t%g\n", index, vect[i*grid_cols+j]);
		fputs(str,fp);
		index++;
	 }

      fclose(fp);
}


void readinput(float * vect, int grid_rows, int grid_cols, char *file){

  	int i,j;
	FILE *fp;
	char str[STR_SIZE];
	float val;

	if( (fp  = fopen(file, "r" )) ==0 )
        printf( "The file was not opened\n" );


	for (i=0; i <= grid_rows-1; i++)
	 for (j=0; j <= grid_cols-1; j++)
	 {
		fgets(str, STR_SIZE, fp);
		if (feof(fp))
			fatal((char *)"not enough lines in file");
		//if ((sscanf(str, "%d%f", &index, &val) != 2) || (index != ((i-1)*(grid_cols-2)+j-1)))
		if ((sscanf(str, "%f", &val) != 1))
			fatal((char *)"invalid file format");
		vect[i*grid_cols+j] = val;
	}

	fclose(fp);

}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))


template<class T>
class HotspotFunctor : public thrust::window_for_each_functor<T>
{
    thrust::Block_2D<T> *MatrixPower;
    int iteration;
    int col;
    int row;
    int borderCols;
    int borderRows;
    float stepDivCap;
    float Rx_1;
    float Ry_1;
    float Rz_1;
public:

    HotspotFunctor (thrust::Block_2D<T> *MatrixPower,int iteration,int col,int row, int borderCols,int borderRows,float stepDivCap,float Rx_1,float Ry_1,float Rz_1)
    {
        this->MatrixPower = MatrixPower;
        this->iteration = iteration;
        this->col = col;
        this->row = row;
        this->borderCols = borderCols;
        this->borderRows = borderRows;
        this->stepDivCap = stepDivCap;
        this->Rx_1 = Rx_1;
        this->Ry_1 = Ry_1;
        this->Rz_1 = Rz_1;
    }

	__device__ void operator() (thrust::window_2D<T> w)
	{
        int ty = w.window_dim_y/2;
        int tx = w.window_dim_x/2;
        int rty = w.start_y + ty;
        int rtx = w.start_x + tx;
        int N = ty-1;
        int S = ty+1;
        int W = tx-1;
        int E = tx+1;

        float my_power = (*MatrixPower)[rtx][rty];
        for (int i=0; i<iteration ; i++)
        {
            w[ty][tx] =  w[ty][tx] + stepDivCap * (my_power + \
                (w[S][tx] + w[N][tx] - 2.0*(w[ty][tx])) * Ry_1 + \
                (w[ty][E] + w[ty][W] - 2.0*(w[ty][tx])) * Rx_1 + \
                (AMBIENT_TEMP - w[ty][tx]) * Rz_1);
            //printf("%d\n",(T) w[ty][tx]);
         }
	}
};/*
   compute N time steps
*/
int thrustCompute(thrust::device_vector<float> MatrixPower,thrust::device_vector<float> MatrixTemp, int col, int row, \
		int total_iterations, int num_iterations, int blockCols, int blockRows, int borderCols, int borderRows, int size)
{

  // dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  // dim3 dimGrid(blockCols, blockRows);

  thrust::counting_iterator<int> it_begin(0);		// used when thread ID is required

	float grid_height = chip_height / row;
	float grid_width = chip_width / col;

	float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
	float Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
	float Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
	float Rz = t_chip / (K_SI * grid_height * grid_width);

	float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
	float step = PRECISION / max_slope;
	float t;
  // float time_elapsed=0.001;
  //int src = 0, dst = 1;

  // float amb_temp = 80.0;
  float step_div_Cap;
  float Rx_1,Ry_1,Rz_1;
  step_div_Cap=step/Cap;

  Rx_1=1/Rx;
  Ry_1=1/Ry;
  Rz_1=1/Rz;

  thrust::Block_2D<float> TemperatureBlock(col,row);
  TemperatureBlock.copy(MatrixTemp.begin(), MatrixTemp.end());

  thrust::Block_2D<float> PowerBlock(col,row);
  PowerBlock.copy(MatrixPower.begin(), MatrixPower.end());

	for (t = 0; t < total_iterations; t+=num_iterations) {
        int requiredIterations = MIN(num_iterations,total_iterations-t);
        //calculate_temp<<<dimGrid, dimBlock>>>(requiredIterations, MatrixPower,MatrixTemp[src],MatrixTemp[dst],\
    col,row,borderCols, borderRows, Cap,Rx,Ry,Rz,step,time_elapsed);
        HotspotFunctor<float> functor(PowerBlock.get_device_pointer(),requiredIterations,col,row,borderCols,borderRows,step_div_Cap,Rx_1,Ry_1,Rz_1);
        thrust::window_for_each(TemperatureBlock.begin(),TemperatureBlock.end(),3,3,3,3,functor);
        //thrust::transform(MatrixTemp[src].begin(), MatrixTemp[src].end(), it_begin,  MatrixTemp[dst].begin(), func_1);

	}
    MatrixTemp=TemperatureBlock.device_data;
        return 0;
}


void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <grid_rows/grid_cols> <pyramid_height> <sim_time> <temp_file> <power_file> <output_file>\n", argv[0]);
	fprintf(stderr, "\t<grid_rows/grid_cols>  - number of rows/cols in the grid (positive integer)\n");
	fprintf(stderr, "\t<pyramid_height> - pyramid heigh(positive integer)\n");
	fprintf(stderr, "\t<sim_time>   - number of iterations\n");
	fprintf(stderr, "\t<temp_file>  - name of the file containing the initial temperature values of each cell\n");
	fprintf(stderr, "\t<power_file> - name of the file containing the dissipated power values of each cell\n");
	fprintf(stderr, "\t<output_file> - name of the output file\n");
	exit(1);
}

int main(int argc, char** argv)
{
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

    run(argc,argv);

    return EXIT_SUCCESS;
}

void run(int argc, char** argv)
{
    int size;
    int grid_rows,grid_cols;
    float *FilesavingTemp,*FilesavingPower,*MatrixOut;
    char *tfile, *pfile, *ofile;

    int total_iterations = 60;
    int pyramid_height = 1; // number of iterations

	  if (argc != 7)
		  usage(argc, argv);
	  if((grid_rows = atoi(argv[1]))<=0||
	   (grid_cols = atoi(argv[1]))<=0||
       (pyramid_height = atoi(argv[2]))<=0||
       (total_iterations = atoi(argv[3]))<=0)
		usage(argc, argv);

	  tfile=argv[4];
    pfile=argv[5];
    ofile=argv[6];

    size=grid_rows*grid_cols;

    /* --------------- pyramid parameters --------------- */
    # define EXPAND_RATE 2// add one iteration will extend the pyramid base by 2 per each borderline
    int borderCols = (pyramid_height)*EXPAND_RATE/2;
    int borderRows = (pyramid_height)*EXPAND_RATE/2;
    int smallBlockCol = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    int smallBlockRow = BLOCK_SIZE-(pyramid_height)*EXPAND_RATE;
    int blockCols = grid_cols/smallBlockCol+((grid_cols%smallBlockCol==0)?0:1);
    int blockRows = grid_rows/smallBlockRow+((grid_rows%smallBlockRow==0)?0:1);

    FilesavingTemp = (float *) malloc(size*sizeof(float));
    FilesavingPower = (float *) malloc(size*sizeof(float));
    MatrixOut = (float *) calloc (size, sizeof(float));

    if( !FilesavingPower || !FilesavingTemp || !MatrixOut)
        fatal((char *)"unable to allocate memory");

    printf("pyramidHeight: %d\ngridSize: [%d, %d]\nborder:[%d, %d]\nblockGrid:[%d, %d]\ntargetBlock:[%d, %d]\n",\
	   pyramid_height, grid_cols, grid_rows, borderCols, borderRows, blockCols, blockRows, smallBlockCol, smallBlockRow);

    readinput(FilesavingTemp, grid_rows, grid_cols, tfile);
    readinput(FilesavingPower, grid_rows, grid_cols, pfile);

    //float *MatrixTemp[2], *MatrixPower;
    //hipMalloc((void**)&MatrixTemp[0], sizeof(float)*size);
    //hipMalloc((void**)&MatrixTemp[1], sizeof(float)*size);
    thrust::device_vector<float> MatrixTemperature;//[2]=
    // {
    //     thrust::device_vector<float> (size),
    //     thrust::device_vector<float> (size),
    // };
    //hipMemcpy(MatrixTemp[0], FilesavingTemp, sizeof(float)*size, hipMemcpyHostToDevice);
    MatrixTemperature.assign(FilesavingTemp,FilesavingTemp+size);
    //hipMalloc((void**)&MatrixPower, sizeof(float)*size);
    thrust::host_vector<float>HostMatrixPowerVector (size);
    thrust::device_vector<float>MatrixPowerVector (size);
    //hipMemcpy(MatrixPower, FilesavingPower, sizeof(float)*size, hipMemcpyHostToDevice);
    HostMatrixPowerVector.assign(FilesavingPower,FilesavingPower+size);
    MatrixPowerVector=HostMatrixPowerVector;
    MatrixPowerVector[0]=1;
    printf("Start computing the transient temperature\n");
    int ret = thrustCompute(MatrixPowerVector,MatrixTemperature,grid_cols,grid_rows, \
	  total_iterations,pyramid_height, blockCols, blockRows, borderCols, borderRows,size);
	  printf("Ending simulation\n");
    //hipMemcpy(MatrixOut, MatrixTemp[ret], sizeof(float)*size, hipMemcpyDeviceToHost);
    //thrust::host_vector<float> MatrixOutput(size);
    //thrust::device_vector<float> resultMatrix(size);
    //resultMatrix.assign(MatrixTemp[ret],MatrixTemp[ret]+size);
    //MatrixOutput=resultMatrix;
    writeoutput(MatrixTemperature,grid_rows, grid_cols, ofile);

    //hipFree(MatrixPower);
    //hipFree(MatrixTemp[0]);
    //hipFree(MatrixTemp[1]);
    //free(MatrixOut);
}
