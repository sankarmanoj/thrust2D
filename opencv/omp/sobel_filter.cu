#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#define PI 3.14159
using namespace cv;
class transFunctor
{
public:
  __device__ uchar operator() (const uchar a,const uchar b) const
  {
    return (uchar) sqrt((float) a*a + b*b);
  }
};

class convolutionFunctor //:public thrust::shared_unary_window_transform_functor<uchar>
{
public:
  int dim;
  thrust::block_2d<float> * kernel;
  convolutionFunctor( thrust::block_2d<float> * kernel,int dim)
  {
    this->dim =dim;
    this->kernel = kernel;
  }
  __device__ uchar operator() (const thrust::window_2d<uchar> & input_window,const thrust::window_2d<uchar> & output_window) const
  {
    uchar temp = 0;
    for(int i = 0; i< dim; i++)
    {
      for(int j = 0; j<dim; j++)
      {
        temp+=input_window[i][j]*(*kernel)[i][j];
      }
    }
    output_window[1][1]=temp;
    return 0.0 ;
  }
};

int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 3;
  image = small;
  thrust::host_block_2d<float> kernelx(dim,dim);
  thrust::host_block_2d<float> kernely(dim,dim);
  thrust::block_2d<float> dkernelx(dim,dim);
  thrust::block_2d<float> dkernely(dim,dim);
  //Sobel Filter
  kernelx[0][0]=-1;
  kernelx[0][1]=0;
  kernelx[0][2]=+1;
  kernelx[1][0]=-2;
  kernelx[1][1]=0;
  kernelx[1][2]=+2;
  kernelx[2][0]=-1;
  kernelx[2][1]=0;
  kernelx[2][2]=+1;
  kernely[0][0]=-1;
  kernely[0][1]=-2;
  kernely[0][2]=-1;
  kernely[1][0]=0;
  kernely[1][1]=0;
  kernely[1][2]=0;
  kernely[2][0]=+1;
  kernely[2][1]=+2;
  kernely[2][2]=+1;
  dkernelx=kernelx;
  dkernely=kernely;
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> convolve1_block (image.cols,image.rows);
  thrust::block_2d<uchar> convolve2_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows);
  thrust::block_2d<uchar> zero_image_block (image.cols,image.rows);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  convolve1_block.upload(img);
  convolve2_block.upload(img);
  thrust::window_vector<uchar> input_wv(&uchar_image_block,dim,dim,1,1);
  thrust::window_vector<uchar> output_wv_x(&convolve1_block,dim,dim,1,1);
  thrust::window_vector<uchar> output_wv_y(&convolve2_block,dim,dim,1,1);

  thrust::transform(input_wv.begin(),input_wv.end(),output_wv_x.begin(),zero_image_block.begin(),convolutionFunctor(dkernelx.device_pointer,dim));
  thrust::transform(input_wv.begin(),input_wv.end(),output_wv_y.begin(),zero_image_block.begin(),convolutionFunctor(dkernely.device_pointer,dim));
  thrust::transform(convolve1_block.begin(),convolve1_block.end(),convolve2_block.begin(),outBlock.begin(),transFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  outBlock.download(&img);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  cudaCheckError();
  imwrite("input.png",image);
  imwrite("output.png",output);

  return 0;
}
