#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }

  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
}
class HarrisIntensityFunctor : public thrust::shared_unary_window_transform_functor<float>
{

public:
  thrust::block_2d<float> * kernel;
  HarrisIntensityFunctor(thrust::block_2d<float> * kernel)
  {
    this->kernel = kernel;
  }
  __device__ void operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {
    float intensityValue;

    for(int xoffset = 1 ; xoffset <=1 ; xoffset++)
    {
      for(int yoffset = 1 ; yoffset <=1 ; yoffset++)
      {
        for(int i = 0; i< 3; i++)
        {
          for(int j = 0; j< 3; j++)
          {
              intensityValue += ((*kernel)[i][j])*(inputWindow[make_int2(j + xoffset,i + yoffset)]-inputWindow[make_int2(i,j)]);
          }
        }
      }
    }
    outputWindow[2][2]=abs(intensityValue);
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  resize(small,image,Size(dim,dim));
  thrust::block_2d<float> float_image_block (image.cols,image.rows,0.0f);
  thrust::block_2d<float> outBlock (image.cols,image.rows,0.0f);
  float * img = (float * )malloc(sizeof(float)*(image.cols*image.rows));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::block_2d<float> kernel(3,3);
  getGaussianKernelBlock(3,5,kernel);
  thrust::window_vector<float> inputVector = thrust::window_vector<float>(&float_image_block,5,5,1,1);
  thrust::window_vector<float> outputVector = thrust::window_vector<float>(&outBlock,5,5,1,1);
  thrust::transform_texture(thrust::cuda::shared,inputVector.begin(),inputVector.end(),outputVector.begin(),HarrisIntensityFunctor(kernel.device_pointer));
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("houtput.png",output);
  return 0;
}
