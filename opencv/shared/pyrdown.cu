#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;

inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }
  float newfloatotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newfloatotal +=  (GaussianKernel)[i][j];
    }
  }
}

// class pyrupfloatransformFunctor
// {
// public:
//   thrust::block_2d<float> *inBlock;
//
// pyrupfloatransformFunctor(thrust::block_2d<float> * inBlock)
//   {
//     this->inBlock = inBlock->device_pointer;
//   }
//   __device__ void operator() (const thrust::window_2d<float> &outputWindow) const
//   {
//     int x_in, y_in;
//     if(outputWindow.start_x%2 && outputWindow.start_y%2)
//     {
//       x_in = outputWindow.start_x*2;
//       y_in = outputWindow.start_y*2;
//       outputWindow[0][0]=(*inBlock)[y_in][x_in];
//     }
//   }
// };

class convolutionFunctor //:public thrust::shared_unary_window_transform_functor<float>
{
public:
  hipTextureObject_t texref;
  convolutionFunctor(   hipTextureObject_t texref)
  {
    this->texref = texref;
  }
  __device__ void operator() (const thrust::window_2d<float> & input_window) const
  {
    input_window[0][0]=0.4*tex2D<float>(texref,input_window.start_x*2,input_window.start_y*2) + \
    0.15*tex2D<float>(texref,input_window.start_x*2+1,input_window.start_y*2) +\
    0.15*tex2D<float>(texref,input_window.start_x*2-1,input_window.start_y*2) +\
    0.15*tex2D<float>(texref,input_window.start_x*2,input_window.start_y*2+1)+\
    0.15*tex2D<float>(texref,input_window.start_x*2,input_window.start_y*2-1);
  }
};

int main(int argc, char const *argv[])
{
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  int dim = 3;
  thrust::block_2d<float> kernel(dim,dim);
  getGaussianKernelBlock(dim,1.0,kernel);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim1 = 512;
  if(argc ==2)
  {
    dim1 = atoi(argv[1]);
  }
  resize(small,image,Size(dim1,dim1));
  thrust::block_2d<float> outBlock (image.cols/2,image.rows/2,0.0f);
  thrust::window_vector<float> output_wv(&outBlock,1,1,1,1);
  float * img = (float * )malloc(sizeof(float)*(image.cols*image.rows));
  float * img1 = (float * )malloc(sizeof(float)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float * aligned_device_memory;
  size_t pitch;
  hipMallocPitch(&aligned_device_memory,&pitch,image.cols*sizeof(float),image.rows);
  hipMemcpy2D(aligned_device_memory,pitch,img,image.cols*sizeof(float),image.cols*sizeof(float),image.rows,hipMemcpyHostToDevice);

  //Create Resource Descriptor
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
  resDesc.res.pitch2D.pitchInBytes=pitch;
  resDesc.res.pitch2D.height = image.rows;
  resDesc.res.pitch2D.width=image.cols;
  resDesc.res.pitch2D.devPtr = aligned_device_memory;
  //Create floatexture Descriptor
  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  //floatexture Object Creation
  hipTextureObject_t texref;
  hipCreateTextureObject(&texref, &resDesc, &texDesc, NULL);

  thrust::for_each(thrust::cuda::shared,output_wv.begin(),output_wv.end(),convolutionFunctor(texref));
  // thrust::window_vector<float> inputVector(&outBlock,1,1,1,1);
  // pyrupfloatransformFunctor ptf(&output_image_block);
  // thrust::for_each(thrust::cuda::shared,inputVector.begin(),inputVector.end(),ptf);
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(outBlock.end()-outBlock.begin()));
  hipMemcpy(img1,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(outBlock.end()-outBlock.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<(outBlock.end()-outBlock.begin());i++)
  {
    outputFloatImageData[i]=(unsigned char)img1[i];
  }
  Mat output (Size(image.cols/2,image.rows/2),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("pyrdown.png",output);
  free (img);
  free (img1);
  free (outputFloatImageData);
  return 0;
}
