#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_for_each.h>
#include <thrust/window_transform.h>
using namespace cv;

class pyrdownTransformFunctor : public thrust::shared_window_for_each_functor<uchar>
{
public:
  thrust::block_2d<uchar> *inBlock;

pyrdownTransformFunctor(thrust::block_2d<uchar> * inBlock)
  {
    this->inBlock = inBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<uchar> &outputWindow) const
  {
    int x_in, y_in;
    // if(outputWindow.start_x%2 && outputWindow.start_y%2)
    {
      x_in = outputWindow.start_x*2;
      y_in = outputWindow.start_y*2;
      outputWindow[0][0]=(*inBlock)[y_in][x_in];
    }
  }
};

int main(int argc, char const *argv[])
{
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  resize(small,image,Size(dim,dim));
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> convolve_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols/2,image.rows/2,0.0f);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  float kernel[5] = {0.0625,0.25,0.375,0.25,0.0625};
  thrust::convolve(thrust::cuda::shared,&uchar_image_block,kernel,kernel,5,&convolve_image_block);
  thrust::window_vector<uchar> outputVector(&outBlock,1,1,1,1);
  pyrdownTransformFunctor ptf(&convolve_image_block);
  thrust::for_each(thrust::cuda::shared,outputVector.begin(),outputVector.end(),ptf);
  outBlock.download(&img);
  Mat output (Size(image.cols/2,image.rows/2),CV_8UC1,img);
  #ifdef OWRITE
  imwrite("input.png",image);
  imwrite("pyrdown.png",output);
  #endif
  #ifdef SHOW
  imshow("input.png",image);
  imshow("pyrdown.png",output);
  waitKey(0);
  #endif
  free (img);
  return 0;
}
