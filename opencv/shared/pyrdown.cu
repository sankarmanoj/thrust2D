#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }

  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
}
class pyrdownTransformFunctor : public thrust::shared_window_for_each_functor<float>
{
public:
  thrust::block_2d<float> *inBlock;

pyrdownTransformFunctor(thrust::block_2d<float> * inBlock)
  {
    this->inBlock = inBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<float> &outputWindow) const
  {
    int x_in, y_in;
    if(outputWindow.start_x%2 && outputWindow.start_y%2)
    {
      x_in = outputWindow.start_x/2;
      y_in = outputWindow.start_y/2;
      outputWindow[0][0]=(*inBlock)[y_in][x_in];
    }
  }
};

int main()
{
  int dim = 5;
  thrust::block_2d<float> kernel(dim,dim);
  getGaussianKernelBlock(dim,1.0,kernel);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image=small;
  hipEvent_t m_start, m_stop;
  hipEventCreate(&m_start);
  hipEventCreate(&m_stop);
  float m_milliseconds;
  hipEventRecord(m_start);
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols*2,image.rows*2,0.0f);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  float * img_out = (float * )malloc(sizeof(float)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  hipEventRecord(m_stop);
  hipEventSynchronize(m_stop);
  hipEventElapsedTime(&m_milliseconds, m_start, m_stop);
  std::cout<<"Time taken from Host to Device = "<<m_milliseconds<<std::endl;
  thrust::window_vector<float> inputVector(&outBlock,1,1,1,1);
  pyrdownTransformFunctor ptf(&float_image_block);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds;
  hipEventRecord(start);
  thrust::for_each(thrust::cuda::shared,inputVector.begin(),inputVector.end(),ptf);
  hipDeviceSynchronize();
  thrust::convolve(outBlock.begin(),outBlock.end(),kernel.begin());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout<<"Time taken on Shared = "<<milliseconds<<std::endl;
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(outBlock.end()-outBlock.begin()));
  hipEventRecord(m_start);
  hipMemcpy(img_out,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(outBlock.end()-outBlock.begin()),hipMemcpyDeviceToHost);
  hipEventRecord(m_stop);
  hipEventSynchronize(m_stop);
  hipEventElapsedTime(&m_milliseconds, m_start, m_stop);
  std::cout<<"Time taken from Device to Host = "<<m_milliseconds<<std::endl;
  for(int i = 0; i<image.cols*image.rows*4;i++)
  {
    outputFloatImageData[i]=(unsigned char)img_out[i];
  }
  Mat output (Size(image.cols*2,image.rows*2),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("pyrdown.png",output);
  return 0;
}
