#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <math.h>
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }
  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
}
class pyrdownTransformFunctor
{
public:
  thrust::block_2d<float> *outBlock;

pyrdownTransformFunctor(thrust::block_2d<float> * outBlock)
  {
    this->outBlock = outBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<float> &inputWindow) const
  {
    int x_out, y_out;
    x_out = inputWindow.start_x*2;
    y_out = inputWindow.start_y*2;
    // if(threadIdx.x%31==0)
    //   printf("%d-%d",inputWindow.start_x,inputWindow.start_y);
    (*outBlock)[y_out][x_out] = inputWindow[0][0];
  }
};

class convolutionFunctor //:public thrust::shared_unary_window_transform_functor<float>
{
public:
  int dim;
  thrust::block_2d<float> * kernel;
  convolutionFunctor( thrust::block_2d<float> * kernel,int dim)
  {
    this->dim =dim;
    this->kernel = kernel;
  }
  __device__ void operator() (const thrust::window_2d<float> & input_window,const thrust::window_2d<float> & output_window) const
  {
    float temp = 0;
    for(int i = 0; i< dim; i++)
    {
      for(int j = 0; j<dim; j++)
      {
        temp+=input_window[i][j]*(*kernel)[i][j];
      }
    }
    output_window[1][1]=temp;
  }
};

int main(int argc, char const *argv[])
{
  int dim = 3;
  thrust::block_2d<float> kernel(dim,dim);
  getGaussianKernelBlock(dim,5.0,kernel);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim1 = 512;
  if(argc ==2)
  {
    dim1 = atoi(argv[1]);
  }
  resize(small,image,Size(dim1,dim1));
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols*2,image.rows*2,0.0f);
  thrust::block_2d<float> output_image_block(image.cols*2,image.rows*2,0.0f);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  float * img_out = (float * )malloc(sizeof(float)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::window_vector<float> inputVector(&float_image_block,1,1,1,1);
  thrust::window_vector<float> inputVector1(&outBlock,dim,dim,1,1);
  pyrdownTransformFunctor ptf(&outBlock);
  thrust::for_each(inputVector.begin(),inputVector.end(),ptf);
  hipDeviceSynchronize();
  thrust::window_vector<float> output_wv(&output_image_block,dim,dim,1,1);
  thrust::transform(thrust::cuda::shared,inputVector1.begin(),inputVector1.end(),output_wv.begin(),convolutionFunctor(kernel.device_pointer,dim));
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(output_image_block.end()-output_image_block.begin()));
  hipMemcpy(img_out,thrust::raw_pointer_cast(output_image_block.data()),sizeof(float)*(output_image_block.end()-output_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows*4;i++)
  {
    outputFloatImageData[i]=(unsigned char)img_out[i];
  }
  Mat output (Size(image.cols*2,image.rows*2),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("pyrdown.png",output);
  return 0;
}
