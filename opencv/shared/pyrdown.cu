#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;

class pyrdownTransformFunctor : public thrust::shared_window_for_each_functor<float>
{
public:
  thrust::block_2d<float> *inBlock;

pyrdownTransformFunctor(thrust::block_2d<float> * inBlock)
  {
    this->inBlock = inBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<float> &outputWindow) const
  {
    int x_in, y_in;
    if(outputWindow.start_x%2 && outputWindow.start_y%2)
    {
      x_in = outputWindow.start_x*2;
      y_in = outputWindow.start_y*2;
      outputWindow[0][0]=(*inBlock)[y_in][x_in];
    }
  }
};

int main()
{
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image=small;
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols/2,image.rows/2,0.0f);
  float * img = (float * )malloc(sizeof(float)*(float_image_block.end()-float_image_block.begin()));
  float * img1 = (float * )malloc(sizeof(float)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  float kernel[3] = {0.25,0.5,0.25};
  thrust::convolve(thrust::cuda::texture,&float_image_block,kernel);
  thrust::window_vector<float> inputVector(&outBlock,1,1,1,1);
  pyrdownTransformFunctor ptf(&float_image_block);
  thrust::for_each(thrust::cuda::texture,inputVector.begin(),inputVector.end(),ptf);
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(outBlock.end()-outBlock.begin()));
  hipMemcpy(img1,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(outBlock.end()-outBlock.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<(outBlock.end()-outBlock.begin());i++)
  {
    outputFloatImageData[i]=(unsigned char)img1[i];
  }
  Mat output (Size(image.cols/2,image.rows/2),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("pyrdown.png",output);
  free (img);
  free (img1);
  free (outputFloatImageData);
  return 0;
}
