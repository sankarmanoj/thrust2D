#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_for_each.h>
#include <thrust/window_transform.h>
using namespace cv;

class pyrdownTransformFunctor : public thrust::shared_window_for_each_functor<uchar>
{
public:
  thrust::block_2d<uchar> *inBlock;

pyrdownTransformFunctor(thrust::block_2d<uchar> * inBlock)
  {
    this->inBlock = inBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<uchar> &outputWindow) const
  {
    int x_in, y_in;
    if(outputWindow.start_x%2 && outputWindow.start_y%2)
    {
      x_in = outputWindow.start_x*2;
      y_in = outputWindow.start_y*2;
      outputWindow[0][0]=(*inBlock)[y_in][x_in];
    }
  }
};

int main(int argc, char const *argv[])
{
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  resize(small,image,Size(dim,dim));
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols/2,image.rows/2,0.0f);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  uchar * img1 = (uchar * )malloc(sizeof(uchar)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  float kernel[3] = {0.25,0.5,0.25};
  thrust::convolve(thrust::cuda::shared,&uchar_image_block,kernel,3,&uchar_image_block);
  thrust::window_vector<uchar> inputVector(&outBlock,1,1,1,1);
  pyrdownTransformFunctor ptf(&uchar_image_block);
  thrust::for_each(thrust::cuda::shared,inputVector.begin(),inputVector.end(),ptf);
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(outBlock.end()-outBlock.begin()));
  outBlock.download(&img);
  for(int i = 0; i<(outBlock.end()-outBlock.begin());i++)
  {
    outputFloatImageData[i]=(unsigned char)img1[i];
  }
  Mat output (Size(image.cols/2,image.rows/2),CV_8UC1,outputFloatImageData);
  #ifdef OWRITE
  imwrite("input.png",image);
  imwrite("pyrdown.png",output);
  #endif
  #ifdef SHOW
  imshow("input.png",image);
  imshow("pyrdown.png",output);
  waitKey(0);
  #endif
  free (img);
  free (img1);
  free (outputFloatImageData);
  return 0;
}
