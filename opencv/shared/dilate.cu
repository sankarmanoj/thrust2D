#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#define PI 3.14159
using namespace cv;
class dilateFunctor : public thrust::shared_unary_window_transform_functor<float>
{
  public:
  __device__ void operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {
    float temp = -1.0;
    for(int i = 0; i<inputWindow.window_dim_y;i++)
    {
      for(int j = 0; j<inputWindow.window_dim_x;j++)
      {
        temp = max(temp,inputWindow[i][j]);
      }
    }
    outputWindow[inputWindow.window_dim_y/2][inputWindow.window_dim_x/2]=temp;
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  image = small;
  hipEvent_t m_start, m_stop;
  hipEventCreate(&m_start);
  hipEventCreate(&m_stop);
  float m_milliseconds;
  hipEventRecord(m_start);
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  hipEventRecord(m_stop);
  hipEventSynchronize(m_stop);
  hipEventElapsedTime(&m_milliseconds, m_start, m_stop);
  std::cout<<"Time taken from Host to Device = "<<m_milliseconds<<std::endl;
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&float_image_block,3,3,1,1);
  thrust::window_vector<float> outputVector = thrust::window_vector<float>(&outBlock,3,3,1,1);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds;
  hipEventRecord(start);
  thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),outputVector.begin(),dilateFunctor());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout<<"Time taken on Shared = "<<milliseconds<<std::endl;
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipEventRecord(m_start);
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  hipEventRecord(m_stop);
  hipEventSynchronize(m_stop);
  hipEventElapsedTime(&m_milliseconds, m_start, m_stop);
  std::cout<<"Time taken from Device to Host = "<<m_milliseconds<<std::endl;
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imshow("input",image);
  imshow("output",output);
  waitKey(0);
  return 0;
}
