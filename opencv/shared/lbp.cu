#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class lbpFunctor : public thrust::shared_unary_window_transform_functor<float>
{
  public:
  __device__ void operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {
    float temp[3][3];
    for(int i = 0; i<inputWindow.window_dim_y;i++)
    {
      for(int j = 0; j<inputWindow.window_dim_x;j++)
      {
        temp[i][j] = inputWindow[make_int2(inputWindow.window_dim_x/2,inputWindow.window_dim_y/2)]>=inputWindow[make_int2(j,i)]?1:0;
      }
    }
    outputWindow[inputWindow.window_dim_y/2][inputWindow.window_dim_x/2]=128*temp[0][1]+64*temp[0][0]+32*temp[1][0]+16*temp[2][0]+8*temp[2][1]+4*temp[2][2]+2*temp[1][2]+1*temp[1][1];
  }
};
int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  resize(small,image,Size(dim,dim));
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&float_image_block,3,3,1,1);
  thrust::window_vector<float> outputVector = thrust::window_vector<float>(&outBlock,3,3,1,1);
  thrust::transform_texture(thrust::cuda::shared,myVector.begin(),myVector.end(),outputVector.begin(),lbpFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("output.png",output);

  return 0;
}
