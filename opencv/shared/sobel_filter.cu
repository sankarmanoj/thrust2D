#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#define PI 3.14159
using namespace cv;

class convolutionFunctor //:public thrust::shared_unary_window_transform_functor<float>
{
public:
  int dim;
  thrust::block_2d<float> * kernel1,*kernel2;
  convolutionFunctor( thrust::block_2d<float> * kernel1,thrust::block_2d<float> * kernel2,int dim)
  {
    this->dim =dim;
    this->kernel1 = kernel1;
    this->kernel2 = kernel2;
  }
  __device__ void operator() (const thrust::window_2d<float> & input_window,const thrust::window_2d<float> & output_window) const
  {
    float temp1 = 0,temp2=0;
    for(int i = 0; i< dim; i++)
    {
      for(int j = 0; j<dim; j++)
      {
        temp1+=input_window[i][j]*(*kernel1)[i][j];
        temp2+=input_window[i][j]*(*kernel2)[i][j];

      }
    }
    output_window[1][1]=sqrt(temp1*temp1 + temp2*temp2);
  }
};

int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 3;
  int dim1 = 512;
  if(argc ==2)
  {
    dim1 = atoi(argv[1]);
  }
  resize(small,image,Size(dim1,dim1));
  thrust::block_2d<float> kernelx(dim,dim);
  thrust::block_2d<float> kernely(dim,dim);
  //Sobel Filter
  kernelx[0][0]=-1;
  kernelx[0][1]=0;
  kernelx[0][2]=+1;
  kernelx[1][0]=-2;
  kernelx[1][1]=0;
  kernelx[1][2]=+2;
  kernelx[2][0]=-1;
  kernelx[2][1]=0;
  kernelx[2][2]=+1;
  kernely[0][0]=-1;
  kernely[0][1]=-2;
  kernely[0][2]=-1;
  kernely[1][0]=0;
  kernely[1][1]=0;
  kernely[1][2]=0;
  kernely[2][0]=+1;
  kernely[2][1]=+2;
  kernely[2][2]=+1;
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> convolve1_block (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::window_vector<float> input_wv(&float_image_block,dim,dim,1,1);
  thrust::window_vector<float> output_wv_x(&convolve1_block,dim,dim,1,1);
  thrust::transform(thrust::cuda::shared,input_wv.begin(),input_wv.end(),output_wv_x.begin(),convolutionFunctor(kernelx.device_pointer,kernely.device_pointer,dim));
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(convolve1_block.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  cudaCheckError();
  imwrite("input.png",image);
  imwrite("output.png",output);

  return 0;
}
