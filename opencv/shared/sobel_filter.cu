#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#define PI 3.14159
using namespace cv;
class transFunctor
{
public:
  __device__ float operator() (const float a,const float b) const
  {
    return sqrt(a*a + b*b);
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("building.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 3;
  image = small;
  thrust::block_2d<float> kernelx(dim,dim);
  thrust::block_2d<float> kernely(dim,dim);
  //Sobel Filter
  kernelx[0][0]=-1;
  kernelx[0][1]=0;
  kernelx[0][2]=+1;
  kernelx[1][0]=-2;
  kernelx[1][1]=0;
  kernelx[1][2]=+2;
  kernelx[2][0]=-1;
  kernelx[2][1]=0;
  kernelx[2][2]=+1;
  kernely[0][0]=-1;
  kernely[0][1]=-2;
  kernely[0][2]=-1;
  kernely[1][0]=0;
  kernely[1][1]=0;
  kernely[1][2]=0;
  kernely[2][0]=+1;
  kernely[2][1]=+2;
  kernely[2][2]=+1;

  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> convolve1_block (image.cols,image.rows);
  thrust::block_2d<float> convolve2_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  convolve1_block.assign(float_image_block.begin(),float_image_block.end());
  convolve2_block.assign(float_image_block.begin(),float_image_block.end());
  thrust::convolve(convolve1_block.begin(),convolve1_block.end(),kernelx.begin());
  thrust::convolve(convolve2_block.begin(),convolve2_block.end(),kernely.begin());
  thrust::transform(convolve1_block.begin(),convolve1_block.end(),convolve2_block.begin(),outBlock.begin(),transFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("sobel.png",output);

  return 0;
}
