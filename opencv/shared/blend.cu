#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class blendFunctor: public thrust::shared_binary_window_transform_functor<uchar>
{
  float alpha;
  hipTextureObject_t t1,t2;
public:
  blendFunctor(float alpha,hipTextureObject_t t1,hipTextureObject_t t2)
  {
    this->alpha = alpha;
    this->t1 =t1;
    this->t2 = t2;
  }
  __device__ void operator() (const thrust::window_2d<uchar> &inputWindow1,const thrust::window_2d<uchar> &inputWindow2,const thrust::window_2d<uchar> &outputWindow) const
  {
    outputWindow[0][0] = alpha*tex2D<float>(t1,outputWindow.start_x,outputWindow.start_y)+(1-alpha)*tex2D<float>(t2,outputWindow.start_x,outputWindow.start_y);
  }
};

int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat input1 = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat input2 = imread("aoutput.png",CV_LOAD_IMAGE_GRAYSCALE);
  Mat temp1;
  resize(input1,temp1,Size(512,512));
  input1 = temp1;
  Mat temp2;
  resize(input2,temp2,Size(512,512));
  input2 = temp2;
  thrust::block_2d<uchar> input_image_block_1 (input1.cols,input1.rows,0.0f);
  thrust::block_2d<uchar> input_image_block_2 (input2.cols,input2.rows,0.0f);
  thrust::block_2d<uchar> output_image_block (input1.cols,input1.rows,0.0f);
  uchar * ucharImageData = (uchar * )malloc(sizeof(uchar)*(input1.cols*input1.rows));
  unsigned char * charImageData = (unsigned char *)malloc(sizeof(unsigned char)*(input_image_block_1.end()-input_image_block_1.begin()));
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    ucharImageData[i]=(uchar)input1.ptr()[i];
  }
  input_image_block_1.assign(ucharImageData,ucharImageData+input1.cols*input1.rows);
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    ucharImageData[i]=(uchar)input2.ptr()[i];
  }
  input_image_block_2.assign(ucharImageData,ucharImageData+input2.cols*input2.rows);
  thrust::window_vector<uchar> inputWindow1 (&input_image_block_1,1,1,1,1);
  thrust::window_vector<uchar> inputWindow2 (&input_image_block_2,1,1,1,1);
  thrust::window_vector<uchar> outputWindow (&output_image_block,1,1,1,1);
  thrust::transform(thrust::cuda::shared,inputWindow1.begin(),inputWindow1.end(),inputWindow2.begin(),outputWindow.begin(),blendFunctor(0.5));
  hipMemcpy(ucharImageData,output_image_block.data().get(),sizeof(uchar)*(output_image_block.end()-output_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    charImageData[i]=(unsigned char)ucharImageData[i];
  }
  Mat output (Size(input1.cols,input1.rows),CV_8UC1,charImageData);
  imwrite("blend-input1.png",input1);
  imwrite("blend-input2.png",input2);
  imwrite("blend-output.png",output);

  return 0;
}
