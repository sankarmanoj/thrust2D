#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class blendFunctor
{
  float alpha;

public:
  blendFunctor(float alpha)
  {
    this->alpha = alpha;
  }
  __device__ void operator() (const thrust::window_2d<uchar> &inputWindow1,const thrust::window_2d<uchar> &inputWindow2,const thrust::window_2d<uchar> &outputWindow) const
  {

    outputWindow[0][0] = 0.5 * inputWindow1[make_int2(0,0)]+0.5 *  inputWindow2[make_int2(0,0)];
  }
};

int main(int argc, char const *argv[]) {
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat input1 = imread("santiago.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat input2 = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat temp1;
  resize(input1,temp1,Size(dim,dim));
  input1 = temp1;
  Mat temp2;
  resize(input2,temp2,Size(dim,dim));
  input2 = temp2;
  thrust::block_2d<uchar> input_image_block_1 (input1.cols,input1.rows);
  thrust::block_2d<uchar> input_image_block_2 (input2.cols,input2.rows);
  thrust::block_2d<uchar> output_image_block (input1.cols,input1.rows);
  uchar * ucharImageData1 = (uchar * )malloc(sizeof(uchar)*(input1.cols*input1.rows));
  uchar * ucharImageData2 = (uchar * )malloc(sizeof(uchar)*(input1.cols*input1.rows));
  unsigned char * charImageData = (unsigned char *)malloc(sizeof(unsigned char)*(input_image_block_1.end()-input_image_block_1.begin()));
    for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    ucharImageData1[i]=(uchar)input1.ptr()[i];
  }
  input_image_block_1.assign(ucharImageData1,ucharImageData1+input1.cols*input1.rows);
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    ucharImageData2[i]=(uchar)input2.ptr()[i];
  }
  input_image_block_2.assign(ucharImageData2,ucharImageData2+input2.cols*input2.rows);
  thrust::window_vector<uchar> inputWindow1 (&input_image_block_1,1,1,1,1);
  thrust::window_vector<uchar> inputWindow2 (&input_image_block_2,1,1,1,1);
  thrust::window_vector<uchar> outputWindow (&output_image_block,1,1,1,1);
  thrust::transform(thrust::cuda::shared,inputWindow1.begin(),inputWindow1.end(),inputWindow2.begin(),outputWindow.begin(),blendFunctor(0));
  hipMemcpy(charImageData,output_image_block.data().get(),sizeof(uchar)*(output_image_block.end()-output_image_block.begin()),hipMemcpyDeviceToHost);
  // for(int i = 0; i<input1.cols*input1.rows;i++)
  // {
  //   charImageData[i]=(unsigned char)ucharImageData[i];
  // }
  Mat output (Size(input1.cols,input1.rows),CV_8UC1,charImageData);
  #ifdef OWRITE
  imwrite("blend-input1.png",input1);
  imwrite("blend-input2.png",input2);
  imwrite("blend-output.png",output);
  #endif
  #ifdef SHOW
  imshow("blend-input1.png",input1);
  imshow("blend-input2.png",input2);
  imshow("blend-output.png",output);
  waitKey(0);
  #endif
  return 0;
}
