#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class blendFunctor
{
  float alpha;
  hipTextureObject_t t1,t2;
public:
  blendFunctor(float alpha,hipTextureObject_t t1,hipTextureObject_t t2)
  {
    this->alpha = alpha;
    this->t1 =t1;
    this->t2 = t2;
  }
  __device__ void operator() (const thrust::window_2d<float> &outputWindow) const
  {
    outputWindow[0][0] = alpha*tex2D<float>(t1,outputWindow.start_x,outputWindow.start_y)+(1-alpha)*tex2D<float>(t2,outputWindow.start_x,outputWindow.start_y);
  }
};

int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat temp1 = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat temp2 = imread("aoutput.png",CV_LOAD_IMAGE_GRAYSCALE);
  Mat input1,input2;
  resize(temp1,input1,Size(512,512));
  resize(temp2,input2,Size(512,512));


  thrust::block_2d<float> output_image_block (input1.cols,input1.rows,0.0f);
  float * floatImageData = (float * )malloc(sizeof(float)*(input1.cols*input1.rows));
  unsigned char * charImageData = (unsigned char *)malloc(sizeof(unsigned char)*(input1.cols*input1.rows));
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    floatImageData[i]=(float)input1.ptr()[i];
  }
  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  size_t pitch;
  float * aligned_device_memory1;
  hipMallocPitch(&aligned_device_memory1,&pitch,input1.cols*sizeof(float),input1.rows);
  hipMemcpy2D(aligned_device_memory1,pitch,floatImageData,input1.cols*sizeof(float),input1.cols*sizeof(float),input1.rows,hipMemcpyHostToDevice);

  //Create Resource Descriptor
  hipResourceDesc resDesc1;
  memset(&resDesc1, 0, sizeof(resDesc1));
  resDesc1.resType = hipResourceTypePitch2D;
  resDesc1.res.pitch2D.desc = hipCreateChannelDesc<float>();
  resDesc1.res.pitch2D.pitchInBytes=pitch;
  resDesc1.res.pitch2D.height = input1.rows;
  resDesc1.res.pitch2D.width=input1.cols;
  resDesc1.res.pitch2D.devPtr = aligned_device_memory1;

  //floatexture Object Creation
  hipTextureObject_t texref1;
  hipCreateTextureObject(&texref1, &resDesc1, &texDesc, NULL);
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    floatImageData[i]=(float)input2.ptr()[i];
  }
  hipMallocPitch(&aligned_device_memory1,&pitch,input1.cols*sizeof(float),input1.rows);
  hipMemcpy2D(aligned_device_memory1,pitch,floatImageData,input1.cols*sizeof(float),input1.cols*sizeof(float),input1.rows,hipMemcpyHostToDevice);

  //Create Resource Descriptor
  hipResourceDesc resDesc2;
  memset(&resDesc2, 0, sizeof(resDesc2));
  resDesc2.resType = hipResourceTypePitch2D;
  resDesc2.res.pitch2D.desc = hipCreateChannelDesc<float>();
  resDesc2.res.pitch2D.pitchInBytes=pitch;
  resDesc2.res.pitch2D.height = input1.rows;
  resDesc2.res.pitch2D.width=input1.cols;
  resDesc2.res.pitch2D.devPtr = aligned_device_memory1;

  //floatexture Object Creation
  hipTextureObject_t texref2;
  hipCreateTextureObject(&texref2, &resDesc2, &texDesc, NULL);

  thrust::window_vector<float> outputWindow (&output_image_block,1,1,1,1);
  thrust::for_each(outputWindow.begin(),outputWindow.end(),blendFunctor(0.5,texref1,texref2));
  hipMemcpy(floatImageData,output_image_block.data().get(),sizeof(float)*(output_image_block.end()-output_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    charImageData[i]=(unsigned char)floatImageData[i];
  }
  Mat output (Size(input1.cols,input1.rows),CV_8UC1,charImageData);
  imwrite("blend-input1.png",input1);
  imwrite("blend-input2.png",input2);
  imwrite("blend-output.png",output);

  return 0;
}
