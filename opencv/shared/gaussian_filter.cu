#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim1 = 512;
  if(argc ==2)
  {
    dim1 = atoi(argv[1]);
  }
  resize(small,image,Size(dim1,dim1));
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  // thrust::window_vector<float> input_wv(&float_image_block,dim,dim,1,1);
  // thrust::window_vector<float> output_wv(&output_image_block,dim,dim,1,1);
  // thrust::transform(thrust::cuda::texture,input_wv.begin(),input_wv.end(),output_wv.begin(),convolutionFunctor(kernel.device_pointer,dim));
  float kernel[3] = {1,1,1};
  thrust::convolve(thrust::cuda::texture,&uchar_image_block,kernel,3);

  unsigned char * outputucharImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  uchar_image_block.download(&img);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputucharImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputucharImageData);
  // cudaCheckError();
  imwrite("input.png",image);
  imwrite("output.png",output);

  return 0;
}
