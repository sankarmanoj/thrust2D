#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }
  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
}
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 5;
  image = small;
  hipEvent_t m_start, m_stop;
  hipEventCreate(&m_start);
  hipEventCreate(&m_stop);
  float m_milliseconds;
  hipEventRecord(m_start);
  thrust::block_2d<float> kernel(dim,dim);
  getGaussianKernelBlock(dim,5,kernel);
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  hipEventRecord(m_stop);
  hipEventSynchronize(m_stop);
  hipEventElapsedTime(&m_milliseconds, m_start, m_stop);
  std::cout<<"Time taken from Host to Device = "<<m_milliseconds<<std::endl;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds;
  hipEventRecord(start);
  thrust::convolve(float_image_block.begin(),float_image_block.end(),kernel.begin());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout<<"Time taken on Shared = "<<milliseconds<<std::endl;
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipEventRecord(m_start);
  hipMemcpy(img,thrust::raw_pointer_cast(float_image_block.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  hipEventRecord(m_stop);
  hipEventSynchronize(m_stop);
  hipEventElapsedTime(&m_milliseconds, m_start, m_stop);
  std::cout<<"Time taken from Device to Host = "<<m_milliseconds<<std::endl;
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  cudaCheckError();
  imshow("input",image);
  imshow("output",output);
  waitKey(0);
  return 0;
}
