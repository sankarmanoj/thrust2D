#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim1 = 512;
  if(argc ==2)
  {
    dim1 = atoi(argv[1]);
  }
  resize(small,image,Size(dim1,dim1));
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(float_image_block.end()-float_image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  // thrust::window_vector<float> input_wv(&float_image_block,dim,dim,1,1);
  // thrust::window_vector<float> output_wv(&output_image_block,dim,dim,1,1);
  // thrust::transform(thrust::cuda::texture,input_wv.begin(),input_wv.end(),output_wv.begin(),convolutionFunctor(kernel.device_pointer,dim));
  float kernel[3] = {0.25,0.5,0.25};
  thrust::convolve(thrust::cuda::texture,&float_image_block,kernel);

  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(float_image_block.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  // cudaCheckError();
  imwrite("input.png",image);
  imwrite("output.png",output);

  return 0;
}
