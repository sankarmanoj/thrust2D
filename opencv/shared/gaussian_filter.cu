#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;

inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,float *GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i*dim + j]=gauss(i,j,mid,sigma);
    }
  }
  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i*dim + j]/=total;
      newTotal +=  (GaussianKernel)[i*dim + j];
    }
  }
}

int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim1 = 512;
  if(argc ==2)
  {
    dim1 = atoi(argv[1]);
  }
  resize(small,image,Size(dim1,dim1));
  // printf("%d\n",dim1);
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> output_image_block (image.cols,image.rows);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  float kernel[9];
  getGaussianKernelBlock(3,5,kernel);
  thrust::convolve(thrust::cuda::texture,&uchar_image_block,kernel,3,&output_image_block);

  unsigned char * outputucharImageData = (unsigned char *)malloc(sizeof(unsigned char)*(output_image_block.end()-output_image_block.begin()));
  output_image_block.download(&img);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputucharImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputucharImageData);
  // cudaCheckError();
  #ifdef OWRITE
  imwrite("input.png",image);
  imwrite("output.png",output);
  #endif
  #ifdef SHOW
  imshow("input.png",image);
  imshow("output.png",output);
  waitKey(0);
  #endif

  return 0;
}
