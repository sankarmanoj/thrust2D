#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }
  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
}

class convolutionFunctor
{
public:
  thrust::block_2d<float> *kernel;
  int dim;
  convolutionFunctor(thrust::block_2d<float> *kernel,int dim)
  {
    this->kernel = kernel;
    this->dim = dim;
  }
  __device__ float operator() (thrust::window_2d<float> input_window,thrust::window_2d<float> output_window)
  {
    float temp = 0;
    for(int i = 0; i< dim; i++)
    {
      for(int j = 0; j<dim; j++)
      {
        temp+=input_window[i][j]*0.1;
      }
    }
    output_window[1][1]=temp;
    return 0.0;
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 3;
  resize(small,image,Size(512,512));
  thrust::block_2d<float> kernel(dim,dim);
  getGaussianKernelBlock(dim,5,kernel);
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> output_image_block(image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::window_vector<float> input_wv(&float_image_block,dim,dim,1,1);
  thrust::window_vector<float> output_wv(&output_image_block,dim,dim,1,1);
  thrust::transform(thrust::cuda::shared,input_wv.begin(),input_wv.end(),output_wv.begin(),convolutionFunctor(kernel.device_pointer,dim));
  // thrust::convolve(float_image_block.begin(),float_image_block.end(),kernel.begin());

  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(output_image_block.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  // cudaCheckError();
  imwrite("input.png",image);
  imwrite("output.png",output);

  return 0;
}
