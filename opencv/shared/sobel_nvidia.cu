#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#define PI 3.14159
using namespace cv;
#define fScale 1
class transFunctor
{
public:
  __device__ void operator() (const thrust::window_2d<uchar> &window, const thrust::window_2d<uchar> &outputWindow) const
  {
    short Horz = window[0][2] + 2*window[1][1] + window[2][2] - window[0][0] - 2*window[1][0] - window[2][0];
    short Vert = window[0][0] + 2*window[0][1] + window[0][2] - window[2][0] - 2*window[2][1] - window[2][2];
    short Sum = (short)(fScale*(abs((int)Horz)+abs((int)Vert)));

    if (Sum < 0)
    {
        outputWindow[1][1]=0;
    }
    else if (Sum > 0xff)
    {
        outputWindow[1][1]=0xff;
    }
    else
      outputWindow[1][1]= Sum;
  }
};
int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  resize(small,image,Size(dim,dim));

  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  thrust::window_vector<uchar> input_vector (&uchar_image_block,3,3,1,1);
  thrust::window_vector<uchar> output_vector (&outBlock,3,3,1,1);
  thrust::transform(thrust::cuda::shared,input_vector.begin(),input_vector.end(),output_vector.begin(),transFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  outBlock.download(&img);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  #ifdef OWRITE
  imwrite("input.png",image);
  imwrite("sobel.png",output);
  #endif
  #ifdef SHOW
  imshow("input.png",image);
  imshow("sobel.png",output);
  waitKey(0);
  #endif

  return 0;
}
