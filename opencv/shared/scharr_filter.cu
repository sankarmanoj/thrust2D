#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#define PI 3.14159
using namespace cv;
class transFunctor
{
public:
  __device__ uchar operator() (const uchar a,const uchar b) const
  {
    return sqrt((float)a*a + b*b);
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  image = small;
  float kernelx[3], kernely[3];
  //Scharr Filter
  kernelx[0]=-3;
  kernelx[1]=0;
  kernelx[2]=+3;
  kernely[0]=+3;
  kernely[1]=+10;
  kernely[2]=+3;
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> convolve1_block (image.cols,image.rows);
  thrust::block_2d<uchar> convolve2_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.assign(img,img+image.cols*image.rows);
  convolve1_block.assign(uchar_image_block.begin(),uchar_image_block.end());
  convolve2_block.assign(uchar_image_block.begin(),uchar_image_block.end());
  thrust::convolve(thrust::cuda::texture,&convolve1_block,kernelx);
  thrust::convolve(thrust::cuda::texture,&convolve2_block,kernely);
  thrust::transform(convolve1_block.begin(),convolve1_block.end(),convolve2_block.begin(),outBlock.begin(),transFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("scharr.png",output);
  return 0;
}
