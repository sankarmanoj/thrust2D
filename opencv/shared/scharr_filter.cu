#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#define PI 3.14159
using namespace cv;
class transFunctor
{
public:
  __device__ float operator() (const float a,const float b) const
  {
    return sqrt(a*a + b*b);
  }
};

class convolutionFunctor //:public thrust::shared_unary_window_transform_functor<float>
{
public:
  int dim;
  thrust::block_2d<float> * kernel;
  convolutionFunctor( thrust::block_2d<float> * kernel,int dim)
  {
    this->dim =dim;
    this->kernel = kernel;
  }
  __device__ void operator() (const thrust::window_2d<float> & input_window,const thrust::window_2d<float> & output_window) const
  {
    float temp = 0;
    for(int i = 0; i< dim; i++)
    {
      for(int j = 0; j<dim; j++)
      {
        temp+=input_window[i][j]*(*kernel)[i][j];
      }
    }
    output_window[1][1]=temp;
  }
};

int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 3;
  image = small;
  thrust::block_2d<float> kernelx(dim,dim);
  thrust::block_2d<float> kernely(dim,dim);
  //Scharr Filter
  kernelx[0][0]=-3;
  kernelx[0][1]=0;
  kernelx[0][2]=+3;
  kernelx[1][0]=-10;
  kernelx[1][1]=0;
  kernelx[1][2]=+10;
  kernelx[2][0]=-3;
  kernelx[2][1]=0;
  kernelx[2][2]=+3;
  kernely[0][0]=-3;
  kernely[0][1]=-10;
  kernely[0][2]=-3;
  kernely[1][0]=0;
  kernely[1][1]=0;
  kernely[1][2]=0;
  kernely[2][0]=+3;
  kernely[2][1]=+10;
  kernely[2][2]=+3;
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> convolve1_block (image.cols,image.rows);
  thrust::block_2d<float> convolve2_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  convolve1_block.assign(float_image_block.begin(),float_image_block.end());
  convolve2_block.assign(float_image_block.begin(),float_image_block.end());

  thrust::window_vector<float> input_wv(&float_image_block,dim,dim,1,1);
  thrust::window_vector<float> output_wv_x(&convolve1_block,dim,dim,1,1);
  thrust::window_vector<float> output_wv_y(&convolve2_block,dim,dim,1,1);

  thrust::transform(thrust::cuda::shared,input_wv.begin(),input_wv.end(),output_wv_x.begin(),convolutionFunctor(kernelx.device_pointer,dim));
  thrust::transform(thrust::cuda::shared,input_wv.begin(),input_wv.end(),output_wv_y.begin(),convolutionFunctor(kernely.device_pointer,dim));
  thrust::transform(convolve1_block.begin(),convolve1_block.end(),convolve2_block.begin(),outBlock.begin(),transFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  cudaCheckError();
  imwrite("input.png",image);
  imwrite("output.png",output);

  return 0;
}
