#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#define PI 3.14159
using namespace cv;
class transFunctor
{
public:
  __device__ uchar operator() (const uchar a,const uchar b) const
  {
    return sqrt((float)a*a + b*b);
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  resize(small,image,Size(dim,dim));
  float kernelx[3], kernely[3];
  //Scharr Filter
  kernelx[0]=-3;
  kernelx[1]=0;
  kernelx[2]=+3;
  kernely[0]=+3;
  kernely[1]=+10;
  kernely[2]=+3;
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> convolve1_block (image.cols,image.rows);
  thrust::block_2d<uchar> convolve2_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  convolve1_block.upload(img);
  convolve2_block.upload(img);
  thrust::convolve(thrust::cuda::texture,&convolve1_block,kernelx,3);
  thrust::convolve(thrust::cuda::texture,&convolve2_block,kernely,3);
  thrust::transform(convolve1_block.begin(),convolve1_block.end(),convolve2_block.begin(),outBlock.begin(),transFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  outBlock.download(&img);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  #ifdef OWRITE
  imwrite("input.png",image);
  imwrite("scharr.png",output);
  #endif
  #ifdef SHOW
  imshow("input.png",image);
  imshow("scharr.png",output);
  waitKey(0);
  #endif
  return 0;
}
