#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <thrust/window_for_each.h>
using namespace cv;
class pyrupTransformFunctor : public thrust::shared_window_for_each_functor<uchar>
{
public:
  thrust::block_2d<uchar> *inBlock;

pyrupTransformFunctor(thrust::block_2d<uchar> * inBlock)
  {
    this->inBlock = inBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<uchar> &outputWindow) const
  {
    int x_in, y_in;
    if(outputWindow.start_x%2 && outputWindow.start_y%2)
    {
      x_in = outputWindow.start_x/2;
      y_in = outputWindow.start_y/2;
      outputWindow[0][0]=(*inBlock)[y_in][x_in];
    }
  }
};

int main(int argc, char const *argv[])
{
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  resize(small,image,Size(dim,dim));
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> intermediate_image_block (image.cols*2,image.rows*2);
  thrust::block_2d<uchar> outBlock (image.cols*2,image.rows*2,0.0f);
  thrust::window_vector<uchar> output_wv(&outBlock,1,1,1,1);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  uchar * img_out = (uchar * )malloc(sizeof(uchar)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  thrust::window_vector<uchar> inputVector(&intermediate_image_block,1,1,1,1);
  pyrupTransformFunctor ptf(&uchar_image_block);
  thrust::for_each(inputVector.begin(),inputVector.end(),ptf);
  hipDeviceSynchronize();
  float kernel[5] = {0.0625*2,0.25*2,0.375*2,0.25*2,0.0625*2};
  thrust::convolve(thrust::cuda::shared,&intermediate_image_block,kernel,kernel,5,&outBlock);
  outBlock.download(&img_out);
  Mat output (Size(image.cols*2,image.rows*2),CV_8UC1,img_out);
  #ifdef OWRITE
  imwrite("input.png",image);
  imwrite("pyrup.png",output);

  #endif
  #ifdef SHOW
  imshow("input.png",image);
  imshow("pyrup.png",output);
  waitKey(0);
  #endif
  return 0;
}
