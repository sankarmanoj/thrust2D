#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <thrust/window_for_each.h>
using namespace cv;
class pyrupTransformFunctor : public thrust::shared_window_for_each_functor<uchar>
{
public:
  thrust::block_2d<uchar> *inBlock;

pyrupTransformFunctor(thrust::block_2d<uchar> * inBlock)
  {
    this->inBlock = inBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<uchar> &outputWindow) const
  {
    int x_in, y_in;
    if(outputWindow.start_x%2 && outputWindow.start_y%2)
    {
      x_in = outputWindow.start_x/2;
      y_in = outputWindow.start_y/2;
      outputWindow[0][0]=(*inBlock)[y_in][x_in];
    }
  }
};

int main(int argc, char const *argv[])
{
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  resize(small,image,Size(dim,dim));
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols*2,image.rows*2,0.0f);
  thrust::window_vector<uchar> output_wv(&outBlock,1,1,1,1);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(image_block.end()-image_block.begin()));
  uchar * img_out = (uchar * )malloc(sizeof(uchar)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  thrust::window_vector<uchar> inputVector(&outBlock,1,1,1,1);
  pyrupTransformFunctor ptf(&uchar_image_block);
  thrust::for_each(thrust::cuda::shared,inputVector.begin(),inputVector.end(),ptf);
  hipDeviceSynchronize();
  float kernel[3] = {0.25,0.5,0.25};
  thrust::convolve(thrust::cuda::shared,&outBlock,kernel,3,&outBlock);
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(outBlock.end()-outBlock.begin()));
  outBlock.download(&img);
  for(int i = 0; i<image.cols*image.rows*4;i++)
  {
    outputFloatImageData[i]=(unsigned char)img_out[i];
  }
  Mat output (Size(image.cols*2,image.rows*2),CV_8UC1,outputFloatImageData);
  #ifdef OWRITE
  imwrite("input.png",image);
  imwrite("pyrup.png",output);

  #endif
  #ifdef SHOW
  imshow("input.png",image);
  imshow("pyrup.png",output);
  waitKey(0);
  #endif
  return 0;
}
