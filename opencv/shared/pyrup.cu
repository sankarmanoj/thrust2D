#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;

inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }
  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
}

class pyrupTransformFunctor
{
public:
  thrust::block_2d<float> *inBlock;

pyrupTransformFunctor(thrust::block_2d<float> * inBlock)
  {
    this->inBlock = inBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<float> &outputWindow) const
  {
    int x_in, y_in;
    if(outputWindow.start_x%2 && outputWindow.start_y%2)
    {
      x_in = outputWindow.start_x*2;
      y_in = outputWindow.start_y*2;
      outputWindow[0][0]=(*inBlock)[y_in][x_in];
    }
  }
};

class convolutionFunctor //:public thrust::shared_unary_window_transform_functor<float>
{
public:
  int dim;
  thrust::block_2d<float> * kernel;
  convolutionFunctor( thrust::block_2d<float> * kernel,int dim)
  {
    this->dim =dim;
    this->kernel = kernel;
  }
  __device__ void operator() (const thrust::window_2d<float> & input_window,const thrust::window_2d<float> & output_window) const
  {
    float temp = 0;
    for(int i = 0; i< dim; i++)
    {
      for(int j = 0; j<dim; j++)
      {
        temp+=input_window[i][j]*(*kernel)[i][j];
      }
    }
    output_window[1][1]=temp;
  }
};

int main(int argc, char const *argv[])
{
  int dim = 3;
  thrust::block_2d<float> kernel(dim,dim);
  getGaussianKernelBlock(dim,1.0,kernel);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim1 = 512;
  if(argc ==2)
  {
    dim1 = atoi(argv[1]);
  }
  resize(small,image,Size(dim1,dim1));
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols/2,image.rows/2,0.0f);
  thrust::block_2d<float> output_image_block(image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  float * img1 = (float * )malloc(sizeof(float)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::window_vector<float> input_wv(&float_image_block,dim,dim,1,1);
  thrust::window_vector<float> output_wv(&output_image_block,dim,dim,1,1);
  thrust::transform(thrust::cuda::shared,input_wv.begin(),input_wv.end(),output_wv.begin(),convolutionFunctor(kernel.device_pointer,dim));
  thrust::window_vector<float> inputVector(&outBlock,1,1,1,1);
  pyrupTransformFunctor ptf(&output_image_block);
  thrust::for_each(thrust::cuda::shared,inputVector.begin(),inputVector.end(),ptf);
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(outBlock.end()-outBlock.begin()));
  hipMemcpy(img1,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(outBlock.end()-outBlock.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<(outBlock.end()-outBlock.begin());i++)
  {
    outputFloatImageData[i]=(unsigned char)img1[i];
  }
  Mat output (Size(image.cols/2,image.rows/2),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("pyrup.png",output);
  free (img);
  free (img1);
  free (outputFloatImageData);
  return 0;
}
