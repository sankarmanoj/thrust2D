#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }

  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
}

class pyrupTransformFunctor : public thrust::shared_window_for_each_functor<float>
{
public:
  thrust::block_2d<float> *inBlock;

pyrupTransformFunctor(thrust::block_2d<float> * inBlock)
  {
    this->inBlock = inBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<float> &outputWindow) const
  {
    int x_in, y_in;
    if(outputWindow.start_x%2 && outputWindow.start_y%2)
    {
      x_in = outputWindow.start_x*2;
      y_in = outputWindow.start_y*2;
      outputWindow[0][0]=(*inBlock)[y_in][x_in];
    }
  }
};

int main()
{
  int dim = 5;
  thrust::block_2d<float> kernel(dim,dim);
  getGaussianKernelBlock(dim,1.0,kernel);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image=small;
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols/2,image.rows/2,0.0f);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  float * img1 = (float * )malloc(sizeof(float)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::convolve(float_image_block.begin(),float_image_block.end(),kernel.begin());
  thrust::window_vector<float> inputVector(&outBlock,1,1,1,1);
  pyrupTransformFunctor ptf(&float_image_block);
  thrust::for_each(inputVector.begin(),inputVector.end(),ptf);
  hipDeviceSynchronize();
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(outBlock.end()-outBlock.begin()));
  hipMemcpy(img1,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(outBlock.end()-outBlock.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<(outBlock.end()-outBlock.begin());i++)
  {
    outputFloatImageData[i]=(unsigned char)img1[i];
  }
  Mat output (Size(image.cols/2,image.rows/2),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("pyrup.png",output);
  free (img);
  free (img1);
  free (outputFloatImageData);
  return 0;
}
