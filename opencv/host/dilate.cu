#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class dilateFunctor //: public thrust::shared_unary_window_transform_functor<float>
{
  public:
  __host__ float operator() (const thrust::host_window_2d<float> &inputWindow,const thrust::host_window_2d<float> &outputWindow) const
  {
    float temp = -1.0;
    for(int i = 0; i<inputWindow.window_dim_y;i++)
    {
      for(int j = 0; j<inputWindow.window_dim_x;j++)
      {
        temp = max(temp,inputWindow[i][j]);
      }
    }
    outputWindow[inputWindow.window_dim_y/2][inputWindow.window_dim_x/2]=temp;
    return 0.0f;
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  image = small;

  std::cout<<"  "<<image.isContinuous()<<std::endl;
  thrust::host_block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::host_block_2d<float> float_image_block (image.cols,image.rows);
  thrust::host_block_2d<float> outBlock (image.cols,image.rows);
  thrust::host_block_2d<float> nullBlock (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::host_window_vector<float> myVector(&float_image_block,3,3,1,1);
  thrust::host_window_vector<float> outputVector(&outBlock,3,3,1,1);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds;
  hipEventRecord(start);
  thrust::transform(thrust::host,myVector.begin(),myVector.end(),outputVector.begin(),nullBlock.begin(),dilateFunctor());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout<<"Time taken on Host = "<<milliseconds<<std::endl;
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyHostToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imshow("input",image);
  imshow("output",output);
  waitKey(0);
  return 0;
}
