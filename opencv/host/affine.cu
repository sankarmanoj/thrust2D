#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class AffineTransformFunctor
{
public:
  thrust::host_block_2d<float> *transformMatrix;
  thrust::host_block_2d<float> *outBlock;

  AffineTransformFunctor(thrust::host_block_2d<float> * tm,thrust::host_block_2d<float> * outBlock)
  {
    this->transformMatrix = tm->device_pointer;
    this->outBlock = outBlock->device_pointer;
  }
  __host__ void operator() (const thrust::host_window_2d<float> &inputWindow) const
  {
    int x_out, y_out;
    x_out = (int)((*transformMatrix)[0][0]*inputWindow.start_x+(*transformMatrix)[0][1]*inputWindow.start_y+(*transformMatrix)[0][2]*1);
    y_out = (int)((*transformMatrix)[1][0]*inputWindow.start_x+(*transformMatrix)[1][1]*inputWindow.start_y+(*transformMatrix)[1][2]*1);

    (*outBlock)[y_out][x_out]=inputWindow[0][0];
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  image = small;
  thrust::host_block_2d<float> float_image_block (image.cols,image.rows,0.0f);
  thrust::host_block_2d<float> outBlock (image.cols,image.rows,0.0f);
  float * img = (float * )malloc(sizeof(float)*(image.cols*image.rows));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  Point2f srcTri[3];
  Point2f dstTri[3];
  Mat warp_mat( 2, 3, CV_32FC1 );
  /// Set your 3 points to calculate the  Affine Transform
  srcTri[0] = Point2f( 0,0 );
  srcTri[1] = Point2f( image.cols - 1, 0 );
  srcTri[2] = Point2f( 0, image.rows - 1 );
  dstTri[0] = Point2f( image.cols*0.0, image.rows*0.5 );
  dstTri[1] = Point2f( image.cols*0.8, image.rows*0.2 );
  dstTri[2] = Point2f( image.cols*0.2, image.rows*0.7 );
  /// Get the Affine Transform
  warp_mat = getAffineTransform( srcTri, dstTri );
  warp_mat.convertTo(warp_mat,CV_32FC1);
  //Move Warp Matrix to Device
  thrust::host_block_2d<float> warp_block(warp_mat.cols,warp_mat.rows);
  for(int i = 0; i< warp_mat.rows;i++)
  {
    for(int j = 0; j<warp_mat.cols;j++)
    {
      warp_block[i][j]=warp_mat.at<float>(i,j);
    }
  }
  //Create Windows For Indexing
  thrust::host_window_vector<float> inputVector(&float_image_block,1,1,1,1);
  AffineTransformFunctor atf(&warp_block,&outBlock);
  thrust::for_each(thrust::host,inputVector.begin(),inputVector.end(),atf);
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyHostToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("aoutput.png",output);
  return 0;
}
