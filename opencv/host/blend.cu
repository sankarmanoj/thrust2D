#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class blendFunctor
{
  float alpha;
public:
  blendFunctor(float alpha)
  {
    this->alpha = alpha;
  }
  __host__ float operator() (const thrust::host_window_2d<float> &inputWindow1,const thrust::host_window_2d<float> &inputWindow2) const
  {
    return alpha*inputWindow1[0][0]+(1-alpha)*inputWindow2[0][0];
  }
};
int main(int argc, char const *argv[]) {
  Mat input1 = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat input2 = imread("aoutput.png",CV_LOAD_IMAGE_GRAYSCALE);
  Mat temp1;
  resize(input1,temp1,Size(500,500));
  input1 = temp1;
  Mat temp2;
  resize(input2,temp2,Size(500,500));
  input2 = temp2;
  thrust::host_block_2d<float> input_image_block_1 (input1.cols,input1.rows,0.0f);
  thrust::host_block_2d<float> input_image_block_2 (input2.cols,input2.rows,0.0f);
  thrust::host_block_2d<float> output_image_block (input1.cols,input1.rows,0.0f);
  float * floatImageData = (float * )malloc(sizeof(float)*(input1.cols*input1.rows));
  unsigned char * charImageData = (unsigned char *)malloc(sizeof(unsigned char)*(input_image_block_1.end()-input_image_block_1.begin()));
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    floatImageData[i]=(float)input1.ptr()[i];
  }
  input_image_block_1.assign(floatImageData,floatImageData+input1.cols*input1.rows);
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    floatImageData[i]=(float)input2.ptr()[i];
  }
  input_image_block_2.assign(floatImageData,floatImageData+input2.cols*input2.rows);
  thrust::host_window_vector<float> inputWindow1 (&input_image_block_1,1,1,1,1);
  thrust::host_window_vector<float> inputWindow2 (&input_image_block_2,1,1,1,1);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds;
  hipEventRecord(start);
  thrust::transform(thrust::host,inputWindow1.begin(),inputWindow1.end(),inputWindow2.begin(),output_image_block.begin(),blendFunctor(0.5));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout<<"Time taken on Host = "<<milliseconds<<std::endl;
  hipMemcpy(floatImageData,output_image_block.data(),sizeof(float)*(output_image_block.end()-output_image_block.begin()),hipMemcpyHostToHost);
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    charImageData[i]=(unsigned char)floatImageData[i];
  }
  Mat output (Size(input1.cols,input1.rows),CV_8UC1,charImageData);
  imshow("blend-input1.png",input1);
  imshow("blend-input2.png",input2);
  imshow("blend-output.png",output);
  waitKey(0);
  return 0;
}
