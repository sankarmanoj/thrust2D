#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_transform.h>
using namespace cv;
#define KERNEL_LENGTH 5
__constant__ float c_kernel[KERNEL_LENGTH*KERNEL_LENGTH];
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,float *GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i*dim + j]=gauss(i,j,mid,sigma);
    }
  }
  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i*dim + j]/=total;
      newTotal +=  (GaussianKernel)[i*dim + j];
    }
  }
}

class convolutionFunctor //:public thrust::shared_unary_window_transform_functor<uchar>
{
public:
  int dim;
  convolutionFunctor(int dim)
  {
    this->dim =dim;
  }
  __device__ void operator() (const thrust::window_2d<uchar> & input_window,const thrust::window_2d<uchar> & output_window) const
  {
    uchar temp = 0;
    for(int i = 0; i< dim; i++)
    {
      for(int j = 0; j<dim; j++)
      {
        temp+=input_window[make_int2(j,i)]*(c_kernel)[i*dim + j];
      }
    }
    output_window[1][1]=temp;
  }
};
int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;

  int dim = 5;
  int dim_image = 512;
  if(argc ==2)
  {
    dim_image = atoi(argv[1]);
  }
  resize(small,image,Size(dim_image,dim_image));
  float *hkernel = (float *) std::malloc(sizeof(float) * dim*dim);
  getGaussianKernelBlock(dim,5,hkernel);
  hipMemcpyToSymbol(HIP_SYMBOL(c_kernel), hkernel, dim*dim * sizeof(float));
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> output_image_block(image.cols,image.rows);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  thrust::window_vector<uchar> input_wv(&uchar_image_block,dim,dim,1,1);
  thrust::window_vector<uchar> output_wv(&output_image_block,dim,dim,1,1);
  thrust::transform(thrust::cuda::shared,input_wv.begin(),input_wv.end(),output_wv.begin(),convolutionFunctor(dim));
  unsigned char * toutputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  output_image_block.download(&img);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    toutputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat toutput (Size(image.cols,image.rows),CV_8UC1,toutputFloatImageData);
  // imwrite("input.png",image);
  // imwrite("output.png",toutput);

  return 0;
}
