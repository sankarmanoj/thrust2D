#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }
  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
}

class convolutionFunctor //:public thrust::shared_unary_window_transform_functor<float>
{
public:
  int dim;
  thrust::block_2d<float> * kernel;
  convolutionFunctor( thrust::block_2d<float> * kernel,int dim)
  {
    this->dim =dim;
    this->kernel = kernel;
  }
  __device__ float operator() (const thrust::window_2d<float> & input_window,const thrust::window_2d<float> & output_window) const
  {
    float temp = 0;
    for(int i = 0; i< dim; i++)
    {
      for(int j = 0; j<dim; j++)
      {
        temp+=input_window[i][j]*(*kernel)[i][j];
      }
    }
    output_window[1][1]=temp;
    return 0.0 ;
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 19;
  image = small;
  thrust::block_2d<float> kernel(dim,dim);
  getGaussianKernelBlock(dim,5,kernel);
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> zero_image_block (image.cols,image.rows);
  thrust::block_2d<float> output_image_block(image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::window_vector<float> input_wv(&float_image_block,dim,dim,1,1);
  thrust::window_vector<float> output_wv(&output_image_block,dim,dim,1,1);
  thrust::counting_iterator<float> zeros(0);
  thrust::transform(input_wv.begin(),input_wv.end(),output_wv.begin(),zero_image_block.begin(),convolutionFunctor(kernel.device_pointer,dim));

  unsigned char * toutputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(output_image_block.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    toutputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat toutput (Size(image.cols,image.rows),CV_8UC1,toutputFloatImageData);

  // cudaCheckError();
  imwrite("input.png",image);
  imwrite("output.png",toutput);

  return 0;
}
