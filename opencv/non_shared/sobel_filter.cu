#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#define PI 3.14159
using namespace cv;
class transFunctor
{
public:
  __device__ uchar operator() (const uchar a,const uchar b) const
  {
    return (uchar) sqrt((float) a*a + b*b);
  }
};

class convolutionFunctor //:public thrust::shared_unary_window_transform_functor<uchar>
{
public:
  int dim;
  thrust::block_2d<float> * kernel;
  convolutionFunctor( thrust::block_2d<float> * kernel,int dim)
  {
    this->dim =dim;
    this->kernel = kernel;
  }
  __device__ uchar operator() (const thrust::window_2d<uchar> & input_window,const thrust::window_2d<uchar> & output_window) const
  {
    uchar temp = 0;
    for(int i = 0; i< dim; i++)
    {
      for(int j = 0; j<dim; j++)
      {
        temp+=input_window[i][j]*(*kernel)[i][j];
      }
    }
    output_window[1][1]=temp;
    return 0.0 ;
  }
};

int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  resize(small,image,Size(dim,dim));
  thrust::host_block_2d<float> kernelx(3,3);
  thrust::host_block_2d<float> kernely(3,3);
  thrust::block_2d<float> dkernelx(3,3);
  thrust::block_2d<float> dkernely(3,3);
  //Sobel Filter
  kernelx[0][0]=-1;
  kernelx[0][1]=0;
  kernelx[0][2]=+1;
  kernelx[1][0]=-2;
  kernelx[1][1]=0;
  kernelx[1][2]=+2;
  kernelx[2][0]=-1;
  kernelx[2][1]=0;
  kernelx[2][2]=+1;
  kernely[0][0]=-1;
  kernely[0][1]=-2;
  kernely[0][2]=-1;
  kernely[1][0]=0;
  kernely[1][1]=0;
  kernely[1][2]=0;
  kernely[2][0]=+1;
  kernely[2][1]=+2;
  kernely[2][2]=+1;
  dkernelx=kernelx;
  dkernely=kernely;
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> convolve1_block (image.cols,image.rows);
  thrust::block_2d<uchar> convolve2_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows);
  thrust::block_2d<uchar> zero_image_block (image.cols,image.rows);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  convolve1_block.upload(img);
  convolve2_block.upload(img);
  thrust::window_vector<uchar> input_wv(&uchar_image_block,3,3,1,1);
  thrust::window_vector<uchar> output_wv_x(&convolve1_block,3,3,1,1);
  thrust::window_vector<uchar> output_wv_y(&convolve2_block,3,3,1,1);

  thrust::transform(input_wv.begin(),input_wv.end(),output_wv_x.begin(),zero_image_block.begin(),convolutionFunctor(dkernelx.device_pointer,3));
  thrust::transform(input_wv.begin(),input_wv.end(),output_wv_y.begin(),zero_image_block.begin(),convolutionFunctor(dkernely.device_pointer,3));
  thrust::transform(convolve1_block.begin(),convolve1_block.end(),convolve2_block.begin(),outBlock.begin(),transFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  outBlock.download(&img);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  // cudaCheckError();
  // imwrite("input.png",image);
  // imwrite("output.png",output);
  return 0;
}
