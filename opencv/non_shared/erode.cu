#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
using namespace cv;
class erodeFunctor //: public thrust::shared_unary_window_transform_functor<uchar>
{
  public:
  __device__ uchar operator() (const thrust::window_2d<uchar> &inputWindow,const thrust::window_2d<uchar> &outputWindow) const
  {
    uchar temp = 255;
    for(int i = 0; i<inputWindow.window_dim_y;i++)
    {
      for(int j = 0; j<inputWindow.window_dim_x;j++)
      {
        temp = min((float)temp,(float)inputWindow[i][j]);
      }
    }
    outputWindow[inputWindow.window_dim_y/2][inputWindow.window_dim_x/2]=temp;
    return 0;
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  image = small;
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows);
  thrust::block_2d<uchar> null_block (image.cols,image.rows);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.assign(img,img+image.cols*image.rows);
  thrust::window_vector<uchar> myVector = thrust::window_vector<uchar>(&uchar_image_block,3,3,1,1);
  thrust::window_vector<uchar> outputVector = thrust::window_vector<uchar>(&outBlock,3,3,1,1);
  thrust::transform(myVector.begin(),myVector.end(),outputVector.begin(),null_block.begin(),erodeFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("output.png",output);

  return 0;
}
