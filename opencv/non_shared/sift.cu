#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class siftTransformFunctor
{
public:
  thrust::block_2d<float> *pyrup;
  thrust::block_2d<float> *pyrdown;

siftTransformFunctor(thrust::block_2d<float> * pyrup,thrust::block_2d<float> * pyrdown)
  {
    this->pyrup=pyrup->device_pointer;
    this->pyrdown=pyrdown->device_pointer;
  }
  __device__ float operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {
    int x = inputWindow.window_dim_x/2;
    int y = inputWindow.window_dim_y/2;
    int pu_x = (inputWindow.start_x + x)/2;
    int pu_y = (inputWindow.start_y + y)/2;
    int pd_x = (inputWindow.start_x + x)*2;
    int pd_y = (inputWindow.start_y + y)*2;
    outputWindow[y][x]=0.0;
    for (int i = 0; i < inputWindow.window_dim_y; i++)
    {
      for (int j = 0; j < inputWindow.window_dim_x; j++)
      {
        outputWindow[y][x]=outputWindow[y][x] + inputWindow[y][x]-inputWindow[i][j];
        outputWindow[y][x]=outputWindow[y][x] + inputWindow[y][x]-(*pyrup)[pu_y][pu_x];
        outputWindow[y][x]=outputWindow[y][x] + inputWindow[y][x]-(*pyrdown)[pd_y][pd_x];
      }
    }
    outputWindow[y][x] = outputWindow[y][x] * 0.05;
    if (outputWindow[y][x] < 0)
    {
      outputWindow[0][0] = 0;
    }
    return 0.0f;
  }
};

int main()
{
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image=small;
  Mat pyrup=imread("pyrup.png",CV_LOAD_IMAGE_GRAYSCALE);
  Mat pyrdown=imread("pyrdown.png",CV_LOAD_IMAGE_GRAYSCALE);
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<unsigned char > pyrup_block (pyrup.cols,pyrup.rows);
  thrust::block_2d<float> float_pyrup_block (pyrup.cols,pyrup.rows);
  thrust::block_2d<unsigned char > pyrdown_block (pyrdown.cols,pyrdown.rows);
  thrust::block_2d<float> float_pyrdown_block (pyrdown.cols,pyrdown.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows,0.0f);
  thrust::block_2d<float> null_block (image.cols,image.rows,0.0f);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  float * imgpu = (float * )malloc(sizeof(float)*(pyrup_block.end()-pyrup_block.begin()));
  float * imgpd = (float * )malloc(sizeof(float)*(pyrdown_block.end()-pyrdown_block.begin()));
  float * img1 = (float * )malloc(sizeof(float)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  for(int i = 0; i<pyrup.cols*pyrup.rows;i++)
  {
    imgpu[i]=(float)pyrup.ptr()[i];
  }
  for(int i = 0; i<pyrdown.cols*pyrdown.rows;i++)
  {
    imgpd[i]=(float)pyrdown.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  float_pyrup_block.assign(imgpu,imgpu+pyrup.cols*pyrup.rows);
  float_pyrdown_block.assign(imgpd,imgpd+pyrdown.cols*pyrdown.rows);
  thrust::window_vector<float> inputVector(&float_image_block,3,3,1,1);
  thrust::window_vector<float> outputVector(&outBlock,3,3,1,1);
  siftTransformFunctor stf(&float_pyrup_block,&float_pyrdown_block);
  thrust::transform(inputVector.begin(),inputVector.end(),outputVector.begin(),null_block.begin(),stf);
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(outBlock.end()-outBlock.begin()));
  hipMemcpy(img1,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(outBlock.end()-outBlock.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<(outBlock.end()-outBlock.begin());i++)
  {
    outputFloatImageData[i]=(unsigned char)img1[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("sift.png",output);
  free (img);
  free (img1);
  free (outputFloatImageData);
  return 0;
}
