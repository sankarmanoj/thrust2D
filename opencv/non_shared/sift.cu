#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class siftTransformFunctor
{
public:
  thrust::block_2d<uchar> *pyrup;
  thrust::block_2d<uchar> *pyrdown;

siftTransformFunctor(thrust::block_2d<uchar> * pyrup,thrust::block_2d<uchar> * pyrdown)
  {
    this->pyrup=pyrup->device_pointer;
    this->pyrdown=pyrdown->device_pointer;
  }
  __device__ uchar operator() (const thrust::window_2d<uchar> &inputWindow,const thrust::window_2d<uchar> &outputWindow) const
  {
    int x = inputWindow.window_dim_x/2;
    int y = inputWindow.window_dim_y/2;
    int pu_x = (inputWindow.start_x + x)/2;
    int pu_y = (inputWindow.start_y + y)/2;
    int pd_x = (inputWindow.start_x + x)*2;
    int pd_y = (inputWindow.start_y + y)*2;
    outputWindow[y][x]=0.0;
    for (int i = 0; i < inputWindow.window_dim_y; i++)
    {
      for (int j = 0; j < inputWindow.window_dim_x; j++)
      {
        outputWindow[y][x]=outputWindow[y][x] + inputWindow[y][x]-inputWindow[i][j];
        outputWindow[y][x]=outputWindow[y][x] + inputWindow[y][x]-(*pyrup)[pu_y][pu_x];
        outputWindow[y][x]=outputWindow[y][x] + inputWindow[y][x]-(*pyrdown)[pd_y][pd_x];
      }
    }
    outputWindow[y][x] = outputWindow[y][x] * 0.05;
    return 0;
  }
};

int main()
{
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image=small;
  Mat pyrup=imread("pyrup.png",CV_LOAD_IMAGE_GRAYSCALE);
  Mat pyrdown=imread("pyrdown.png",CV_LOAD_IMAGE_GRAYSCALE);
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<unsigned char > pyrup_block (pyrup.cols,pyrup.rows);
  thrust::block_2d<uchar> uchar_pyrup_block (pyrup.cols,pyrup.rows);
  thrust::block_2d<unsigned char > pyrdown_block (pyrdown.cols,pyrdown.rows);
  thrust::block_2d<uchar> uchar_pyrdown_block (pyrdown.cols,pyrdown.rows);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows,0.0f);
  thrust::block_2d<uchar> null_block (image.cols,image.rows,0.0f);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(image_block.end()-image_block.begin()));
  uchar * imgpu = (uchar * )malloc(sizeof(uchar)*(pyrup_block.end()-pyrup_block.begin()));
  uchar * imgpd = (uchar * )malloc(sizeof(uchar)*(pyrdown_block.end()-pyrdown_block.begin()));
  uchar * img1 = (uchar * )malloc(sizeof(uchar)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  for(int i = 0; i<pyrup.cols*pyrup.rows;i++)
  {
    imgpu[i]=(uchar)pyrup.ptr()[i];
  }
  for(int i = 0; i<pyrdown.cols*pyrdown.rows;i++)
  {
    imgpd[i]=(uchar)pyrdown.ptr()[i];
  }
  uchar_image_block.assign(img,img+image.cols*image.rows);
  uchar_pyrup_block.assign(imgpu,imgpu+pyrup.cols*pyrup.rows);
  uchar_pyrdown_block.assign(imgpd,imgpd+pyrdown.cols*pyrdown.rows);
  thrust::window_vector<uchar> inputVector(&uchar_image_block,3,3,1,1);
  thrust::window_vector<uchar> outputVector(&outBlock,3,3,1,1);
  siftTransformFunctor stf(&uchar_pyrup_block,&uchar_pyrdown_block);
  thrust::transform(inputVector.begin(),inputVector.end(),outputVector.begin(),null_block.begin(),stf);
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(outBlock.end()-outBlock.begin()));
  hipMemcpy(img1,thrust::raw_pointer_cast(outBlock.data()),sizeof(uchar)*(outBlock.end()-outBlock.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<(outBlock.end()-outBlock.begin());i++)
  {
    outputFloatImageData[i]=(unsigned char)img1[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("sift.png",output);
  free (img);
  free (img1);
  free (outputFloatImageData);
  return 0;
}
