#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_2d.h>
using namespace cv;
class blendFunctor
{
  float alpha;
public:
  blendFunctor(float alpha)
  {
    this->alpha = alpha;
  }
  __device__ uchar operator() (const thrust::window_2d<uchar> &inputWindow1,const thrust::window_2d<uchar> &inputWindow2) const
  {
    return alpha*inputWindow1[0][0]+(1-alpha)*inputWindow2[0][0];
  }
};
int main(int argc, char const *argv[]) {
  Mat input1 = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat input2 = imread("aoutput.png",CV_LOAD_IMAGE_GRAYSCALE);
  Mat temp1;
  resize(input1,temp1,Size(500,500));
  input1 = temp1;
  Mat temp2;
  resize(input2,temp2,Size(500,500));
  input2 = temp2;
  thrust::block_2d<uchar> input_image_block_1 (input1.cols,input1.rows,0.0f);
  thrust::block_2d<uchar> input_image_block_2 (input2.cols,input2.rows,0.0f);
  thrust::block_2d<uchar> output_image_block (input1.cols,input1.rows,0.0f);
  uchar * ucharImageData = (uchar * )malloc(sizeof(uchar)*(input1.cols*input1.rows));
  unsigned char * charImageData = (unsigned char *)malloc(sizeof(unsigned char)*(input_image_block_1.end()-input_image_block_1.begin()));
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    ucharImageData[i]=(uchar)input1.ptr()[i];
  }
  input_image_block_1.assign(ucharImageData,ucharImageData+input1.cols*input1.rows);
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    ucharImageData[i]=(uchar)input2.ptr()[i];
  }
  input_image_block_2.assign(ucharImageData,ucharImageData+input2.cols*input2.rows);
  thrust::window_vector<uchar> inputWindow1 (&input_image_block_1,1,1,1,1);
  thrust::window_vector<uchar> inputWindow2 (&input_image_block_2,1,1,1,1);
  thrust::transform(inputWindow1.begin(),inputWindow1.end(),inputWindow2.begin(),output_image_block.begin(),blendFunctor(0.5));
  hipMemcpy(ucharImageData,output_image_block.data().get(),sizeof(uchar)*(output_image_block.end()-output_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    charImageData[i]=(unsigned char)ucharImageData[i];
  }
  Mat output (Size(input1.cols,input1.rows),CV_8UC1,charImageData);
  imwrite("blend-input1.png",input1);
  imwrite("blend-input2.png",input2);
  imwrite("blend-output.png",output);

  return 0;
}
