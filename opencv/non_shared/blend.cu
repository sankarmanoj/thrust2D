#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class blendFunctor
{
  float alpha;
public:
  blendFunctor(float alpha)
  {
    this->alpha = alpha;
  }
  __device__ float operator() (const thrust::window_2d<float> &inputWindow1,const thrust::window_2d<float> &inputWindow2) const
  {
    return alpha*inputWindow1[0][0]+(1-alpha)*inputWindow2[0][0];
  }
};
int main(int argc, char const *argv[]) {
  Mat input1 = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat input2 = imread("aoutput.png",CV_LOAD_IMAGE_GRAYSCALE);
  Mat temp1;
  resize(input1,temp1,Size(500,500));
  input1 = temp1;
  Mat temp2;
  resize(input2,temp2,Size(500,500));
  input2 = temp2;
  thrust::block_2d<float> input_image_block_1 (input1.cols,input1.rows,0.0f);
  thrust::block_2d<float> input_image_block_2 (input2.cols,input2.rows,0.0f);
  thrust::block_2d<float> output_image_block (input1.cols,input1.rows,0.0f);
  float * floatImageData = (float * )malloc(sizeof(float)*(input1.cols*input1.rows));
  unsigned char * charImageData = (unsigned char *)malloc(sizeof(unsigned char)*(input_image_block_1.end()-input_image_block_1.begin()));
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    floatImageData[i]=(float)input1.ptr()[i];
  }
  input_image_block_1.assign(floatImageData,floatImageData+input1.cols*input1.rows);
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    floatImageData[i]=(float)input2.ptr()[i];
  }
  input_image_block_2.assign(floatImageData,floatImageData+input2.cols*input2.rows);
  thrust::window_vector<float> inputWindow1 (&input_image_block_1,1,1,1,1);
  thrust::window_vector<float> inputWindow2 (&input_image_block_2,1,1,1,1);
  thrust::transform(inputWindow1.begin(),inputWindow1.end(),inputWindow2.begin(),output_image_block.begin(),blendFunctor(0.5));
  hipMemcpy(floatImageData,output_image_block.data().get(),sizeof(float)*(output_image_block.end()-output_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<input1.cols*input1.rows;i++)
  {
    charImageData[i]=(unsigned char)floatImageData[i];
  }
  Mat output (Size(input1.cols,input1.rows),CV_8UC1,charImageData);
  imwrite("blend-input1.png",input1);
  imwrite("blend-input2.png",input2);
  imwrite("blend-output.png",output);
  
  return 0;
}
