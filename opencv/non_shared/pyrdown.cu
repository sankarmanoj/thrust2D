#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_transform.h>
using namespace cv;
#define KERNEL_LENGTH 5
__constant__ float c_kernel[KERNEL_LENGTH*KERNEL_LENGTH];
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,float *GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i*dim + j]=gauss(i,j,mid,sigma);
    }
  }
  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i*dim + j]/=total;
      newTotal +=  (GaussianKernel)[i*dim + j];
    }
  }
}

class convolutionFunctor //:public thrust::shared_unary_window_transform_functor<uchar>
{
public:
  int dim;
  convolutionFunctor(int dim)
  {
    this->dim =dim;
  }
  __device__ void operator() (const thrust::window_2d<uchar> & input_window,const thrust::window_2d<uchar> & output_window) const
  {
    uchar temp = 0;
    for(int i = 0; i< dim; i++)
    {
      for(int j = 0; j<dim; j++)
      {
        temp+=input_window[make_int2(j,i)]*(c_kernel)[i*dim + j];
      }
    }
    output_window[1][1]=temp;
  }
};
class pyrdownTransformFunctor
{
public:
  thrust::block_2d<uchar> *outBlock;

pyrdownTransformFunctor(thrust::block_2d<uchar> * outBlock)
  {
    this->outBlock = outBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<uchar> &inputWindow) const
  {
    int x_out, y_out;
    x_out = inputWindow.start_x*2;
    y_out = inputWindow.start_y*2;
    (*outBlock)[y_out][x_out] = inputWindow[0][0];
  }
};
int main(int argc, char const *argv[])
{
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;

  int dim = 5;
  int dim_image = 512;
  if(argc ==2)
  {
    dim_image = atoi(argv[1]);
  }
  resize(small,image,Size(dim_image,dim_image));
  float *hkernel = (float *) std::malloc(sizeof(float) * dim*dim);
  getGaussianKernelBlock(dim,5,hkernel);
  hipMemcpyToSymbol(HIP_SYMBOL(c_kernel), hkernel, dim*dim * sizeof(float));
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols*2,image.rows*2,0.0f);
  thrust::block_2d<uchar> output_image_block(image.cols*2,image.rows*2,0.0f);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()));
  uchar * img_out = (uchar * )malloc(sizeof(uchar)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  thrust::window_vector<uchar> inputVector(&uchar_image_block,1,1,1,1);
  thrust::window_vector<uchar> input_wv(&outBlock,dim,dim,1,1);
  pyrdownTransformFunctor ptf(&outBlock);
  thrust::for_each(inputVector.begin(),inputVector.end(),ptf);
  hipDeviceSynchronize();
  thrust::window_vector<uchar> output_wv(&output_image_block,dim,dim,1,1);
  thrust::transform(thrust::cuda::shared,input_wv.begin(),input_wv.end(),output_wv.begin(),convolutionFunctor(dim));
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(output_image_block.end()-output_image_block.begin()));
  output_image_block.download(&img);
  for(int i = 0; i<image.cols*image.rows*4;i++)
  {
    outputFloatImageData[i]=(unsigned char)img_out[i];
  }
  Mat output (Size(image.cols*2,image.rows*2),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("pyrdown.png",output);
  return 0;
}
