#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#define PI 3.14159
using namespace cv;
class transFunctor
{
public:
  __device__ float operator() (const float a,const float b) const
  {
    return sqrt(a*a + b*b);
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 3;
  image = small;
  thrust::block_2d<float> kernelx(dim,dim);
  thrust::block_2d<float> kernely(dim,dim);
  //Scharr Filter
  kernelx[0][0]=-3;
  kernelx[0][1]=0;
  kernelx[0][2]=+3;
  kernelx[1][0]=-10;
  kernelx[1][1]=0;
  kernelx[1][2]=+10;
  kernelx[2][0]=-3;
  kernelx[2][1]=0;
  kernelx[2][2]=+3;
  kernely[0][0]=-3;
  kernely[0][1]=-10;
  kernely[0][2]=-3;
  kernely[1][0]=0;
  kernely[1][1]=0;
  kernely[1][2]=0;
  kernely[2][0]=+3;
  kernely[2][1]=+10;
  kernely[2][2]=+3;
  hipEvent_t m_start, m_stop;
  hipEventCreate(&m_start);
  hipEventCreate(&m_stop);
  float m_milliseconds;
  hipEventRecord(m_start);
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> convolve1_block (image.cols,image.rows);
  thrust::block_2d<float> convolve2_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  convolve1_block.assign(float_image_block.begin(),float_image_block.end());
  convolve2_block.assign(float_image_block.begin(),float_image_block.end());
  hipEventRecord(m_stop);
  hipEventSynchronize(m_stop);
  hipEventElapsedTime(&m_milliseconds, m_start, m_stop);
  std::cout<<"Time taken from Host to Device = "<<m_milliseconds<<std::endl;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds;
  hipEventRecord(start);
  thrust::convolve(convolve1_block.begin(),convolve1_block.end(),kernelx.begin());
  thrust::convolve(convolve2_block.begin(),convolve2_block.end(),kernely.begin());
  thrust::transform(convolve1_block.begin(),convolve1_block.end(),convolve2_block.begin(),outBlock.begin(),transFunctor());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout<<"Time taken on Non Shared = "<<milliseconds<<std::endl;
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipEventRecord(m_start);
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  hipEventRecord(m_stop);
  hipEventSynchronize(m_stop);
  hipEventElapsedTime(&m_milliseconds, m_start, m_stop);
  std::cout<<"Time taken from Device to Host = "<<m_milliseconds<<std::endl;
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  cudaCheckError();
  imshow("input",image);
  imshow("output",output);
  waitKey(0);
  return 0;
}
