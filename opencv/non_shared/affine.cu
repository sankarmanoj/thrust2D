#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
using namespace cv;
class AffineTransformFunctor
{
public:
  thrust::block_2d<float> *transformMatrix;
  thrust::block_2d<uchar> *outBlock;

  AffineTransformFunctor(thrust::block_2d<float> * tm,thrust::block_2d<uchar> * outBlock)
  {
    this->transformMatrix = tm->device_pointer;
    this->outBlock = outBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<uchar> &inputWindow) const
  {
    int x_out, y_out;
    x_out = (int)((*transformMatrix)[0][0]*inputWindow.start_x+(*transformMatrix)[0][1]*inputWindow.start_y+(*transformMatrix)[0][2]*1);
    y_out = (int)((*transformMatrix)[1][0]*inputWindow.start_x+(*transformMatrix)[1][1]*inputWindow.start_y+(*transformMatrix)[1][2]*1);

    (*outBlock)[y_out][x_out]=inputWindow[0][0];
  }
};
int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  resize(small,image,Size(dim,dim));
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows,0.0f);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows,0.0f);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(image.cols*image.rows));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  Point2f srcTri[3];
  Point2f dstTri[3];
  Mat warp_mat( 2, 3, CV_32FC1 );
  /// Set your 3 points to calculate the  Affine Transform
  srcTri[0] = Point2f( 0,0 );
  srcTri[1] = Point2f( image.cols - 1, 0 );
  srcTri[2] = Point2f( 0, image.rows - 1 );
  dstTri[0] = Point2f( image.cols*0.0, image.rows*0.5 );
  dstTri[1] = Point2f( image.cols*0.8, image.rows*0.2 );
  dstTri[2] = Point2f( image.cols*0.2, image.rows*0.7 );
  /// Get the Affine Transform
  warp_mat = getAffineTransform( srcTri, dstTri );
  warp_mat.convertTo(warp_mat,CV_32FC1);
  thrust::host_block_2d<float> host_warp_block(warp_mat.cols,warp_mat.rows);
  thrust::block_2d<float> warp_block(warp_mat.cols,warp_mat.rows);
  for(int i = 0; i< warp_mat.rows;i++)
  {
    for(int j = 0; j<warp_mat.cols;j++)
    {
      host_warp_block[i][j]=warp_mat.at<float>(i,j);
    }
  }
  warp_block = host_warp_block;
  //Create Windows For Indexing
  thrust::window_vector<uchar> inputVector(&uchar_image_block,1,1,1,1);
  AffineTransformFunctor atf(&warp_block,&outBlock);
  thrust::for_each(inputVector.begin(),inputVector.end(),atf);
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  outBlock.download(&img);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  #ifdef OWRITE
  cv::imwrite("ainput.png",image);
  cv::imwrite("aoutput.png",output);
  #endif
  #ifdef SHOW
  cv::imshow("ainput.png",image);
  cv::imshow("aoutput.png",output);
    cv::waitKey(0);
  #endif
  return 0;
}
