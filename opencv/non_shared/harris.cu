#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }
  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
}
class HarrisIntensityThrustFunctor
{
public:
  thrust::block_2d<float> * kernel;
  HarrisIntensityThrustFunctor(thrust::block_2d<float> * kernel)
  {
    this->kernel = kernel;
  }
  __device__ float operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {
    float intensityValue;

    for(int xoffset = 1 ; xoffset <=1 ; xoffset++)
    {
      for(int yoffset = 1 ; yoffset <=1 ; yoffset++)
      {
        for(int i = 0; i< 3; i++)
        {
          for(int j = 0; j< 3; j++)
          {
              intensityValue += ((*kernel)[i][j])*(inputWindow[i + yoffset][j + xoffset]-inputWindow[i][j]);
          }
        }
      }
    }
    outputWindow[2][2]=intensityValue*0.08;
    return 0.0;
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  image = small;
  hipEvent_t m_start, m_stop;
  hipEventCreate(&m_start);
  hipEventCreate(&m_stop);
  float m_milliseconds;
  hipEventRecord(m_start);
  thrust::block_2d<float> float_image_block (image.cols,image.rows,0.0f);
  thrust::block_2d<float> null_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows,0.0f);
  float * img = (float * )malloc(sizeof(float)*(image.cols*image.rows));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  hipEventRecord(m_stop);
  hipEventSynchronize(m_stop);
  hipEventElapsedTime(&m_milliseconds, m_start, m_stop);
  std::cout<<"Time taken from Host to Device = "<<m_milliseconds<<std::endl;
  thrust::block_2d<float> kernel(3,3);
  getGaussianKernelBlock(3,5,kernel);
  thrust::window_vector<float> inputVector = thrust::window_vector<float>(&float_image_block,5,5,1,1);
  thrust::window_vector<float> outputVector = thrust::window_vector<float>(&outBlock,5,5,1,1);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds;
  hipEventRecord(start);
  thrust::transform(inputVector.begin(),inputVector.end(),outputVector.begin(),null_block.begin(),HarrisIntensityThrustFunctor(kernel.device_pointer));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout<<"Time taken on Non Shared = "<<milliseconds<<std::endl;
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipEventRecord(m_start);
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  hipEventRecord(m_stop);
  hipEventSynchronize(m_stop);
  hipEventElapsedTime(&m_milliseconds, m_start, m_stop);
  std::cout<<"Time taken from Device to Host = "<<m_milliseconds<<std::endl;
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("ainput.png",image);
  imwrite("aoutput.png",output);
  return 0;
}
