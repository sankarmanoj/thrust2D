#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }
  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
}
class HarrisIntensityThrustFunctor
{
public:
  thrust::block_2d<float> * kernel;
  HarrisIntensityThrustFunctor(thrust::block_2d<float> * kernel)
  {
    this->kernel = kernel;
  }
  __device__ uchar operator() (const thrust::window_2d<uchar> &inputWindow,const thrust::window_2d<uchar> &outputWindow) const
  {
    uchar intensityValue;

    for(int xoffset = 1 ; xoffset <=1 ; xoffset++)
    {
      for(int yoffset = 1 ; yoffset <=1 ; yoffset++)
      {
        for(int i = 0; i< 3; i++)
        {
          for(int j = 0; j< 3; j++)
          {
              intensityValue += ((*kernel)[i][j])*(inputWindow[i + yoffset][j + xoffset]-inputWindow[i][j]);
          }
        }
      }
    }
    outputWindow[2][2]=intensityValue*0.08;
    return 0.0;
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  image = small;
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows,0.0f);
  thrust::block_2d<uchar> null_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows,0.0f);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(image.cols*image.rows));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.assign(img,img+image.cols*image.rows);
  thrust::block_2d<float> kernel(3,3);
  getGaussianKernelBlock(3,5,kernel);
  thrust::window_vector<uchar> inputVector = thrust::window_vector<uchar>(&uchar_image_block,5,5,1,1);
  thrust::window_vector<uchar> outputVector = thrust::window_vector<uchar>(&outBlock,5,5,1,1);
  thrust::transform(inputVector.begin(),inputVector.end(),outputVector.begin(),null_block.begin(),HarrisIntensityThrustFunctor(kernel.device_pointer));
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(uchar)*(uchar_image_block.end()-uchar_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("houtput.png",output);
  return 0;
}
