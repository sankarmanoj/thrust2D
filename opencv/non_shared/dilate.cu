#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class dilateFunctor //: public thrust::shared_unary_window_transform_functor<float>
{
  public:
  __device__ float operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {
    float temp = -1.0;
    for(int i = 0; i<inputWindow.window_dim_y;i++)
    {
      for(int j = 0; j<inputWindow.window_dim_x;j++)
      {
        temp = max(temp,inputWindow[i][j]);
      }
    }
    outputWindow[inputWindow.window_dim_y/2][inputWindow.window_dim_x/2]=temp;
    return 0.0f;
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  resize(small,image,Size(512,512));
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  thrust::block_2d<float> null_block (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&float_image_block,3,3,1,1);
  thrust::window_vector<float> outputVector = thrust::window_vector<float>(&outBlock,3,3,1,1);
  thrust::transform(myVector.begin(),myVector.end(),outputVector.begin(),null_block.begin(),dilateFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("output.png",output);

  return 0;
}
