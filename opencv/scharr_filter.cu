#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <math.h>


#define PI 3.14159
using namespace cv;
class transFunctor : public thrust::shared_binary_window_transform_functor<float>
{
public:
  __device__ void operator() (const thrust::window_2d<float> &inputWindow1,const thrust::window_2d<float> &inputWindow2,const thrust::window_2d<float> &outputWindow) const
  {
    (outputWindow)[outputWindow.window_dim_y/2][outputWindow.window_dim_x/2] = sqrt((inputWindow1)[inputWindow1.window_dim_y/2][inputWindow1.window_dim_x/2]*(inputWindow1)[inputWindow1.window_dim_y/2][inputWindow1.window_dim_x/2] + (inputWindow2)[inputWindow2.window_dim_y/2][inputWindow2.window_dim_x/2]*(inputWindow2)[inputWindow2.window_dim_y/2][inputWindow2.window_dim_x/2]);
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 3;
  image = small;
  // resize(small,image,Size(50,50));
  thrust::block_2d<float> kernelx(dim,dim);
  thrust::block_2d<float> kernely(dim,dim);
  //Sobel Filter
  kernelx[0][0]=-1;
  kernelx[0][1]=0;
  kernelx[0][2]=+1;
  kernelx[1][0]=-2;
  kernelx[1][1]=0;
  kernelx[1][2]=+2;
  kernelx[2][0]=-1;
  kernelx[2][1]=0;
  kernelx[2][2]=+1;
  kernely[0][0]=-1;
  kernely[0][1]=-2;
  kernely[0][2]=-1;
  kernely[1][0]=0;
  kernely[1][1]=0;
  kernely[1][2]=0;
  kernely[2][0]=+1;
  kernely[2][1]=+2;
  kernely[2][2]=+1;

  //Scharr Filter
  // kernelx[0][0]=-3;
  // kernelx[0][1]=0;
  // kernelx[0][2]=+3;
  // kernelx[1][0]=-10;
  // kernelx[1][1]=0;
  // kernelx[1][2]=+10;
  // kernelx[2][0]=-3;
  // kernelx[2][1]=0;
  // kernelx[2][2]=+3;
  // kernely[0][0]=-3;
  // kernely[0][1]=-10;
  // kernely[0][2]=-3;
  // kernely[1][0]=0;
  // kernely[1][1]=0;
  // kernely[1][2]=0;
  // kernely[2][0]=+3;
  // kernely[2][1]=+10;
  // kernely[2][2]=+3;

  std::cout<<dim<<"  "<<image.isContinuous()<<std::endl;
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> convolve1_block (image.cols,image.rows);
  thrust::block_2d<float> convolve2_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  convolve1_block.assign(float_image_block.begin(),float_image_block.end());
  convolve2_block.assign(float_image_block.begin(),float_image_block.end());


  thrust::convolve(convolve1_block.begin(),convolve1_block.end(),kernelx.begin());
  thrust::convolve(convolve2_block.begin(),convolve2_block.end(),kernely.begin());
  thrust::window_vector<float> myVector1 = thrust::window_vector<float>(&convolve1_block,3,3,1,1);
  thrust::window_vector<float> myVector2 = thrust::window_vector<float>(&convolve2_block,3,3,1,1);
  thrust::window_vector<float> myVector3 = thrust::window_vector<float>(&outBlock,3,3,1,1);
  thrust::transform(thrust::cuda::shared,myVector1.begin(),myVector1.end(),myVector2.begin(),myVector3.begin(),transFunctor());

  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  // std::cout<<output;
  cudaCheckError();
  // std::cout<<output.type()<<"  "<<Size(image.cols,image.rows)<<"="<<image_block.end()-image_block.begin()<<"\n";
  imshow("input",image);
  imshow("output",output);
  // imwrite("output.png",output);

  waitKey(0);
  // std::cout<<float_image<<"\n";
  return 0;
}
