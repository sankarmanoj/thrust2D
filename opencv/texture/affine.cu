#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
__constant__ float constantTransformMatrix [6];
class AffineTransformFunctor : public thrust::shared_unary_window_transform_functor<uchar>
{
public:
  thrust::block_2d<uchar> *outBlock;

  AffineTransformFunctor(thrust::block_2d<uchar> * outBlock)
  {

    this->outBlock = outBlock;
  }
  __device__ void operator() (const thrust::window_2d<uchar> &inputWindow, const thrust::window_2d<uchar> &outputWindow) const
  {
    int x_out, y_out;
    x_out = (int)(constantTransformMatrix[0]*inputWindow.start_x+constantTransformMatrix[1]*inputWindow.start_y+constantTransformMatrix[2]*1);
    y_out = (int)(constantTransformMatrix[0+3]*inputWindow.start_x+constantTransformMatrix[1+3]*inputWindow.start_y+constantTransformMatrix[2+3]*1);
    (*outBlock)[y_out][x_out]=inputWindow[make_int2(0,0)];
  }
};
int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  resize(small,image,Size(dim,dim));
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows,0.0f);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows,0.0f);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(image.cols*image.rows));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  Point2f srcTri[3];
  Point2f dstTri[3];
  Mat warp_mat( 2, 3, CV_32FC1 );
  /// Set your 3 points to calculate the  Affine Transform
  srcTri[0] = Point2f( 0,0 );
  srcTri[1] = Point2f( image.cols - 1, 0 );
  srcTri[2] = Point2f( 0, image.rows - 1 );
  dstTri[0] = Point2f( image.cols*0.0, image.rows*0.5 );
  dstTri[1] = Point2f( image.cols*0.8, image.rows*0.2 );
  dstTri[2] = Point2f( image.cols*0.2, image.rows*0.7 );
  /// Get the Affine Transform
  warp_mat = getAffineTransform( srcTri, dstTri );
  warp_mat.convertTo(warp_mat,CV_32FC1);

  hipMemcpyToSymbol(HIP_SYMBOL(constantTransformMatrix),warp_mat.ptr(),sizeof(float)*warp_mat.rows*warp_mat.cols);
  //Create Windows For Indexing
  thrust::window_vector<uchar> inputVector(&uchar_image_block,1,1,1,1);
  AffineTransformFunctor atf(outBlock.device_pointer);
  thrust::transform(thrust::cuda::texture,inputVector.begin(),inputVector.end(),inputVector.begin(),atf);
  // hipDeviceSynchronize();
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  outBlock.download(&img);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  #ifdef OWRITE
  imwrite("input.png",image);
  imwrite("aoutput.png",output);
  #endif
  #ifdef SHOW
  imshow("input.png",image);
  imshow("aoutput.png",output);
  waitKey(0);
  #endif
  return 0;
}
