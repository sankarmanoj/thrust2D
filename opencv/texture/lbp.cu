#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
using namespace cv;
class lbpFunctor : public thrust::shared_unary_window_transform_functor<uchar>
{
  public:
  __device__ void operator() (const thrust::window_2d<uchar> &inputWindow,const thrust::window_2d<uchar> &outputWindow) const
  {
    uchar temp[3][3];
    for(int i = 0; i<3;i++)
    {
      for(int j = 0; j<3;j++)
      {
        temp[i][j] = inputWindow[make_int2(3/2,3/2)]>=inputWindow[make_int2(j,i)]?1:0;
      }
    }
    outputWindow[3/2][3/2]=128*temp[0][1]+64*temp[0][0]+32*temp[1][0]+16*temp[2][0]+8*temp[2][1]+4*temp[2][2]+2*temp[1][2]+1*temp[1][1];
  }
};
int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  resize(small,image,Size(dim,dim));
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  thrust::window_vector<uchar> myVector = thrust::window_vector<uchar>(&uchar_image_block,3,3,1,1);
  thrust::window_vector<uchar> outputVector = thrust::window_vector<uchar>(&outBlock,3,3,1,1);
  thrust::transform(thrust::cuda::texture,myVector.begin(),myVector.end(),outputVector.begin(),lbpFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  outBlock.download(&img);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
#ifdef OWRITE
imwrite("input.png",image);
imwrite("output.png",output);
#endif
#ifdef SHOW
  imshow("input.png",image);
  imshow("output.png",output);
  waitKey(0);
#endif
  return 0;
}
