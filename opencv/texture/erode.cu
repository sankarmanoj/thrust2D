#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#define PI 3.14159
using namespace cv;
class erodeFunctor : public thrust::shared_unary_window_transform_functor<uchar>
{
  public:
  __device__ void operator() (const thrust::window_2d<uchar> &inputWindow,const thrust::window_2d<uchar> &outputWindow) const
  {
    uchar temp = 255;
    for(int i = 0; i<3;i++)
    {
      for(int j = 0; j<3;j++)
      {
        temp = min((float)temp,(float)inputWindow[make_int2(j,i)]);
      }
    }
    outputWindow[1][1]=temp;
  }
};
int main(int argc, char const *argv[]) {
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  resize(small,image,Size(dim,dim));
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<uchar> uchar_image_block (image.cols,image.rows);
  thrust::block_2d<uchar> outBlock (image.cols,image.rows);
  uchar * img = (uchar * )malloc(sizeof(uchar)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(uchar)image.ptr()[i];
  }
  uchar_image_block.upload(img);
  thrust::window_vector<uchar> myVector = thrust::window_vector<uchar>(&uchar_image_block,3,3,1,1);
  thrust::window_vector<uchar> outputVector = thrust::window_vector<uchar>(&outBlock,3,3,1,1);
  thrust::transform(thrust::cuda::texture,myVector.begin(),myVector.end(),outputVector.begin(),erodeFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(uchar_image_block.end()-uchar_image_block.begin()));
  outBlock.download(&img);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  #ifdef OWRITE
  imwrite("input.png",image);
  imwrite("output.png",output);
  #endif
  #ifdef SHOW
  imshow("input.png",image);
  imshow("output.png",output);
  waitKey(0);
  #endif
  return 0;
}
