#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <math.h>


#define PI 3.14159
using namespace cv;
class lbpFunctor //: public thrust::shared_unary_window_transform_functor<float>
{
  public:
  __device__ void operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {
    float temp[3][3];
    for(int i = 0; i<inputWindow.window_dim_y;i++)
    {
      for(int j = 0; j<inputWindow.window_dim_x;j++)
      {
        temp[i][j] = inputWindow[inputWindow.window_dim_y/2][inputWindow.window_dim_x/2]>=inputWindow[i][j]?1:0;
      }
    }
    outputWindow[inputWindow.window_dim_y/2][inputWindow.window_dim_x/2]=128*temp[0][1]+64*temp[0][0]+32*temp[1][0]+16*temp[2][0]+8*temp[2][1]+4*temp[2][2]+2*temp[1][2]+1*temp[1][1];
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  image = small;

  std::cout<<"  "<<image.isContinuous()<<std::endl;
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&float_image_block,3,3,1,1);
  thrust::window_vector<float> outputVector = thrust::window_vector<float>(&outBlock,3,3,1,1);
  thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),outputVector.begin(),lbpFunctor());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  // std::cout<<output;
  cudaCheckError();
  // std::cout<<output.type()<<"  "<<Size(image.cols,image.rows)<<"="<<image_block.end()-image_block.begin()<<"\n";
  imshow("input",image);
  imshow("output",output);
  // imwrite("output.png",output);

  waitKey(0);
  // std::cout<<float_image<<"\n";
  return 0;
}
