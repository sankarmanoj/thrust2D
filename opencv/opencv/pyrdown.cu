#include <opencv2/opencv.hpp>
#include <opencv2/cudawarping.hpp>
int main( int argc, char** argv )
{
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  cv::Mat src, dst,r;
  cv::cuda::GpuMat src_d, dst_d;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  r = cv::imread( "car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  cv::resize(r,src,cv::Size(dim,dim));
  src_d.upload(src);
  cv::cuda::pyrDown( src_d, dst_d);
  dst_d.download(dst);
  #ifdef OWRITE
  cv::imwrite( "pyrdown.png", dst );
  #endif

  #ifdef SHOW
  cv::imshow( "pyrdown.png", dst );
    cv::waitKey(0);
  #endif

  return 0;
}
