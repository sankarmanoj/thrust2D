#include <opencv2/opencv.hpp>
#include <opencv2/cudawarping.hpp>
int main( int argc, char** argv )
{
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  cv::Mat src, dst,r;
  cv::cuda::GpuMat src_d, dst_d;
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  r = cv::imread( "car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  cv::resize(r,src,cv::Size(dim,dim));
  src_d.upload(src);
  cv::cuda::pyrUp( src_d, dst_d);
  dst_d.download(dst);
  #ifdef OWRITE
  cv::imwrite( "pyrup.png", dst );
  #endif

  #ifdef SHOW
  cv::imshow( "pyrup.png", dst );
    cv::waitKey(0);
  #endif
  return 0;
}
