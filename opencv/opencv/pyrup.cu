#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
using namespace cv;
Mat src, dst, tmp;
int main( int argc, char** argv )
{
  src = imread( "car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  if( !src.data )
    { printf(" No data! -- Exiting the program \n");
      return -1; }
  dst = tmp;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds;
  hipEventRecord(start);
  pyrUp( src, dst, Size( tmp.cols*2, tmp.rows*2));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout<<"Time taken on OpenCV = "<<milliseconds<<std::endl;
  imshow( "pyrdown", dst );
  waitKey(0);
  return 0;
}
