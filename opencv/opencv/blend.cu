#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include <iostream>

using namespace cv;

int main( int argc, char** argv )
{
 double alpha = 0.5; double beta;

 Mat src1, src2, dst;

 /// Ask the user enter alpha
 alpha = 0.5;

 /// Read image ( same size, same type )
 src1 = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
 src2 = imread("ainput.png",CV_LOAD_IMAGE_GRAYSCALE);

 if( !src1.data ) { printf("Error loading src1 \n"); return -1; }
 if( !src2.data ) { printf("Error loading src2 \n"); return -1; }

 /// Create Windows
 namedWindow("Linear Blend", 1);

 beta = ( 1.0 - alpha );
 hipEvent_t start, stop;
 hipEventCreate(&start);
 hipEventCreate(&stop);
 float milliseconds;
 hipEventRecord(start);
 addWeighted( src1, alpha, src2, beta, 0.0, dst);
 hipEventRecord(stop);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&milliseconds, start, stop);
 std::cout<<"Time taken on OpenCV = "<<milliseconds<<std::endl;

 imshow( "Linear Blend", dst );

 waitKey(0);
 return 0;
}
