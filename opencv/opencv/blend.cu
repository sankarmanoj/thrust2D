#include <opencv2/opencv.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudaarithm.hpp>

int main( int argc, char** argv )
{
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
 double alpha = 0.5; double beta;

 cv::Mat t, src1, src2, dst;
 cv::cuda::GpuMat src1_d, src2_d,dst_d;

 /// Ask the user enter alpha
 alpha = 0.5;
 //Set Image Dimension
 int dim = 512;
 if(argc ==2)
 {
   dim = atoi(argv[1]);
 }
 /// Read image ( same size, same type )
 t = cv::imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
 cv::resize(t,src1,cv::Size(dim,dim));
 t = cv::imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
 cv::resize(t,src2,cv::Size(dim,dim));
 if( !src1.data ) { printf("Error loading src1 \n"); return -1; }
 if( !src2.data ) { printf("Error loading src2 \n"); return -1; }

 src1_d.upload(src1);
 src2_d.upload(src2);

 beta = ( 1.0 - alpha );
 cv::cuda::addWeighted( src1_d, alpha, src2_d, beta, 0.0, dst_d);
 dst_d.download(dst);
 #ifdef OWRITE
  imwrite( "blend.png", dst );
 #endif
 #ifdef SHOW
  imshow( "blend.png", dst );
    cv::waitKey(0);
 #endif

 return 0;
}
