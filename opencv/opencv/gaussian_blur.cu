#include <opencv2/highgui/highgui.hpp>
#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/core/core.hpp>
#include <iostream>
using namespace cv;
using namespace std;
int main()
{
  Mat image = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  if (image.empty())
  {
    cout << "Cannot load image!" << endl;
    return -1;
  }
  imshow("Image", image);//displaying image
  Mat image1=image.clone();//cloning image
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds;
  hipEventRecord(start);
  GaussianBlur( image, image1, Size( 7, 7), 0, 0 );//applying Gaussian filter
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout<<"Time taken on OpenCV = "<<milliseconds<<std::endl;
  imshow("image1:",image1);//displaying image1
  waitKey(0);
}
