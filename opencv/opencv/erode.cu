#include <opencv2/opencv.hpp>
#include <opencv2/cudafilters.hpp>

int main ( int argc, char **argv )
{
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
    cv::Mat img_final;
    cv::Mat t,im_rgb;
    int dim = 512;
    if(argc ==2)
    {
      dim = atoi(argv[1]);
    }
    t  = cv::imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE );
    cv::resize(t,im_rgb,cv::Size(dim,dim));
    cv::cuda::GpuMat im_rgb_d, img_final_d;
    im_rgb_d.upload(im_rgb);
    cv::Ptr<cv::cuda::Filter> erode = cv::cuda::createMorphologyFilter(cv::MORPH_ERODE, im_rgb_d.type(), cv::Mat());
    erode->apply(im_rgb_d, img_final_d);
    img_final_d.download(img_final);
    #ifdef OWRITE
        cv::imwrite("erode.png",img_final);
    #endif
    #ifdef SHOW
        cv::imshow("erode.png",img_final);
          cv::waitKey(0);
    #endif

    return 0;
}
