#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
using namespace cv;

int main ( int argc, char **argv )
{
    Mat im_gray;
    Mat img_bw;
    Mat img_final;
    Mat im_rgb  = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE );
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;
    hipEventRecord(start);
    erode(im_rgb, img_final, Mat(), Point(-1, -1), 2, 1, 1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout<<"Time taken on OpenCV = "<<milliseconds<<std::endl;
    imshow("dilate",img_final);
    waitKey(0);
    return 0;
}
