#include <opencv2/opencv.hpp>
#include <opencv2/cudafilters.hpp>

int main ( int argc, char **argv )
{
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
    cv::Mat img_final;
    cv::Mat im_rgb;
    int dim = 512;
    if(argc ==2)
    {
      dim = atoi(argv[1]);
    }

    cv::Mat im_rgb_t  = cv::imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE );
    cv::resize(im_rgb_t,im_rgb,cv::Size(dim,dim));
    cv::cuda::GpuMat im_rgb_d, img_final_d;
    im_rgb_d.upload(im_rgb);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    cv::Ptr<cv::cuda::Filter> dilate = cv::cuda::createMorphologyFilter(cv::MORPH_DILATE, im_rgb_d.type(), cv::Mat());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_in_ms;
    hipEventElapsedTime(&time_in_ms,start,stop);
    printf("%f\n",time_in_ms);
    dilate->apply(im_rgb_d, img_final_d);
    img_final_d.download(img_final);
    #ifdef OWRITE
      cv::imwrite("dilate.png",img_final);
    #endif
    #ifdef SHOW
      cv::imshow("dilate.png",img_final);
        cv::waitKey(0);
    #endif

    return 0;
}
