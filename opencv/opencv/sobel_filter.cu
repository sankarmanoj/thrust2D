#include <opencv2/opencv.hpp>
#include <opencv2/cudafilters.hpp>
#include <opencv2/cudaarithm.hpp>
int main( int argc, char** argv )
{
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  cv::Mat t,image;
  t = cv::imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  cv::resize(t,image,cv::Size(dim,dim));
  cv::cuda::GpuMat image_d, grad_x_d,grad_y_d,grad_d;
  // cv::imwrite("input.png", image);//displaying image
  image_d.upload(image);
  /// Generate grad_x and grad_y
  cv::Mat grad_x, grad_y,grad;
  cv::Mat abs_grad_x, abs_grad_y;

  /// Gradient X
  cv::Ptr<cv::cuda::Filter> sobel_x = cv::cuda::createSobelFilter(image_d.type(),grad_x_d.type(),1,0);
  sobel_x->apply(image_d, grad_x_d);
  grad_x_d.download(grad_x);
  cv::convertScaleAbs( grad_x, abs_grad_x );

  /// Gradient Y
  cv::Ptr<cv::cuda::Filter> sobel_y = cv::cuda::createSobelFilter(image_d.type(),grad_y_d.type(),0,1);
  sobel_y->apply(image_d, grad_y_d);
  grad_y_d.download(grad_y);
  cv::convertScaleAbs(grad_y,abs_grad_y);

  /// Total Gradient (approximate)
  cv::addWeighted(abs_grad_x, 0.5, abs_grad_y, 0.5, 0, grad);
  #ifdef OWRITE
  cv::imwrite("sobel.png",grad);//displaying image1
  #endif

  #ifdef SHOW
  cv::imshow("sobel.png",grad);//displaying image1
    cv::waitKey(0);
  #endif
}
