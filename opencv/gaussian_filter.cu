#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <math.h>


#define PI 3.14159
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }

  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
  printf("Total = %f,newTotal=%f\n",total,newTotal);
}
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 13;
  image = small;
  // resize(small,image,Size(50,50));
  thrust::block_2d<float> kernel(dim,dim);
  getGaussianKernelBlock(dim,5,kernel);
  // thrust::fill(kernel.begin(),kernel.end(),0.0f);
  //
  // for(int i = 0; i<dim;i++)
  // {
  //   for(int j = 0; j<dim;j++)
  //   {
  //     float x = (kernel)[i][j];
  //     printf("%f ",x);
  //   }
  //   printf("\n");
  // }

  std::cout<<dim<<"  "<<image.isContinuous()<<std::endl;
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  Mat cvGB;
  GaussianBlur(image,cvGB,Size(3,3),3);
  thrust::convolve(float_image_block.begin(),float_image_block.end(),kernel.begin());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(float_image_block.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  // std::cout<<output;
  cudaCheckError();
  // std::cout<<output.type()<<"  "<<Size(image.cols,image.rows)<<"="<<image_block.end()-image_block.begin()<<"\n";
  imshow("input",image);
  imshow("output",output);
  // imwrite("output.png",output);

  waitKey(0);
  // std::cout<<float_image<<"\n";
  return 0;
}
