#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <math.h>
using namespace cv;
// #define AFFINE

#define PI 3.14159
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
#define HARRIS
class AffineTransformFunctor : public thrust::shared_window_for_each_functor<float>
{
public:
  thrust::block_2d<float> *transformMatrix;
  thrust::block_2d<float> *outBlock;

  AffineTransformFunctor(thrust::block_2d<float> * tm,thrust::block_2d<float> * outBlock)
  {
    this->transformMatrix = tm->device_pointer;
    this->outBlock = outBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<float> &inputWindow) const
  {
    int x_out, y_out;
    x_out = (int)((*transformMatrix)[0][0]*inputWindow.start_x+(*transformMatrix)[0][1]*inputWindow.start_y+(*transformMatrix)[0][2]*1);
    y_out = (int)((*transformMatrix)[1][0]*inputWindow.start_x+(*transformMatrix)[1][1]*inputWindow.start_y+(*transformMatrix)[1][2]*1);

    (*outBlock)[y_out][x_out]=inputWindow[0][0];
  }
};
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }

  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
  printf("Total = %f,newTotal=%f\n",total,newTotal);
}
class HarrisIntensityFunctor : public thrust::shared_unary_window_transform_functor<float>
{

public:
  thrust::block_2d<float> * kernel;
  HarrisIntensityFunctor(thrust::block_2d<float> * kernel)
  {
    this->kernel = kernel;
  }
  __device__ void operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {
    float intensityValue;

    for(int xoffset = 1 ; xoffset <=1 ; xoffset++)
    {
      for(int yoffset = 1 ; yoffset <=1 ; yoffset++)
      {
        for(int i = 0; i< 3; i++)
        {
          for(int j = 0; j< 3; j++)
          {
              intensityValue += ((*kernel)[i][j])*(inputWindow[i + yoffset][j + xoffset]-inputWindow[i][j]);
          }
        }
      }
    }
    outputWindow[2][2]=intensityValue*0.08;
  }
};
class HarrisIntensityThrustFunctor
{

public:
  thrust::block_2d<float> * kernel;
  HarrisIntensityThrustFunctor(thrust::block_2d<float> * kernel)
  {
    this->kernel = kernel;
  }
  __device__ float operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {
    float intensityValue;

    for(int xoffset = 1 ; xoffset <=1 ; xoffset++)
    {
      for(int yoffset = 1 ; yoffset <=1 ; yoffset++)
      {
        for(int i = 0; i< 3; i++)
        {
          for(int j = 0; j< 3; j++)
          {
              intensityValue += ((*kernel)[i][j])*(inputWindow[i + yoffset][j + xoffset]-inputWindow[i][j]);
          }
        }
      }
    }
    outputWindow[2][2]=intensityValue*0.08;
    return 0.0;
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  image = small;
  // resize(small,image,Size(50,50));
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds;
  std::cout<<image.isContinuous()<<image.type()<<std::endl;
  thrust::block_2d<float> float_image_block (image.cols,image.rows);

  thrust::block_2d<float> null_block (image.cols,image.rows);
  thrust::fill(float_image_block.begin(),float_image_block.end(),0.0f);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  thrust::fill(outBlock.begin(),outBlock.end(),0.0f);
  float * img = (float * )malloc(sizeof(float)*(image.cols*image.rows));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);

  thrust::block_2d<float> kernel(3,3);
  getGaussianKernelBlock(3,5,kernel);
  thrust::window_vector<float> inputVector = thrust::window_vector<float>(&float_image_block,5,5,1,1);
  thrust::window_vector<float> outputVector = thrust::window_vector<float>(&outBlock,5,5,1,1);
  hipEventRecord(start);
  thrust::transform(inputVector.begin(),inputVector.end(),outputVector.begin(),null_block.begin(),HarrisIntensityThrustFunctor(kernel.device_pointer));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout<<"Time taken on Shared = "<<milliseconds<<std::endl;
  hipEventRecord(start);
  thrust::transform(thrust::cuda::shared,inputVector.begin(),inputVector.end(),outputVector.begin(),HarrisIntensityFunctor(kernel.device_pointer));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout<<"Time taken on Thrust = "<<milliseconds<<std::endl;
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  // std::cout<<output;
  // cudaCheckError();
  // std::cout<<output.type()<<"  "<<Size(image.cols,image.rows)<<"="<<image_block.end()-image_block.begin()<<"\n";
  imwrite("ainput.png",image);
  // imwrite("output",output);
  imwrite("aoutput.png",output);

;
  return 0;
}
