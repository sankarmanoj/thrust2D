#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <math.h>

#define PI 3.14159
using namespace cv;

void createFilter(float multiplier,float sigma,int dim,thrust::block_2d<float> &gKernel)
{
  // double r, s = 2.0 * sigma * sigma;
  int hdim = dim/2;
  // sum is for normalization
  double sum = 0.0;

  // generate dimxdim kernel
  // for (int x = -hdim; x <= hdim; x++)
  // {
  //     for(int y = -hdim; y <= hdim; y++)
  //     {
  //         r = sqrt(x*x + y*y);
  //         (gKernel)[x + hdim][y + hdim] = (exp(-(r*r)/s))/(M_PI * s);
  //         sum += (gKernel)[x + hdim][y + hdim];
  //     }
  // }
  (gKernel)[0][0] = 1.0/16.0;
  (gKernel)[0][1] = 4.0/16.0;
  (gKernel)[0][2] = 6.0/16.0;
  (gKernel)[0][3] = 4.0/16.0;
  (gKernel)[0][4] = 1.0/16.0;
  (gKernel)[1][0] = 4.0/16.0;
  (gKernel)[1][1] = 16.0/16.0;
  (gKernel)[1][2] = 24.0/16.0;
  (gKernel)[1][3] = 16.0/16.0;
  (gKernel)[1][4] = 4.0/16.0;
  (gKernel)[2][0] = 6.0/16.0;
  (gKernel)[2][1] = 24.0/16.0;
  (gKernel)[2][2] = 36.0/16.0;
  (gKernel)[2][3] = 24.0/16.0;
  (gKernel)[2][4] = 6.0/16.0;
  (gKernel)[3][0] = 4.0/16.0;
  (gKernel)[3][1] = 16.0/16.0;
  (gKernel)[3][2] = 24.0/16.0;
  (gKernel)[3][3] = 16.0/16.0;
  (gKernel)[3][4] = 4.0/16.0;
  (gKernel)[4][0] = 1.0/16.0;
  (gKernel)[4][1] = 4.0/16.0;
  (gKernel)[4][2] = 6.0/16.0;
  (gKernel)[4][3] = 4.0/16.0;
  (gKernel)[4][4] = 1.0/16.0;
  for (int x = -hdim; x <= hdim; x++)
  {
      for(int y = -hdim; y <= hdim; y++)
      {
          (gKernel)[x + hdim][y + hdim] *= multiplier;
          sum += (gKernel)[x + hdim][y + hdim];
      }
  }

  // normalize the Kernel
  for(int i = 0; i < dim; ++i)
  {
    for(int j = 0; j < dim; ++j)
    {
      (gKernel)[i][j] /= sum;
    }
  }
}

class pyrdownTransformFunctor : public thrust::shared_window_for_each_functor<float>
{
public:
  thrust::block_2d<float> *outBlock;

pyrdownTransformFunctor(thrust::block_2d<float> * outBlock)
  {
    this->outBlock = outBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<float> &inputWindow) const
  {
    int x_out, y_out;
    if(inputWindow.start_x%2 && inputWindow.start_y%2)
    {
      x_out = inputWindow.start_x/2;
      y_out = inputWindow.start_y/2;
      (*outBlock)[y_out][x_out]=inputWindow[0][0];
    }
  }
};

int main()
{
  int dim = 5;
  thrust::block_2d<float> kernel(dim,dim);
  createFilter(1.0,1.0,dim,kernel);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image=small;
  // resize(small,image,Size(1500,1500));
  std::cout<<dim<<"  "<<image.isContinuous()<<std::endl;
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  thrust::fill(outBlock.begin(),outBlock.end(),0.0f);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::convolve(float_image_block.begin(),float_image_block.end(),kernel.begin());
  thrust::window_vector<float> inputVector(&float_image_block,1,1,1,1);
  pyrdownTransformFunctor ptf(&outBlock);
  thrust::for_each(inputVector.begin(),inputVector.end(),ptf);
  hipDeviceSynchronize();
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(outBlock.end()-outBlock.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("output.png",output);
  return 0;
}
