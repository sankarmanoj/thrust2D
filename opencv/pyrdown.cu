#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <math.h>

#define PI 3.14159
using namespace cv;

inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }

  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
  printf("Total = %f,newTotal=%f\n",total,newTotal);
}

class pyrdownTransformFunctor : public thrust::shared_window_for_each_functor<float>
{
public:
  thrust::block_2d<float> *inBlock;

pyrdownTransformFunctor(thrust::block_2d<float> * inBlock)
  {
    this->inBlock = inBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<float> &outputWindow) const
  {
    int x_in, y_in;
    if(outputWindow.start_x%2 && outputWindow.start_y%2)
    {
      x_in = outputWindow.start_x/2;
      y_in = outputWindow.start_y/2;
      outputWindow[0][0]=(*inBlock)[y_in][x_in];
    }
  }
};

int main()
{
  int dim = 5;
  thrust::block_2d<float> kernel(dim,dim);
  getGaussianKernelBlock(dim,1.0,kernel);
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image=small;
  // resize(small,image,Size(1500,1500));
  std::cout<<dim<<"  "<<image.isContinuous()<<std::endl;
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols*2,image.rows*2);
  thrust::fill(outBlock.begin(),outBlock.end(),0.0f);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  float * img_out = (float * )malloc(sizeof(float)*(outBlock.end()-outBlock.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  thrust::window_vector<float> inputVector(&outBlock,1,1,1,1);
  pyrdownTransformFunctor ptf(&float_image_block);
  thrust::for_each(inputVector.begin(),inputVector.end(),ptf);
  hipDeviceSynchronize();
  thrust::convolve(outBlock.begin(),outBlock.end(),kernel.begin());
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(outBlock.end()-outBlock.begin()));
  hipMemcpy(img_out,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(outBlock.end()-outBlock.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows*4;i++)
  {
    outputFloatImageData[i]=(unsigned char)img_out[i];
  }
  Mat output (Size(image.cols*2,image.rows*2),CV_8UC1,outputFloatImageData);
  imwrite("input.png",image);
  imwrite("pyrdown.png",output);
  return 0;
}
