#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>

#define STR_SIZE 256
/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5

#define AMBIENT_TEMP 80

/* chip parameters	*/
float t_chip = 0.0005;
float chip_height = 0.016;
float chip_width = 0.016;
/* ambient temperature, assuming no package at all	*/
float amb_temp = 80.0;

void run(int argc, char** argv);

void fatal(char *s)
{
	fprintf(stderr, "error: %s\n", s);

}

void writeoutput(float *vect, int grid_rows, int grid_cols, char *file){
	int i,j, index=0;
	FILE *fp;
	char str[STR_SIZE];

	if( (fp = fopen(file, "w" )) == 0 )
	printf( "The file was not opened\n" );


	for (i=0; i < grid_rows; i++)
	for (j=0; j < grid_cols; j++)
	{

		sprintf(str, "%f\n", vect[i*grid_cols+j]);
		fputs(str,fp);
		index++;
	}

	fclose(fp);
}


void readinput(float * vect, int grid_rows, int grid_cols, char *file){

	int i,j;
	FILE *fp;
	char str[STR_SIZE];
	float val;

	if( (fp  = fopen(file, "r" )) ==0 )
	printf( "The file was not opened\n" );
	for (i=0; i <= grid_rows-1; i++)
	for (j=0; j <= grid_cols-1; j++)
	{
		fgets(str, STR_SIZE, fp);
		if (feof(fp))
		fatal((char *)"not enough lines in file");
		if ((sscanf(str, "%f", &val) != 1))
		fatal((char *)"invalid file format");
		vect[i*grid_cols+j] = val;
	}

	fclose(fp);

}
#define MIN(a, b) ((a)<=(b) ? (a) : (b))


class HotspotFunctor
{
	int iteration;
	int cols;
	int rows;
	float stepDivCap;
	float Rx_1;
	float Ry_1;
	float Rz_1;
	thrust::block_2d<float> *MatrixPower;
public:

	HotspotFunctor (thrust::block_2d<float> *PowerBlock,int iteration,int cols,int rows,float stepDivCap,float Rx_1,float Ry_1,float Rz_1)
	{
		this->MatrixPower = PowerBlock;
		this->iteration = iteration;
		this->cols = cols;
		this->rows = rows;
		this->stepDivCap = stepDivCap;
		this->Rx_1 = Rx_1;
		this->Ry_1 = Ry_1;
		this->Rz_1 = Rz_1;
	}

	__device__ void operator() (const thrust::window_2d<float> &w) const
	{
		int ty = w.window_dim_y/2;
		int tx = w.window_dim_x/2;
		int rty = w.start_y + ty;
		int rtx = w.start_x +tx;
		int N = ty-1;
		int S = ty+1;
		int W = tx-1;
		int E = tx+1;
		float myPower = (*MatrixPower)[rty][rtx];
		for (int i=0; i<iteration ; i++)
		{
			w[ty][tx] =  w[ty][tx] + stepDivCap * (myPower + \
				(w[S][tx] + w[N][tx] - 2.0*(w[ty][tx])) * Ry_1 + \
				(w[ty][E] + w[ty][W] - 2.0*(w[ty][tx])) * Rx_1 + \
				(AMBIENT_TEMP - w[ty][tx]) * Rz_1);
			}
			// Boundary Condtions - causes warp divergence.
			if(w.start_y == 0)
			w[N][tx] = w[ty][tx];
			if(w.start_y == rows - w.window_dim_y)
			w[S][tx] = w[ty][tx];
			if(w.start_x == 0)
			w[ty][W] = w[ty][tx];
			if(w.start_x == cols - w.window_dim_x)
			w[ty][E] = w[ty][tx];
			if(w.start_y == 0 && w.start_x == 0)
			w[N][W] = w[ty][tx];
			if(w.start_y == rows - w.window_dim_y && w.start_x == cols - w.window_dim_x)
			w[S][E] = w[ty][tx];
			if(w.start_x == 0 && w.start_y == rows - w.window_dim_y)
			w[S][W] = w[ty][tx];
			if(w.start_x == cols - w.window_dim_x && w.start_y == 0)
			w[N][E] = w[ty][tx];
		}
	};

	void usage(int argc, char **argv)
	{
		fprintf(stderr, "Usage: %s <grid_rows/grid_cols> <pyramid_height> <sim_time> <temp_file> <power_file> <output_file>\n", argv[0]);
		fprintf(stderr, "\t<grid_rows/grid_cols>  - number of rows/cols in the grid (positive integer)\n");
		fprintf(stderr, "\t<pyramid_height> - pyramid heigh(positive integer)\n");
		fprintf(stderr, "\t<sim_time>   - number of iterations\n");
		fprintf(stderr, "\t<temp_file>  - name of the file containing the initial temperature values of each cell\n");
		fprintf(stderr, "\t<power_file> - name of the file containing the dissipated power values of each cell\n");
		fprintf(stderr, "\t<output_file> - name of the output file\n");
		exit(1);
	}

	int main(int argc, char** argv)
	{
		// printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

		run(argc,argv);

		return EXIT_SUCCESS;
	}

	void run(int argc, char** argv)
	{
		int size;
		int grid_rows,grid_cols;
		float *FilesavingTemp,*FilesavingPower;
		char *tfile, *pfile, *ofile;

		int total_iterations = 60;
		int pyramid_height = 1; // number of iterations

		if (argc != 7)
		usage(argc, argv);
		if((grid_rows = atoi(argv[1]))<=0||
		(grid_cols = atoi(argv[1]))<=0||
		(pyramid_height = atoi(argv[2]))<=0||
		(total_iterations = atoi(argv[3]))<=0)
		usage(argc, argv);

		tfile=argv[4];
		pfile=argv[5];
		ofile=argv[6];

		size=grid_rows*grid_cols;

		FilesavingTemp = (float *) malloc(size*sizeof(float));
		FilesavingPower = (float *) malloc(size*sizeof(float));

		if( !FilesavingPower || !FilesavingTemp)
		fatal((char *)"unable to allocate memory");

		readinput(FilesavingTemp, grid_rows, grid_cols, tfile);
		readinput(FilesavingPower, grid_rows, grid_cols, pfile);
		thrust::block_2d<float> TemperatureBlock(grid_rows,grid_cols);
		thrust::block_2d<float> PowerBlock(grid_rows,grid_cols);
		TemperatureBlock.assign(FilesavingTemp,FilesavingTemp+size);
		PowerBlock.assign(FilesavingPower,FilesavingPower+size);
		printf("Start computing the transient temperature\n");
		float grid_height = chip_height / grid_rows;
		float grid_width = chip_width / grid_cols;

		float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
		float Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
		float Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
		float Rz = t_chip / (K_SI * grid_height * grid_width);

		float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
		float step = PRECISION / max_slope;
		int t;
		float step_div_Cap;
		float Rx_1,Ry_1,Rz_1;
		step_div_Cap=step/Cap;
		Rx_1=1/Rx;
		Ry_1=1/Ry;
		Rz_1=1/Rz;
		printf("step_div_Cap = %f\nrx,ry,rz = %f,%f,%f\n",step_div_Cap,Rx_1,Ry_1,Rz_1);
		hipEvent_t tstart, tstop;
		hipEventCreate(&tstart);
		hipEventCreate(&tstop);
		hipEventRecord(tstart);
		for (t = 0; t < total_iterations; t+=pyramid_height)
		{
			int required_iterations = MIN(pyramid_height,total_iterations-t);
			HotspotFunctor functor(PowerBlock.device_pointer,required_iterations,grid_cols,grid_rows,step_div_Cap,Rx_1,Ry_1,Rz_1);
			thrust::window_vector<float> wv = thrust::window_vector<float>(&(TemperatureBlock),3,3,1,1);
			// thrust::window_vector<float> wp = thrust::window_vector<float>(&(PowerBlock),3,3,1,1);
			// thrust::device_vector<int> null_vector(grid_rows*grid_cols);
			// thrust::transform(wv.begin(),wv.end(),wp.begin(),null_vector.begin(),functor);
			thrust::for_each(wv.begin(),wv.end(),functor);
		}
		printf("Ending simulation\n");
		hipEventRecord(tstop);
		hipEventSynchronize(tstop);
		float timeTaken;
		hipEventElapsedTime(&timeTaken,tstart,tstop);
		printf("Thrust Time = %f \n",timeTaken);
		hipMemcpy(FilesavingTemp,thrust::raw_pointer_cast(TemperatureBlock.data()),size*sizeof(float),hipMemcpyDeviceToHost);
		writeoutput(FilesavingTemp,grid_rows, grid_cols, ofile);
	}
