
#include "primary.h"
void frame_calculate(int *frame, int rows, int cols,int niter, int nErode, float threshold,float lambda)
{
  float  q0sqr, sum, sum2,meanROI,varROI;
	unsigned int r1, r2, c1, c2,size_R;
  int iter,size_I = rows*cols;
  r1 = 0;
  r2 = rows - 1;
  c1 = 0;
  c2 = cols - 1;

  size_R = (r2-r1+1)*(c2-c1+1);
  thrust::Block_2D<int> J_cuda (cols,rows);
  thrust::Block_2D<float> J_square(cols,rows);
  thrust::Block_2D<float> d_c(cols,rows);
  thrust::Block_2D<float> J_floatcuda(cols,rows);
  thrust::fill(d_c.begin(),d_c.end(),0.0f);
  J_cuda.assign(frame[0],frame[size_I]);
  thrust::transform(J_cuda.begin(),J_cuda.end(),J_floatcuda.begin(),extractFunctor());


  printf("Start the SRAD main loop\n");
    for (iter=0; iter< niter; iter++)
  {
    thrust::copy(J_floatcuda.begin(),J_floatcuda.end(),J_square.begin());
    thrust::for_each(J_square.begin(),J_square.end(),square());
    sum = thrust::reduce(J_floatcuda.begin(),J_floatcuda.end());
    sum2 = thrust::reduce(J_square.begin(),J_square.end());
    meanROI = sum / size_R;
    varROI  = (sum2 / size_R) - meanROI*meanROI;
    q0sqr   = varROI / (meanROI*meanROI);
    SRADFunctor1 functor1(cols,rows,q0sqr);
    SRADFunctor2 functor2(cols,rows,lambda,q0sqr);
    thrust::window_vector<float> wv = thrust::window_vector<float>(&(J_floatcuda),3,3,1,1);
    thrust::window_vector<float> d_cwv = thrust::window_vector<float>(&(d_c),3,3,1,1);
    thrust::transform(wv.begin(),wv.end(),d_cwv.begin(),J_square.begin(),functor1);
    thrust::transform(wv.begin(),wv.end(),d_cwv.begin(),J_square.begin(),functor2);
  }
  printf("Binarize\n");
  thrust::transform(J_floatcuda.begin(),J_floatcuda.end(),J_cuda.begin(),binarizeFunctor(threshold));
  printf("Erode And Dilate\n");
  thrust::window_vector<int> erodeInputWindow = thrust::window_vector<int>(&(J_cuda),3,3,1,1);
  for(int erodeTimes = 0; erodeTimes < nErode ; erodeTimes++)
  {
    //Erode
      thrust::for_each(erodeInputWindow.begin(),erodeInputWindow.end(),erodeFunctor());
  }
  for(int erodeTimes = 0; erodeTimes < nErode ; erodeTimes++)
  {
    //Dilate
      thrust::for_each(erodeInputWindow.begin(),erodeInputWindow.end(),dilateFunctor());
  }

  printf("Computation Done\n");
  thrust::for_each(J_cuda.begin(),J_cuda.end(),compressFunctor());
  hipMemcpy(frame,thrust::raw_pointer_cast(J_cuda.data()),size_I*sizeof(int),hipMemcpyDeviceToHost);

}
