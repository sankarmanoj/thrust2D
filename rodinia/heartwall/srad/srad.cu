// includes, system
extern "C"
{
#include "ffmpeg.h"
}
#include <stdio.h>
#include <stdlib.h>
#include "primary.h"

int runTest( int argc, char** argv);
void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <frames> <lambda> <niter> <in video> <out video>\n", argv[0]);
	exit(1);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv)
{
	time_t t;
	srand((unsigned) time(&t));

  runTest( argc, argv);
  return EXIT_SUCCESS;
}


int runTest( int argc, char** argv)
{
  unsigned int rows, cols, size_I, size_R, niter = 10, iter,nErode;
  float lambda, q0sqr, sum, sum2,meanROI,varROI,threshold;
	int ret;
	AVPacket packet;
	AVFrame *frame = NULL;
	enum AVMediaType type;
	unsigned int stream_index;
	unsigned int i;
	int got_frame;
	int (*dec_func)(AVCodecContext *, AVFrame *, int *, const AVPacket *);

	char *in,*out;

	if (argc == 7)
	{
		threshold = atof(argv[1]);
		lambda = atof(argv[2]); //Lambda value
		niter = atoi(argv[3]); //number of iterations
		nErode = atoi(argv[4]);
		in = argv[5];
		out = argv[6];
	}
  else
	{
		usage(argc, argv);
  }

	printf(" %s Threshold = %f \n",argv[1],threshold);
	av_register_all();
	ret = open_input_file(in);
	printf("Input File Opened\n");
	ret = open_output_file(out);
	printf("Output File Opened - %d\n",ret);
	/* read all packets */
	while (1) {
			printf("Beginning of Loop\n");
			if ((ret = av_read_frame(ifmt_ctx, &packet)) < 0)
					break;
			stream_index = packet.stream_index;
			type = ifmt_ctx->streams[packet.stream_index]->codec->codec_type;
			rows = ifmt_ctx->streams[packet.stream_index]->codec->height;
			cols = ifmt_ctx->streams[packet.stream_index]->codec->width;


			size_I = cols * rows;
			av_log(NULL, AV_LOG_DEBUG, "Demuxer gave frame of stream_index %u\n",stream_index);
			frame = av_frame_alloc();
			if (!frame) {
					ret = AVERROR(ENOMEM);
					break;
			}
			ifmt_ctx->streams[stream_index]->codec->refcounted_frames = 1;
			av_packet_rescale_ts(&packet,ifmt_ctx->streams[stream_index]->time_base,ifmt_ctx->streams[stream_index]->codec->time_base);
			dec_func = avcodec_decode_video2;
			ret = dec_func(ifmt_ctx->streams[stream_index]->codec, frame,&got_frame, &packet);
			printf(" Got Frame = %d\n",got_frame);
			ret = av_frame_make_writable (frame);
			if (ret < 0) {
					// av_frame_free(&frame);
					av_log(NULL, AV_LOG_ERROR, "Decoding failed - Make Writable\n");
					break;
			}
			if (got_frame) {
					frame->pts = av_frame_get_best_effort_timestamp(frame);
					printf("Key Frame = %d\n",frame->key_frame);


					//MAIN CALCULATION
					int *temp = (int *) malloc(size_I * sizeof(int));
					for (int i = 0; i<size_I; i++)
						temp[i] = frame->data[0][i];

					frame_calculate(temp,rows,cols,niter,nErode,threshold,lambda);


					for (int i = 0; i<size_I; i++)
						frame->data[0][i] = 255;



					AVPacket enc_pkt;
			    int (*enc_func)(AVCodecContext *, AVPacket *, const AVFrame *, int *) =
			        (ifmt_ctx->streams[stream_index]->codec->codec_type ==
			         AVMEDIA_TYPE_VIDEO) ? avcodec_encode_video2 : avcodec_encode_audio2;
			    // av_log(NULL, AV_LOG_INFO, "Encoding frame\n");
			    /* encode filtered frame */
			    enc_pkt.data = NULL;
			    enc_pkt.size = 0;
			    av_init_packet(&enc_pkt);
			    ret = enc_func(ofmt_ctx->streams[stream_index]->codec, &enc_pkt,
			            frame, &got_frame);
			    av_frame_free(&frame);
			    if (ret < 0)
			        return ret;
			    if (!(got_frame))
			        return 0;
			    /* prepare packet for muxing */
			    enc_pkt.stream_index = stream_index;
			    av_packet_rescale_ts(&enc_pkt,
			                         ofmt_ctx->streams[stream_index]->codec->time_base,
			                         ofmt_ctx->streams[stream_index]->time_base);
			    // av_log(NULL, AV_LOG_DEBUG, "Muxing frame\n");
			    /* mux encoded frame */
					ret = av_write_frame(ofmt_ctx, &enc_pkt);

			}
			av_frame_unref(frame);
	}
	av_write_trailer(ofmt_ctx);

	av_packet_unref(&packet);
	av_frame_free(&frame);
	for (i = 0; i < ifmt_ctx->nb_streams; i++) {
			avcodec_close(ifmt_ctx->streams[i]->codec);
			if (ofmt_ctx && ofmt_ctx->nb_streams > i && ofmt_ctx->streams[i] && ofmt_ctx->streams[i]->codec)
					avcodec_close(ofmt_ctx->streams[i]->codec);
	}
	avformat_close_input(&ifmt_ctx);
	if (ofmt_ctx && !(ofmt_ctx->oformat->flags & AVFMT_NOFILE))
			avio_closep(&ofmt_ctx->pb);
	avformat_free_context(ofmt_ctx);
	return 0;

}
