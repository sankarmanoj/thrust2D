// includes, system
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/window_2d.h>
#include <stdlib.h>
// includes, kernels
#include "srad_kernel.cu"

extern "C" {
#include "ffmpeg.c"
}

int runTest( int argc, char** argv);
void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <frames> <lambda> <niter> <in video> <out video>\n", argv[0]);
	exit(1);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv)
{
	time_t t;
	srand((unsigned) time(&t));

  runTest( argc, argv);
  return EXIT_SUCCESS;
}


int runTest( int argc, char** argv)
{
  unsigned int rows, cols, size_I, size_R, niter = 10, iter,nErode;
  float lambda, q0sqr, sum, sum2,meanROI,varROI,threshold,maxThreshold ;
	unsigned int r1, r2, c1, c2;
	int ret;
	AVPacket packet;
	AVFrame *frame = NULL;
	enum AVMediaType type;
	unsigned int stream_index;
	unsigned int i;
	int got_frame;
	int (*dec_func)(AVCodecContext *, AVFrame *, int *, const AVPacket *);

	char *in,*out;

	if (argc == 8)
	{
		threshold = atof(argv[1]);
		maxThreshold = atof(argv[2]);
		lambda = atof(argv[3]); //Lambda value
		niter = atoi(argv[4]); //number of iterations
		nErode = atoi(argv[5]);
		in = argv[6];
		out = argv[7];
	}
  else
	{
		usage(argc, argv);
  }

	printf(" %s Threshold = %f \n",argv[1],threshold);
	av_register_all();
	ret = open_input_file(in);
	printf("Input File Opened\n");
	ret = open_output_file(out);
	printf("Output File Opened - %d\n",ret);
	/* read all packets */
	while (1) {
			printf("Beginning of Loop\n");
			if ((ret = av_read_frame(ifmt_ctx, &packet)) < 0)
					break;
			stream_index = packet.stream_index;
			printf("Stream Index =%d",stream_index);
			type = ifmt_ctx->streams[packet.stream_index]->codec->codec_type;
			rows = ifmt_ctx->streams[packet.stream_index]->codec->height;
			cols = ifmt_ctx->streams[packet.stream_index]->codec->width;
			r1 = 0;
			r2 = rows - 1;
			c1 = 0;
			c2 = cols - 1;

			size_R = (r2-r1+1)*(c2-c1+1);

			size_I = cols * rows;
			av_log(NULL, AV_LOG_DEBUG, "Demuxer gave frame of stream_index %u\n",stream_index);
			frame = av_frame_alloc();
			if (!frame) {
					ret = AVERROR(ENOMEM);
					break;
			}
			ifmt_ctx->streams[stream_index]->codec->refcounted_frames = 1;
			av_packet_rescale_ts(&packet,ifmt_ctx->streams[stream_index]->time_base,ifmt_ctx->streams[stream_index]->codec->time_base);
			dec_func = avcodec_decode_video2;
			ret = dec_func(ifmt_ctx->streams[stream_index]->codec, frame,&got_frame, &packet);
			printf(" Got Frame = %d\n",got_frame);
			ret = av_frame_make_writable (frame);
			if (ret < 0) {
					// av_frame_free(&frame);
					av_log(NULL, AV_LOG_ERROR, "Decoding failed - Make Writable\n");
					break;
			}
			if (got_frame) {
					frame->pts = av_frame_get_best_effort_timestamp(frame);
					AVPixelFormat tempFormat = (AVPixelFormat)frame->format;
					const AVPixFmtDescriptor *x =  av_pix_fmt_desc_get(tempFormat);
					printf("Pixel Format = %s\n",av_pix_fmt_desc_get(tempFormat)->name);
					thrust::block_2d<int> J_cuda (cols,rows);
					thrust::block_2d<float> J_square(cols,rows);
					thrust::block_2d<float> d_c(cols,rows);
					thrust::block_2d<float> J_floatcuda(cols,rows);
					thrust::fill(d_c.begin(),d_c.end(),0.0f);
					J_cuda.assign(&frame->data[0][0],&frame->data[0][size_I]);
					thrust::transform(J_cuda.begin(),J_cuda.end(),J_floatcuda.begin(),extractFunctor());





					printf("Start the SRAD main loop\n");
						for (iter=0; iter< niter; iter++)
					{
						thrust::copy(J_floatcuda.begin(),J_floatcuda.end(),J_square.begin());
						thrust::for_each(J_square.begin(),J_square.end(),square());
						sum = thrust::reduce(J_floatcuda.begin(),J_floatcuda.end());
						sum2 = thrust::reduce(J_square.begin(),J_square.end());
					  meanROI = sum / size_R;
					  varROI  = (sum2 / size_R) - meanROI*meanROI;
					  q0sqr   = varROI / (meanROI*meanROI);
						SRADFunctor1 functor1(cols,rows,q0sqr);
						SRADFunctor2 functor2(cols,rows,lambda,q0sqr);
						thrust::window_vector<float> wv = thrust::window_vector<float>(&(J_floatcuda),3,3,1,1);
						thrust::window_vector<float> d_cwv = thrust::window_vector<float>(&(d_c),3,3,1,1);
						thrust::transform(wv.begin(),wv.end(),d_cwv.begin(),J_square.begin(),functor1);
						thrust::transform(wv.begin(),wv.end(),d_cwv.begin(),J_square.begin(),functor2);
					}
					printf("Binarize\n");
					thrust::transform(J_floatcuda.begin(),J_floatcuda.end(),J_cuda.begin(),binarizeFunctor(threshold,maxThreshold));
					printf("Erode And Dilate\n");
					thrust::window_vector<int> erodeInputWindow = thrust::window_vector<int>(&(J_cuda),3,3,1,1);
					for(int erodeTimes = 0; erodeTimes < nErode ; erodeTimes++)
					{
						//Erode
							thrust::for_each(erodeInputWindow.begin(),erodeInputWindow.end(),erodeFunctor());
							thrust::for_each(erodeInputWindow.begin(),erodeInputWindow.end(),dilateFunctor());
					}
					for(int erodeTimes = 0; erodeTimes < nErode ; erodeTimes++)
					{
						//Erode
							thrust::for_each(erodeInputWindow.begin(),erodeInputWindow.end(),erodeFunctor());
					}
					for(int erodeTimes = 0; erodeTimes < nErode ; erodeTimes++)
					{
						//Dilate
							thrust::for_each(erodeInputWindow.begin(),erodeInputWindow.end(),dilateFunctor());
					}

					printf("Computation Done\n");
					thrust::for_each(J_cuda.begin(),J_cuda.end(),compressFunctor());
					int *temp = (int *) malloc(size_I * sizeof(int));
					hipMemcpy(temp,thrust::raw_pointer_cast(J_cuda.data()),size_I*sizeof(int),hipMemcpyDeviceToHost);
					for (int i = 0; i<size_I; i++)
						frame->data[0][i] = temp[i];
					AVPacket enc_pkt;
			    int (*enc_func)(AVCodecContext *, AVPacket *, const AVFrame *, int *) =
			        (ifmt_ctx->streams[stream_index]->codec->codec_type ==
			         AVMEDIA_TYPE_VIDEO) ? avcodec_encode_video2 : avcodec_encode_audio2;
			    // av_log(NULL, AV_LOG_INFO, "Encoding frame\n");
			    /* encode filtered frame */
			    enc_pkt.data = NULL;
			    enc_pkt.size = 0;
			    av_init_packet(&enc_pkt);
			    ret = enc_func(ofmt_ctx->streams[stream_index]->codec, &enc_pkt,
			            frame, &got_frame);
			    av_frame_free(&frame);

			    if (ret < 0)
			        return ret;
			    if (!(got_frame))
			        return 0;
			    /* prepare packet for muxing */
			    enc_pkt.stream_index = stream_index;
			    av_packet_rescale_ts(&enc_pkt,
			                         ofmt_ctx->streams[stream_index]->codec->time_base,
			                         ofmt_ctx->streams[stream_index]->time_base);
			    // av_log(NULL, AV_LOG_DEBUG, "Muxing frame\n");
			    /* mux encoded frame */
					ret = av_write_frame(ofmt_ctx, &enc_pkt);

			}
			av_frame_unref(frame);
	}
	av_write_trailer(ofmt_ctx);

	av_packet_unref(&packet);
	av_frame_free(&frame);
	for (i = 0; i < ifmt_ctx->nb_streams; i++) {
			avcodec_close(ifmt_ctx->streams[i]->codec);
			if (ofmt_ctx && ofmt_ctx->nb_streams > i && ofmt_ctx->streams[i] && ofmt_ctx->streams[i]->codec)
					avcodec_close(ofmt_ctx->streams[i]->codec);
	}
	avformat_close_input(&ifmt_ctx);
	if (ofmt_ctx && !(ofmt_ctx->oformat->flags & AVFMT_NOFILE))
			avio_closep(&ofmt_ctx->pb);
	avformat_free_context(ofmt_ctx);
	return 0;

}
