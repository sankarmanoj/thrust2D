#include "hip/hip_runtime.h"
class compressFunctor
{
public:
	__device__  void operator() (int &x) const
	{
		x =  x*255;
	}
};
class extractFunctor
{
public:
	__device__ float operator() (int &x) const
	{

		float y  = exp((float)x/255);
		return y;
	}
};

class binarizeFunctor
{
	float threshold;
	float maxValue = 3.0f;
	public:
	binarizeFunctor(float t)
	{
		this->threshold = t;
	}
	binarizeFunctor(float t, float max)
	{
			this->threshold = t;
			this->maxValue = max;
	}
	__device__ int operator() (float &x) const
	{
		if(x>threshold&&x<maxValue)
		{
			return 1;
		}
		else{
			return 0;
		}
	}
};
class erodeTransformFunctor
{
public:
	__device__ int operator() (const thrust::window_2D<int> &w,const thrust::window_2D<int> &v) const
	{
		int minvalue = 1;
		for(int i = 0; i<w.window_dim_x;i++)
		{
			for(int j = 0; j<w.window_dim_y;j++)
			{
				minvalue *=w[i][j];
			}
		}

		v[(v.window_dim_y-1)/2][(v.window_dim_x-1)/2]=minvalue;
		return 4;
	}
};
class erodeFunctor
{
public:
	__device__ void operator() (const thrust::window_2D<int> &w) const
	{
		int minvalue =w[0][1]*w[2][1]*w[1][0]*w[1][2]*w[1][1];
		w[(w.window_dim_y-1)/2][(w.window_dim_x-1)/2]=minvalue;
	}
};
class dilateFunctor
{
public:
	__device__ void operator() (const thrust::window_2D<int> &w) const
	{
		int maxvalue = w[0][1]+w[2][1]+w[1][0]+w[1][2]+w[1][1];
		if(maxvalue>1)
		maxvalue =1;
		w[(w.window_dim_y-1)/2][(w.window_dim_x-1)/2]=maxvalue;
	}
};
class square
{

public:
 __host__ __device__ void operator() (float &lhs) const
  {
		lhs = lhs*lhs;
	}
};
class SRADFunctor1
{
public:
	int cols;
	int rows;
	float q0sqr;

	SRADFunctor1 (int cols, int rows, float q0sqr)
	{
		this->cols = cols;
		this->rows = rows;
		this->q0sqr = q0sqr;
	}
	__device__ int operator() (const thrust::window_2D<float> &w,const thrust::window_2D<float> &v) const
	{
		int ty = w.window_dim_y/2;
		int tx = w.window_dim_x/2;
		int N = ty-1;
		int S = ty+1;
		int W = tx-1;
		int E = tx+1;

		float jc,n,s,we,e,g2,l,num,den,qsqr,c;
		jc = (float) w[ty][tx];
		n  = (float) w[N][tx] - jc;
    s  = (float) w[S][tx] - jc;
    we = (float) w[ty][W]  - jc;
    e  = (float) w[ty][E] - jc;


    g2 = ( n * n + s * s + we * we + e * e ) / (jc * jc);

    l = ( n + s + we + e ) / jc;

		num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
		den  = 1 + (.25*l);
		qsqr = num/(den*den);
		// diffusion coefficent (equ 33)
		den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
		c = 1.0 / (1.0+den) ;


	  // saturate diffusion coefficent
		if(c<0)
		{
			c=0;
		}
		else if(c>1)
		{
			c=1;
		}
		v[ty][tx] = c;

		if(w.start_y == 0)
			w[N][tx] = w[ty][tx];
		if(w.start_y == rows - w.window_dim_y)
			w[S][tx] = w[ty][tx];
		if(w.start_x == 0)
			w[ty][W] = w[ty][tx];
		if(w.start_x == cols - w.window_dim_x)
			w[ty][E] = w[ty][tx];
		if(w.start_y == 0 && w.start_x == 0)
			w[N][W] = w[ty][tx];
		if(w.start_y == rows - w.window_dim_y && w.start_x == cols - w.window_dim_x)
			w[S][E] = w[ty][tx];
		if(w.start_x == 0 && w.start_y == rows - w.window_dim_y)
			w[S][W] = w[ty][tx];
		if(w.start_x == cols - w.window_dim_x && w.start_y == 0)
			w[N][E] = w[ty][tx];

		return 0.0;

	}

};
class printFunctor
{
public:
	__device__ void operator() (const int &x) const
	{
		printf(" %d \n",x);
	}
};

class SRADFunctor2
{
public:
	int cols;
	int rows;
	float lambda;
	float q0sqr;

	SRADFunctor2 (int cols,int rows,float lambda,float q0sqr)
	{
		this->cols = cols;
		this->rows = rows;
		this->lambda = lambda;
		this->q0sqr = q0sqr;
	}

	__device__ int operator() (const thrust::window_2D<float> &w, const thrust::window_2D<float> &c) const
	{
		int ty = w.window_dim_y/2;
		int tx = w.window_dim_x/2;
		int N = ty-1;
		int S = ty+1;
		int W = tx-1;
		int E = tx+1;

		float cc,cn,cs,cw,ce;

		cc = (float) c[ty][tx];

		cn  = cc;
    cs  = (float) c[S][tx];
    cw  = cc;
    ce  = (float) c[ty][E];

		float jc,n,s,we,e;
		jc = (float) w[ty][tx];
		n  = (float) w[N][tx] - jc;
		s  = (float) w[S][tx] - jc;
		we = (float) w[ty][W]  - jc;
		e  = (float) w[ty][E] - jc;

		// divergence (equ 58)
		float d_D = cn*n +cs*s + ce*e + cw*we;
		// image update (equ 61)
		// w[ty][tx];
		w[ty][tx] =  w[ty][tx] + 0.25 * lambda * d_D;

		if(w.start_y == 0)
			w[N][tx] = w[ty][tx];
		if(w.start_y == rows - w.window_dim_y)
			w[S][tx] = w[ty][tx];
		if(w.start_x == 0)
			w[ty][W] = w[ty][tx];
		if(w.start_x == cols - w.window_dim_x)
			w[ty][E] = w[ty][tx];
		if(w.start_y == 0 && w.start_x == 0)
			w[N][W] = w[ty][tx];
		if(w.start_y == rows - w.window_dim_y && w.start_x == cols - w.window_dim_x)
			w[S][E] = w[ty][tx];
		if(w.start_x == 0 && w.start_y == rows - w.window_dim_y)
			w[S][W] = w[ty][tx];
		if(w.start_x == cols - w.window_dim_x && w.start_y == 0)
			w[N][E] = w[ty][tx];

		// printf("%f\n", (int) w[ty][tx]);
		return 0.0f;
	}

};
