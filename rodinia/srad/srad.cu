#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "srad.h"
// includes, project
#include <hip/hip_runtime.h>

// includes, kernels
#include "srad_kernel.cu"

void random_matrix(float* I, int rows, int cols);
void runTest( int argc, char** argv);
void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <rows> <cols> <y1> <y2> <x1> <x2> <lamda> <no. of iter>\n", argv[0]);
	fprintf(stderr, "\t<rows>   - number of rows\n");
	fprintf(stderr, "\t<cols>    - number of cols\n");
	fprintf(stderr, "\t<y1> 	 - y1 value of the speckle\n");
	fprintf(stderr, "\t<y2>      - y2 value of the speckle\n");
	fprintf(stderr, "\t<x1>       - x1 value of the speckle\n");
	fprintf(stderr, "\t<x2>       - x2 value of the speckle\n");
	fprintf(stderr, "\t<lamda>   - lambda (0,1)\n");
	fprintf(stderr, "\t<no. of iter>   - number of iterations\n");

	exit(1);
}
//
void writeoutput(thrust::host_vector<float> vect, int grid_rows, int grid_cols, char *file){
	int i,j, index=0;
	FILE *fp;
	char str[STR_SIZE];

	if( (fp = fopen(file, "w" )) == 0 )
        printf( "The file was not opened\n" );


	for (i=0; i < grid_rows; i++)
	{
	 for (j=0; j < grid_cols; j++)
	 {
		sprintf(str, "%g ", vect[i*grid_cols+j]);
		// printf("%g ",vect[i*grid_cols+j]);
		fputs(str,fp);
		index++;
	 }
	 sprintf(str, "\n");
	//  printf("\n");
	 fputs(str,fp);
 }

      fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv)
{
	time_t t;
	srand((unsigned) time(&t));
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);
  runTest( argc, argv);

  return EXIT_SUCCESS;
}


void
runTest( int argc, char** argv)
{
  unsigned int rows, cols, size_I, size_R, niter = 10, iter;
  float *J,*I,lambda, q0sqr, sum, sum2, tmp, meanROI,varROI ;
	unsigned int r1, r2, c1, c2;

	if (argc == 9)
	{
		rows = atoi(argv[1]);  //number of rows in the domain
		cols = atoi(argv[2]);  //number of cols in the domain
		if ((rows%16!=0) || (cols%16!=0)){
		fprintf(stderr, "rows and cols must be multiples of 16\n");
		exit(1);
		}
		r1   = atoi(argv[3]);  //y1 position of the speckle
		r2   = atoi(argv[4]);  //y2 position of the speckle
		c1   = atoi(argv[5]);  //x1 position of the speckle
		c2   = atoi(argv[6]);  //x2 position of the speckle
		lambda = atof(argv[7]); //Lambda value
		niter = atoi(argv[8]); //number of iterations

	}
  else
	{
		usage(argc, argv);
  }



	size_I = cols * rows;
  size_R = (r2-r1+1)*(c2-c1+1);

	J = (float *) malloc(size_I*sizeof(float));
	I = (float *) malloc(size_I*sizeof(float));

	thrust::Block_2D<float> J_cuda (rows,cols);

	printf("Randomizing the input matrix\n");
	//Generate a random matrix
	random_matrix(I, rows, cols);

  for (int k = 0;  k < size_I; k++ )
	{
   	J[k] = (float) exp(I[k]) ;
  }

	J_cuda.device_data.assign(J,J+size_I);

	writeoutput(J_cuda.device_data,rows,cols,"input.out");

	printf("Start the SRAD main loop\n");
 	for (iter=0; iter< niter; iter++)
	{
		printf("Iteration Started\n");
		sum=0; sum2=0;
    for (int i=r1; i<=r2; i++)
		{
        for (int j=c1; j<=c2; j++)
				{
					// printf("%f ", (float) J_cuda[i][j]);
          tmp   = J_cuda[i][j];
          sum  += tmp ;
          sum2 += tmp*tmp;
        }
    }
	  meanROI = sum / size_R;
	  varROI  = (sum2 / size_R) - meanROI*meanROI;
	  q0sqr   = varROI / (meanROI*meanROI);
		// SRADFunctor0 functor0;
		SRADFunctor1 functor1(cols,rows,q0sqr);
		SRADFunctor2 functor2(cols,rows,lambda,q0sqr);
		thrust::window_vector<float> wv = thrust::window_vector<float>(&(J_cuda),3,3,1,1);
		// thrust::for_each(wv.begin(),wv.end(),functor0);
		thrust::for_each(wv.begin(),wv.end(),functor1);
		hipDeviceSynchronize();
		thrust::for_each(wv.begin(),wv.end(),functor2);
		hipDeviceSynchronize();
		printf("Iteration Ended\n");
	}
	printf("Printing Output:\n");
  writeoutput(J_cuda.device_data,rows,cols,"result.out");
	printf("Computation Done\n");
	// hipDeviceReset();
}


void random_matrix(float *I, int rows, int cols){
	for( int i = 0 ; i < rows ; i++){
		for ( int j = 0 ; j < cols ; j++){
		 I[i * cols + j] = rand()/(float)RAND_MAX ;
		}
	}

}
