// includes, system
#include "srad.h"
#include "graphics.c"
#include "resize.c"

// includes, kernels
#include "srad_kernel.cu"

void random_matrix(float* I, int rows, int cols);
void runTest( int argc, char** argv);
void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <rows> <cols> <lamda> <no. of iter>\n", argv[0]);
	fprintf(stderr, "\t<rows>   - number of rows\n");
	fprintf(stderr, "\t<cols>    - number of cols\n");
	fprintf(stderr, "\t<lamda>   - lambda (0,1)\n");
	fprintf(stderr, "\t<no. of iter>   - number of iterations\n");
	fprintf(stderr, "\t<input_file>   - input file\n");
	fprintf(stderr, "\t<output file>   - output file\n");

	exit(1);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv)
{
	time_t t;
	srand((unsigned) time(&t));

  runTest( argc, argv);
  return EXIT_SUCCESS;
}


void
runTest( int argc, char** argv)
{
  unsigned int rows, cols, size_I, size_R, niter = 10, iter;
  float *J,lambda, q0sqr, sum, sum2,meanROI,varROI ;
	unsigned int r1, r2, c1, c2;
	char *in,*out;
	if (argc == 7)
	{
		rows = atoi(argv[1]);  //number of rows in the domain
		cols = atoi(argv[2]);  //number of cols in the domain
		lambda = atof(argv[3]); //Lambda value
		niter = atoi(argv[4]); //number of iterations
		in = argv[5];
		out = argv[6];
	}
  else
	{
		usage(argc, argv);
  }

	r1 = 0;
	r2 = rows - 1;
	c1 = 0;
	c2 = cols - 1;

	size_R = (r2-r1+1)*(c2-c1+1);

	int image_ori_rows = rows;
	int image_ori_cols = cols;
	// long image_ori_elem = image_ori_rows * image_ori_cols;

	size_I = cols * rows;

	J = (float*) malloc(sizeof(float) * size_I);

	read_graphics(in,J,image_ori_rows,image_ori_cols,0);

	// resize(	image_ori,image_ori_rows,image_ori_cols,J,rows,cols,0);

	thrust::Block_2D<float> J_cuda (cols,rows);
	thrust::Block_2D<float> J_square(cols,rows);
	thrust::Block_2D<float> d_c(cols,rows);
	thrust::Block_2D<float> nullBlock(cols,rows);
	thrust::fill(d_c.begin(),d_c.end(),0);
	J_cuda.device_data.assign(J,J+size_I);
	thrust::for_each(J_cuda.begin(),J_cuda.end(),extractFunctor());
	printf("Start the SRAD main loop\n");
 	for (iter=0; iter< niter; iter++)
	{
		J_square.copy(J_cuda.begin(),J_cuda.end());
		thrust::for_each(J_square.begin(),J_square.end(),square());
		sum = thrust::reduce(J_cuda.begin(),J_cuda.end());
		sum2 = thrust::reduce(J_square.begin(),J_square.end());
	  meanROI = sum / size_R;
	  varROI  = (sum2 / size_R) - meanROI*meanROI;
	  q0sqr   = varROI / (meanROI*meanROI);
		SRADFunctor1 functor1(cols,rows,q0sqr);
		SRADFunctor2 functor2(cols,rows,lambda,q0sqr);
		thrust::window_vector<float> wv = thrust::window_vector<float>(&(J_cuda),3,3,1,1);
		thrust::window_vector<float> d_cwv = thrust::window_vector<float>(&(d_c),3,3,1,1);
		thrust::transform(wv.begin(),wv.end(),d_cwv.begin(),nullBlock.begin(),functor1);
		thrust::transform(wv.begin(),wv.end(),d_cwv.begin(),nullBlock.begin(),functor2);
	}
	printf("Computation Done\n");
	thrust::for_each(J_cuda.begin(),J_cuda.end(),compressFunctor());
	hipMemcpy(J,thrust::raw_pointer_cast(J_cuda.device_data.data()),size_I*sizeof(float),hipMemcpyDeviceToHost);
	write_graphics(out,J,rows,cols,0,255);
}
