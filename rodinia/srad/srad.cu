// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "srad.h"
#include "graphics.c"
#include "resize.c"
// includes, project
#include <hip/hip_runtime.h>

// includes, kernels
#include "srad_kernel.cu"

void random_matrix(float* I, int rows, int cols);
void runTest( int argc, char** argv);
void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <rows> <cols> <lamda> <no. of iter>\n", argv[0]);
	fprintf(stderr, "\t<rows>   - number of rows\n");
	fprintf(stderr, "\t<cols>    - number of cols\n");
	fprintf(stderr, "\t<lamda>   - lambda (0,1)\n");
	fprintf(stderr, "\t<no. of iter>   - number of iterations\n");

	exit(1);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv)
{
	time_t t;
	srand((unsigned) time(&t));
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);
  runTest( argc, argv);
  return EXIT_SUCCESS;
}


void
runTest( int argc, char** argv)
{
  unsigned int rows, cols, size_I, size_R, niter = 10, iter;
  float *J,*image_ori,lambda, q0sqr, sum, sum2, tmp, meanROI,varROI ;
	unsigned int r1, r2, c1, c2;

	if (argc == 5)
	{
		rows = atoi(argv[1]);  //number of rows in the domain
		cols = atoi(argv[2]);  //number of cols in the domain
		lambda = atof(argv[3]); //Lambda value
		niter = atoi(argv[4]); //number of iterations

	}
  else
	{
		usage(argc, argv);
  }

	r1 = 0;
	r2 = rows - 1;
	c1 = 0;
	c2 = cols - 1;

	size_R = (r2-r1+1)*(c2-c1+1);

	int image_ori_rows = 502;
	int image_ori_cols = 458;
	long image_ori_elem = image_ori_rows * image_ori_cols;

	image_ori = (float*)malloc(sizeof(float) * image_ori_elem);


	size_I = cols * rows;

	J = (float*) malloc(sizeof(float) * size_I);

	read_graphics("image.pgm",J,image_ori_rows,image_ori_cols,0);

	// resize(	image_ori,image_ori_rows,image_ori_cols,J,rows,cols,0);

	thrust::Block_2D<float> J_cuda (cols,rows);

	J_cuda.device_data.assign(J,J+size_I);

	printf("Start the SRAD main loop\n");
 	for (iter=0; iter< niter; iter++)
	{
		printf("Iteration Started\n");
		sum=0; sum2=0;
    for (int i=r1; i<=r2; i++)
		{
        for (int j=c1; j<=c2; j++)
				{
					// printf("%f ", (float) J_cuda[i][j]);
          tmp   = J[i*cols+j];
          sum  += tmp ;
          sum2 += tmp*tmp;
        }
    }
	  meanROI = sum / size_R;
	  varROI  = (sum2 / size_R) - meanROI*meanROI;
	  q0sqr   = varROI / (meanROI*meanROI);
		// SRADFunctor0 functor0;
		SRADFunctor1 functor1(cols,rows,q0sqr,lambda);
		// SRADFunctor2 functor2(cols,rows,lambda,q0sqr);
		thrust::window_vector<float> wv = thrust::window_vector<float>(&(J_cuda),3,3,1,1);
		// thrust::for_each(wv.begin(),wv.end(),functor0);
		thrust::for_each(wv.begin(),wv.end(),functor1);
		// hipDeviceSynchronize();
		// thrust::for_each(wv.begin(),wv.end(),functor2);
		// hipDeviceSynchronize();
		printf("Iteration Ended\n");
	}
	printf("Computation Done\n");
	hipMemcpy(J,thrust::raw_pointer_cast(J_cuda.device_data.data()),size_I,hipMemcpyDeviceToHost);
	write_graphics(	"image_out.pgm",J,rows,cols,0,255);
	// hipDeviceReset();
}
