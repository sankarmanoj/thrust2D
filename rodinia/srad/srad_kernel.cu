#include "hip/hip_runtime.h"
#include "srad.h"
#include <stdio.h>

class compressFunctor
{
public:
	__device__ void operator() (float &x)
	{
		float y  = 255*log(x);
		x=y;
	}
};
class extractFunctor
{
public:
	__device__ void operator() (float &x)
	{
		x = exp((float)x/255);
	}
};

class square
{

public:
	__host__ __device__ float operator() (float &lhs)
	{
		return lhs*lhs;
	}
};
class SRADFunctor1
{
public:
	int cols;
	int rows;
	float q0sqr;

	SRADFunctor1 (int cols, int rows, float q0sqr)
	{
		this->cols = cols;
		this->rows = rows;
		this->q0sqr = q0sqr;
	}
	__device__ float operator() (thrust::window_2d<float> &w, thrust::window_2d<float> &v)
	{
		int ty = w.window_dim_y/2;
		int tx = w.window_dim_x/2;
		int N = ty-1;
		int S = ty+1;
		int W = tx-1;
		int E = tx+1;
		float jc,n,s,we,e,g2,l,num,den,qsqr,c;
		jc = (float) w[ty][tx];
		n  = (float) w[N][tx] - jc;
		s  = (float) w[S][tx] - jc;
		we = (float) w[ty][W]  - jc;
		e  = (float) w[ty][E] - jc;
		g2 = ( n * n + s * s + we * we + e * e ) / (jc * jc);
		l = ( n + s + we + e ) / jc;
		num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
		den  = 1 + (.25*l);
		qsqr = num/(den*den);
		// printf("%f\n",qsqr);
		// diffusion coefficent (equ 33)
		den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
		c = 1.0 / (1.0+den) ;
		// saturate diffusion coefficent
		if(c<0)
		{
			c=0;
		}
		else if(c>1)
		{
			c=1;
		}
		v[ty][tx] = c;
		return 0.0f;
	}
};
class printFunctor
{
public:
	__device__ void operator() (float &x) const
	{
		printf(" %f \n",x);
	}
};

class SRADFunctor2
{
public:
	int cols;
	int rows;
	float lambda;
	float q0sqr;

	SRADFunctor2 (int cols,int rows,float lambda,float q0sqr)
	{
		this->cols = cols;
		this->rows = rows;
		this->lambda = lambda;
		this->q0sqr = q0sqr;
	}

	__device__ float operator() (thrust::window_2d<float> &c, thrust::window_2d<float> &w)
	{
		int ty = w.window_dim_y/2;
		int tx = w.window_dim_x/2;
		int N = ty-1;
		int S = ty+1;
		int W = tx-1;
		int E = tx+1;
		float cc,cn,cs,cw,ce;
		cc = (float) c[ty][tx];
		cn  = cc;
		cs  = (float) c[S][tx];
		cw  = cc;
		ce  = (float) c[ty][E];
		float jc,n,s,we,e;
		jc = (float) w[ty][tx];
		n  = (float) w[N][tx] - jc;
		s  = (float) w[S][tx] - jc;
		we = (float) w[ty][W]  - jc;
		e  = (float) w[ty][E] - jc;
		// divergence (equ 58)
		float d_D = cn*n +cs*s + ce*e + cw*we;
		// image update (equ 61)
		// w[ty][tx];
		w[ty][tx] = (float) w[ty][tx] + 0.25 * lambda * d_D;
		return 0.0f;
	}
};
