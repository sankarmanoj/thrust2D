#include "hip/hip_runtime.h"
#include "srad.h"
#include <stdio.h>

class compressFunctor
{
public:
	__device__ void operator() (float &x) const
	{
		float y  = 255*log(x);
		// printf("%f ,, %f \n",y,x);
		x=y;
	}
};
class extractFunctor
{
public:
	__device__ void operator() (float &x) const
	{

		x = exp((float)x/255);

	}
};

class square
{

public:
 __host__ __device__ void operator() (float &lhs) const
  {
		lhs = lhs*lhs;
	}
};
class SRADFunctor1
{
public:
	int cols;
	int rows;
	float q0sqr;

	SRADFunctor1 (int cols, int rows, float q0sqr)
	{
		this->cols = cols;
		this->rows = rows;
		this->q0sqr = q0sqr;
	}
	__device__ float operator() (const thrust::window_2D<float> &w,const thrust::window_2D<float> &v) const
	{
		int ty = w.window_dim_y/2;
		int tx = w.window_dim_x/2;
		int N = ty-1;
		int S = ty+1;
		int W = tx-1;
		int E = tx+1;

		float jc,n,s,we,e,g2,l,num,den,qsqr,c;
		jc = (float) w[ty][tx];
		n  = (float) w[N][tx] - jc;
    s  = (float) w[S][tx] - jc;
    we = (float) w[ty][W]  - jc;
    e  = (float) w[ty][E] - jc;


    g2 = ( n * n + s * s + we * we + e * e ) / (jc * jc);

    l = ( n + s + we + e ) / jc;

		num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
		den  = 1 + (.25*l);
		qsqr = num/(den*den);
		// diffusion coefficent (equ 33)
		den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
		c = 1.0 / (1.0+den) ;


	  // saturate diffusion coefficent
		if(c<0)
		{
			c=0;
		}
		else if(c>1)
		{
			c=1;
		}
		v[ty][tx] = c;

		if(w.start_y == 0)
			w[N][tx] = w[ty][tx];
		if(w.start_y == rows - w.window_dim_y)
			w[S][tx] = w[ty][tx];
		if(w.start_x == 0)
			w[ty][W] = w[ty][tx];
		if(w.start_x == cols - w.window_dim_x)
			w[ty][E] = w[ty][tx];
		if(w.start_y == 0 && w.start_x == 0)
			w[N][W] = w[ty][tx];
		if(w.start_y == rows - w.window_dim_y && w.start_x == cols - w.window_dim_x)
			w[S][E] = w[ty][tx];
		if(w.start_x == 0 && w.start_y == rows - w.window_dim_y)
			w[S][W] = w[ty][tx];
		if(w.start_x == cols - w.window_dim_x && w.start_y == 0)
			w[N][E] = w[ty][tx];

		return 0.0;

	}

};
class printFunctor
{
public:
	__device__ void operator() (const float &x) const
	{
		printf(" %f \n",x);
	}
};

class SRADFunctor2
{
public:
	int cols;
	int rows;
	float lambda;
	float q0sqr;

	SRADFunctor2 (int cols,int rows,float lambda,float q0sqr)
	{
		this->cols = cols;
		this->rows = rows;
		this->lambda = lambda;
		this->q0sqr = q0sqr;
	}

	__device__ float operator() (const thrust::window_2D<float> &w, const thrust::window_2D<float> &c) const
	{
		int ty = w.window_dim_y/2;
		int tx = w.window_dim_x/2;
		int N = ty-1;
		int S = ty+1;
		int W = tx-1;
		int E = tx+1;

		float cc,cn,cs,cw,ce;

		cc = (float) c[ty][tx];

		cn  = cc;
    cs  = (float) c[S][tx];
    cw  = cc;
    ce  = (float) c[ty][E];

		float jc,n,s,we,e;
		jc = (float) w[ty][tx];
		n  = (float) w[N][tx] - jc;
		s  = (float) w[S][tx] - jc;
		we = (float) w[ty][W]  - jc;
		e  = (float) w[ty][E] - jc;

		// divergence (equ 58)
		float d_D = cn*n +cs*s + ce*e + cw*we;
		// image update (equ 61)
		// w[ty][tx];
		w[ty][tx] = (float) w[ty][tx] + 0.25 * lambda * d_D;

		if(w.start_y == 0)
			w[N][tx] = w[ty][tx];
		if(w.start_y == rows - w.window_dim_y)
			w[S][tx] = w[ty][tx];
		if(w.start_x == 0)
			w[ty][W] = w[ty][tx];
		if(w.start_x == cols - w.window_dim_x)
			w[ty][E] = w[ty][tx];
		if(w.start_y == 0 && w.start_x == 0)
			w[N][W] = w[ty][tx];
		if(w.start_y == rows - w.window_dim_y && w.start_x == cols - w.window_dim_x)
			w[S][E] = w[ty][tx];
		if(w.start_x == 0 && w.start_y == rows - w.window_dim_y)
			w[S][W] = w[ty][tx];
		if(w.start_x == cols - w.window_dim_x && w.start_y == 0)
			w[N][E] = w[ty][tx];

		// printf("%f\n", (float) w[ty][tx]);
		return 0.0f;
	}

};
