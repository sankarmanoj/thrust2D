#include "srad.h"
#include <stdio.h>

class SRADFunctor0
{
public:
	int cols;
	int rows;
	float q0sqr;

	SRADFunctor0 ()
	{
	}

	__device__ void operator() (thrust::window_2D<float> w)
	{
		printf("%f\n", (float) w[0][0]);
	}

};

class SRADFunctor1
{
public:
	int cols;
	int rows;
	float q0sqr;

	SRADFunctor1 (int cols, int rows, float q0sqr)
	{
		this->cols = cols;
		this->rows = rows;
		this->q0sqr = q0sqr;
	}

	__device__ void operator() (thrust::window_2D<float> w)
	{
		int ty = w.window_dim_y/2;
		int tx = w.window_dim_x/2;
		// int rty = w.start_y + ty;
		// int rtx = w.start_x + tx;
		int W = ty-1;
		int E = ty+1;
		int N = tx-1;
		int S = tx+1;

		// printf("functor %d %d \n",w.start_x,w.start_y);
		// printf("%f\n", (float) w[tx][ty]);

		float jc,n,s,we,e,g2,l,num,den,qsqr,c;
		jc = (float) w[tx][ty];
		n  = (float) w[N][ty] - jc;
    s  = (float) w[S][ty] - jc;
    we = (float) w[tx][W]  - jc;
    e  = (float) w[tx][E] - jc;


    g2 = ( n * n + s * s + we * we + e * e ) / (jc * jc);

    l = ( n + s + we + e ) / jc;

		num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
		den  = 1 + (.25*l);
		qsqr = num/(den*den);
		// diffusion coefficent (equ 33)
		den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
		c = 1.0 / (1.0+den) ;

	  // saturate diffusion coefficent
		if (c < 0){w[tx][ty] = 0;}
		else if (c > 1) {w[tx][ty] = 1;}
		else {w[tx][ty] = c;}

		// printf("%f\n", (float) w[tx][ty]);
	}

};

class SRADFunctor2
{
public:
	int cols;
	int rows;
	float lambda;
	float q0sqr;

	SRADFunctor2 (int cols,int rows,float lambda,float q0sqr)
	{
		this->cols = cols;
		this->rows = rows;
		this->lambda = lambda;
		this->q0sqr = q0sqr;
	}

	__device__ void operator() (thrust::window_2D<float> w)
	{
		// printf("functor2\n");
		int ty = w.window_dim_y/2;
		int tx = w.window_dim_x/2;
		// int rty = w.start_y + ty;
		// int rtx = w.start_x + tx;
		int W = ty-1;
		int E = ty+1;
		int N = tx-1;
		int S = tx+1;

		float cc,cn,cs,cw,ce,d_sum;

		cc = (float) w[tx][ty];

		cn  = cc;
    cs  = (float) w[S][ty];
    cw  = cc;
    ce  = (float) w[tx][E];

		// divergence (equ 58)
		d_sum = cn * ((float) w[N][ty]) + cs * ((float) w[S][ty]) + cw * ((float) w[tx][W]) + ce * ((float) w[tx][E]);

		// image update (equ 61)
		w[tx][ty] = (float) w[tx][ty] + 0.25 * lambda * d_sum;
		// printf("%f\n", (float) w[tx][ty]);
	}

};
