#include "srad.h"
#include <stdio.h>

class SRADFunctor0
{
public:
	int cols;
	int rows;
	float q0sqr;

	SRADFunctor0 ()
	{
	}

	__device__ void operator() (thrust::window_2D<float> &w)
	{
		printf("%f\n", (float) w[0][0]);
	}

};

class SRADFunctor1
{
public:
	int cols;
	int rows;
	float q0sqr;

	SRADFunctor1 (int cols, int rows, float q0sqr)
	{
		this->cols = cols;
		this->rows = rows;
		this->q0sqr = q0sqr;
	}
	// __device__ float operator() (float x, float y)
	// {
	// 	return x + y;
	// }
	__device__ float operator() (thrust::window_2D<float> &w,thrust::window_2D<float> &yolo)
	{
		int ty = w.window_dim_y/2;
		int tx = w.window_dim_x/2;
		// int rty = w.start_y + ty;
		// int rtx = w.start_x + tx;
		int S = ty-1;
		int N = ty+1;
		int W = tx-1;
		int E = tx+1;

		// printf("functor %d %d \n",w.start_x,w.start_y);
		// printf("%f\n", (float) w[ty][tx]);

		float jc,n,s,we,e,g2,l,num,den,qsqr,c;
		jc = (float) w[ty][tx];
		n  = (float) w[N][tx] - jc;
    s  = (float) w[S][tx] - jc;
    we = (float) w[ty][W]  - jc;
    e  = (float) w[ty][E] - jc;


    g2 = ( n * n + s * s + we * we + e * e ) / (jc * jc);

    l = ( n + s + we + e ) / jc;

		num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
		den  = 1 + (.25*l);
		qsqr = num/(den*den);
		// diffusion coefficent (equ 33)
		den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
		c = 1.0 / (1.0+den) ;


	  // saturate diffusion coefficent
		if(c<0)
		{
			c=0;
		}
		else if(c>1)
		{
			c=1;
		}
		yolo[ty][tx] = c;
		return 0.0;

		// printf("%f\n", (float) w[ty][tx]);
	}

};

class SRADFunctor2
{
public:
	int cols;
	int rows;
	float lambda;
	float q0sqr;

	SRADFunctor2 (int cols,int rows,float lambda,float q0sqr)
	{
		this->cols = cols;
		this->rows = rows;
		this->lambda = lambda;
		this->q0sqr = q0sqr;
	}

	__device__ void operator() (thrust::window_2D<float> w)
	{
		// printf("functor2\n");
		int ty = w.window_dim_y/2;
		int tx = w.window_dim_x/2;
		// int rty = w.start_y + ty;
		// int rtx = w.start_x + tx;
		int S = ty-1;
		int N = ty+1;
		int W = tx-1;
		int E = tx+1;

		float cc,cn,cs,cw,ce,d_sum;

		cc = (float) w[ty][tx];

		cn  = cc;
    cs  = (float) w[S][tx];
    cw  = cc;
    ce  = (float) w[ty][E];
		float jc,n,s,we,e,g2,l,num,den,qsqr,c;
		jc = (float) w[ty][tx];
		n  = (float) w[N][tx] - jc;
		s  = (float) w[S][tx] - jc;
		we = (float) w[ty][W]  - jc;
		e  = (float) w[ty][E] - jc;

		// divergence (equ 58)
		d_sum = cn * ((float) w[N][tx]) + cs * ((float) w[S][tx]) + cw * ((float) w[ty][W]) + ce * ((float) w[ty][E]);

		// image update (equ 61)
		w[ty][tx] = (float) w[ty][tx] + 0.25 * lambda * d_sum;
		// printf("%f\n", (float) w[ty][tx]);
	}

};
