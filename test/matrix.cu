#include "hip/hip_runtime.h"
#include <thrust/window_transform.h>
int main()
{
  thrust::Block_2D<int> a(3,5,1),b(5,3,1),c(5,5,0);
  // thrust::transpose(&b);
  c = thrust::matrix_multiply(&a,&b);
  for (int i=0; i<5; i++)
  {
    for (int j=0; j<3; j++)
    {
      printf("%d ",a[make_int2(j,i)]);
    }
    printf("\n");
  }
  for (int i=0; i<3; i++)
  {
    for (int j=0; j<5; j++)
    {
      printf("%d ",b[make_int2(j,i)]);
    }
    printf("\n");
  }
  for (int i=0; i<5; i++)
  {
    for (int j=0; j<5; j++)
    {
      printf("%d ",c[make_int2(j,i)]);
    }
    printf("\n");
  }
  return 0;
}
