#include "hip/hip_runtime.h"
#include <thrust/block_2d.h>
#include <thrust/matrix_operations.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main()
{
  thrust::block_2d<float> a(3,3,1),b(3,3,1);
  // thrust::transpose(&b);
  // c = thrust::matrix_multiply(&a,&b);
  hipblasHandle_t handle;
  hipblasStatus_t status;
  hipblasCreate(&handle);
  thrust::sequence(a.begin(),a.end());

  thrust::host_block_2d<float> host_a(3,5,1);
  host_a = b;

  for (int i=0; i<5; i++)
  {
    for (int j=0; j<3; j++)
    {
      printf("%f ",host_a[make_int2(j,i)]);
    }
    printf("\n");
  }
  // for (int i=0; i<3; i++)
  // {
  //   for (int j=0; j<5; j++)
  //   {
  //     printf("%d ",b[make_int2(j,i)]);
  //   }
  //   printf("\n");
  // }
  // for (int i=0; i<5; i++)
  // {
  //   for (int j=0; j<5; j++)
  //   {
  //     printf("%d ",c[make_int2(j,i)]);
  //   }
  //   printf("\n");
  // }
  return 0;
}
