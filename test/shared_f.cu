#include <thrust/device_vector.h>

// #include <thrust/for_each.h>
#include <thrust/sequence.h>
#include <thrust/scan.h>
#include <thrust/shared_for_each.h>
// #include <thrust/shared_reduce.h>
using namespace thrust;
class printFunctor
{
public:

__device__  void  operator() ( int  &a)
  {
    printf("%d ",a);
  }
};
class copyFunctor
{
public:

__device__  int  operator() ( int  &a)
  {
    return a*a;
  }
};
class binaryFunctor
{
public:

__device__  int  operator() ( int  &a,int &b)
  {
    return a*b;
  }
};
int main()
{
  device_vector<int> a(12);
  device_vector<int> b(12);
  // device_vector<int> c(1250);
  //
  sequence(a.begin(),a.end());
  // sequence(b.begin(),b.end());
  // printf("%d ",reduce(cuda::shared,a.begin(),a.end()));
  // printf("%d ",reduce(a.begin(),a.end()));
  exclusive_scan(cuda::shared,a.begin(),a.end(),b.begin());
  hipDeviceSynchronize();
  for_each(a.begin(),a.end(),printFunctor());
  hipDeviceSynchronize();
  printf("\n");
  for_each(b.begin(),b.end(),printFunctor());
  hipDeviceSynchronize();
  printf("\n");
}
