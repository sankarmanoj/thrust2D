// #include <thrust/device_vector.h>

// #include <thrust/for_each.h>
#include <thrust/sequence.h>
#include <thrust/constant_memory.h>
#include <thrust/scan.h>
#include <thrust/transform_reduce.h>
#include <thrust/shared_algorithms.h>
// #include <thrust/shared_reduce.h>
using namespace thrust;
class printFunctor
{
public:
__device__  void  operator() ( float &a)
  {
    printf("%f\n",a);
  }
};
class copyFunctor
{
public:

__device__  float  operator() ( float  &a)
  {
    return a*a;
  }
};
class binaryFunctor
{
public:

__device__  float  operator() ( float  &a,float &b)
  {
    return a*b;
  }
};
int main(int argc, char ** argv)
{
  if(argc!=2)
    exit(0);
  device_vector<float> a(atoi(argv[1]));
  device_vector<float> b(atoi(argv[1]));
  device_vector<float> c(atoi(argv[1]));
  // device_vector<int> c(1200);
  //
  sequence(a.begin(),a.end());
  sequence(b.begin(),b.end());

  // printf("%d ",reduce(cuda::shared,a.begin(),a.end()));
  // printf("%d ",reduce(a.begin(),a.end()));
  // exclusive_scan(cuda::shared,a.begin(),a.end(),b.begin());
  // hipDeviceSynchronize();
  transform(cuda::shared_first,a.begin(),a.end(),b.begin(),c.begin(),binaryFunctor());

  // hipDeviceSynchronize();
  // printf("\n");
  for_each(cuda::shared,c.begin(),c.end(),printFunctor());
  // hipDeviceSynchronize();
  // printf("\n");

  // printf("Thrust = %f\n",transform_reduce(a.begin(),a.end(),copyFunctor(),0.0f, thrust::plus<float>()));
  // printf("Shared = %f\n",transform_reduce(cuda::shared,a.begin(),a.end(),b.begin(),binaryFunctor()));
  //   printf("Shared = %f \n",reduce(cuda::shared,a.begin(),a.end()-10));
}
