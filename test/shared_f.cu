// #include <thrust/device_vector.h>

// #include <thrust/for_each.h>
#include <thrust/sequence.h>
#include <thrust/constant_memory.h>
#include <thrust/scan.h>
#include <thrust/transform_reduce.h>
#include <thrust/shared_for_each.h>
// #include <thrust/shared_reduce.h>
using namespace thrust;
class printFunctor
{
public:
  float * data;
printFunctor(float * data)
{
  this->data = data;
}
__device__  void  operator() ( float &a)
  {
    printf("%f\n",data[(int)a]);
  }
};
class copyFunctor
{
public:

__device__  float  operator() ( float  &a)
  {
    return a*a;
  }
};
class binaryFunctor
{
public:

__device__  float  operator() ( float  &a,float &b)
  {
    return a*b;
  }
};
int main(int argc, char ** argv)
{
  if(argc!=2)
    exit(0);
  device_vector<float> a(atoi(argv[1]));
  device_vector<float> b(atoi(argv[1]));
  device_vector<float> c(atoi(argv[1]));
  // device_vector<int> c(1200);
  //
  sequence(a.begin(),a.end());
  sequence(b.begin(),b.end());

  float * c_a = get_constant_memory_pointer(a.begin(),a.end());
  // printf("%d ",reduce(cuda::shared,a.begin(),a.end()));
  // printf("%d ",reduce(a.begin(),a.end()));
  // exclusive_scan(cuda::shared,a.begin(),a.end(),b.begin());
  // hipDeviceSynchronize();
  transform(cuda::shared,a.begin(),a.end(),b.begin(),copyFunctor());
  float * c_b = get_constant_memory_pointer(b.begin(),b.end());
  for_each(cuda::shared,a.begin(),a.end(),printFunctor(c_b));
  for_each(cuda::shared,a.begin(),a.end(),printFunctor(c_a));

  // hipDeviceSynchronize();
  // printf("\n");
  // for_each(cuda::shared,c.begin(),c.end(),printFunctor());
  // hipDeviceSynchronize();
  // printf("\n");

  // printf("Thrust = %f\n",transform_reduce(a.begin(),a.end(),copyFunctor(),0.0f, thrust::plus<float>()));
  // printf("Shared = %f\n",transform_reduce(cuda::shared,a.begin(),a.end(),b.begin(),binaryFunctor()));
  //   printf("Shared = %f \n",reduce(cuda::shared,a.begin(),a.end()-10));
}
