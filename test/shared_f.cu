// #include <thrust/device_vector.h>

// #include <thrust/for_each.h>
#include <thrust/sequence.h>
#include <thrust/scan.h>
#include <thrust/transform_reduce.h>
#include <thrust/shared_for_each.h>
// #include <thrust/shared_reduce.h>
using namespace thrust;
class printFunctor
{
public:

__device__  void  operator() ( int  &a)
  {
    printf("%d\n",a);
  }
};
class copyFunctor
{
public:

__device__  int  operator() ( int  &a)
  {
    return a*a;
  }
};
class binaryFunctor
{
public:

__device__  int  operator() ( int  &a,int &b)
  {
    return a*b;
  }
};
int main()
{
  device_vector<int> a(32);
  device_vector<int> b(32);
  // device_vector<int> c(1200);
  //
  sequence(a.begin(),a.end());
  sequence(b.begin(),b.end());
  // printf("%d ",reduce(cuda::shared,a.begin(),a.end()));
  // printf("%d ",reduce(a.begin(),a.end()));
  // exclusive_scan(cuda::shared,a.begin(),a.end(),b.begin());
  // hipDeviceSynchronize();
  // for_each(cuda::shared,a.begin(),a.end(),printFunctor());
  // transform(cuda::shared,a.begin(),a.end(),b.begin(),c.begin(),binaryFunctor());
  // hipDeviceSynchronize();
  // printf("\n");
  // for_each(cuda::shared,c.begin(),c.end(),printFunctor());
  // hipDeviceSynchronize();
  // printf("\n");
  printf("Thrust = %d\n",transform_reduce(a.begin(),a.end(),copyFunctor(),0, thrust::plus<int>()));
  printf("Shared = %d\n",transform_reduce(cuda::shared,a.begin(),a.end(),b.begin(),binaryFunctor()));
}
