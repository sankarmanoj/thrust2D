// #include <thrust/device_vector.h>

// #include <thrust/for_each.h>
#include <thrust/sequence.h>
#include <thrust/scan.h>
#include <thrust/shared_for_each.h>
// #include <thrust/shared_reduce.h>
using namespace thrust;
class printFunctor
{
public:

__device__  void  operator() ( int  &a)
  {
    printf("%d\n",a);
  }
};
class copyFunctor
{
public:

__device__  int  operator() ( int  &a)
  {
    return a*a;
  }
};
class binaryFunctor
{
public:

__device__  int  operator() ( int  &a,int &b)
  {
    return a*b;
  }
};
int main()
{
  device_vector<int> a(1200);
  device_vector<int> b(1200);
  device_vector<int> c(1200);
  //
  sequence(a.begin(),a.end());
  sequence(b.begin(),b.end());
  // printf("%d ",reduce(cuda::shared,a.begin(),a.end()));
  // printf("%d ",reduce(a.begin(),a.end()));
  // exclusive_scan(cuda::shared,a.begin(),a.end(),b.begin());
  // hipDeviceSynchronize();
  // for_each(cuda::shared,a.begin(),a.end(),printFunctor());
  transform(cuda::shared,a.begin(),a.end(),b.begin(),c.begin(),binaryFunctor());
  hipDeviceSynchronize();
  printf("\n");
  for_each(cuda::shared,c.begin(),c.end(),printFunctor());
  hipDeviceSynchronize();
  // printf("\n");
}
