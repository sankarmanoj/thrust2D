#include "hip/hip_runtime.h"
#include <thrust/window_transform.h>
#include <thrust/generate.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
int main(int argc, char** argv)
{
    int x;
    x = atoi(argv[1]);
    thrust::block_2d<float> inBlock(x,x);
    thrust::sequence(inBlock.begin(),inBlock.end());
    for (int i=0; i<x;i++)
    {
      for (int j=0;j<x  ;j++)
      {
          std::cout<<inBlock[make_int2(j,i)]<<" ";
      }
      std::cout<<"\n";
    }
    float kernel[3] = {0.25,1,0.25};
    thrust::convolve(thrust::cuda::texture,&inBlock,kernel);
    for (int i=0; i<x;i++)
    {
      for (int j=0;j<x;j++)
      {
            std::cout<<inBlock[make_int2(j,i)]<<" ";
      }
      std::cout<<"\n";
    }
}
