#include "hip/hip_runtime.h"
#include<thrust/window_transform.h>
#include <thrust/generate.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
int main(int argc, char** argv)
{
    int x,y;
    x = atoi(argv[1]);
    y = atoi(argv[2]);
    srand(13);
    thrust::Block_2D<float> inBlock(x,x);
    thrust::Block_2D<float> kernel(y,y);
    thrust::device_vector<float> a((long long int)x*x);
    thrust::sequence(a.begin(),a.end());
    thrust::copy(a.begin(),a.end(),inBlock.begin());
    thrust::fill(kernel.begin(),kernel.end(),1.0);
    // for (int i=0; i<Y;i++)
    // {
    //   for (int j=0;j<X  ;j++)
    //   {
    //       std::cout<<inBlock[make_int2(j,i)]<<" ";
    //   }
    //   std::cout<<"\n";
    // }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    thrust::convolve(inBlock.begin(), inBlock.end(), kernel.begin());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f\n",milliseconds);
    // //
    // for (int i=0; i<Y;i++)
    // {
    //   for (int j=0;j<X;j++)
    //   {
    //         std::cout<<inBlock[make_int2(j,i)]<<" ";
    //   }
    //   std::cout<<"\n";
    // }
}
