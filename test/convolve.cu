#include "hip/hip_runtime.h"
#include<thrust/window_2d.h>
#include <thrust/generate.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#define X 10000
#define Y 500
int main()
{
  srand(13);
  thrust::Block_2D<int> inBlock(X,Y);
  thrust::Block_2D<int> kernel(7,7);
  thrust::device_vector<int> a(X*Y);
  thrust::sequence(a.begin(),a.end());
  thrust::copy(a.begin(),a.end(),inBlock.begin());
  thrust::fill(kernel.begin(),kernel.end(),1);
  // for (int i=0; i<Y;i++)
  // {
  //   for (int j=0;j<X  ;j++)
  //   {
  //       std::cout<<inBlock[make_int2(j,i)]<<" ";
  //   }
  //   std::cout<<"\n";
  // }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::convolve(&inBlock,&kernel);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time Taken = %f\n",milliseconds);
  // //
  // for (int i=0; i<Y;i++)
  // {
  //   for (int j=0;j<X;j++)
  //   {
  //         std::cout<<inBlock[make_int2(j,i)]<<" ";
  //   }
  //   std::cout<<"\n";
  // }
}
