#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <math.h>


#define PI 3.14159
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  float inverseSigmaSquare;
  inverseSigmaSquare = 1/pow(sigma,2);
  float pi2SigSquare;
  pi2SigSquare = inverseSigmaSquare*2*PI;
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }

  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
  printf("Total = %f,newTotal=%f\n",total,newTotal);
}
class transFormFunctor //: public thrust::shared_unary_window_transform_functor<float>
{
  public:


  __device__ float operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {
    float temp = 255;
    for(int i = 0; i<inputWindow.window_dim_y;i++)
    {
      for(int j = 0; j<inputWindow.window_dim_x;j++)
      {
        temp = min(temp,inputWindow[i][j]);
      }
    }
    outputWindow[inputWindow.window_dim_y/2][inputWindow.window_dim_x/2]=temp;
    return 0.0f;

  }
};
class forEachFunctor : public thrust::shared_window_for_each_functor<float>
{
  public:


  __device__ void operator() (const thrust::window_2d<float> &inputWindow) const
  {
    float temp = 255;
    for(int i = 0; i<inputWindow.window_dim_y;i++)
    {
      for(int j = 0; j<inputWindow.window_dim_x;j++)
      {
        temp = min(temp,inputWindow[i][j]);
      }
    }
    inputWindow[inputWindow.window_dim_y/2][inputWindow.window_dim_x/2]=threadIdx.x%255;

  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 17;
  image = small;
  resize(small,image,Size(1500,1500));
  thrust::block_2d<float> kernel(dim,dim);
  getGaussianKernelBlock(dim,5,kernel);
  // thrust::fill(kernel.begin(),kernel.end(),0.0f);
  //
  // for(int i = 0; i<dim;i++)
  // {
  //   for(int j = 0; j<dim;j++)
  //   {
  //     float x = (kernel)[i][j];
  //     printf("%f ",x);
  //   }
  //   printf("\n");
  // }

  std::cout<<dim<<"  "<<image.isContinuous()<<std::endl;
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image_block.end()-image_block.begin()));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  // image_block.assign(image.ptr(),image.ptr()+image.cols*image.rows);
  Mat cvGB;
  GaussianBlur(image,cvGB,Size(3,3),3);
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&float_image_block,9,9,1,1);
  thrust::window_vector<float> outputVector = thrust::window_vector<float>(&outBlock,9,9,1,1);
  // thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),outputVector.begin(),transFormFunctor());
  // thrust::transform(myVector.begin(),myVector.end(),outputVector.begin(),image_block.begin(),transFormFunctor());
  // thrust::for_each(thrust::cuda::shared,myVector.begin(),myVector.end(),forEachFunctor());
  thrust::convolve(float_image_block.begin(),float_image_block.end(),kernel.begin());
  // unsigned char * outputImageData = (unsigned char *)malloc(sizeof(unsigned char)*(image_block.end()-image_block.begin()));
  // hipMemcpy(outputImageData,thrust::raw_pointer_cast(image_block.data()),sizeof(unsigned char)*(image_block.end()-image_block.begin()),hipMemcpyDeviceToHost);

  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(float_image_block.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  // std::cout<<output;
  // cudaCheckError();
  // std::cout<<output.type()<<"  "<<Size(image.cols,image.rows)<<"="<<image_block.end()-image_block.begin()<<"\n";
  imwrite("input.png",image);
  // imshow("output",output);
  imwrite("output.png",output);

  waitKey(0);
  // std::cout<<float_image<<"\n";
  return 0;
}
