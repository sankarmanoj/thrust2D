#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
using namespace cv;
int main(int argc, char const *argv[]) {
  Mat image = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  std::cout<<image.type()<<"  "<<image.isContinuous()<<std::endl;
  thrust::block_2d<unsigned char > image_block (image.cols,image.rows);
  image_block.assign(image.ptr(),image.ptr()+image.cols*image.rows);
  unsigned char * outputImageData = (unsigned char *)malloc(sizeof(unsigned char)*(image_block.end()-image_block.begin()));
  hipMemcpy(outputImageData,thrust::raw_pointer_cast(image_block.data()),sizeof(unsigned char)*(image_block.end()-image_block.begin()),hipMemcpyDeviceToHost);
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputImageData);
  std::cout<<output.type()<<"  "<<Size(image.cols,image.rows)<<"="<<image_block.end()-image_block.begin()<<"\n";
  imshow("input",image);
  imshow("output",output);
  waitKey(0);
  // std::cout<<image<<"\n";
  return 0;
}
