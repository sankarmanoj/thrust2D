#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <iostream>
#include <typeinfo>
using namespace thrust;

// testing basic block functions
class windowPrintFunctor
{
public:

__device__  void operator() (const window_2D<int> &a) const
  {
    a[0][0] = 666;
    // int value = a[0][0];
    // printf("%d \n",value);
  }
};
class printFunctor
{
public:

__device__  void  operator() ( int  &a)
  {
    printf("%d \n",a);
    // return 10;
  }
};
class printFunctor2
{
public:

__device__  void operator() (const int  &a) const
  {
    printf("%d \n",a);
  }
};
int main()
{
  // int *temp;
  // temp = (int *) malloc(25 * sizeof(int));
  Block_2D<int> a1(5,5);
  Block_2D<int> b = a1;
  device_vector<int> a(5*5);
  device_vector<int> c(5*5);

  sequence(b.begin(),b.end());
  thrust::detail::normal_iterator<thrust::device_ptr<int> > adf = a.begin();
  std::cout<<typeid(thrust::detail::normal_iterator<thrust::device_ptr<int> >::reference).name()<<"\n";

  // copy(a.begin(),a.end(),b.begin());
  // for(int i = 0; i<25; i++)
  // {
  //   printf("%f\n",a[i/5][i%5]);
  // }
  // // hipDeviceSynchronize();

  // thrust::window_vector<int> wv = window_vector<int>(&(b),3,3,1,1);
  // printf("Start\n");
  // // thrust::window_iterator<int> wi = wv.end();
  // // int num = wv.begin().operator-(wv.end());
  // // printf("%d\n", num);
  // thrust::for_each(wv.begin(),wv.end(),windowPrintFunctor());
  // hipDeviceSynchronize();
  // // hipMemcpy(temp,thrust::raw_pointer_cast(b.data()),25*sizeof(int),hipMemcpyDeviceToHost);
  // // b.assign(temp,temp+25);
  thrust::for_each(b.begin(),b.end(),printFunctor());
  return 0;
}
