#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <iostream>
#include <typeinfo>
using namespace thrust;

// testing basic block functions
class windowPrintFunctor
{
public:

__device__  void operator() (const window_2D<int> &a) const
  {
    a[0][0]=3;
    int value = a[0][0];
    printf("%d \n",value);
  }
};
class printFunctor
{
public:

__device__  void operator() (const int  & a) const
  {
    printf("%d \n",a);
  }
};

int main()
{
  Block_2D<int> a1(5,5);
  Block_2D<int> b = a1;
  device_vector<int> a(5*5);
  sequence(a.begin(),a.end());
  copy(a.begin(),a.end(),b.begin());
  thrust::for_each(b.begin(),b.end(),printFunctor());
  hipDeviceSynchronize();
  thrust::window_vector<int> wv = window_vector<int>(&(b),3,3,1,1);
  printf("Start\n");
  // thrust::window_iterator<int> wi = wv.end();
  // int num = wv.begin().operator-(wv.end());
  // printf("%d\n", num);
  thrust::for_each(wv.begin(),wv.end(),windowPrintFunctor());
  hipDeviceSynchronize();
  thrust::for_each(b.begin(),b.end(),printFunctor());
  return 0;
}
