#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <thrust/block_2d.h>
using namespace thrust;
#define X 6
#define Y 6
class printFunctor
{
public:

__device__  void  operator() ( int  &a)
  {
    // a=threadIdx.x;
    printf("%d \n",a);
    // return 10;
  }
};
class printFunctor2
{
public:

__device__  __host__ int operator() (int  &a)
  {
    return 2*a;
  }
};
int main()
{
  block_2d<int> a(X,Y,99);
  // block_2d<int> b(X,Y);
  // printf("Last value = %d",(int)a[1][2]);
  sequence(a.begin(),a.end());

  // transform(a.begin(),a.end(),b.begin(),printFunctor2());
  for_each(a.begin(),a.end(),printFunctor());
  // host_block_2d<int> b(5,5);
  // sequence(b.begin(),b.end());
  // for_each(thrust::host,b.begin(),b.end(),printFunctor2());

  // copy(a.begin(),a.end(),b.begin());
  // for(int i = 0; i<25; i++)
  // {
  //   printf("%f\n",a[i/5][i%5]);
  // }
  hipDeviceSynchronize();
  int *b = (int *) malloc(X*Y);
  hipMemcpy2D(b,X*sizeof(int),a.data_pointer,a.pitch,X,Y,hipMemcpyDeviceToHost);
  for (int i=0; i<Y;i++)
  {
    for (int j=0;j<X;j++)
    {
      std::cout<<b[i*X+j]<< " ";
    }
    std::cout<<"\n";
  }

  // thrust::window_vector<int> wv = window_vector<int>(&(b),3,3,1,1);
  // printf("Start\n");
  // // thrust::window_iterator<int> wi = wv.end();
  // // int num = wv.begin().operator-(wv.end());
  // // printf("%d\n", num);
  // thrust::for_each(wv.begin(),wv.end(),windowPrintFunctor());
  // hipDeviceSynchronize();
  // // hipMemcpy(temp,thrust::raw_pointer_cast(b.data()),25*sizeof(int),hipMemcpyDeviceToHost);
  // // b.assign(temp,temp+25);
  // thrust::for_each(b.begin(),b.end(),printFunctor());
  return 0;
}
