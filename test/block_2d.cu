#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/block_2d.h>
using namespace thrust;


class printFunctor
{
public:

__device__  void  operator() ( int  &a)
  {
    printf("%d \n",a);
    // return 10;
  }
};
class printFunctor2
{
public:

__device__  __host__ void operator() (const int  &a) const
  {
    printf("%d \n",a);
  }
};
int main()
{
  block_2d<int> a(5,5);
  sequence(a.begin(),a.end());
  for_each(thrust::host,a.begin(),a.end(),printFunctor2());


  // copy(a.begin(),a.end(),b.begin());
  // for(int i = 0; i<25; i++)
  // {
  //   printf("%f\n",a[i/5][i%5]);
  // }
  // // hipDeviceSynchronize();

  // thrust::window_vector<int> wv = window_vector<int>(&(b),3,3,1,1);
  // printf("Start\n");
  // // thrust::window_iterator<int> wi = wv.end();
  // // int num = wv.begin().operator-(wv.end());
  // // printf("%d\n", num);
  // thrust::for_each(wv.begin(),wv.end(),windowPrintFunctor());
  // hipDeviceSynchronize();
  // // hipMemcpy(temp,thrust::raw_pointer_cast(b.data()),25*sizeof(int),hipMemcpyDeviceToHost);
  // // b.assign(temp,temp+25);
  // thrust::for_each(b.begin(),b.end(),printFunctor());
  return 0;
}
