#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/window_2d.h>
#define X 6
#define Y 6
using namespace thrust;
// testing window indexing
class printFunctor
{
public:
  __host__ __device__ void operator() (const window_2d<int> &myWindow) const
  {
    int value = myWindow[0][0];
    myWindow[0][0]=666;
    printf(" %ld , %ld , %d\n",myWindow.start_x, myWindow.start_y,value);
  }
};

class printFunctor2
{
public:
  __host__ __device__ int operator() (const window_2d<int> &myWindow) const
  {
    int value = myWindow[0][0];
    myWindow[0][0]=666;
    printf(" %ld , %ld , %d\n",myWindow.start_x, myWindow.start_y,value);
    return value;
  }
};

class printFunctor1
{
public:
  __device__ void operator() (const window_2d<int> &myWindow) const
  {
    int value = myWindow[0][0];

    myWindow[0][0]=666;
    printf(" %d , %d , %d\n",myWindow.start_x, myWindow.start_y,value);

  }
};
int main()
{
  block_2d<int> a(X,Y,0);
  sequence(a.begin(),a.end());
  window_vector<int> myVector(&a,3,3,3,3);
  std::cout<<"Size ="<<myVector.end()-myVector.begin()<<"\n Pitch = "<<a.pitch<<std::endl;
  // for_each(myVector.begin(),myVector.end(),printFunctor());
  hipDeviceSynchronize();
  int *b = (int *) malloc(a.pitch*Y);
  hipMemcpy2D(b,a.pitch,a.data_pointer,a.pitch,X,Y,hipMemcpyDeviceToHost);
  for (int i=0; i<Y;i++)
  {
    for (int j=0;j<X;j++)
    {
      std::cout<<b[i*a.pitch/sizeof(int)+j]<< " ";
    }
    std::cout<<"\n";
  }
  // for(int i=0; i<myVector.end()-myVector.begin(); i++)
  //   std::cout<<b[i]<< " ";
  // std::cout<<"\n";
  return 0;
}
