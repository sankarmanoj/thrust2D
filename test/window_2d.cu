#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/window_transform.h>
#include <thrust/window_for_each.h>
#include <thrust/device_vector.h>
#include <thrust/window_2d.h>
#define X 100
#define Y 100
using namespace thrust;
// testing window indexing
class bob
{
public:
	int cols;
	int rows;
	int lambda;

  bob ()
	{
		this->cols = X;
		this->rows = Y;
		this->lambda = 1;
	}

	__device__ void operator() (const thrust::window_2d<int> &c) const
	{
  	for(int i = 0; i< 3; i++)
			for(int j = 0; j<3;j++)
				c[i][j]=i*3+j;
	}
};
class printFunctor
{
public:
  __host__ __device__ void operator() (const window_2d<int> &myWindow) const
  {
    int value = myWindow[0][0];
    myWindow[0][0]=666;
    printf(" %ld , %ld , %d\n",myWindow.start_x, myWindow.start_y,value);
  }
};

class printFunctor2
{
public:
  __host__ __device__ int operator() (const window_2d<int> &myWindow) const
  {
    int value = myWindow[0][0];
    myWindow[0][0]=666;
    printf(" %ld , %ld , %d\n",myWindow.start_x, myWindow.start_y,value);
    return value;
  }
};

class printFunctor1
{
public:
  __device__ void operator() (const window_2d<int> &myWindow) const
  {
    int value = myWindow[0][0];

    myWindow[0][0]=666;
    printf(" %d , %d , %d\n",myWindow.start_x, myWindow.start_y,value);

  }
};
int main()
{
  block_2d<int> a(X,Y,1);
  block_2d<int> c(X,Y,0);
  device_vector<int> hello(X*Y);
	fill(a.begin(),a.end(),1);
  // window_vector<int> myVector(&a,3,3,1,1);
  // std::cout<<"Size ="<<myVector.end()-myVector.begin()<<"\n Pitch = "<<a.pitch<<std::endl;
  // for_each(myVector.begin(),myVector.end(),printFunctor());
  // hipDeviceSynchronize();
  // int *b = (int *) malloc(a.pitch*Y);
  // hipMemcpy2D(b,a.pitch,a.data_pointer,a.pitch,X,Y,hipMemcpyDeviceToHost);
  host_block_2d<int> b = a;
  for (int i=0; i<Y;i++)
  {
    for (int j=0;j<X;j++)
    {
      std::cout<<b[i][j]<< " ";
    }
    std::cout<<"\n";
  }
  thrust::window_vector<int> wv(&a,3,3,1,1);
  thrust::window_vector<int> wv2(&c,3,3,1,1);

  thrust::for_each(thrust::cuda::shared,wv.begin(),wv.end(),bob());

  b = a;
  for (int i=0; i<Y;i++)
  {
    for (int j=0;j<X;j++)
    {
      printf("%4d ",b[i][j]);
    }
    std::cout<<"\n";
  }
  return 0;
}
