#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#define X 7
#define Y 6
using namespace thrust;
// testing window indexing
class printFunctor
{
public:
  __host__ __device__ void operator() (window_2d<int,std::allocator<int> > myWindow)
  {
    int value = myWindow[0][0];

    myWindow[0][0]=666;
    printf(" %d , %d , %d\n",myWindow.start_x, myWindow.start_y,value);

  }
};
int main()
{
  block_2d<int,std::allocator<int> > a(X,Y,0);
  sequence(a.begin(),a.end());
  window_vector<int,std::allocator<int> > myVector(&a,3,3,3,3);
  std::cout<<"Size ="<<myVector.end()-myVector.begin()<<std::endl;
  for_each(host,myVector.begin(),myVector.end(),printFunctor());
  hipDeviceSynchronize();
  for (int i=0; i<Y;i++)
  {
    for (int j=0;j<X;j++)
    {
      std::cout<<a[i][j]<< " ";
    }
    std::cout<<"\n";
  }
  return 0;
}
