#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#define X 7
#define Y 6
using namespace thrust;
// testing window indexing
class printFunctor
{
public:
  __host__ __device__ void operator() (const window_2d<int,std::allocator<int> > &myWindow) const
  {
    int value = myWindow[0][0];
    myWindow[0][0]=666;
    printf(" %d , %d , %d\n",myWindow.start_x, myWindow.start_y,value);
  }
};

class printFunctor1 : shared_window_for_each_functor<int>
{
public:
  __device__ void operator() (const window_2d<int> &myWindow) const
  {
    int value = myWindow[0][0];

    myWindow[0][0]=666;
    printf(" %d , %d , %d\n",myWindow.start_x, myWindow.start_y,value);

  }
};
int main()
{
  block_2d<int,std::allocator<int> > a(X,Y,0);
  sequence(a.begin(),a.end());
  window_vector<int,std::allocator<int> > myVector(&a,3,3,3,3);
  std::cout<<"Size ="<<myVector.end()-myVector.begin()<<std::endl;
  for_each(host,myVector.begin(),myVector.end(),printFunctor());
  hipDeviceSynchronize();
  for (int i=0; i<Y;i++)
  {
    for (int j=0;j<X;j++)
    {
      std::cout<<a[i][j]<< " ";
    }
    std::cout<<"\n";
  }
  return 0;
}
