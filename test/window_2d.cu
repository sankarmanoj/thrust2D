#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/window_2d.h>
#define X 10
#define Y 10
using namespace thrust;
// testing window indexing
class bob
{
public:
	int cols;
	int rows;
	int lambda;

  bob ()
	{
		this->cols = X;
		this->rows = Y;
		this->lambda = 1;
	}

	__device__ int operator() (thrust::window_2d<int> &c, thrust::window_2d<int> &w)
	{
    w[0][0]=c[0][0];
    printf("%d , %d = %d \n",c.start_x,c.start_y,&(c[0][0])- c.data);
    return 0;
	}
};
class printFunctor
{
public:
  __host__ __device__ void operator() (const window_2d<int> &myWindow) const
  {
    int value = myWindow[0][0];
    myWindow[0][0]=666;
    printf(" %ld , %ld , %d\n",myWindow.start_x, myWindow.start_y,value);
  }
};

class printFunctor2
{
public:
  __host__ __device__ int operator() (const window_2d<int> &myWindow) const
  {
    int value = myWindow[0][0];
    myWindow[0][0]=666;
    printf(" %ld , %ld , %d\n",myWindow.start_x, myWindow.start_y,value);
    return value;
  }
};

class printFunctor1
{
public:
  __device__ void operator() (const window_2d<int> &myWindow) const
  {
    int value = myWindow[0][0];

    myWindow[0][0]=666;
    printf(" %d , %d , %d\n",myWindow.start_x, myWindow.start_y,value);

  }
};
int main()
{
  block_2d<int> a(X,Y,0);
  block_2d<int> c(X,Y,0);
  device_vector<int> hello(X*Y);
  sequence(a.begin(),a.end());
  window_vector<int> myVector(&a,3,3,1,1);
  std::cout<<"Size ="<<myVector.end()-myVector.begin()<<"\n Pitch = "<<a.pitch<<std::endl;
  // for_each(myVector.begin(),myVector.end(),printFunctor());
  hipDeviceSynchronize();
  // int *b = (int *) malloc(a.pitch*Y);
  // hipMemcpy2D(b,a.pitch,a.data_pointer,a.pitch,X,Y,hipMemcpyDeviceToHost);
  host_block_2d<int> b = a;
  for (int i=0; i<Y;i++)
  {
    for (int j=0;j<X;j++)
    {
      std::cout<<b[i][j]<< " ";
    }
    std::cout<<"\n";
  }
  thrust::window_vector<int> wv(&a,1,1,1,1);
  thrust::window_vector<int> wv2(&c,1,1,1,1);

  printf("%d -%d  %d\n",wv.begin().windows_along_x,wv.begin().windows_along_y,wv.end()-wv.end());
  thrust::transform(wv.begin(),wv.end(),wv2.begin(),hello.begin(),bob());

  b = c ;
  for (int i=0; i<Y;i++)
  {
    for (int j=0;j<X;j++)
    {
      std::cout<<b[i][j]<< " ";
    }
    std::cout<<"\n";
  }
  return 0;
}
