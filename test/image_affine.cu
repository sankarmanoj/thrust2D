#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <math.h>
using namespace cv;

class AffineTransformFunctor : public thrust::shared_window_for_each_functor<float>
{
public:
  thrust::block_2d<float> *transformMatrix;
  thrust::block_2d<float> *outBlock;

  AffineTransformFunctor(thrust::block_2d<float> * tm,thrust::block_2d<float> * outBlock)
  {
    this->transformMatrix = tm->device_pointer;
    this->outBlock = outBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<float> &inputWindow) const
  {
    int x_out, y_out;
    x_out = (int)((*transformMatrix)[0][0]*inputWindow.start_x+(*transformMatrix)[0][1]*inputWindow.start_y+(*transformMatrix)[0][2]*1);
    y_out = (int)((*transformMatrix)[1][0]*inputWindow.start_x+(*transformMatrix)[1][1]*inputWindow.start_y+(*transformMatrix)[1][2]*1);

    (*outBlock)[y_out][x_out]=inputWindow[0][0];
  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  image = small;
  // resize(small,image,Size(50,50));

  std::cout<<image.isContinuous()<<image.type()<<std::endl;
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::fill(float_image_block.begin(),float_image_block.end(),0.0f);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  thrust::fill(outBlock.begin(),outBlock.end(),0.0f);
  float * img = (float * )malloc(sizeof(float)*(image.cols*image.rows));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);


    Point2f srcTri[3];
    Point2f dstTri[3];

    Mat rot_mat( 2, 3, CV_32FC1 );
    Mat warp_mat( 2, 3, CV_32FC1 );
    Mat warp_dst;
    /// Set the dst image the same type and size as src
    warp_dst = Mat::zeros( image.rows, image.cols, image.type() );

    /// Set your 3 points to calculate the  Affine Transform
    srcTri[0] = Point2f( 0,0 );
    srcTri[1] = Point2f( image.cols - 1, 0 );
    srcTri[2] = Point2f( 0, image.rows - 1 );

    dstTri[0] = Point2f( image.cols*0.0, image.rows*0.5 );
    dstTri[1] = Point2f( image.cols*0.8, image.rows*0.2 );
    dstTri[2] = Point2f( image.cols*0.2, image.rows*0.7 );

    /// Get the Affine Transform
    warp_mat = getAffineTransform( srcTri, dstTri );
    warp_mat.convertTo(warp_mat,CV_32FC1);
    std::cout<<warp_mat.isContinuous()<<"Warp matrix \n"<<warp_mat;

    //Move Warp Matrix to Device
    thrust::block_2d<float> warp_block(warp_mat.cols,warp_mat.rows);

    printf("\n");
    for(int i = 0; i< warp_mat.rows;i++)
    {
      for(int j = 0; j<warp_mat.cols;j++)
      {
        warp_block[i][j]=warp_mat.at<float>(i,j);
      }
    }
    printf("Device Transformation Matrix\n");
    for(int i = 0; i< warp_mat.rows;i++)
    {
      for(int j = 0; j<warp_mat.cols;j++)
      {
        printf(" %f ",(float)warp_block[i][j]);
      }
      printf("\n");
    }
  //Create Windows For Indexing
  thrust::window_vector<float> inputVector(&float_image_block,1,1,1,1);
  AffineTransformFunctor atf(&warp_block,&outBlock);
  thrust::for_each(inputVector.begin(),inputVector.end(),atf);
  hipDeviceSynchronize();
  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  // std::cout<<output;
  // cudaCheckError();
  // std::cout<<output.type()<<"  "<<Size(image.cols,image.rows)<<"="<<image_block.end()-image_block.begin()<<"\n";
  imwrite("ainput.png",image);
  // imshow("output",output);
  imwrite("aoutput.png",output);

  // waitKey(0);
  // std::cout<<float_image<<"\n";
  return 0;
}
