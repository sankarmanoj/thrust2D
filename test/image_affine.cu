#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <math.h>
using namespace cv;
// #define AFFINE

#define PI 3.14159
using namespace cv;
inline float gauss(int x, int y, int mid, float sigma )
{
  float temp = (pow(x-mid,2)+pow(y-mid,2))/sigma;
  temp= exp(-temp);
  return temp;
}
#define HARRIS
class AffineTransformFunctor : public thrust::shared_window_for_each_functor<float>
{
public:
  thrust::block_2d<float> *transformMatrix;
  thrust::block_2d<float> *outBlock;

  AffineTransformFunctor(thrust::block_2d<float> * tm,thrust::block_2d<float> * outBlock)
  {
    this->transformMatrix = tm->device_pointer;
    this->outBlock = outBlock->device_pointer;
  }
  __device__ void operator() (const thrust::window_2d<float> &inputWindow) const
  {
    int x_out, y_out;
    x_out = (int)((*transformMatrix)[0][0]*inputWindow.start_x+(*transformMatrix)[0][1]*inputWindow.start_y+(*transformMatrix)[0][2]*1);
    y_out = (int)((*transformMatrix)[1][0]*inputWindow.start_x+(*transformMatrix)[1][1]*inputWindow.start_y+(*transformMatrix)[1][2]*1);

    (*outBlock)[y_out][x_out]=inputWindow[0][0];
  }
};
void getGaussianKernelBlock(int dim, float sigma,thrust::block_2d<float> &GaussianKernel )
{
  assert(dim%2);
  float inverseSigmaSquare;
  inverseSigmaSquare = 1/pow(sigma,2);
  float pi2SigSquare;
  pi2SigSquare = inverseSigmaSquare*2*PI;
  int mid = (dim-1)/2;
  float total = 0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      total+=gauss(i,j,mid,sigma);
      (GaussianKernel)[i][j]=gauss(i,j,mid,sigma);
    }
  }

  float newTotal=0;
  for(int i = 0; i<dim;i++)
  {
    for(int j = 0; j<dim;j++)
    {
      (GaussianKernel)[i][j]/=total;
      newTotal +=  (GaussianKernel)[i][j];
    }
  }
  printf("Total = %f,newTotal=%f\n",total,newTotal);
}
class HarrisIntensityFunctor : public thrust::shared_unary_window_transform_functor<float>
{

public:
  thrust::block_2d<float> * kernel;
  HarrisIntensityFunctor(thrust::block_2d<float> * kernel)
  {
    this->kernel = kernel;
  }
  __device__ void operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {
    float intensityValue;

    for(int xoffset = 1 ; xoffset <=1 ; xoffset++)
    {
      for(int yoffset = 1 ; yoffset <=1 ; yoffset++)
      {
        for(int i = 0; i< 3; i++)
        {
          for(int j = 0; j< 3; j++)
          {
              intensityValue += ((*kernel)[i][j])*(inputWindow[i + yoffset][j + xoffset]-inputWindow[i][j]);
          }
        }
      }
    }
    outputWindow[2][2]=intensityValue*0.08;
  }
};

int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  image = small;
  // resize(small,image,Size(50,50));

  std::cout<<image.isContinuous()<<image.type()<<std::endl;
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::fill(float_image_block.begin(),float_image_block.end(),0.0f);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  thrust::fill(outBlock.begin(),outBlock.end(),0.0f);
  float * img = (float * )malloc(sizeof(float)*(image.cols*image.rows));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);
  #ifdef AFFINE

    Point2f srcTri[3];
    Point2f dstTri[3];

    Mat rot_mat( 2, 3, CV_32FC1 );
    Mat warp_mat( 2, 3, CV_32FC1 );
    Mat warp_dst;
    /// Set the dst image the same type and size as src
    warp_dst = Mat::zeros( image.rows, image.cols, image.type() );

    /// Set your 3 points to calculate the  Affine Transform
    srcTri[0] = Point2f( 0,0 );
    srcTri[1] = Point2f( image.cols - 1, 0 );
    srcTri[2] = Point2f( 0, image.rows - 1 );

    dstTri[0] = Point2f( image.cols*0.0, image.rows*0.5 );
    dstTri[1] = Point2f( image.cols*0.8, image.rows*0.2 );
    dstTri[2] = Point2f( image.cols*0.2, image.rows*0.7 );

    /// Get the Affine Transform
    warp_mat = getAffineTransform( srcTri, dstTri );
    warp_mat.convertTo(warp_mat,CV_32FC1);
    std::cout<<warp_mat.isContinuous()<<"Warp matrix \n"<<warp_mat;

    //Move Warp Matrix to Device
    thrust::block_2d<float> warp_block(warp_mat.cols,warp_mat.rows);

    printf("\n");
    for(int i = 0; i< warp_mat.rows;i++)
    {
      for(int j = 0; j<warp_mat.cols;j++)
      {
        warp_block[i][j]=warp_mat.at<float>(i,j);
      }
    }
    printf("Device Transformation Matrix\n");
    for(int i = 0; i< warp_mat.rows;i++)
    {
      for(int j = 0; j<warp_mat.cols;j++)
      {
        printf(" %f ",(float)warp_block[i][j]);
      }
      printf("\n");
    }
  //Create Windows For Indexing
  thrust::window_vector<float> inputVector(&float_image_block,1,1,1,1);
  AffineTransformFunctor atf(&warp_block,&outBlock);
  thrust::for_each(inputVector.begin(),inputVector.end(),atf);
  hipDeviceSynchronize();

    unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
    hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
    for(int i = 0; i<image.cols*image.rows;i++)
    {
      outputFloatImageData[i]=(unsigned char)img[i];
    }
    Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
    // std::cout<<output;
    // cudaCheckError();
    // std::cout<<output.type()<<"  "<<Size(image.cols,image.rows)<<"="<<image_block.end()-image_block.begin()<<"\n";
    imwrite("ainput.png",image);
    // imshow("output",output);
    imwrite("aoutput.png",output);
  #endif

  #ifdef SOBELDERIV
  thrust::block_2d<float> XDeriv (image.cols,image.rows);
  thrust::block_2d<float> YDeriv (image.cols,image.rows);
  thrust::copy(float_image_block.begin(),float_image_block.end(),XDeriv.begin());
  thrust::copy(float_image_block.begin(),float_image_block.end(),YDeriv.begin());
  thrust::block_2d<float> SobelXKernel (3,3,0);
  thrust::block_2d<float> SobelYKernel (3,3,0);
  SobelXKernel[0][0]= -1;
  SobelXKernel[0][1]= -2;
  SobelXKernel[0][2]= -1;
  SobelXKernel[2][0]= 1;
  SobelXKernel[2][1]= 2;
  SobelXKernel[2][2]= 1;
  SobelYKernel[0][0]= -1;
  SobelYKernel[1][0]= -2;
  SobelYKernel[2][0]= -1;
  SobelYKernel[0][2]= 1;
  SobelYKernel[1][2]= 2;
  SobelYKernel[2][2]= 1;
  thrust::convolve(XDeriv.begin(),XDeriv.end(),SobelXKernel.begin());
  thrust::convolve(YDeriv.begin(),YDeriv.end(),SobelYKernel.begin());

  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(XDeriv.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat xout (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("harris_input.png",image);
  imwrite("xout.png",xout);
  hipMemcpy(img,thrust::raw_pointer_cast(YDeriv.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat yout (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  imwrite("yout.png",yout);
  #endif

  #ifdef HARRIS
  thrust::block_2d<float> kernel(3,3);
  getGaussianKernelBlock(3,5,kernel);
  thrust::window_vector<float> inputVector = thrust::window_vector<float>(&float_image_block,5,5,1,1);
  thrust::window_vector<float> outputVector = thrust::window_vector<float>(&outBlock,5,5,1,1);
  thrust::transform(thrust::cuda::shared,inputVector.begin(),inputVector.end(),outputVector.begin(),HarrisIntensityFunctor(kernel.device_pointer));

  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  // std::cout<<output;
  // cudaCheckError();
  // std::cout<<output.type()<<"  "<<Size(image.cols,image.rows)<<"="<<image_block.end()-image_block.begin()<<"\n";
  imshow("ainput.png",image);
  // imshow("output",output);
  imshow("aoutput.png",output);
  // waitKey(0);
  #endif
  // waitKey(0);
  // std::cout<<float_image<<"\n";
  return 0;
}
