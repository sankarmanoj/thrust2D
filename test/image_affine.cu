#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
#include <math.h>
using namespace cv;

class AffineTransformFunctor : public thrust::shared_unary_window_transform_functor<float>
{
public:
  thrust::block_2d<float> *transformMatrix;
  thrust::block_2d<float> *transformMatrix;
  
  AffineTransformFunctor(thrust::block_2d<float> * tm)
  {
    this->transformMatrix = tm;
  }
  __device__ void operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &outputWindow) const
  {

  }
};
int main(int argc, char const *argv[]) {
  Mat small = imread("car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat image;
  int dim = 17;
  image = small;
  // resize(small,image,Size(1500,1500));

  std::cout<<dim<<"  "<<image.isContinuous()<<std::endl;
  thrust::block_2d<float> float_image_block (image.cols,image.rows);
  thrust::block_2d<float> outBlock (image.cols,image.rows);
  float * img = (float * )malloc(sizeof(float)*(image.cols*image.rows));
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    img[i]=(float)image.ptr()[i];
  }
  float_image_block.assign(img,img+image.cols*image.rows);


    Point2f srcTri[3];
    Point2f dstTri[3];

    Mat rot_mat( 2, 3, CV_32FC1 );
    Mat warp_mat( 2, 3, CV_32FC1 );
    Mat warp_dst;
    /// Set the dst image the same type and size as src
    warp_dst = Mat::zeros( image.rows, image.cols, image.type() );

    /// Set your 3 points to calculate the  Affine Transform
    srcTri[0] = Point2f( 0,0 );
    srcTri[1] = Point2f( image.cols - 1, 0 );
    srcTri[2] = Point2f( 0, image.rows - 1 );

    dstTri[0] = Point2f( image.cols*0.0, image.rows*0.33 );
    dstTri[1] = Point2f( image.cols*0.85, image.rows*0.25 );
    dstTri[2] = Point2f( image.cols*0.15, image.rows*0.7 );

    /// Get the Affine Transform
    warp_mat = getAffineTransform( srcTri, dstTri );
    std::cout<<"Warp matrix \n"<<warp_mat;

    //Move Warp Matrix to Device
    thrust::block_2d<float> warp_block(warp_mat.cols,warp_mat.rows);
    for(int i = 0; i<warp_mat.cols*warp_mat.rows;i++)
    {
      warp_block[i/warp_block.dim_x][i%warp_block.dim_x]=(float)warp_mat.ptr()[i];
    }

  //Create Windows For Indexing
  thrust::window_vector<float> inputVector(&float_image_block,1,1,1,1);
  thrust::window_vector<float> outputVector(&outBlock,1,1,1,1);
  thrust::transform(thrust::cuda::shared,inputVector.begin(),inputVector.end(),outputVector.begin(),AffineTransformFunctor(warp_block.device_pointer));

  unsigned char * outputFloatImageData = (unsigned char *)malloc(sizeof(unsigned char)*(float_image_block.end()-float_image_block.begin()));
  hipMemcpy(img,thrust::raw_pointer_cast(outBlock.data()),sizeof(float)*(float_image_block.end()-float_image_block.begin()),hipMemcpyDeviceToHost);
  for(int i = 0; i<image.cols*image.rows;i++)
  {
    outputFloatImageData[i]=(unsigned char)img[i];
  }
  Mat output (Size(image.cols,image.rows),CV_8UC1,outputFloatImageData);
  // std::cout<<output;
  // cudaCheckError();
  // std::cout<<output.type()<<"  "<<Size(image.cols,image.rows)<<"="<<image_block.end()-image_block.begin()<<"\n";
  imwrite("ainput.png",image);
  // imshow("output",output);
  imwrite("aoutput.png",output);

  // waitKey(0);
  // std::cout<<float_image<<"\n";
  return 0;
}
