// #include <thrust/device_vector.h>

// #include <thrust/for_each.h>
#include <thrust/sequence.h>
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
// #include <thrust/reduce.h>
// #include <thrust/system/cuda/window_structures.h>
#include <thrust/shared_for_each.h>
// #include <thrust/shared_reduce.h>
using namespace thrust;
class printFunctor
{
public:

__device__  void  operator() ( int  &a)
  {
    printf("%d\n",a);
  }
};
int main()
{
  hipEvent_t start,stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);
  hipEventRecord(start);
  device_vector<long int> a(1024*1024);
  device_vector<int> b(63);
  //
  sequence(a.begin(),a.end());
  inclusive_scan(a.begin(),a.end(),b.begin());
  // // for (int i =0;i<1;i++)
  // // {
  //   printf("Shared = %ld \n",reduce(cuda::shared,a.begin(),a.end()));
  //   // hipDeviceSynchronize();
  //   // printf("Thrust = %ld \n",reduce(a.begin(),a.end()));
  // // }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("%f\n", milliseconds);
  // hipDeviceSynchronize();
  // printf("\n");
  // for_each(cuda::shared,a.begin(),a.end(),printFunctor());
  // inclusive_scan(a.begin(),a.end(),b.begin());
  // hipDeviceSynchronize();
  // printf("\n");
  // for_each(cuda::shared,b.begin(),b.end(),printFunctor());
  // inclusive_scan(cuda::shared,a.begin(),a.end(),b.begin());
  // hipDeviceSynchronize();
  // printf("\n");
  // for_each(cuda::shared,b.begin(),b.end(),printFunctor());
  // hipDeviceSynchronize();
  // printf("\n");
}
