// #include <thrust/device_vector.h>

// #include <thrust/for_each.h>
#include <thrust/sequence.h>
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
// #include <thrust/reduce.h>
// #include <thrust/system/cuda/window_structures.h>
#include <thrust/shared_algorithms.h>
#include <thrust/block_2d.h>
// #include <thrust/shared_reduce.h>
using namespace thrust;
class printFunctor
{
public:

__device__  void  operator() ( int  &a)
  {
    printf("%d\n",a);
  }
};
int main()
{
  block_2d<int> a(100,100);
  //
  sequence(a.begin(),a.end());
  // inclusive_scan(a.begin(),a.end(),b.begin());
  // // for (int i =0;i<1;i++)
  // // {
    printf("Shared = %d \n",reduce(cuda::shared,a.begin(),a.end(),0,thrust::plus<int>()));
    printf("Thrust = %d \n",reduce(a.begin(),a.end()));
  //   // hipDeviceSynchronize();
  //   // printf("Thrust = %ld \n",reduce(a.begin(),a.end()));
  // // }
  // hipDeviceSynchronize();
  // printf("\n");
  // for_each(cuda::shared,a.begin(),a.end(),printFunctor());
  // inclusive_scan(a.begin(),a.end(),b.begin());
  // hipDeviceSynchronize();
  // printf("\n");
  // for_each(cuda::shared,b.begin(),b.end(),printFunctor());
  // inclusive_scan(cuda::shared,a.begin(),a.end(),b.begin());
  // hipDeviceSynchronize();
  // printf("\n");
  // for_each(cuda::shared,b.begin(),b.end(),printFunctor());
  // hipDeviceSynchronize();
  // printf("\n");
}
