// #include <thrust/device_vector.h>

// #include <thrust/for_each.h>
#include <thrust/sequence.h>
#include <thrust/scan.h>
#include <thrust/shared_for_each.h>
// #include <thrust/shared_reduce.h>
using namespace thrust;
class printFunctor
{
public:

__device__  void  operator() ( int  &a)
  {
    printf("%d\n",a);
  }
};
int main()
{
  device_vector<int> a(102*100+1 );
  // device_vector<int> b(63);
  //
  sequence(a.begin(),a.end());
  printf("Shared = %d \n",reduce(cuda::shared,a.begin(),a.end()));
  hipDeviceSynchronize();
  printf("Thrust = %d \n",reduce(a.begin(),a.end()));
  hipDeviceSynchronize();
  // printf("\n");
  // for_each(cuda::shared,a.begin(),a.end(),printFunctor());
  // inclusive_scan(a.begin(),a.end(),b.begin());
  // hipDeviceSynchronize();
  // printf("\n");
  // for_each(cuda::shared,b.begin(),b.end(),printFunctor());
  // inclusive_scan(cuda::shared,a.begin(),a.end(),b.begin());
  // hipDeviceSynchronize();
  // printf("\n");
  // for_each(cuda::shared,b.begin(),b.end(),printFunctor());
  // hipDeviceSynchronize();
  // printf("\n");
}
