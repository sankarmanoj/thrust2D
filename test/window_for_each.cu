#include "hip/hip_runtime.h"
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <thrust/window_transform.h>
#define X 32
#define Y 32
#define XSTART 10
#define XRANGE 20
#define YSTART 10
#define YRANGE 20

class printFunctor
{
public:
  __device__ void operator() (const thrust::window_2d<int> &inputWindow,const thrust::window_2d<int> &inputWindow1, const thrust::window_2d<int> &outputWindow) const
  {
  printf("%f=%f+%f\n",(int)outputWindow[0][0],(int) inputWindow[0][0],(int)inputWindow1[0][0]);
     outputWindow[0][0]=inputWindow[0][0] + inputWindow1[0][0];
  }
};

class printFunctor1
{
public:
  __device__ void operator() (const thrust::window_2d<int> &inputWindow, const thrust::window_2d<int> &outputWindow) const
  {
    // printf("%f - %d - %d\n",tex2D<int>(inputWindow.texref,blockIdx.x*blockDim.x + threadIdx.x,blockIdx.y*blockDim.y + threadIdx.y),blockIdx.x*blockDim.x + threadIdx.x,blockIdx.y*blockDim.y + threadIdx.y);
    outputWindow[0][0]=inputWindow[make_int2(1,1)];
  }
};

class forEachFunctor
{
public:

  __device__ void operator() (const thrust::window_2d<int> &inputWindow) const
  {
     inputWindow[0][0]=934;
    //  printf("%d %d %d\n",outputWindow[0][0], inputWindow[0][0],inputWindow1[0][0]);
  }
};
int main()
{
  srand(13);
  thrust::block_2d<int> inBlock(X,Y);
  thrust::block_2d<int> outBlock(X,Y);
  thrust::host_block_2d<int> h_inBlock(X,Y);
  thrust::host_block_2d<int> h_outBlock(X,Y);
  thrust::sequence(inBlock.begin(),inBlock.end());
  thrust::fill(outBlock.begin(),outBlock.end(),777.0f);
  h_inBlock =inBlock;
  thrust::window_vector<int> myVector = thrust::window_vector<int>(&inBlock,3,3,1,1);
  thrust::window_vector<int> mySecondVector = thrust::window_vector<int>(&outBlock,3,3,1,1);
  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%d ",h_inBlock[pos]);
    }
    std::cout<<"\n";
  }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  // thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),myVector1.begin(),mySecondVector.begin(),printFunctor());
  thrust::transform(thrust::cuda::texture,myVector.begin(),myVector.end(),mySecondVector.begin(),printFunctor1());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  h_outBlock = outBlock;
  printf("\nTime Taken = %f\n",milliseconds);

  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%d  ",(int)h_outBlock[pos]);
    }
    std::cout<<"\n";
  }
}
