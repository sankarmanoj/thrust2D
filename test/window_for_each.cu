#include "hip/hip_runtime.h"
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <iostream>
#include <thrust/window_transform.h>
#define X 120
#define Y 100
class printFunctor
{
public:
  __device__ void operator() (thrust::window_2D<float> &w)
  {
  printf("%f",w[0][0]);
}
};
int main()
{
  srand(13);
  thrust::Block_2D<float> inBlock(X,Y);
  thrust::Block_2D<float> kernel(3,3);
  thrust::device_vector<float> a((long long int)X*Y);
  thrust::sequence(a.begin(),a.end());
  thrust::copy(a.begin(),a.end(),inBlock.begin());
  thrust::fill(kernel.begin(),kernel.end(),1.0);
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&inBlock,3,3 ,3,3);
  printf("Windows Along X,Y = %d,%d \n",myVector.begin().windows_along_x,myVector.begin().windows_along_y);
  // for (int i=0; i<Y;i++)
  // {
  //   for (int j=0;j<X  ;j++)
  //   {
  //       printf("%5.0f ",inBlock[make_int2(j,i)]);
  //   }
  //   std::cout<<"\n";
  // }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::window_for_each(myVector.begin(),myVector.end(),printFunctor());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time Taken = %f\n",milliseconds);
  //
  // for (int i=0; i<Y;i++)
  // {
  //   for (int j=0;j<X;j++)
  //   {
  //         printf("%5.0f ",inBlock[make_int2(j,i)]);
  //   }
  //   std::cout<<"\n";
  // }
}
