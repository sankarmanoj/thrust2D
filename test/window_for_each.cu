#include "hip/hip_runtime.h"
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <thrust/window_transform.h>
#define X 10
#define Y 10
#define XSTART 0
#define XRANGE 10
#define YSTART 0
#define YRANGE 10

class printFunctor
{
public:
  __device__ void operator() (const thrust::window_2D<float> &inputWindow,const thrust::window_2D<float> &inputWindow1, const thrust::window_2D<float> & outputWindow) const
  {
     outputWindow[0][0]=inputWindow[0][0] + inputWindow1[0][0];
     printf("%f=%f+%f\n",(float)outputWindow[0][0],(float) inputWindow[0][0],(float)inputWindow1[0][0]);
  }
};

class printFunctor1
{
public:
  __device__ void operator() (const thrust::window_2D<float> &inputWindow, const thrust::window_2D<float> & outputWindow) const
  {
     outputWindow[0][0]=inputWindow[0][0];
    //  printf("%d %d %d\n",outputWindow[0][0], inputWindow[0][0],inputWindow1[0][0]);
  }
};

int main()
{
  srand(13);
  thrust::Block_2D<float> inBlock(X,Y);
  thrust::Block_2D<float> inBlock1(X,Y);
  thrust::Block_2D<float> outBlock(X,Y);
  thrust::Block_2D<float> kernel(3,3);
  thrust::device_vector<float> a((long long int)X*Y);
  thrust::sequence(a.begin(),a.end());
  thrust::copy(a.begin(),a.end(),inBlock.begin());
  thrust::copy(a.begin(),a.end(),inBlock1.begin());
  thrust::fill(kernel.begin(),kernel.end(),1.0);
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&inBlock,3,3,3,3);
  thrust::window_vector<float> myVector1 = thrust::window_vector<float>(&inBlock1,3,3,3,3);
  thrust::window_vector<float> mySecondVector = thrust::window_vector<float>(&outBlock,3,3,3,3);
  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%5.0f ",inBlock[pos]);
    }
    std::cout<<"\n";
  }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),myVector1.begin(),mySecondVector.begin(),printFunctor());
  // thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),mySecondVector.begin(),printFunctor1());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("\nTime Taken = %f\n",milliseconds);

  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%5.0f  ",(float)outBlock[pos]);
    }
    std::cout<<"\n";
  }
}
