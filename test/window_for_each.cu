#include "hip/hip_runtime.h"
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <thrust/window_for_each.h>
#include <thrust/window_transform.h>
#define X 70
#define Y 70
#define XSTART 25
#define XRANGE 40
#define YSTART 25
#define YRANGE 40

class printFunctor
{
public:
  __device__ void operator() (const thrust::window_2d<int> &inputWindow,const thrust::window_2d<int> &inputWindow1, const thrust::window_2d<int> &outputWindow) const
  {
  printf("%f=%f+%f\n",(int)outputWindow[0][0],(int) inputWindow[0][0],(int)inputWindow1[0][0]);
     outputWindow[0][0]=inputWindow[0][0] + inputWindow1[0][0];
  }
};

class printFunctor1
{
public:
  __device__ void operator() (const thrust::window_2d<int> &inputWindow, const thrust::window_2d<int> &outputWindow) const
  {
    // printf("%f - %d - %d\n",tex2D<int>(inputWindow.texref,blockIdx.x*blockDim.x + threadIdx.x,blockIdx.y*blockDim.y + threadIdx.y),blockIdx.x*blockDim.x + threadIdx.x,blockIdx.y*blockDim.y + threadIdx.y);
    outputWindow[1][1]=inputWindow[make_int2(0,0)]+inputWindow[make_int2(0,1)]+inputWindow[make_int2(2,1)]+
                inputWindow[make_int2(1,0)]+inputWindow[make_int2(1,2)]+
                inputWindow[make_int2(0,0)]+inputWindow[make_int2(2,2)]+
                inputWindow[make_int2(0,2)]+inputWindow[make_int2(2,0)];
    printf("@ %dx%d, val = %d (%d x %d):(%d x %d)\n",outputWindow.start_x,outputWindow.start_y,outputWindow[1][1],blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);

  }
};

class forEachFunctor
{
public:

  __device__ void operator() (const thrust::window_2d<int> &inputWindow) const
  {
     inputWindow[0][0]=inputWindow.start_y*inputWindow.start_x;
     printf("%d x %d\n",inputWindow.start_y,inputWindow.start_x);
  }
};
int main()
{
  srand(13);
  thrust::block_2d<int> inBlock(X,Y);
  thrust::block_2d<int> outBlock(X,Y);
  thrust::host_block_2d<int> h_inBlock(X,Y);
  thrust::host_block_2d<int> h_outBlock(X,Y);
  thrust::fill(inBlock.begin(),inBlock.end(),1);

  thrust::fill(outBlock.begin(),outBlock.end(),777.0f);
  h_inBlock =inBlock;
  thrust::window_vector<int> myVector = thrust::window_vector<int>(&inBlock,3,3,1,1);
  thrust::window_vector<int> mySecondVector = thrust::window_vector<int>(&outBlock,3,3,1,1);
  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%d ",h_inBlock[pos]);
    }
    std::cout<<"\n";
  }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),mySecondVector.begin(),printFunctor1());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  h_outBlock = outBlock;
  printf("\nTime Taken = %f\n",milliseconds);

  printf("   " );
  for (int i=XSTART; i<XSTART + XRANGE;i++)
  {
      printf("%d  ",i);
  }
  std::cout<<"\n";

  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {printf("%d  ",j);
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%d   ",(int)h_outBlock[pos]);

    }
    std::cout<<"\n";
  }
}
