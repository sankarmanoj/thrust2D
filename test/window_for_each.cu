#include "hip/hip_runtime.h"
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <thrust/window_transform.h>
#define X 10
#define Y 10
#define XSTART 0
#define XRANGE 10
#define YSTART 0
#define YRANGE 10

class printFunctor : public thrust::shared_binary_window_transform_functor<float>
{
public:
  __device__ void operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &inputWindow1, const thrust::window_2d<float> &outputWindow) const
  {
  printf("%f=%f+%f\n",(float)outputWindow[0][0],(float) inputWindow[0][0],(float)inputWindow1[0][0]);
     outputWindow[0][0]=inputWindow[0][0] + inputWindow1[0][0];
  }
};

class printFunctor1 : public thrust::shared_unary_window_transform_functor<float>
{
public:
  __device__ void operator() (const thrust::window_2d<float> &inputWindow, const thrust::window_2d<float> &outputWindow) const
  {
     outputWindow[0][0]=inputWindow[0][0];
    //  printf("%d %d %d\n",outputWindow[0][0], inputWindow[0][0],inputWindow1[0][0]);
  }
};

int main()
{
  srand(13);
  thrust::block_2d<float> inBlock(X,Y);
  thrust::block_2d<float> inBlock1(X,Y);
  thrust::block_2d<float> outBlock(X,Y);
  thrust::block_2d<float> kernel(3,3);
  thrust::device_vector<float> a((long long int)X*Y);
  thrust::sequence(a.begin(),a.end());
  thrust::copy(a.begin(),a.end(),inBlock.begin());
  thrust::fill(inBlock1.begin(),inBlock1.end(),456.0f);
  thrust::fill(kernel.begin(),kernel.end(),1.0);
  thrust::fill(outBlock.begin(),outBlock.end(),777.0f);
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&inBlock,3,3,3,3);
  thrust::window_vector<float> myVector1 = thrust::window_vector<float>(&inBlock1,3,3,3,3);
  thrust::window_vector<float> mySecondVector = thrust::window_vector<float>(&outBlock,3,3,3,3);
  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%5.0f ",inBlock[pos]);
    }
    std::cout<<"\n";
  }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),myVector1.begin(),mySecondVector.begin(),printFunctor());
  // thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),mySecondVector.begin(),printFunctor1());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("\nTime Taken = %f\n",milliseconds);

  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%5.0f  ",(float)outBlock[pos]);
    }
    std::cout<<"\n";
  }
}
