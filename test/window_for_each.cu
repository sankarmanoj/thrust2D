#include "hip/hip_runtime.h"
#include<thrust/window_2d.h>
#include <thrust/generate.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#define X 10000
#define Y 10000
int main()
{
  srand(13);
  thrust::Block_2D<float> inBlock(X,Y);
  thrust::Block_2D<float> kernel(3,3);
  thrust::device_vector<float> a((long long int)X*Y);
  thrust::sequence(a.begin(),a.end());
  thrust::copy(a.begin(),a.end(),inBlock.begin());
  thrust::fill(kernel.begin(),kernel.end(),1.0);
  // for (int i=0; i<Y;i++)
  // {
  //   for (int j=0;j<X  ;j++)
  //   {
  //       std::cout<<inBlock[make_int2(j,i)]<<" ";
  //   }
  //   std::cout<<"\n";
  // }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  thrust::convolve(inBlock.begin(), inBlock.end(), kernel.begin());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time Taken = %f\n",milliseconds);
  // //
  // for (int i=0; i<Y;i++)
  // {
  //   for (int j=0;j<X;j++)
  //   {
  //         std::cout<<inBlock[make_int2(j,i)]<<" ";
  //   }
  //   std::cout<<"\n";
  // }
}
