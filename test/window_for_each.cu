#include "hip/hip_runtime.h"
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <iostream>
#include <thrust/window_transform.h>
#define X 10
#define Y 10
#define XSTART 0
#define XRANGE 10
#define YSTART 0
#define YRANGE 10

class printFunctor
{
public:
  __device__ void operator() (const thrust::window_2D<float> &w) const
  {
     w[1][1]=666;
  }
};
int main()
{
  srand(13);
  thrust::Block_2D<float> inBlock(X,Y);
  thrust::Block_2D<float> kernel(3,3);
  thrust::device_vector<float> a((long long int)X*Y);
  thrust::sequence(a.begin(),a.end());
  thrust::copy(a.begin(),a.end(),inBlock.begin());
  thrust::fill(kernel.begin(),kernel.end(),1.0);
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&inBlock,3,3 ,3,3);
  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%5.0f ",inBlock[pos]);
    }
    std::cout<<"\n";
  }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::for_each(thrust::cuda::shared,myVector.begin(),myVector.end(),printFunctor());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("\nTime Taken = %f\n",milliseconds);

  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%5.0f  ",inBlock[pos]);
    }
    std::cout<<"\n";
  }
}
