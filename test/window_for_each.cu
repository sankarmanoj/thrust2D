#include "hip/hip_runtime.h"
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <iostream>
#include <thrust/window_transform.h>
#define X 30
#define Y 30
class printFunctor
{
public:
  __device__ void operator() (thrust::window_2D<float> &w)
  {
  printf("%f",w[0][0]);
}
};
int main()
{
  srand(13);
  thrust::Block_2D<float> inBlock(X,Y);
  thrust::Block_2D<float> kernel(3,3);
  thrust::device_vector<float> a((long long int)X*Y);
  thrust::sequence(a.begin(),a.end());
  thrust::copy(a.begin(),a.end(),inBlock.begin());
  thrust::fill(kernel.begin(),kernel.end(),1.0);
<<<<<<< HEAD
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&inBlock,3,3 ,4,4);

  for (int i=0; i<Y;i++)
  {
    for (int j=0;j<X  ;j++)
    {
        printf("%5.0f ",inBlock[make_int2(j,i)]);
    }
    std::cout<<"\n";
  }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::for_each(shared_policy,myVector.begin(),myVector.end(),printFunctor());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time Taken = %f\n",milliseconds);

  for (int i=0; i<Y;i++)
  {
    for (int j=0;j<X;j++)
    {
          printf("%5.0f ",inBlock[make_int2(j,i)]);
    }
    std::cout<<"\n";
  }
}
