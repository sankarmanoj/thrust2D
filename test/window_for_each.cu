#include "hip/hip_runtime.h"
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <iostream>
#include <thrust/window_transform.h>
#define X 10
#define Y 10
#define XSTART 0
#define XRANGE 10
#define YSTART 0
#define YRANGE 10

class printFunctor
{
public:
  __device__ void operator() (const thrust::window_2D<float> &inputWindow,const thrust::window_2D<float> & outputWindow) const
  {
     outputWindow[0][0]=inputWindow[0][0];
  }
};
int main()
{
  srand(13);
  thrust::Block_2D<float> inBlock(X,Y);
  thrust::Block_2D<float> outBlock(X,Y);
  thrust::Block_2D<float> kernel(3,3);
  thrust::device_vector<float> a((long long int)X*Y);
  thrust::sequence(a.begin(),a.end());
  thrust::copy(a.begin(),a.end(),inBlock.begin());
  thrust::fill(kernel.begin(),kernel.end(),1.0);
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&inBlock,3,3 ,3,3);
  thrust::window_vector<float> mySecondVector = thrust::window_vector<float>(&outBlock,3,3,3,3);
  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%5.0f ",inBlock[pos]);
    }
    std::cout<<"\n";
  }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),mySecondVector.begin(),printFunctor());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("\nTime Taken = %f\n",milliseconds);

  for (int j=YSTART;j<YSTART + YRANGE;j++)
  {
    for (int i=XSTART; i<XSTART + XRANGE;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%5.0f  ",inBlock[pos]);
    }
    std::cout<<"\n";
  }
}
