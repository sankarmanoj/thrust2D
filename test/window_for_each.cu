#include "hip/hip_runtime.h"
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/sequence.h>
#include <iostream>
#include <thrust/window_transform.h>
#define X 100
#define Y 100
class printFunctor
{
public:
  __device__ void operator() (thrust::window_2D<float> &w)
  {
  printf("%f",w[0][0]);
}
};
int main()
{
  srand(13);
  thrust::Block_2D<float> inBlock(X,Y);
  thrust::Block_2D<float> kernel(3,3);
  thrust::device_vector<float> a((long long int)X*Y);
  thrust::sequence(a.begin(),a.end());
  thrust::copy(a.begin(),a.end(),inBlock.begin());
  thrust::fill(kernel.begin(),kernel.end(),1.0);
  thrust::window_vector<float> myVector = thrust::window_vector<float>(&inBlock,2,3 ,2,3);
  // for (int i=0; i<Y;i++)
  // {
  //   for (int j=0;j<X  ;j++)
  //   {
  //       std::cout<<inBlock[make_int2(j,i)]<<" ";
  //   }
  //   std::cout<<"\n";
  // }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::window_for_each(myVector.begin(),myVector.end(),printFunctor());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time Taken = %f\n",milliseconds);
  // //
  // for (int i=0; i<Y;i++)
  // {
  //   for (int j=0;j<X;j++)
  //   {
  //         std::cout<<inBlock[make_int2(j,i)]<<" ";
  //   }
  //   std::cout<<"\n";
  // }
}
