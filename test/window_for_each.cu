#include "hip/hip_runtime.h"
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#define X 10
#define Y 10
#define XSTART 9990
#define XRANGE 10
#define YSTART 990
#define YRANGE 10

class printFunctor
{
public:
  __device__ void operator() (const thrust::window_2d<float> &inputWindow,const thrust::window_2d<float> &inputWindow1, const thrust::window_2d<float> &outputWindow) const
  {
  printf("%f=%f+%f\n",(float)outputWindow[0][0],(float) inputWindow[0][0],(float)inputWindow1[0][0]);
     outputWindow[0][0]=inputWindow[0][0] + inputWindow1[0][0];
  }
};

class printFunctor1
{
public:
  __device__ void operator() (const thrust::window_2d<float> &inputWindow, const thrust::window_2d<float> &outputWindow) const
  {
     outputWindow[0][0]=inputWindow[0][0];
    //  printf("%d %d %d\n",outputWindow[0][0], inputWindow[0][0],inputWindow1[0][0]);
  }
};

class forEachFunctor
{
public:

  __device__ void operator() (const thrust::window_2d<float> &inputWindow) const
  {
     inputWindow[0][0]=934;
    //  printf("%d %d %d\n",outputWindow[0][0], inputWindow[0][0],inputWindow1[0][0]);
  }
};
int main()
{
  thrust::block_2d<float> outBlock(X,Y,0.0);
  thrust::window_vector<float> mySecondVector = thrust::window_vector<float>(&outBlock,3,3,1,1);
  // thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),myVector1.begin(),mySecondVector.begin(),printFunctor());
  // thrust::transform(thrust::cuda::shared,myVector.begin(),myVector.end(),mySecondVector.begin(),printFunctor1());
  thrust::for_each(mySecondVector.begin(),mySecondVector.end(),forEachFunctor());

  for (int j=0;j<Y;j++)
  {
    for (int i=0;i<X;i++)
    {
        int2 pos = make_int2(i,j);
        printf("%5.0f  ",(float)outBlock[pos]);
    }
    std::cout<<"\n";
  }
}
