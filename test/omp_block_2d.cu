#define THRUST_DEVICE_SYSTEM 2
#include <thrust/sequence.h>
#include <thrust/block_2d.h>
#include <thrust/window_2d.h>
#include <thrust/device_vector.h>
#include <thrust/system/omp/execution_policy.h>
#include <iostream>
using namespace thrust;
#define X 30
#define Y 30
class printFunctor
{
public:

  __host__ __device__  void  operator() ( int  &a)
  {
    // a=threadIdx.x;
    printf("%d \n",a);
    // return 10;
  }
};
class printFunctorW
{
public:
  __host__ __device__ void operator() (const window_2d<int> &myWindow) const
  {
    int value = myWindow[0][0];
    myWindow[0][0]=666;
    printf(" %ld , %ld , %d\n",myWindow.start_x, myWindow.start_y,value);
  }
};
int main()
{
  block_2d<int> a(X,Y,99);
  sequence(thrust::omp::par,a.begin(),a.end());
  for_each(thrust::omp::par,a.begin(),a.end(),printFunctor());
  window_vector<int> myVector(&a,3,3,3,3);
  for_each(myVector.begin(),myVector.end(),printFunctorW());
  hipDeviceSynchronize();
  return 0;
}
