#include <stdio.h>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/block_2d.h>
#include <thrust/window_transform.h>
#define SRAND_VALUE 1985
#define BLOCK_SIZE 16

class ghostRowsFunctor
{
public:
  __device__ void operator() (const thrust::window_2d<bool> &inputWindow) const
  {
      inputWindow[inputWindow.block_dim_y-1][0]=inputWindow[1][0];
      inputWindow[0][0]=inputWindow[inputWindow.block_dim_y-2][0];
  }
};
class ghostColsFunctor
{
public:
  __device__ void operator() (const thrust::window_2d<bool> &inputWindow) const
  {
      inputWindow[0][inputWindow.block_dim_x-1]=inputWindow[0][1];
      inputWindow[0][0]=inputWindow[0][inputWindow.block_dim_x-2];
  }
};

class GOLFunctor
{

public:
  __device__ int operator() (thrust::window_2d<bool> &inputWindow, thrust::window_2d<bool> &outputWindow) const
  {
      int numNeighbors;
      // Get the number of neighbors for a given grid point
      numNeighbors = inputWindow[0][1]+inputWindow[2][1]+
                  inputWindow[1][0]+inputWindow[1][2]+
                  inputWindow[0][0]+inputWindow[2][2]+
                  inputWindow[0][2]+inputWindow[2][0];
      int cell = inputWindow[1][1];
      // Here we have explicitly all of the game rules
      if (cell == 1 && numNeighbors < 2)
          outputWindow[1][1] = 0;
      else if (cell == 1 && (numNeighbors == 2 || numNeighbors == 3))
          outputWindow[1][1] = 1;
      else if (cell == 1 && numNeighbors > 3)
          outputWindow[1][1] = 0;
      else if (cell == 0 && numNeighbors == 3)
          outputWindow[1][1] = 1;
      else
          outputWindow[1][1] = cell;

      return 0;
  }
};
int main(int argc, char* argv[])
{
    int i,j,iter;


    int dim;
    if (argc==2)
     dim = atoi(argv[1]);
    else
     dim = 1024;
    int maxIter = 1<<10; //Number of game steps

    thrust::block_2d<bool> *d_grid = new thrust::block_2d<bool>(dim+2,dim+2);
    thrust::block_2d<bool> *d_new_grid = new thrust::block_2d<bool>(dim+2,dim+2);
    thrust::block_2d<bool> *d_temp_grid;
    thrust::device_vector<bool> null_vector(dim*dim);
    thrust::host_block_2d<bool> h_grid(dim+2,dim+2);

    // Assign initial population randomly
    srand(SRAND_VALUE);
    for(i = 1; i<=dim; i++) {
        for(j = 1; j<=dim; j++) {
            h_grid[i][j] = rand() % 2;
        }
    }

    // Copy over initial game grid (Dim-1 threads)
    *d_grid = h_grid;




    // Main game loop
    for (iter = 0; iter<maxIter; iter++) {

        thrust::window_vector<bool> ghostRowsWindows (d_grid,1,d_grid->dim_y,1,d_grid->dim_y);
        thrust::window_vector<bool> ghostColsWindows (d_grid,d_grid->dim_x,1,d_grid->dim_x,1);
        thrust::for_each(ghostRowsWindows.begin(),ghostRowsWindows.end(),ghostRowsFunctor());
        thrust::for_each(ghostColsWindows.begin(),ghostColsWindows.end(),ghostColsFunctor());
        thrust::window_vector<bool>GOLInputVector(d_grid,3,3,1,1);
        thrust::window_vector<bool>GOLOutputVector(d_new_grid,3,3,1,1);
        thrust::transform(GOLInputVector.begin(),GOLInputVector.end(),GOLOutputVector.begin(),null_vector.begin(),GOLFunctor());
        // ghostRows<<<cpyGridRowsGridSize, cpyBlockSize>>>(dim, d_grid);
        // ghostCols<<<cpyGridColsGridSize, cpyBlockSize>>>(dim, d_grid);
        // GOL<<<gridSize, blockSize>>>(dim, d_grid, d_newGrid);

        // Swap our grids and iterate again
        d_temp_grid = d_grid;
        d_grid = d_new_grid;
        d_new_grid = d_temp_grid;
    }//iter loop

    // Copy back results and sum
    // hipMemcpy(h_grid, d_grid, bytes, hipMemcpyDeviceToHost);

    // Sum up alive cells and print results
    h_grid = *d_grid;
    int total = 0;
    for (i = 1; i<=dim; i++) {
        for (j = 1; j<=dim; j++) {
           total += h_grid[i][j];
        }
    }
    printf("Total Alive: %d\n", total);

    // Release memory

    return 0;
}
