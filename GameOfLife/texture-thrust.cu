#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/block_2d.h>
#include <thrust/window_transform.h>
#define SRAND_VALUE 1985
#define BLOCK_SIZE 16

class ghostRowsFunctor
{
public:
  __device__ void operator() (const thrust::window_2d<int> &inputWindow) const
  {
      inputWindow[inputWindow.block_dim_y-1][0]=inputWindow[1][0];
      inputWindow[0][0]=inputWindow[inputWindow.block_dim_y-2][0];
  }
};
class ghostColsFunctor
{
public:
  __device__ void operator() (const thrust::window_2d<int> &inputWindow) const
  {
      inputWindow[0][inputWindow.block_dim_x-1]=inputWindow[0][1];
      inputWindow[0][0]=inputWindow[0][inputWindow.block_dim_x-2];
  }
};

class GOLFunctor
{

public:
  __device__ int operator() (thrust::window_2d<int> &inputWindow, thrust::window_2d<int> &outputWindow) const
  {
      int numNeighbors;
      // Get the number of neighbors for a given grid point
      numNeighbors = inputWindow[make_int2(0,1)]+inputWindow[make_int2(2,1)]+
                  inputWindow[make_int2(1,0)]+inputWindow[make_int2(1,2)]+
                  inputWindow[make_int2(0,0)]+inputWindow[make_int2(2,2)]+
                  inputWindow[make_int2(0,2)]+inputWindow[make_int2(2,0)];
      outputWindow[1][1]=(numNeighbors==3)||(inputWindow[make_int2(1,1)]&&(numNeighbors==2));

      return 0;
  }
};
int main(int argc, char* argv[])
{
    int i,j,iter;


    int dim;
    if (argc==2)
     dim = atoi(argv[1]);
    else
     dim = 1024;
    int maxIter = 1<<10; //Number of game steps

    thrust::block_2d<int> *d_grid = new thrust::block_2d<int>(dim+2,dim+2);
    thrust::block_2d<int> *d_new_grid = new thrust::block_2d<int>(dim+2,dim+2);
    thrust::block_2d<int> *d_temp_grid;
    thrust::host_block_2d<int> h_grid(dim+2,dim+2);

    // Assign initial population randomly
    srand(SRAND_VALUE);
    for(i = 1; i<=dim; i++) {
        for(j = 1; j<=dim; j++) {
            h_grid[i][j] = rand() % 2;
        }
    }

    // Copy over initial game grid (Dim-1 threads)
    *d_grid = h_grid;




    // Main game loop
    for (iter = 0; iter<maxIter; iter++) {

        thrust::window_vector<int> ghostRowsWindows (d_grid,1,d_grid->dim_y,1,d_grid->dim_y);
        thrust::window_vector<int> ghostColsWindows (d_grid,d_grid->dim_x,1,d_grid->dim_x,1);
        thrust::for_each(ghostRowsWindows.begin(),ghostRowsWindows.end(),ghostRowsFunctor());
        thrust::for_each(ghostColsWindows.begin(),ghostColsWindows.end(),ghostColsFunctor());
        thrust::window_vector<int>GOLInputVector(d_grid,3,3,1,1);
        thrust::window_vector<int>GOLOutputVector(d_new_grid,3,3,1,1);
        thrust::transform(thrust::cuda::texture,GOLInputVector.begin(),GOLInputVector.end(),GOLOutputVector.begin(),GOLFunctor());
        // ghostRows<<<cpyGridRowsGridSize, cpyBlockSize>>>(dim, d_grid);
        // ghostCols<<<cpyGridColsGridSize, cpyBlockSize>>>(dim, d_grid);
        // GOL<<<gridSize, blockSize>>>(dim, d_grid, d_newGrid);

        // Swap our grids and iterate again
        d_temp_grid = d_grid;
        d_grid = d_new_grid;
        d_new_grid = d_temp_grid;
    }//iter loop

    // Copy back results and sum
    // hipMemcpy(h_grid, d_grid, bytes, hipMemcpyDeviceToHost);

    // Sum up alive cells and print results
    h_grid = *d_grid;
    int total = 0;
    for (i = 1; i<=dim; i++) {
        for (j = 1; j<=dim; j++) {
           total += h_grid[i][j];
        }
    }
    printf("Total Alive: %d\n", total);

    // Release memory

    return 0;
}
