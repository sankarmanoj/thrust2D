#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/window_2d.h>
#include <thrust/block_2d.h>
#include <thrust/window_transform.h>
#define SRAND_VALUE 1985
#define BLOCK_SIZE 16

class ghostRowsFunctor
{
public:
  __device__ void operator() (const thrust::window_2d<short> &inputWindow) const
  {
      inputWindow[inputWindow.block_dim_y-1][0]=inputWindow[make_int2(0,1)];
      inputWindow[0][0]=inputWindow[make_int2(0,inputWindow.block_dim_y-2)];
  }
};
class ghostColsFunctor
{
public:
  __device__ void operator() (const thrust::window_2d<short> &inputWindow) const
  {
      inputWindow[0][inputWindow.block_dim_x-1]=inputWindow[make_int2(1,0)];
      inputWindow[0][0]=inputWindow[make_int2(inputWindow.block_dim_x-2,0)];
  }
};

class GOLFunctor
{

public:
  __device__ void operator() (hipTextureObject_t texref,thrust::block_2d<short> &output) const
  {
      int numNeighbors;
      // Get the number of neighbors for a given grid point
      numNeighbors = tex2D<short>(texref,blockIdx.x*blockDim.x + threadIdx.x+ 0,threadIdx.y + blockIdx.y*blockDim.y +1)+tex2D<short>(texref,blockIdx.x*blockDim.x + threadIdx.x+ 2,threadIdx.y + blockIdx.y*blockDim.y +1)+
                  tex2D<short>(texref,blockIdx.x*blockDim.x + threadIdx.x+ 1,threadIdx.y + blockIdx.y*blockDim.y +0)+tex2D<short>(texref,blockIdx.x*blockDim.x + threadIdx.x+ 1,threadIdx.y + blockIdx.y*blockDim.y +2)+
                  tex2D<short>(texref,blockIdx.x*blockDim.x + threadIdx.x+ 0,threadIdx.y + blockIdx.y*blockDim.y +0)+tex2D<short>(texref,blockIdx.x*blockDim.x + threadIdx.x+ 2,threadIdx.y + blockIdx.y*blockDim.y +2)+
                  tex2D<short>(texref,blockIdx.x*blockDim.x + threadIdx.x+ 0,threadIdx.y + blockIdx.y*blockDim.y +2)+tex2D<short>(texref,blockIdx.x*blockDim.x + threadIdx.x+ 2,threadIdx.y + blockIdx.y*blockDim.y +0);
      output[threadIdx.y + blockIdx.y*blockDim.y +1][blockIdx.x*blockDim.x + 1]=(numNeighbors==3)||(tex2D<short>(texref,blockIdx.x*blockDim.x + 1,threadIdx.y + blockIdx.y*blockDim.y +1)&&(numNeighbors==2));

  }
};int main(int argc, char* argv[])
{
    int i,j,iter;


    int dim;
    int maxIter = 1<<10; //Number of game steps
    if (argc==2)
     dim = atoi(argv[1]);
    else if (argc==3)
    {
      dim = atoi(argv[1]);
      maxIter = atoi(argv[2]);
    }
    else
     dim = 1024;

    thrust::block_2d<short> *d_grid = new thrust::block_2d<short>(dim+2,dim+2);
    thrust::block_2d<short> *d_new_grid = new thrust::block_2d<short>(dim+2,dim+2);
    thrust::block_2d<short> *d_temp_grid;
    thrust::host_block_2d<short> h_grid(dim+2,dim+2);

    // Assign initial population randomly
    srand(SRAND_VALUE);
    for(i = 1; i<=dim; i++) {
        for(j = 1; j<=dim; j++) {
            h_grid[i][j] = rand() % 2;
            // printf("%d ",h_grid[i][j]);

        }
        // printf("\n");
    }

    // Copy over initial game grid (Dim-1 threads)
    *d_grid = h_grid;




    // Main game loop
    for (iter = 0; iter<maxIter; iter++) {

        thrust::window_vector<short> ghostRowsWindows (d_grid,1,d_grid->dim_y,1,d_grid->dim_y);
        thrust::window_vector<short> ghostColsWindows (d_grid,d_grid->dim_x,1,d_grid->dim_x,1);
        thrust::for_each(ghostRowsWindows.begin(),ghostRowsWindows.end(),ghostRowsFunctor());
        thrust::for_each(ghostColsWindows.begin(),ghostColsWindows.end(),ghostColsFunctor());
        thrust::window_vector<short>GOLInputVector(d_grid,3,3,1,1);
        thrust::window_vector<short>GOLOutputVector(d_new_grid,3,3,1,1);
        thrust::transform(thrust::cuda::texture,GOLInputVector.begin(),GOLInputVector.end(),GOLOutputVector.begin(),GOLFunctor());
        // ghostRows<<<cpyGridRowsGridSize, cpyBlockSize>>>(dim, d_grid);
        // ghostCols<<<cpyGridColsGridSize, cpyBlockSize>>>(dim, d_grid);
        // GOL<<<gridSize, blockSize>>>(dim, d_grid, d_newGrid);

        // Swap our grids and iterate again
        d_temp_grid = d_grid;
        d_grid = d_new_grid;
        d_new_grid = d_temp_grid;
    }//iter loop

    // Copy back results and sum
    // hipMemcpy(h_grid, d_grid, bytes, hipMemcpyDeviceToHost);

    // Sum up alive cells and print results
    printf("\n\nOutput \n\n\n");
    h_grid = *d_grid;
    int total = 0;
    for (i = 1; i<=dim; i++) {
        for (j = 1; j<=dim; j++) {
          // printf("%d ",h_grid[i][j]);
           total += h_grid[i][j];
        }
        // printf("\n");
    }
    printf("Total Alive: %d\n", total);

    // Release memory

    return 0;
}
