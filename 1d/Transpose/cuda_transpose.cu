
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

// Uses shared memory to achieve coalesing in both reads and writes
// Tile width == #banks causes shared memory bank conflicts.
__global__ void transposeCoalesced(int *odata, const int *idata)
{
	__shared__ float tile[TILE_DIM][TILE_DIM+1];
    
	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	int width = gridDim.x * TILE_DIM;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
   	tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

	__syncthreads();

	x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
	y = blockIdx.x * TILE_DIM + threadIdx.y;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
   	odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

const int row = 10240;
const int col = 10240;

int a[row][col];

int main(int argc, char **argv)
{

    int *d_v_i; // Our Device Input Vector
    int *d_v_o; // Our Device Output Vector

	for (int i=0;i<row;i++)
	  for (int j=0; j<col;j++)
		a[i][j] = i*col+j;

	// Allocate And Copy The Memory In To d_v

	hipMalloc ((int**)&d_v_i,(row*col*sizeof(int)));
	hipMalloc ((int**)&d_v_o,(row*col*sizeof(int)));

	hipMemcpy (d_v_i,&a[0][0],row*col*sizeof(int),hipMemcpyHostToDevice);

	// Lets Do The Transpose

	dim3 dimGrid  (row/TILE_DIM, col/TILE_DIM, 1);
  	dim3 dimBlock (TILE_DIM, BLOCK_ROWS, 1);

/*  	const int num_iterations = 1000; // This Is For Profiling The Code To Get An Average Value
  	float elapsedTime = 0.0f;
	float total_time  = 0.0f;

	cudaEvent_t start, stop;

  	for (int i=0;i<num_iterations;i++)
  	{
  		cudaEventCreate(&start);
	    cudaEventCreate(&stop);
	    cudaEventRecord(start);
*/
  		transposeCoalesced<<<dimGrid, dimBlock>>>(d_v_o, d_v_i);
/*
  		cudaEventRecord(stop);
	    cudaEventSynchronize(stop);

	    cudaEventElapsedTime(&elapsedTime, start, stop);

	    total_time += elapsedTime;
  	}

	

	

	printf("Average Elapsed Time Is %lf\n", total_time/num_iterations);
*/

	hipMemcpy (&a[0][0],d_v_o,row*col*sizeof(int),hipMemcpyDeviceToHost);
	printf ("Element at Index 100 is %d\n",a[0][100]);


}

