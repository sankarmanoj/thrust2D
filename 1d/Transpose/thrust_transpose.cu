/* 
* This Example Shows The Matrix Transpose Operation
*/
#include <cstdio>
#include <iostream>
#include <thrust/shared_vector.h>

const int row = 10240;
const int col = 10240;

int a[row][col];

int main ()
{

	thrust::hybrid::shared_vector<int> out (&a[0][0],&a[row-1][col-1]); // Initialised to 0

	for (int i=0;i<row;i++)
	  for (int j=0; j<col;j++)
		a[i][j] = i*col+j;

	thrust::hybrid::shared_vector<int> inp (&a[0][0],&a[row-1][col-1]); // Initialised inp[i] = i

/*	const int num_iterations = 1000; // This Is For Profiling The Code To Get An Average Value
  	float elapsedTime = 0.0f;
	float total_time  = 0.0f;

	hipEvent_t start, stop;
	for (int i=0;i<num_iterations;i++)
	{
		hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventRecord(start);
*/
		thrust::hybrid::transpose (inp.begin(),out.begin(),10240);

/*		hipEventRecord(stop);
	    hipEventSynchronize(stop);

	    hipEventElapsedTime(&elapsedTime, start, stop);

	    total_time += elapsedTime;
	}
		
	printf("Average Elapsed Time Is %lf\n", total_time/num_iterations);
*/
	std::cout<<out[100]<<"\n";

	return 0;
}
