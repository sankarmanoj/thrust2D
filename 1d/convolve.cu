
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define RG 10
#define USECPSEC 1000000ULL
#define nTPB 256
#define DSIZE (32768*1024)
#define FSIZE 17

//cuda error checking macros
#ifdef DEBUG
#define CUDA_CALL(F)  if( (F) != cudaSuccess ) \
  {printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
   __FILE__,__LINE__); exit(-1);}
#define CUDA_CHECK()  if( (cudaPeekAtLastError()) != cudaSuccess ) \
  {printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);}
#else
#define CUDA_CALL(F) (F)
#define CUDA_CHECK()
#endif


typedef float mytype;
// host function to compute convolution reference results
void conv(const mytype *A, const mytype *B, mytype* out, int N, int P) {
// P is assumed to be odd, and greater than 1
    int rb = (P-1)/2;
    int lb = -rb;
    for (int i = rb; i < N-rb; ++i)
        for (int j = lb; j <= rb; ++j)
            out[i] += A[i+j] * B[j+rb];
}
// timing measurement function
unsigned long long dtime_usec(unsigned long long prev){
  timeval tv1;
  gettimeofday(&tv1,0);
  return ((tv1.tv_sec * USECPSEC)+tv1.tv_usec) - prev;
}
// convolution GPU kernel - not using shared memory
// Task 2
__global__ void conv_Kernel(const mytype * __restrict__ A, const mytype * __restrict__ B, mytype *C, const int N, const int P){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    int radius = (P-1)/2;
    if ((idx < (N-radius)) && (idx >= radius)){
      mytype my_sum = 0;
      for (int j = -radius; j <= radius; j++)
        my_sum += A[idx+j]*B[j+radius];
      C[idx] = my_sum;
    }
}

__global__ void conv_shared_Kernel(const mytype * __restrict__ A, const mytype * __restrict__ B, mytype *C, const int N, const int P){
    __shared__ mytype sA[nTPB+FSIZE];
    __shared__ mytype sB[FSIZE];
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    int radius = (P-1)/2;
    int lidx = threadIdx.x + radius;
    if (threadIdx.x < P) sB[threadIdx.x] = B[threadIdx.x];
    if (idx < N){
      sA[lidx] = A[idx];
      if (threadIdx.x < radius) {
        if (idx >= radius)   sA[threadIdx.x] = A[idx - radius];
        if ((idx + nTPB)< N) sA[nTPB + lidx] = A[idx + nTPB];}}
    __syncthreads();
    if ((idx < (N-radius)) && (idx >= radius)){
      mytype my_sum = 0;
      for (int j = -radius; j <= radius; j++)
        my_sum += sA[lidx+j] * sB[j+radius];
      C[idx] = my_sum;
    }
}


int main(int argc, char *argv[]){

  mytype *d_A, *A, *d_B, *B, *d_C, *C, *h_C;
  int my_N = DSIZE;
  int my_P = FSIZE;
// allocate host data
  A   = (mytype *)malloc(my_N*sizeof(mytype));
  B   = (mytype *)malloc(my_P*sizeof(mytype));
  C   = (mytype *)malloc(my_N*sizeof(mytype));
  h_C = (mytype *)malloc(my_N*sizeof(mytype));
// allocate device data
  CUDA_CALL(hipMalloc(&d_A, my_N*sizeof(mytype)));
  CUDA_CALL(hipMalloc(&d_B, my_P*sizeof(mytype)));
  CUDA_CALL(hipMalloc(&d_C, my_N*sizeof(mytype)));
//initialize host input data
  for (int i=0; i < my_N; i++)
    A[i] = rand()%RG;
  for (int i=0; i < my_P; i++)
    B[i] = 1;
//zero out host result data
  for (int i=0; i < my_N; i++){
    C[i]   = 0;
    h_C[i] = 0;}
//begin timing for host reference function
  unsigned long long cpu_time = dtime_usec(0);
  conv(A, B, C, my_N, my_P);
  cpu_time = dtime_usec(cpu_time);
//initialize device result data
  CUDA_CALL(hipMemset(d_C, 0, my_N*sizeof(mytype)));
//begin timing for device function
  unsigned long long gpu_time = dtime_usec(0);
//copy host input data to device
  CUDA_CALL(hipMemcpy(d_A, A, my_N*sizeof(mytype), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_B, B, my_P*sizeof(mytype), hipMemcpyHostToDevice));
//run convolution kernel on GPU
  conv_shared_Kernel<<<(my_N+nTPB-1)/nTPB,nTPB>>>(d_A, d_B, d_C, my_N, my_P);
  CUDA_CHECK();
//copy results from device to host
  CUDA_CALL(hipMemcpy(h_C, d_C, my_N*sizeof(mytype), hipMemcpyDeviceToHost));
  gpu_time = dtime_usec(gpu_time);
//check validity of results
  for (int i = 0; i < my_N; i++) if (C[i] != h_C[i]) {printf("FAIL at %d, cpu: %f, gpu %f\n", i, C[i], h_C[i]); return 1;}
//print timing and speed comparison
  printf("PASS.  cpu time: %ldus, gpu time: %ldus\n", cpu_time, gpu_time);
  printf("Speedup: cpu/gpu = %f\n", cpu_time/(float)gpu_time);
//all host and device allocated data will be implicitly freed at program termination
  return 0;
} 
