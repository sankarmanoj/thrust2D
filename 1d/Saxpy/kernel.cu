
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>



__global__ void saxpy(int N, float a, float *d_x, float *d_y, float *d_z)
{
	
	__shared__ float sm_x[1024];
	__shared__ float sm_y[1024];

	int idx =  blockIdx.x * blockDim.x + threadIdx.x;
	if(idx >=N)
		return;

	sm_x[threadIdx.x] = d_x[idx];
	sm_y[threadIdx.x] = d_y[idx];
	__syncthreads();

	d_z[idx] = a*sm_x[threadIdx.x] + sm_y[threadIdx.x];
	__syncthreads();

	d_z[idx] +=  + d_y[idx];
}

int main () 
{

	int N = 15485863;

	float *h_x, *h_y, *h_z;
	float *d_x, *d_y, *d_z;
	
	h_x = (float*) malloc (N * sizeof(float));
	h_y = (float*) malloc (N * sizeof(float));
	h_z = (float*) malloc (N * sizeof(float));
	
	hipMalloc ((float**) &d_x, N * sizeof(float));
	hipMalloc ((float**) &d_y, N * sizeof(float));
	hipMalloc ((float**) &d_z, N * sizeof(float));

	for(int i = 0; i < N; i++)
	{
		h_x[i] = 2.0f*(float)i;
		h_y[i] = 3.0f*(float)i;
	}
	
	hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemset(d_z, 0, N * sizeof(float));

	int threads_block = 1024;
	int num_blocks = (N/threads_block) + (N%threads_block == 0?0:1);

	saxpy <<< num_blocks, threads_block>>> (N, 9.0, d_x, d_y, d_z);

	hipMemcpy(h_z, d_z, N * sizeof(float), hipMemcpyDeviceToHost);

	printf("Element in %d position is %f\n",100, h_z[100]);
	
	hipFree (d_x);	
	hipFree (d_y);
	hipFree (d_z);

	free (h_x);	
	free (h_y);	
	free (h_z);
	hipDeviceReset();
	return 0;
}