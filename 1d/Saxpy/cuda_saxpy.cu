
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

__global__
void saxpy (int N,float a,float *d_x,float *d_y)
{	
	__shared__ float sm_x[1024];
	__shared__ float sm_y[1024];
			
	int idx =  blockIdx.x * blockDim.x + threadIdx.x;
	int num_threads = gridDim.x * blockDim.x;
	int id = threadIdx.x;
	
	for(int i = idx; i < N; i += num_threads) {
	
		sm_x[id] = d_x[i];
		sm_y[id] = d_y[i];
		
		sm_y[id] += sm_x[id] * a;
		
		d_y[i] = sm_y[id];	
	}	
}


int main () {

	int N = 15485863;

	float *d_x, *d_y;
	float *h_x, *h_y;
	
	h_x = (float*) malloc (N * sizeof(float));
	h_y = (float*) malloc (N * sizeof(float));
	
	hipMalloc ((float**) &d_x, N * sizeof(float));
	hipMalloc ((float**) &d_y, N * sizeof(float));
	
	for(int i = 0; i < N; i++)
	{
		h_x[i] = 2.0;
		h_y[i] = 3.0;
	}
	
	hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);


	saxpy <<< 26, 1024 >>> (N, 9.0, d_x,d_y);

	hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

	printf("%lf\n", h_y[100]);
	
	hipFree (d_x);	
	hipFree (d_y);
	free (h_x);	
	free (h_y);	

	hipDeviceReset();
	return 0;
}

