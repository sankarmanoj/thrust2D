#include "hip/hip_runtime.h"
/*
* This Exmaple Shows The Working Of shared_vector With Thrust Code Without Breaking The Overloadings
* The shared_vector And thrust::transform Is Our Code
* The device_vector Is Using The Built-In Functions Of Thrust like fill,transform etc.
*/

#include <cstdio>
#include <iostream>
#include <vector>
#include <thrust/shared_vector.h>

namespace thrust
{

	namespace hybrid
	{
		 __device__
		void synchronize_device ()
		{
			__syncthreads ();
		}

		__host__  __device__  
		void synchronize_host ()
		{
			;
		}

		__device__
		void synchronize ()
		{
			#if __BULK_HAS_CUDART__
			  synchronize_device ();
			#else
			  synchronize_host ();
			#endif
		}
	}
}

class my_functor
{
	float a;
	public :

		my_functor(float b)
		{
			a = b;
		}


		
		__device__ float operator() (float x, float y)
		{

			float z;

			thrust::hybrid::synchronize ();

			z = a*x + y;

			thrust::hybrid::synchronize ();

			z += y;

			return z;
		}
};

int main ()
{
	int N = 15485863;

	my_functor functor(9.0);

	std::vector<float> a(N);	
	std::vector<float> b(N);

	for (int i=0;i<N;i++)
	{
		a[i] = 2.0f * (float)i;
		b[i] = 3.0f * (float)i;
	}	

	thrust::hybrid::shared_vector<float> d1 (a.begin(),a.end());
	thrust::hybrid::shared_vector<float> d2 (b.begin(),b.end());

	thrust::hybrid::transform (d1.begin (),d1.end (),d2.begin (),d2.begin (),functor);
		
	std::cout<<"Element at position 100 is "<<d2[100]<<"\n";

	return 0;
}


