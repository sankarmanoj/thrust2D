
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__
void reduce_kernel (int *start,int *output,int N)
{

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = gridDim.x * blockDim.x;

    extern __shared__ int partial_sum[]; // Dynamic Shared Memory

    int sum = 0; // For Generic Function, Replace By Idendity Element

    for (int i=idx;i<N;i+=num_threads)
        sum += start[i]; // For Generic Function, Replace By Functor

    partial_sum[threadIdx.x] = sum;

    __syncthreads ();

    // This Step Computes The Reduction Function In A Bottom's Up Manner

    for (int activeThreads = blockDim.x>>1;activeThreads != 0;activeThreads >>= 1)
    {
        if (threadIdx.x < activeThreads)
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x+activeThreads]; // For Generic Function, Replace += with functor

        __syncthreads ();
    }

    // When You Reach The Root Of The Tree Store It In Output Vector
    if (threadIdx.x == 0)
        output[blockIdx.x] = partial_sum[0];
}

int a[1<<26];
int main ()
{
    int *d_v,*partial_output,*d_answer;
    int *answer;
    int N = 1<<26;

    for (int i=0;i<N;i++)
        a[i] = 1;

    hipMalloc ((int**)&d_v,N*sizeof(int));

    hipMalloc ((int**)&d_answer,sizeof(int));

    hipMemcpy (d_v,a,N*sizeof(int),hipMemcpyHostToDevice);

    const int numBlocks = 26;
    const int numThreads = 1024;    
    const int sharedSize = numThreads*sizeof (int);

    answer = (int*)malloc (sizeof(int));
    *answer = 0;

/*   const int num_iterations = 1000; // This Is For Profiling The Code To Get An Average Value
    float elapsedTime = 0.0f;
    float total_time  = 0.0f;
    cudaEvent_t start, stop;

    for (int i=0;i<num_iterations;i++)
    {
        cudaEventCreate(&start);
        cudaEventCreate(&stop);
        cudaEventRecord(start);

        
*/      
        hipMalloc ((int**)&partial_output,N*sizeof(int));
      
        reduce_kernel<<<numBlocks,numThreads,sharedSize>>> (d_v,partial_output,N);
        reduce_kernel<<<1,numThreads,sharedSize>>> (partial_output,d_answer,numBlocks);

        hipMemcpy (answer,d_answer,sizeof(int),hipMemcpyDeviceToHost);
        hipFree (partial_output);
/*
        cudaEventRecord(stop);
        cudaEventSynchronize(stop);

        cudaEventElapsedTime(&elapsedTime, start, stop);

        total_time += elapsedTime;


    }

    printf("Average Elapsed Time Is %lf\n", total_time/num_iterations);
*/
    printf ("Answer is %d\n",*answer);

    return 0;
}
