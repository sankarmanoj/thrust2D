/* 
* This Example Show The Working Of Reduce Operation
*/

#include <thrust/device_vector.h>
#include <vector>

int main ()
{
	int N = 1<<26;
	int answer;
	std::vector<int> a(N);

	for (int i=0;i<N;i++)
		a[i] = 1;

	thrust::device_vector<int> d(a.begin(),a.end());

/*	const int num_iterations = 1000; // This Is For Profiling The Code To Get An Average Value
	float elapsedTime = 0.0f;
	float total_time  = 0.0f;

	hipEvent_t start, stop;
    
	for (int i=0;i<num_iterations;i++)
	{
		hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventRecord(start);
*/

		answer = thrust::reduce (d.begin(),d.end());

/*		hipEventRecord(stop);
	    hipEventSynchronize(stop);

	    hipEventElapsedTime(&elapsedTime, start, stop);

	    total_time += elapsedTime;
	}

    printf("Average Elapsed Time Is %lf\n", total_time/num_iterations);
*/
	printf ("Answer is %d\n",answer);

	return 0;
}
