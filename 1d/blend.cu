#include <opencv2/opencv.hpp>
#include <thrust/device_vector.h>
#include <thrust/shared_for_each.h>
using namespace cv;
class blendFunctor
{
  float alpha;

public:
  blendFunctor(float alpha)
  {
    this->alpha = alpha;
  }
  __device__ uchar operator() (uchar &input1,uchar &input2) const
  {
    return alpha * input1+ (1-alpha) *  input2;
  }
};

int main(int argc, char const *argv[]) {
  int dim = 512;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat input1 = imread("../opencv/shared/santiago.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat input2 = imread("../opencv/shared/car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat temp1;
  resize(input1,temp1,Size(dim,dim));
  input1 = temp1;
  Mat temp2;
  resize(input2,temp2,Size(dim,dim));
  input2 = temp2;

  thrust::device_vector<uchar>input_vector1(input1.ptr(),input1.ptr()+input1.cols*input1.rows);
  thrust::device_vector<uchar>input_vector2(input2.ptr(),input2.ptr()+input2.cols*input2.rows);
  thrust::device_vector<uchar>output_vector(input1.cols*input1.rows);
  thrust::transform(thrust::cuda::shared,input_vector1.begin(),input_vector1.end(),input_vector2.begin(),output_vector.begin(),blendFunctor(0.3));
  thrust::host_vector<uchar>host_output_vector(input1.cols*input1.rows);
  host_output_vector = output_vector;
  Mat output (Size(input1.cols,input1.rows),CV_8UC1,host_output_vector.data());
  #ifdef OWRITE
  imwrite("blend-input1.png",input1);
  imwrite("blend-input2.png",input2);
  imwrite("blend-output.png",output);
  #endif
  #ifdef SHOW
  imshow("blend-input1.png",input1);
  imshow("blend-input2.png",input2);
  imshow("blend-output.png",output);
  waitKey(0);
  #endif
  return 0;
}
