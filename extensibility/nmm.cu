
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <iostream>
 void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
     // Create a pseudo-random number generator
     hiprandGenerator_t prng;
     hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

     // Set the seed for the random number generator using the system clock
     hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

     // Fill the array with random numbers on the device
     hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
 }
 void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
       int lda=m,ldb=k,ldc=m;
       const float alf = 1;
      const float bet = 0;
      const float *alpha = &alf;
      const float *beta = &bet;

      // Create a handle for CUBLAS
      hipblasHandle_t handle;
      hipblasCreate(&handle);

      // Do the actual multiplication
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

      // Destroy the handle
      hipblasDestroy(handle);
  }
 int main(int argc, char **argv) {
     // Allocate 3 arrays on CPU
     int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

     if (argc ==2)
     nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = atoi(argv[1]);
     else
     nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 100;

     float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
     float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
    float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

     // Allocate 3 arrays on GPU
     float *d_A, *d_B, *d_C;
     hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
     hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
     hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

     // Fill the arrays A and B on GPU with random numbers
     GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
     GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

     // Optionally we can copy the data back on CPU and print the arrays
     hipMemcpy(h_A,d_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost);
     hipMemcpy(h_B,d_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyDeviceToHost);


     // Multiply A and B on GPU
     gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

   // Copy (and print) the result on host memory
   hipMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost);

     //Free GPU memory
     hipFree(d_A);
     hipFree(d_B);
   hipFree(d_C);

     // Free CPU memory
     free(h_A);
 free(h_B);
     free(h_C);

     return 0;
 }
