#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <iostream>
#include <thrust/block_2d.h>
#include <thrust/sequence.h>
//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
using namespace thrust;
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

  for(int i = 0; i < nr_rows_A; ++i){
    for(int j = 0; j < nr_cols_A; ++j){
      std::cout << A[j * nr_rows_A + i] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}
class pointwise_product
{
public:
  float div;
  pointwise_product(float div)
  {
    this->div = div;
  }
  __device__ hipfftComplex operator() (const hipfftComplex &a, const hipfftComplex &b) const
  {
    hipfftComplex c;
    c.x = a.x*b.x - a.y*b.y;
    c.y = a.x*b.y + a.y*b.x;
    c.x *= div;
    c.y *= div;
    return c;
  }
};
void fft_convolve(block_2d<float> *x, block_2d<float> *h, block_2d<float> *y)
{
  assert(x->dim_x == h->dim_x);
  assert(x->dim_y == h->dim_y);
  hipfftReal* d_inA, *d_inB;
  hipfftComplex* d_outA, *d_outB;

  size_t real_size = x->dim_x * x->dim_y * sizeof(hipfftReal);
  size_t complex_size = x->dim_x * (x->dim_y/2+1) * sizeof(hipfftComplex);

  hipMalloc((void**)&d_inA, real_size);
  hipMalloc((void**)&d_inB, real_size);

  hipMalloc((void**)&d_outA, complex_size);
  hipMalloc((void**)&d_outB, complex_size);

  hipMemset(d_inA,0,real_size);
  hipMemset(d_inB,0,real_size);

  hipMemcpy2D(d_inA,0, x->data_pointer,x->pitch, x->dim_x*sizeof(hipfftReal),x->dim_y, hipMemcpyHostToDevice);
  hipMemcpy2D(d_inB,0, h->data_pointer,h->pitch, h->dim_x*sizeof(hipfftReal),h->dim_y, hipMemcpyHostToDevice);


  hipfftHandle fwplanA, fwplanB, bwplan;
  hipfftPlan2d(&fwplanA, x->dim_y, x->dim_x, HIPFFT_R2C);
  hipfftPlan2d(&fwplanB, x->dim_y, x->dim_x, HIPFFT_R2C);
  hipfftPlan2d(&bwplan, x->dim_y, x->dim_x, HIPFFT_C2R);

  // cufftSetCompatibilityMode(fwplanA,CUFFT_COMPATIBILITY_NATIVE);
  // cufftSetCompatibilityMode(fwplanB,CUFFT_COMPATIBILITY_NATIVE);
  // cufftSetCompatibilityMode(bwplan,CUFFT_COMPATIBILITY_NATIVE);

  hipfftExecR2C(fwplanA, d_inA, d_outA);
  hipfftExecR2C(fwplanB, d_inB, d_outB);

  int blocksx = ceil((x->dim_x*(x->dim_y/2+1 )) / 256.0f);
  dim3 threads(256);
  dim3 grid(blocksx);
  // One complex product for each thread, scaled by the inverse of the
  // number of elements involved in the FFT
  thrust::transform(d_outA,d_outA + (x->dim_x * (x->dim_y/2+1)),d_outB,d_outA,pointwise_product(1.0f/(x->dim_x*x->dim_y)));
  hipfftExecC2R(bwplan, d_outA, d_inA);
  hipMemcpy2D(y->data_pointer,y->pitch, d_inA,0, y->dim_x*sizeof(hipfftReal),y->dim_y,hipMemcpyDeviceToHost);
}
int main()
{
  // Allocate 3 arrays on CPU
  int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

  // for simplicity we are going to use square arrays
  nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;

  float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
  float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
  float *h_C = (float *)malloc((nr_rows_C/2+1) * nr_cols_C*2 * sizeof(float));

  // Allocate 3 arrays on GPU
  thrust::block_2d<float> d_A(nr_rows_A,nr_cols_A),d_B(nr_rows_B,nr_cols_B),d_C((nr_rows_C/2+1),nr_cols_C*2);

  thrust::sequence(d_A.begin(),d_A.end());
  thrust::sequence(d_B.begin(),d_B.end());

  // Optionally we can copy the data back on CPU and print the arrays
  d_A.download(&h_A);
  d_B.download(&h_B);
  std::cout << "A =" << std::endl;
  print_matrix(h_A, nr_rows_A, nr_cols_A);
  std::cout << "B =" << std::endl;
  print_matrix(h_B, nr_rows_B, nr_cols_B);

  // Multiply A and B on GPU
  fft_convolve(&d_A, &d_B, &d_C);

  // Copy (and print) the result on host memory
  d_C.download(&h_C);
  std::cout << "C =" << std::endl;
  print_matrix(h_C, nr_rows_C, nr_cols_C);

  free(h_A);
  free(h_B);
  free(h_C);
  return 0;
}
