#include <hipblas.h>
#include <iostream>
#include <thrust/block_2d.h>
#include <thrust/sequence.h>
void gpu_blas_mmul(hipblasHandle_t &handle, thrust::block_2d<float> *A, thrust::block_2d<float> *B, thrust::block_2d<float> *C, const int m, const int k, const int n) {
  int lda=A->pitch/sizeof(float),ldb=B->pitch/sizeof(float),ldc=C->pitch/sizeof(float);
  const float alf = 1;
  const float bet = 0;
  const float *alpha = &alf;
  const float *beta = &bet;

  // Do the actual multiplication
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A->data_pointer, lda, B->data_pointer, ldb, beta, C->data_pointer, ldc);
}
//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

  for(int i = 0; i < nr_rows_A; ++i){
    for(int j = 0; j < nr_cols_A; ++j){
      std::cout << A[j * nr_rows_A + i] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}
int main()
{
  // Allocate 3 arrays on CPU
  int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

  // for simplicity we are going to use square arrays
  nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 100;

  float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
  float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
  float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

  // Allocate 3 arrays on GPU
  thrust::block_2d<float> d_A(nr_rows_A,nr_cols_A),d_B(nr_rows_B,nr_cols_B),d_C(nr_rows_C,nr_cols_C);

  thrust::sequence(d_A.begin(),d_A.end());
  thrust::sequence(d_B.begin(),d_B.end());

  // Optionally we can copy the data back on CPU and print the arrays
  d_A.download(&h_A);
  d_B.download(&h_B);
  std::cout << "A =" << std::endl;
  print_matrix(h_A, nr_rows_A, nr_cols_A);
  std::cout << "B =" << std::endl;
  print_matrix(h_B, nr_rows_B, nr_cols_B);

  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Multiply A and B on GPU
  gpu_blas_mmul(handle, &d_A, &d_B, &d_C, nr_rows_A, nr_cols_A, nr_cols_B);

  // Copy (and print) the result on host memory
  d_C.download(&h_C);
  std::cout << "C =" << std::endl;
  print_matrix(h_C, nr_rows_C, nr_cols_C);

  // //Free GPU memory
  // hipFree(d_A);
  // hipFree(d_B);
  // hipFree(d_C);
  // Free CPU memory
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
