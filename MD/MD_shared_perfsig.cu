#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/shared_algorithms.h>
#include <stdio.h>
#include "cputime.h"

// float *accnew_gpu;
// float *velnew_gpu;
// float *parforce_gpu;
// float *parpot_gpu;
// float *parvel_gpu;
// float *acc_gpu;
// float *force_gpu;
// float *pos_gpu;
// float *vel_gpu;


thrust::device_vector<float> *parforce_gpuX;
thrust::device_vector<float> *parforce_gpuY;
thrust::device_vector<float> *parforce_gpuZ;

thrust::device_vector<float> *parpot_gpu;

thrust::device_vector<float> *acc_gpuX;
thrust::device_vector<float> *acc_gpuY;
thrust::device_vector<float> *acc_gpuZ;

thrust::device_vector<float> *force_gpuX;
thrust::device_vector<float> *force_gpuY;
thrust::device_vector<float> *force_gpuZ;


thrust::device_vector<float> *pos_gpuX;
thrust::device_vector<float> *pos_gpuY;
thrust::device_vector<float> *pos_gpuZ;

thrust::device_vector<float> *vel_gpuX;
thrust::device_vector<float> *vel_gpuY;
thrust::device_vector<float> *vel_gpuZ;


/*
extern "C"
double cputime()
{
	struct timeval tp;
	int rtn;
	rtn=gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec+(1.e-6)*tp.tv_usec);
}
*/

extern "C"
void allocMemOnGPU(int nd, int np)
{


	parforce_gpuX = (new thrust::device_vector<float>(np));
	parforce_gpuY = (new thrust::device_vector<float>(np));
	parforce_gpuZ = (new thrust::device_vector<float>(np));

	parpot_gpu = (new thrust::device_vector<float>(np) );

  acc_gpuX = (new thrust::device_vector<float>(np));
  acc_gpuY = (new thrust::device_vector<float>(np));
  acc_gpuZ = (new thrust::device_vector<float>(np));

  force_gpuX = (new thrust::device_vector<float>(np));
  force_gpuY = (new thrust::device_vector<float>(np));
  force_gpuZ = (new thrust::device_vector<float>(np));


	pos_gpuX = (new thrust::device_vector<float>(np));
  pos_gpuY = (new thrust::device_vector<float>(np));
  pos_gpuZ = (new thrust::device_vector<float>(np));

  vel_gpuX = (new thrust::device_vector<float>(np));
  vel_gpuY = (new thrust::device_vector<float>(np));
  vel_gpuZ = (new thrust::device_vector<float>(np));

}

extern "C"
void copyDataToGPU(float *h_acc, float *h_force, float *h_vel, float *h_pos, int nd, int np)
{
  acc_gpuX->assign( h_acc        ,h_acc + np);
  acc_gpuY->assign( h_acc +np    ,h_acc + 2*np);
  acc_gpuZ->assign( h_acc +2*np  ,h_acc + nd*np);

  force_gpuX->assign( h_force        ,h_force + np);
  force_gpuY->assign( h_force +np    ,h_force + 2*np);
  force_gpuZ->assign( h_force +2*np  ,h_force + nd*np);

  vel_gpuX->assign( h_vel        ,h_vel + np);
  vel_gpuY->assign( h_vel +np    ,h_vel + 2*np);
  vel_gpuZ->assign( h_vel +2*np  ,h_vel + nd*np);

	pos_gpuX->assign( h_pos        ,h_pos + np);
  pos_gpuY->assign( h_pos +np    ,h_pos + 2*np);
  pos_gpuZ->assign( h_pos +2*np  ,h_pos + nd*np);

}


// __global__ void dummyCopy(float *g_idata, float *g_odata)
// {
// 	int idx = blockIdx.x * blockDim.x + threadIdx.x ;
//
// 	g_odata[idx] = g_idata[idx];
// 	__syncthreads();
// }


// START K1 - Compute Force on Particle

class compute_forceonparticle_functor
{
  float PI2;
  float currentposx,currentposy,currentposz;
  int currentMoleculeIndex;
public:
  compute_forceonparticle_functor(float PI2,  float currentposx, float currentposy, float currentposz,int currentMoleculeIndex)
  {
    this->PI2 = PI2;
    this->currentposx = currentposx;
    this->currentposy = currentposy;
    this->currentposz = currentposz;
    this->currentMoleculeIndex = currentMoleculeIndex;
  }
  template <typename TupleInput, typename TupleOutput>
  __device__ float operator() (TupleInput input, TupleOutput output)
  {
    float px = this->currentposx - thrust::get<0>(input);
    float py = this->currentposy - thrust::get<1>(input);
    float pz = this->currentposz - thrust::get<2>(input);
    float dist = px*px + py*py + pz*pz;
    dist = sqrt(dist);
    float dist2 = (dist < PI2) ? dist : PI2;
    if(thrust::get<3>(input)==this->currentMoleculeIndex)
    {
        thrust::get<0>(output)= 0;
        thrust::get<1>(output)= 0;
        thrust::get<2>(output)= 0;
        return 0;
    }
    thrust::get<0>(output)=- (px * sin(2.0 * dist2) / dist);
    thrust::get<1>(output)=- (py * sin(2.0 * dist2) / dist);
    thrust::get<2>(output)=- (pz * sin(2.0 * dist2) / dist);
  	return 0.5 * sin(dist2) * sin(dist2);

  }

};
extern "C"
void GPU_compute_forceonparticle(int nd, int np, int currentMoleculeIndex, const float PI2, int step, double *time_elapsed)
{

thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(pos_gpuX->begin(),pos_gpuY->begin(),pos_gpuZ->begin(),thrust::counting_iterator<int>(0))),
                  thrust::make_zip_iterator(thrust::make_tuple(pos_gpuX->end(),pos_gpuY->end(),pos_gpuZ->end(),thrust::counting_iterator<int>(pos_gpuX->end()-pos_gpuX->begin()))),
                  thrust::make_zip_iterator(thrust::make_tuple(parforce_gpuX->begin(),parforce_gpuY->begin(),parforce_gpuZ->begin())),
                  parpot_gpu->begin(),compute_forceonparticle_functor(PI2,(*pos_gpuX)[currentMoleculeIndex],(*pos_gpuY)[currentMoleculeIndex],(*pos_gpuZ)[currentMoleculeIndex],currentMoleculeIndex));
// if(currentMoleculeIndex<100)
// printf("%f \n",(float) parforce_gpuX[0]);
}



float GPU_accumulate_parpot_wShrdMem(int nd, int np, int step, double *time_elapsed)
{
	return thrust::reduce(parpot_gpu->begin(),parpot_gpu->end());
}

//END K2 - Accumulate PE with/without shared memory


//START K3 - Accumulate Force with/without shared memory
void GPU_accumulate_parforce_wShrdMem(int nd, int np, int currentMoleculeIndex, int step, double *time_elapsed)
{
    (*force_gpuX)[currentMoleculeIndex] = thrust::reduce(thrust::cuda::shared,parforce_gpuX->begin(),parforce_gpuX->end());
    // printf("%f\n",(float) force_gpuX[currentMoleculeIndex]);
    (*force_gpuY)[currentMoleculeIndex] = thrust::reduce(thrust::cuda::shared,parforce_gpuY->begin(),parforce_gpuY->end());
    (*force_gpuZ)[currentMoleculeIndex] = thrust::reduce(thrust::cuda::shared,parforce_gpuZ->begin(),parforce_gpuZ->end());

}
//END K3 - Accumulate Force with/without shared memory

//Accumulates PE and Force using K2 and K3
extern "C"
float GPU_seq_wShrdMem_accumulate_parpot_and_parforce(int nd, int np, int currentMoleculeIndex, int step, double *time_elap1, double *time_elap2)
{
	GPU_accumulate_parforce_wShrdMem(nd,np,currentMoleculeIndex, step, time_elap1);
	return GPU_accumulate_parpot_wShrdMem(nd, np, step, time_elap2);
}

//START K4 - Compute and accumulate KE without shared memory
//Compute KE with shared memory

struct squareOp
{
  __device__ float operator() ( const float input) const
  {
    return input*input;
  }
};

extern "C"
float GPU_accumulate_KE_wShrdMem(int nd, int np, float mass, int step, double *time_elapsed)
{
  // for(int i = 0; i<100;i++)
  // {
  //   printf("%f \n",(float) vel_gpuX[i]);
  // }

    float sum = thrust::transform_reduce(thrust::cuda::shared,vel_gpuX->begin(),vel_gpuX->end(),squareOp())
              + thrust::transform_reduce(thrust::cuda::shared,vel_gpuY->begin(),vel_gpuY->end(),squareOp())
              + thrust::transform_reduce(thrust::cuda::shared,vel_gpuZ->begin(),vel_gpuZ->end(),squareOp());
	return 0.5 * mass * sum;
}

//END K4 - Compute and accumulate KE with shared memory


//START K5 - Update position
__global__ void GPU_updatePos(int numberOfThreads, float dt, float * pos_gpu, const float * vel_gpu, const float * acc_gpu)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x ;
	if (idx >= numberOfThreads)
		return ;
	pos_gpu[idx] += vel_gpu[idx] * dt + 0.5 * acc_gpu[idx] * dt * dt;
}

class updateOp
{
  float dt;
public:
  updateOp(float dt)
  {
    this->dt = dt;
  }
  template <typename Tuple>
  __device__ float operator() (Tuple input, float current_value)
  {
    return thrust::get<0>(input) * (this->dt) + (this->dt) * 0.5 * (this->dt) * thrust::get<1>(input) + current_value;
  }
};
extern "C"
void GPU_updatePos(int nd, int np, float dt, int step, double *time_elapsedCPU, float *time_elapsedGPU)
{

	thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(vel_gpuX->begin(),acc_gpuX->begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(vel_gpuX->end(),acc_gpuX->end())),
                    pos_gpuX->begin(),pos_gpuX->begin(),updateOp(dt));
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(vel_gpuY->begin(),acc_gpuY->begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(vel_gpuY->end(),acc_gpuY->end())),
                    pos_gpuY->begin(),pos_gpuY->begin(),updateOp(dt));
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(vel_gpuZ->begin(),acc_gpuZ->begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(vel_gpuZ->end(),acc_gpuZ->end())),
                    pos_gpuZ->begin(),pos_gpuZ->begin(),updateOp(dt));


}
//END K5 - Update position


//START K6 - Update velocity
__global__ void GPU_updateVel(int numberOfThreads, float dt, float rmass, float * vel_gpu, const float * force_gpu, const float * acc_gpu)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x ;
	if (idx >= numberOfThreads)
		return ;
	vel_gpu[idx] += 0.5 * dt * (force_gpu[idx] * rmass + acc_gpu[idx]);
}

class updateVelOp
{
  float dt, rmass;
public:
  updateVelOp(float dt, float rmass)
  {
    this->dt = dt;
    this->rmass =rmass;
  }
  template <typename Tuple>
  __device__ float operator() (Tuple input, float current_value)
  {
    return current_value + ( 0.5*(this->dt)*(thrust::get<0>(input)*(this->rmass) + thrust::get<1>(input)) );
  }
};
extern "C"
void GPU_updateVel(int nd, int np, float dt, float rmass, int step, double *time_elapsedCPU, float *time_elapsedGPU)
{
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(force_gpuX->begin(),acc_gpuX->begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(force_gpuX->end(),acc_gpuX->end())),
                    vel_gpuX->begin(),vel_gpuX->begin(),updateVelOp(dt,rmass));
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(force_gpuY->begin(),acc_gpuY->begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(force_gpuY->end(),acc_gpuY->end())),
                    vel_gpuY->begin(),vel_gpuY->begin(),updateVelOp(dt,rmass));
  thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(force_gpuZ->begin(),acc_gpuZ->begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(force_gpuZ->end(),acc_gpuZ->end())),
                    vel_gpuZ->begin(),vel_gpuZ->begin(),updateVelOp(dt,rmass));
}
//END K6 - Update velocity


//START K7 - Update acceleration

class updateAccOp
{
  float rmass;
public:
  updateAccOp(float rmass)
  {
    this->rmass = rmass;
  }
  __device__ float operator() (float input)
  {
    return input*(this->rmass);
  }
};

extern "C"
void GPU_updateAcc(int nd, int np, float rmass, int step, double *time_elapsedCPU, float *time_elapsedGPU)
{

  thrust::transform(force_gpuX->begin(),force_gpuX->end(),acc_gpuX->begin(),updateAccOp(rmass));
  thrust::transform(force_gpuY->begin(),force_gpuY->end(),acc_gpuY->begin(),updateAccOp(rmass));
  thrust::transform(force_gpuZ->begin(),force_gpuZ->end(),acc_gpuZ->begin(),updateAccOp(rmass));

}
//END K7 - Update acceleration
