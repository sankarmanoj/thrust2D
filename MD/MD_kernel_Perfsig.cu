#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "cputime.h"

// float *accnew_gpu;
// float *velnew_gpu;
float *parforce_gpu;
float *parpot_gpu;
float *parvel_gpu;
float *acc_gpu;
float *force_gpu;
float *pos_gpu;
float *vel_gpu;

/*
extern "C"
double cputime()
{
	struct timeval tp;
	int rtn;
	rtn=gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec+(1.e-6)*tp.tv_usec);
}
*/

extern "C"
void allocMemOnGPU(int nd, int np)
{
	// hipMalloc ((void**)(&accnew_gpu), nd*np*sizeof(float));
	// hipMalloc ((void**)(&velnew_gpu), nd*np*sizeof(float));
	hipMalloc ((void**)(&parforce_gpu), nd*np*sizeof(float));
	hipMalloc ((void**)(&parpot_gpu), np*sizeof(float));
	hipMalloc ((void**)(&parvel_gpu), nd*np*sizeof(float));
	hipMalloc ((void**)(&acc_gpu), nd*np*sizeof(float));
	hipMalloc ((void**)(&force_gpu), nd*np*sizeof(float));
	hipMalloc ((void**)(&pos_gpu), nd*np*sizeof(float));
	hipMalloc ((void**)(&vel_gpu), nd*np*sizeof(float));
}

extern "C"
void copyDataToGPU(float *h_acc, float *h_force, float *h_vel, float *h_pos, int nd, int np)
{
	hipMemcpy(acc_gpu, h_acc, nd*np*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(force_gpu, h_force, nd*np*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(vel_gpu, h_vel, nd*np*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pos_gpu, h_pos, nd*np*sizeof(float), hipMemcpyHostToDevice);
}


__global__ void dummyCopy(float *g_idata, float *g_odata)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x ;

	g_odata[idx] = g_idata[idx];
	__syncthreads();
}

// START K1 - Compute Force on Particle
__global__ void GPU_compute_forceonparticle_KERNEL(int np, int numberOfThreads, int currentMoleculeIndex, const float PI2,
													float * pos_gpu, float * parforce_gpu, float * parpot_gpu)
{
	float posx, posy, posz, dist, dist2;
    __shared__ float currentposx,currentposy,currentposz;

    int idx = blockIdx.x * blockDim.x + threadIdx.x ;

    if (idx >= numberOfThreads) return ;

    posx = pos_gpu[idx];
    posy = pos_gpu[idx+np];
    posz = pos_gpu[idx+np+np];
    currentposx = pos_gpu[currentMoleculeIndex];
    currentposy = pos_gpu[currentMoleculeIndex+np];
    currentposz = pos_gpu[currentMoleculeIndex+np+np];

    posx = currentposx - posx;
    posy = currentposy - posy;
    posz = currentposz - posz;

    dist = posx*posx + posy*posy + posz*posz;
    dist = sqrt(dist);


    dist2 = (dist < PI2) ? dist : PI2;

    if(idx==currentMoleculeIndex){
			posx=0.0;
    	posy=0.0;
    	posz=0.0;
    	dist=0.0;
    }else{
    	posx= - (posx * sin(2.0 * dist2) / dist);
    	posy= - (posy * sin(2.0 * dist2) / dist);
    	posz= - (posz * sin(2.0 * dist2) / dist);
    	dist= 0.5 * sin(dist2) * sin(dist2);
    }

    parforce_gpu[idx] = posx;
    parforce_gpu[idx+np] = posy;
    parforce_gpu[idx+np+np] = posz;
    parpot_gpu[idx] = dist;
}


extern "C"
void GPU_compute_forceonparticle(int nd, int np, int currentMoleculeIndex, const float PI2, int step, double *time_elapsed)
{
	int BLOCK_SIZE = 128;
	int numberOfThreads = np;
	int numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;

	dim3 dimGrid(numBlocks) ;
	dim3 dimBlock(BLOCK_SIZE) ;


/*	if(step == 4)
	{
		for(i=0; i<iter; i++)
		{
			//printf("Molecule index is %d in iter %d\n",currentMoleculeIndex, i);
			t0=cputime();
			GPU_compute_forceonparticle_KERNEL<<<dimGrid,dimBlock>>>(np, numberOfThreads, currentMoleculeIndex, PI2, pos_gpu, parforce_gpu, parpot_gpu);
			hipDeviceSynchronize();
			t1=cputime();
			time_sum += (t1-t0)*iCPS*1000*1000;
		}
		*time_elapsed = time_sum/iter;
	}
	else
*/	{
		//t0=cputime();
		//printf("Molecule index is %d in step %d\n", currentMoleculeIndex, step);
		GPU_compute_forceonparticle_KERNEL<<<dimGrid,dimBlock>>>(np, numberOfThreads, currentMoleculeIndex, PI2, pos_gpu, parforce_gpu, parpot_gpu);
		hipDeviceSynchronize();
		//t1=cputime();
		//*time_elapsed = (t1-t0);
	}
}

//END K1 - Compute Force on Particle

//START - GENERAL REDUCTION KERNEL THAT USES SHARED MEMORY
template<class T>
struct SharedMemory
{
    __device__ inline operator       T*()
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }

    __device__ inline operator const T*() const
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }
};

template <class T>
__global__ void reduce_wShrdMem(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;

    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
	__syncthreads();
}
//END - GENERAL REDUCTION KERNEL THAT USES SHARED MEMORY

float GPU_accumulate_parpot_wShrdMem(int nd, int np, int step, double *time_elapsed)
{
	int BLOCK_SIZE = 128;
	int numberOfThreads = np;
	int numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;
    	float *parpot, sum;

	dim3 dimBlock = dim3(BLOCK_SIZE, 1, 1);
	dim3 dimGrid;
	int smemSize = (BLOCK_SIZE <= 32) ? 2 * BLOCK_SIZE * sizeof(float) : BLOCK_SIZE * sizeof(float);


/*	if(step==4)
	{
		numBlocksT = numBlocks;
		numberOfThreadsT = numberOfThreads;
		hipMalloc ((void**)(&parpotT_gpu), np*sizeof(float));
		hipMemcpy(parpotT_gpu, parpot_gpu, np *sizeof(float), hipMemcpyDeviceToDevice);
		for(i=0; i<iter; i++)
		{
			hipMemcpy(parpot_gpu, parpotT_gpu, np *sizeof(float), hipMemcpyDeviceToDevice);
			numBlocks = numBlocksT;
			numberOfThreads = numberOfThreadsT;
			t0=cputime();
			while(numberOfThreads>512)
			{
				dimGrid = dim3(numBlocks, 1, 1);
				reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parpot_gpu, parpot_gpu, numberOfThreads);
				hipDeviceSynchronize();
				numberOfThreads = numBlocks;
				numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;
			}
			t1=cputime();
			time_sum1 += (t1-t0)*iCPS*1000*1000;
		}
		hipFree(parpotT_gpu);

		parpot = (float*) malloc(numberOfThreads * sizeof (float));
		hipMemcpy(parpot, parpot_gpu, numberOfThreads*sizeof(float), hipMemcpyDeviceToHost);

		for(i=0; i<iter; i++)
		{
			sum = 0.0;
			t0=cputime();
			for(int j=0; j<numberOfThreads; j++)
				sum += parpot[j];
			t1=cputime();
			time_sum2 += (t1-t0)*iCPS*1000*1000;
		}
		*time_elapsed = time_sum1/iter + time_sum2/iter;
	}
	else
*/	{
		while(numBlocks > 1)
		//while(numberOfThreads>512)
		{
			dimGrid = dim3(numBlocks, 1, 1);
			reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parpot_gpu, parpot_gpu, numberOfThreads);
			hipDeviceSynchronize();
			numberOfThreads = numBlocks;
			numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;
		}

		parpot = (float*) malloc(numberOfThreads * sizeof (float));
		hipMemcpy(parpot, parpot_gpu, numberOfThreads*sizeof(float), hipMemcpyDeviceToHost);
		sum = 0.0;
		for(int i=0; i<numberOfThreads; i++)
			sum += parpot[i];
	}
	free(parpot);
	return sum;
}

//END K2 - Accumulate PE with/without shared memory


//START K3 - Accumulate Force with/without shared memory
void GPU_accumulate_parforce_wShrdMem(int nd, int np, int currentMoleculeIndex, int step, double *time_elapsed)
{
	int BLOCK_SIZE = 128;
	int numberOfThreads = np;
	int numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;
	int smemSize = (BLOCK_SIZE <= 32) ? 2 * BLOCK_SIZE * sizeof(float) : BLOCK_SIZE * sizeof(float);

	dim3 dimBlock = dim3(BLOCK_SIZE, 1, 1);
	dim3 dimGrid;


/*	if(step ==4)
	{
		numBlocksT = numBlocks;
		numberOfThreadsT = numberOfThreads;
		hipMalloc ((void**)(&parforceT_gpu), nd*np*sizeof(float));
		hipMemcpy(parforceT_gpu, parforce_gpu, nd * np *sizeof(float), hipMemcpyDeviceToDevice);
		for(i=0; i<iter; i++)
		{
			numBlocks = numBlocksT;
			numberOfThreads = numberOfThreadsT;
			hipMemcpy(parforce_gpu, parforceT_gpu, nd * np *sizeof(float), hipMemcpyDeviceToDevice);
			t0=cputime();
			while(numberOfThreads>1)
			{
				dimGrid = dim3(numBlocks, 1);
				reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu, parforce_gpu, 													numberOfThreads);
				hipDeviceSynchronize();
				reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu+(np), parforce_gpu+(np), 												numberOfThreads);
				hipDeviceSynchronize();
				reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu+(np*2),
											parforce_gpu+(np*2),numberOfThreads);
				hipDeviceSynchronize();
				numberOfThreads = numBlocks;
				numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1);
			}

			dimGrid = dim3(numBlocks, 1);
			reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu, force_gpu+currentMoleculeIndex, 											numberOfThreads);
			hipDeviceSynchronize();
			reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu+(np), force_gpu+
										(np+currentMoleculeIndex), numberOfThreads);
			hipDeviceSynchronize();
			reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu+(np*2), force_gpu+
										(np+np+currentMoleculeIndex), numberOfThreads);
			hipDeviceSynchronize();
			t1=cputime();
			time_sum += (t1-t0)*iCPS*1000*1000;
		}
		*time_elapsed = time_sum/iter;
		hipFree(parforceT_gpu);
	}
	else
*/	{
		while(numBlocks > 1)
		//while(numberOfThreads>1)
		{
			dimGrid = dim3(numBlocks, 1);
			reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu, parforce_gpu, 												numberOfThreads);
			hipDeviceSynchronize();
			reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu+(np), parforce_gpu+(np), 											numberOfThreads);
			hipDeviceSynchronize();
			reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu+(np*2),
										parforce_gpu+(np*2),numberOfThreads);
			hipDeviceSynchronize();
			numberOfThreads = numBlocks;
			numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1);
		}

		dimGrid = dim3(numBlocks, 1);
		reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu, force_gpu+currentMoleculeIndex, 										numberOfThreads);
		// float x;
		// hipMemcpy(&x,force_gpu + currentMoleculeIndex,sizeof(float),hipMemcpyDeviceToHost);
		// printf("%f \n",x);
		hipDeviceSynchronize();
		reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu+(np), force_gpu+
									(np+currentMoleculeIndex), numberOfThreads);
		hipDeviceSynchronize();
		reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parforce_gpu+(np*2), force_gpu+
									(np+np+currentMoleculeIndex), numberOfThreads);
		hipDeviceSynchronize();

	}
}
//END K3 - Accumulate Force with/without shared memory

//Accumulates PE and Force using K2 and K3
extern "C"
float GPU_seq_wShrdMem_accumulate_parpot_and_parforce(int nd, int np, int currentMoleculeIndex, int step, double *time_elap1, double *time_elap2)
{
	GPU_accumulate_parforce_wShrdMem(nd,np,currentMoleculeIndex, step, time_elap1);
	return GPU_accumulate_parpot_wShrdMem(nd, np, step, time_elap2);
}

//START K4 - Compute and accumulate KE without shared memory
//Compute KE with shared memory
template <class T>
__global__ void  GPU_compute_KE_wShrdMem(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;
    __syncthreads();

    sdata[tid] *= sdata[tid];
	__syncthreads();
    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
	__syncthreads();
}

extern "C"
float GPU_accumulate_KE_wShrdMem(int nd, int np, float mass, int step, double *time_elapsed)
{
	int BLOCK_SIZE = 128;
	int numberOfThreads = nd*np;
	int numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;
	float* parvel, sum;
	dim3 dimBlock = dim3(BLOCK_SIZE, 1, 1);
	dim3 dimGrid;
	int smemSize = (BLOCK_SIZE <= 32) ? 2 * BLOCK_SIZE * sizeof(double) : BLOCK_SIZE * sizeof(double);
	dimGrid = dim3(numBlocks, 1, 1);
	float val[100];
	hipMemcpy(val,vel_gpu,100*sizeof(float),hipMemcpyDeviceToHost);


/*	if(step == 4)
	{
		t0 = cputime();
		for(i=0; i<iter; i++)
		{
			GPU_compute_KE_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(vel_gpu, parvel_gpu, numberOfThreads);
			hipDeviceSynchronize();

		}
		t1=cputime();
		time_sum1 = (t1-t0)*iCPS*1000*1000/iter;
		numberOfThreads = numBlocks;
		numberOfThreadsT = numberOfThreads;
		numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;
		numBlocksT = numBlocks;
		hipMalloc ((void**)(&parvelT_gpu), nd*np*sizeof(float));
		hipMemcpy(parvelT_gpu, parvel_gpu, nd * np *sizeof(float), hipMemcpyDeviceToDevice);
		for(i=0; i<iter; i++)
		{
			numBlocks = numBlocksT;
			numberOfThreads = numberOfThreadsT;
			hipMemcpy(parvel_gpu, parvelT_gpu, nd * np *sizeof(float), hipMemcpyDeviceToDevice);
			t0=cputime();
			while(numBlocks>1)
			{
				dimGrid = dim3(numBlocks, 1, 1);
				reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parvel_gpu, parvel_gpu, numberOfThreads);
				hipDeviceSynchronize();
				numberOfThreads = numBlocks;
				numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;
			}
			t1=cputime();
			time_sum2 += (t1-t0)*iCPS*1000*1000;
		}
		hipFree(parvelT_gpu);

		parvel = (float*) malloc(numberOfThreads * sizeof (float));
		hipMemcpy(parvel, parvel_gpu, numberOfThreads*sizeof(float), hipMemcpyDeviceToHost);

		for(i=0; i<iter; i++)
		{
			t0=cputime();
			sum = 0.0;
			for(int j=0; j<numberOfThreads; j++)
			{
				sum += parvel[j];
			}
			t1=cputime();
			time_sum3 += (t1-t0)*iCPS*1000*1000;
		}
		*time_elapsed = time_sum1 + time_sum2/iter + time_sum3/iter;
	}
	else
*/	{
		GPU_compute_KE_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(vel_gpu, parvel_gpu, numberOfThreads);
		hipDeviceSynchronize();

		numberOfThreads = numBlocks;
		numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;

		while(numBlocks>1)
		{
			dimGrid = dim3(numBlocks, 1, 1);
			reduce_wShrdMem<float><<< dimGrid, dimBlock, smemSize >>>(parvel_gpu, parvel_gpu, numberOfThreads);
			hipDeviceSynchronize();
			numberOfThreads = numBlocks;
			numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;
		}

		parvel = (float*) malloc(numberOfThreads * sizeof (float));
		hipMemcpy(parvel, parvel_gpu, numberOfThreads*sizeof(float), hipMemcpyDeviceToHost);

		sum = 0.0;
		for(int i=0; i<numberOfThreads; i++)
			sum += parvel[i];
	}
	free(parvel);
	return 0.5 * mass * sum;
}

//END K4 - Compute and accumulate KE with shared memory


//START K5 - Update position
__global__ void GPU_updatePos(int numberOfThreads, float dt, float * pos_gpu, const float * vel_gpu, const float * acc_gpu)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x ;
	if (idx >= numberOfThreads)
		return ;
	pos_gpu[idx] += vel_gpu[idx] * dt + 0.5 * acc_gpu[idx] * dt * dt;
}

extern "C"
void GPU_updatePos(int nd, int np, float dt, int step, double *time_elapsedCPU, float *time_elapsedGPU)
{
	int BLOCK_SIZE = 128;
	int numberOfThreads = nd*np;
	int numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;

	dim3 dimGrid(numBlocks) ;
	dim3 dimBlock(BLOCK_SIZE) ;


/*	if(step==4)
	{
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipMalloc ((void**)(&posT_gpu), nd*np*sizeof(float));
		for(i=0; i<iter; i++)
		{
			dummyCopy<<<dimGrid,dimBlock>>>(pos_gpu, posT_gpu); //(input, output)
			t0=cputime();
			//hipEventRecord(start,0);
			GPU_updatePos<<<dimGrid,dimBlock>>>(numberOfThreads, dt, pos_gpu, vel_gpu, acc_gpu);
			hipDeviceSynchronize();
			//hipEventRecord(stop,0);
			//hipEventSynchronize(stop);
			t1=cputime();
			//hipEventElapsedTime(&time_diff, start, stop);
			if(i<(iter-1))
				dummyCopy<<<dimGrid,dimBlock>>>(posT_gpu, pos_gpu);
			time_sumCPU += (t1-t0)*iCPS*1000*1000;
			//time_diff /= 1000;
			//time_sumGPU += time_diff;
		}
		*time_elapsedCPU=time_sumCPU/iter;
		//*time_elapsedGPU=time_sumGPU/iter;
		hipFree(posT_gpu);
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
	else
*/	{
		GPU_updatePos<<<dimGrid,dimBlock>>>(numberOfThreads, dt, pos_gpu, vel_gpu, acc_gpu) ;
		hipDeviceSynchronize();
	}
}
//END K5 - Update position


//START K6 - Update velocity
__global__ void GPU_updateVel(int numberOfThreads, float dt, float rmass, float * vel_gpu, const float * force_gpu, const float * acc_gpu)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x ;
	if (idx >= numberOfThreads)
		return ;
	vel_gpu[idx] += 0.5 * dt * (force_gpu[idx] * rmass + acc_gpu[idx]);
}

extern "C"
void GPU_updateVel(int nd, int np, float dt, float rmass, int step, double *time_elapsedCPU, float *time_elapsedGPU)
{
	int BLOCK_SIZE = 128;
	int numberOfThreads = nd*np;
	int numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;

	dim3 dimGrid(numBlocks) ;
	dim3 dimBlock(BLOCK_SIZE) ;


/*	if(step==4)
	{
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipMalloc ((void**)(&velT_gpu), nd*np*sizeof(float));
		for(i=0; i<iter; i++)
		{
			dummyCopy<<<dimGrid,dimBlock>>>(vel_gpu, velT_gpu); //(input, output)
			t0=cputime();
			//hipEventRecord(start,0);
			GPU_updateVel<<<dimGrid,dimBlock>>>(numberOfThreads, dt, rmass, vel_gpu, force_gpu, acc_gpu) ;
			hipDeviceSynchronize();
			//hipEventRecord(stop,0);
			//hipEventSynchronize(stop);
			t1=cputime();
			//hipEventElapsedTime(&time_diff, start, stop);
			if(i<(iter-1))
				dummyCopy<<<dimGrid,dimBlock>>>(velT_gpu, vel_gpu);
			time_sumCPU += (t1-t0)*iCPS*1000*1000;
			//time_diff /= 1000;
			//time_sumGPU += time_diff;
		}
		*time_elapsedCPU=time_sumCPU/iter;
		//*time_elapsedGPU=time_sumGPU/iter;
		hipFree(velT_gpu);
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
	else
*/	{
		GPU_updateVel<<<dimGrid,dimBlock>>>(numberOfThreads, dt, rmass, vel_gpu, force_gpu, acc_gpu) ;
		hipDeviceSynchronize();
	}
}
//END K6 - Update velocity


//START K7 - Update acceleration
__global__ void GPU_updateAcc(int numberOfThreads, float rmass, float * acc_gpu, const float * force_gpu)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x ;
	if (idx >= numberOfThreads)
		return ;
	acc_gpu[idx] = force_gpu[idx] * rmass;
}

extern "C"
void GPU_updateAcc(int nd, int np, float rmass, int step, double *time_elapsedCPU, float *time_elapsedGPU)
{
	int BLOCK_SIZE = 128;
	int numberOfThreads = nd*np;
	int numBlocks = numberOfThreads / BLOCK_SIZE + (numberOfThreads % BLOCK_SIZE == 0 ? 0 : 1) ;

	dim3 dimGrid(numBlocks) ;
	dim3 dimBlock(BLOCK_SIZE) ;


/*	if(step==4)
	{
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		for(i=0; i<iter; i++)
		{
			t0=cputime();
			//hipEventRecord(start,0);
			GPU_updateAcc<<<dimGrid,dimBlock>>>(numberOfThreads, rmass, acc_gpu, force_gpu) ;
			hipDeviceSynchronize();
			//hipEventRecord(stop,0);
			//hipEventSynchronize(stop);
			t1=cputime();
			//hipEventElapsedTime(&time_diff, start, stop);
			time_sumCPU += (t1-t0)*iCPS*1000*1000;
			//time_diff /= 1000;
			//time_sumGPU += time_diff;
		}
		*time_elapsedCPU=time_sumCPU/iter;
		//*time_elapsedGPU = time_sumGPU/iter;
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}
	else
*/	{
		GPU_updateAcc<<<dimGrid,dimBlock>>>(numberOfThreads, rmass, acc_gpu, force_gpu) ;
		hipDeviceSynchronize();
	}
}
//END K7 - Update acceleration
