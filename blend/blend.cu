#include <opencv2/opencv.hpp>
#include <thrust/device_vector.h>
#include <thrust/shared_for_each.h>
using namespace cv;
class blendFunctor
{
  float alpha;
public:
  blendFunctor(float alpha)
  {
    this->alpha = alpha;
  }
  __device__ uchar operator() (uchar &input1,uchar &input2) const
  {
    return alpha * input1+ (1-alpha) *  input2;
  }
};

int main(int argc, char const *argv[]) {
  int dim = 4096;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat input1 = imread("../opencv/shared/santiago.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat input2 = imread("../opencv/shared/car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat temp1;
  resize(input1,temp1,Size(dim,dim));
  input1 = temp1;
  Mat temp2;
  resize(input2,temp2,Size(dim,dim));
  input2 = temp2;

  thrust::host_vector<uchar>host_input_vector1(input1.ptr(),input1.ptr()+input1.cols*input1.rows);
  thrust::host_vector<uchar>host_input_vector2(input2.ptr(),input2.ptr()+input2.cols*input2.rows);

  thrust::device_vector<uchar>input_vector1(input1.ptr(),input1.ptr()+input1.cols*input1.rows);
  thrust::device_vector<uchar>input_vector2(input2.ptr(),input2.ptr()+input2.cols*input2.rows);
  thrust::device_vector<uchar>output_vector(input1.cols*input1.rows);
  thrust::host_vector<uchar>host_output_vector(input1.cols*input1.rows);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  for(int i = 0; i<10;i++)
  {
    input_vector1 = host_input_vector1;
    input_vector2 = host_input_vector2;
    thrust::transform(input_vector1.begin(),input_vector1.end(),input_vector2.begin(),output_vector.begin(),blendFunctor(0.3));
    host_output_vector = output_vector;
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float time_in_ms;
  hipEventElapsedTime(&time_in_ms,start,stop);
  printf("Non Shared Blend = %f\n",time_in_ms);
  Mat output (Size(input1.cols,input1.rows),CV_8UC1,host_output_vector.data());
  #ifdef OWRITE
  imwrite("blend-input1.png",input1);
  imwrite("blend-input2.png",input2);
  imwrite("blend-output.png",output);
  #endif
  #ifdef SHOW
  imshow("blend-input1.png",input1);
  imshow("blend-input2.png",input2);
  imshow("blend-output.png",output);
  waitKey(0);
  #endif
  return 0;
}
