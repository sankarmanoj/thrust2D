#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/device_vector.h>
#include <thrust/shared_for_each.h>
using namespace cv;
int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}
__global__ void blendKernel(uchar * input1,uchar * input2, uchar * output,float alpha, int size)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  int stride = blockDim.x*gridDim.x;
  while(index<size)
  {
  output[index] = input1[index]*alpha + input2[index]*(1-alpha);
  index+=stride;
  }
}

int main(int argc, char const *argv[]) {
  int dim = 4096;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat input1 = imread("../opencv/shared/santiago.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat input2 = imread("../opencv/shared/car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat temp1;
  resize(input1,temp1,Size(dim,dim));
  input1 = temp1;
  Mat temp2;
  resize(input2,temp2,Size(dim,dim));
  input2 = temp2;

  uchar * d_input1, *d_input2,*d_output;
  hipMalloc((void **)&d_input1,sizeof(uchar)*dim*dim);
  hipMalloc((void **)&d_input2,sizeof(uchar)*dim*dim);
  hipMalloc((void **)&d_output,sizeof(uchar)*dim*dim);
  uchar * h_output = new uchar[dim*dim];
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  for(int i = 0; i<10;i++)
  {
  hipMemcpy(d_input1,input1.ptr(),sizeof(uchar)*dim*dim,hipMemcpyHostToDevice);
  hipMemcpy(d_input2,input2.ptr(),sizeof(uchar)*dim*dim,hipMemcpyHostToDevice);
  blendKernel<<<30,1024>>>(d_input1,d_input2,d_output,0.3,dim*dim);
  hipMemcpy(h_output,d_output,sizeof(uchar)*dim*dim,hipMemcpyDeviceToHost);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float time_in_ms;
  hipEventElapsedTime(&time_in_ms,start,stop);
  printf("Native Blend = %f\n",time_in_ms);
  Mat output (Size(input1.cols,input1.rows),CV_8UC1,h_output);
  #ifdef OWRITE
  imwrite("blend-input1.png",input1);
  imwrite("blend-input2.png",input2);
  imwrite("blend-output.png",output);
  #endif
  #ifdef SHOW
  Mat out;
  resize(input1,temp1,Size(512,512));
  imshow("blend-input1.png",temp1);
  resize(input2,temp2,Size(512,512));
  imshow("blend-input2.png",temp2);
  resize(output,out,Size(512,512));
  imshow("blend-output.png",out);
  waitKey(0);
  #endif
  return 0;
}
