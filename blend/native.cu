#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <thrust/device_vector.h>
#include <thrust/shared_for_each.h>
using namespace cv;
int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}
__global__ void blendKernel(uchar * input1,uchar * input2, uchar * output,float alpha, int size)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  if(index>=size)
    return;
  output[index] = input1[index]*alpha + input2[index]*(1-alpha);
}
class blendFunctor
{
  float alpha;

public:
  blendFunctor(float alpha)
  {
    this->alpha = alpha;
  }
  __device__ uchar operator() (uchar &input1,float &input2) const
  {
    return alpha * input1+ (1-alpha) *  input2;
  }
};

int main(int argc, char const *argv[]) {
  int dim = 4096;
  if(argc ==2)
  {
    dim = atoi(argv[1]);
  }
  hipDeviceProp_t dev_prop;
  hipGetDeviceProperties(&dev_prop,0);
  Mat input1 = imread("../opencv/shared/santiago.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat input2 = imread("../opencv/shared/car.jpg",CV_LOAD_IMAGE_GRAYSCALE);
  Mat temp1;
  resize(input1,temp1,Size(dim,dim));
  input1 = temp1;
  Mat temp2;
  resize(input2,temp2,Size(dim,dim));
  input2 = temp2;

  uchar * d_input1, *d_input2,*d_output;
  hipMalloc((void **)&d_input1,sizeof(uchar)*dim*dim);
  hipMalloc((void **)&d_input2,sizeof(uchar)*dim*dim);
  hipMalloc((void **)&d_output,sizeof(uchar)*dim*dim);
  hipMemcpy(d_input1,input1.ptr(),sizeof(uchar)*dim*dim,hipMemcpyHostToDevice);
  hipMemcpy(d_input2,input2.ptr(),sizeof(uchar)*dim*dim,hipMemcpyHostToDevice);
  for(int i = 0; i<100;i++)
  blendKernel<<<iDivUp(dim*dim,1024),1024>>>(d_input1,d_input2,d_output,0.3,dim*dim);
  uchar * h_output = new uchar[dim*dim];
  hipMemcpy(h_output,d_output,sizeof(uchar)*dim*dim,hipMemcpyDeviceToHost);
  Mat output (Size(input1.cols,input1.rows),CV_8UC1,h_output);
  #ifdef OWRITE
  imwrite("blend-input1.png",input1);
  imwrite("blend-input2.png",input2);
  imwrite("blend-output.png",output);
  #endif
  #ifdef SHOW
  imshow("blend-input1.png",input1);
  imshow("blend-input2.png",input2);
  imshow("blend-output.png",output);
  waitKey(0);
  #endif
  return 0;
}
