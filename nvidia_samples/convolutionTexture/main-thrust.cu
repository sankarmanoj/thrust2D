#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


/*
 * This sample implements the same algorithm as the convolutionSeparable
 * CUDA Sample, but without using the shared memory at all.
 * Instead, it uses textures in exactly the same way an OpenGL-based
 * implementation would do.
 * Refer to the "Performance" section of convolutionSeparable whitepaper.
 */




#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "convolutionTexture_common.h"
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    float
    *h_Kernel,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;



    hipChannelFormatDesc floatTex = hipCreateChannelDesc<float>();


    float
    gpuTime;

    StopWatchInterface *hTimer = NULL;

    const int imageW = 3072;
    const int imageH = 3072 / 2;
    const unsigned int iterations = 1;

    printf("[%s] - Starting...\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaDevice(argc, (const char **)argv);

    sdkCreateTimer(&hTimer);

    printf("Initializing data...\n");
    h_Kernel    = (float *)malloc(KERNEL_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));
    thrust::block_2d<float> src_block(imageW,imageH);
    thrust::block_2d<float> output_block(imageW,imageH);

    srand(2009);

    for (unsigned int i = 0; i < KERNEL_LENGTH; i++)
    {
        h_Kernel[i] = (float)(rand() % 16);
    }

    for (unsigned int i = 0; i < imageW * imageH; i++)
    {
        h_Input[i] = (float)(rand() % 16);
    }

    src_block.upload(h_Input);


    printf("Running GPU rows convolution (%u identical iterations)...\n", iterations);
    checkCudaErrors(hipDeviceSynchronize());
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);

    for (unsigned int i = 0; i < iterations; i++)
    {
      thrust::convolve(thrust::cuda::texture,&src_block,h_Kernel,KERNEL_LENGTH,&output_block);
    }

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    gpuTime = sdkGetTimerValue(&hTimer) / (float)iterations;
    printf("Average convolutionRowsGPU() time: %f msecs; //%f Mpix/s\n", gpuTime, imageW * imageH * 1e-6 / (0.001 * gpuTime));

    //While CUDA kernels can't write to textures directly, this copy is inevitable
    printf("Copying convolutionRowGPU() output back to the texture...\n");
    checkCudaErrors(hipDeviceSynchronize());
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);
    sdkStopTimer(&hTimer);
    gpuTime = sdkGetTimerValue(&hTimer);
    printf("hipMemcpyToArray() time: %f msecs; //%f Mpix/s\n", gpuTime, imageW * imageH * 1e-6 / (0.001 * gpuTime));



    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    gpuTime = sdkGetTimerValue(&hTimer) / (float)iterations;
    printf("Average convolutionColumnsGPU() time: %f msecs; //%f Mpix/s\n", gpuTime, imageW * imageH * 1e-6 / (0.001 * gpuTime));

    printf("Reading back GPU results...\n");
    output_block.download(&h_OutputGPU);

    printf("Checking the results...\n");
    printf("...running convolutionRowsCPU()\n");
    convolutionRowsCPU(
        h_Buffer,
        h_Input,
        h_Kernel,
        imageW,
        imageH,
        KERNEL_RADIUS
    );

    printf("...running convolutionColumnsCPU()\n");
    convolutionColumnsCPU(
        h_OutputCPU,
        h_Buffer,
        h_Kernel,
        imageW,
        imageH,
        KERNEL_RADIUS
    );

    double delta = 0;
    double sum = 0;

    for (unsigned int i = 0; i < imageW * imageH; i++)
    {
        sum += h_OutputCPU[i] * h_OutputCPU[i];
        delta += (h_OutputGPU[i] - h_OutputCPU[i]) * (h_OutputGPU[i] - h_OutputCPU[i]);
    }

    double L2norm = sqrt(delta / sum);
    printf("Relative L2 norm: %E\n", L2norm);
    printf("Shutting down...\n");


    free(h_OutputGPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Kernel);

    sdkDeleteTimer(&hTimer);

    if (L2norm > 1e-6)
    {
        printf("Test failed!\n");
        exit(EXIT_FAILURE);
    }

    printf("Test passed\n");
    exit(EXIT_SUCCESS);
}
