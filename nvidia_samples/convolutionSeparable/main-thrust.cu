#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
* This sample implements a separable convolution filter
* of a 2D image with an arbitrary kernel.
*/

// CUDA runtime
#include <hip/hip_runtime.h>
#include <thrust/block_2d.h>
#include <thrust/window_transform.h>
// Utilities and system includes
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "convolutionSeparable_common.h"
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
////////////////////////////////////////////////////////////////////////////////
// Reference CPU convolution
////////////////////////////////////////////////////////////////////////////////
extern "C" void convolutionRowCPU(
    float *h_Result,
    float *h_Data,
    float *h_Kernel,
    int imageW,
    int imageH,
    int kernelR
);

extern "C" void convolutionColumnCPU(
    float *h_Result,
    float *h_Data,
    float *h_Kernel,
    int imageW,
    int imageH,
    int kernelR
);




////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // start logs
    printf("[%s] - Starting...\n", argv[0]);

    float
    *h_Kernel,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;


    const int imageW = 3072;
    const int imageH = 3072;
    const int iterations = 16;

    StopWatchInterface *hTimer = NULL;

    //Use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaDevice(argc, (const char **)argv);

    sdkCreateTimer(&hTimer);

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    h_Kernel    = (float *)malloc(KERNEL_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));
    srand(200);

    for (unsigned int i = 0; i < KERNEL_LENGTH; i++)
    {
        h_Kernel[i] = (float)(rand() % 16);
    }

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
        h_Input[i] = (float)(rand() % 16);
    }

    printf("Allocating and initializing CUDA arrays...\n");
    thrust::block_2d<float> input_block (imageW,imageH);
    thrust::block_2d<float> output_block (imageW,imageH);

    input_block.upload(h_Input);

    printf("Running GPU convolution (%u identical iterations)...\n\n", iterations);

    for (int i = -1; i < iterations; i++)
    {
        //i == -1 -- warmup iteration
        if (i == 0)
        {
            checkCudaErrors(hipDeviceSynchronize());
            sdkResetTimer(&hTimer);
            sdkStartTimer(&hTimer);
        }

        thrust::convolve(thrust::cuda::shared,&input_block,h_Kernel,KERNEL_LENGTH,&output_block);
    }

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    double gpuTime = 0.001 * sdkGetTimerValue(&hTimer) / (double)iterations;
    printf("convolutionSeparable, Throughput = %.4f MPixels/sec, Time = %.5f s, Size = %u Pixels, NumDevsUsed = %i, Workgroup = %u\n",
           (1.0e-6 * (double)(imageW * imageH)/ gpuTime), gpuTime, (imageW * imageH), 1, 0);

    printf("\nReading back GPU results...\n\n");
    output_block.download(&h_OutputGPU);

    printf("Checking the results...\n");
    printf(" ...running convolutionRowCPU()\n");
    convolutionRowCPU(
        h_Buffer,
        h_Input,
        h_Kernel,
        imageW,
        imageH,
        KERNEL_RADIUS
    );

    printf(" ...running convolutionColumnCPU()\n");
    convolutionColumnCPU(
        h_OutputCPU,
        h_Buffer,
        h_Kernel,
        imageW,
        imageH,
        KERNEL_RADIUS
    );

    printf(" ...comparing the results`\n");
    double sum = 0, delta = 0;

    for (unsigned i = 0; i < imageW * imageH; i++)
    {
        delta += (h_OutputGPU[i] - h_OutputCPU[i]) * (h_OutputGPU[i] - h_OutputCPU[i]);
        sum   += h_OutputCPU[i] * h_OutputCPU[i];
    }

    double L2norm = sqrt(delta / sum);
    printf(" ...Relative L2 norm: %E\n\n", L2norm);
    printf("Shutting down...\n");



    free(h_OutputGPU);
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Kernel);

    sdkDeleteTimer(&hTimer);

    if (L2norm > 1e-6)
    {
        printf("Test failed!\n");
        exit(EXIT_FAILURE);
    }

    printf("Test passed\n");
    exit(EXIT_SUCCESS);
}
