#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>       // CUDA device initialization helper functions
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
__constant__ float cGaussian[64];   //gaussian array in device side
texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;

uint *dImage  = NULL;   //original image
uint *dTemp   = NULL;   //temp array for iterations
size_t pitch;

/*
    Perform a simple bilateral filter.

    Bilateral filter is a nonlinear filter that is a mixture of range
    filter and domain filter, the previous one preserves crisp edges and
    the latter one filters noise. The intensity value at each pixel in
    an image is replaced by a weighted average of intensity values from
    nearby pixels.

    The weight factor is calculated by the product of domain filter
    component(using the gaussian distribution as a spatial distance) as
    well as range filter component(Euclidean distance between center pixel
    and the current neighbor pixel). Because this process is nonlinear,
    the sample just uses a simple pixel by pixel step.

    Texture fetches automatically clamp to edge of image. 1D gaussian array
    is mapped to a 1D texture instead of using shared memory, which may
    cause severe bank conflict.

    Threads are y-pass(column-pass), because the output is coalesced.

    Parameters
    od - pointer to output data in global memory
    d_f - pointer to the 1D gaussian array
    e_d - euclidean delta
    w  - image width
    h  - image height
    r  - filter radius
*/

//Euclidean Distance (x, y, d) = exp((|x - y| / d)^2 / 2)
__device__ float euclideanLen(float4 a, float4 b, float d)
{

    float mod = (b.x - a.x) * (b.x - a.x) +
                (b.y - a.y) * (b.y - a.y) +
                (b.z - a.z) * (b.z - a.z);

    return __expf(-mod / (2.f * d * d));
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(fabs(rgba.x));   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(fabs(rgba.y));
    rgba.z = __saturatef(fabs(rgba.z));
    rgba.w = __saturatef(fabs(rgba.w));
    return (uint(rgba.w * 255.0f) << 24) | (uint(rgba.z * 255.0f) << 16) | (uint(rgba.y * 255.0f) << 8) | uint(rgba.x * 255.0f);
}

__device__ float4 rgbaIntToFloat(uint c)
{
    float4 rgba;
    rgba.x = (c & 0xff) * 0.003921568627f;       //  /255.0f;
    rgba.y = ((c>>8) & 0xff) * 0.003921568627f;  //  /255.0f;
    rgba.z = ((c>>16) & 0xff) * 0.003921568627f; //  /255.0f;
    rgba.w = ((c>>24) & 0xff) * 0.003921568627f; //  /255.0f;
    return rgba;
}

//column pass using coalesced global memory reads
__global__ void
d_bilateral_filter(uint *od, int w, int h,
                   float e_d,  int r)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= w || y >= h)
    {
        return;
    }

    float sum = 0.0f;
    float factor;
    float4 t = {0.f, 0.f, 0.f, 0.f};
    float4 center = tex2D(rgbaTex, x, y);

    for (int i = -r; i <= r; i++)
    {
        for (int j = -r; j <= r; j++)
        {
            float4 curPix = tex2D(rgbaTex, x + j, y + i);
            factor = cGaussian[i + r] * cGaussian[j + r] *     //domain factor
                     euclideanLen(curPix, center, e_d);             //range factor

            t += factor * curPix;
            sum += factor;
        }
    }

    od[y * w + x] = rgbaFloatToInt(t/sum);
}
class d_bilateral_filter_functor
{
  float e_d;
  int r;
public:
  d_bilateral_filter_functor(float e_d,  int r)
  {
      this->e_d = e_d;
      this->r = r;
  }
  __device__ int operator() ( const thrust::window_2d<uchar4> &input,  const thrust::window_2d<uint> &output ) const
  {


    float sum = 0.0f;
    float factor;
    float4 t = {0.f, 0.f, 0.f, 0.f};
    uchar4 center_int = input[r][r];
    float4 center = {(float)center_int.x,(float)center_int.y,(float)center_int.z,(float)center_int.w};
    for (int i = 0; i <= 2*r; i++)
    {
        for (int j = 0; j <= 2*r; j++)
        {
            uchar4 temp = input[j][i];
            float4 curPix = {(float)temp.x,(float)temp.y,(float)temp.z,(float)temp.w,};
            factor = cGaussian[i + r] * cGaussian[j + r] *     //domain factor
                     euclideanLen(curPix, center, e_d);             //range factor

            t += factor * curPix;
            sum += factor;
        }
    }

    output[r][r]=rgbaFloatToInt(t/sum);\
    return 1;
  }
};
extern "C"
void initTexture(int width, int height, uint *hImage)
{
    // copy image data to array
    checkCudaErrors(hipMallocPitch(&dImage, &pitch, sizeof(uint)*width, height));
    checkCudaErrors(hipMallocPitch(&dTemp,  &pitch, sizeof(uint)*width, height));
    checkCudaErrors(hipMemcpy2D(dImage, pitch, hImage, sizeof(uint)*width,
                                 sizeof(uint)*width, height, hipMemcpyHostToDevice));
}

extern "C"
void freeTextures()
{
    checkCudaErrors(hipFree(dImage));
    checkCudaErrors(hipFree(dTemp));
}

/*
    Because a 2D gaussian mask is symmetry in row and column,
    here only generate a 1D mask, and use the product by row
    and column index later.

    1D gaussian distribution :
        g(x, d) -- C * exp(-x^2/d^2), C is a constant amplifier

    parameters:
    og - output gaussian array in global memory
    delta - the 2nd parameter 'd' in the above function
    radius - half of the filter size
             (total filter size = 2 * radius + 1)
*/
extern "C"
void updateGaussian(float delta, int radius)
{
    float  fGaussian[64];

    for (int i = 0; i < 2*radius + 1; ++i)
    {
        float x = i-radius;
        fGaussian[i] = expf(-(x*x) / (2*delta*delta));
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, sizeof(float)*(2*radius+1)));
}

/*
    Perform 2D bilateral filter on image using CUDA

    Parameters:
    d_dest - pointer to destination image in device memory
    width  - image width
    height - image height
    e_d    - euclidean delta
    radius - filter radius
    iterations - number of iterations
*/

// RGBA version
extern "C"
double bilateralFilterRGBA(uint *dDest,
                           int width, int height,
                           float e_d, int radius, int iterations,
                           StopWatchInterface *timer)
{
    // var for kernel computation timing
    double dKernelTime;

    // Bind the array to the texture
    hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
    checkCudaErrors(hipBindTexture2D(0, rgbaTex, dImage, desc, width, height, pitch));
    thrust::block_2d<uchar4> d_image_block(width,height);
    d_image_block.upload((uchar4*)dImage,hipMemoryTypeDevice);
    thrust::block_2d<uint> d_dest_block(width,height);
    thrust::device_vector<int> nulla(width*height);
    thrust::window_vector<uchar4> input_wv(&d_image_block,2*radius+1,2*radius+1,1,1);
    thrust::window_vector<uint> output_wv(&d_dest_block,2*radius+1,2*radius+1,1,1);
    for (int i=0; i<iterations; i++)
    {
        // sync host and start kernel computation timer
        dKernelTime = 0.0;
        checkCudaErrors(hipDeviceSynchronize());
        sdkResetTimer(&timer);

        dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
        dim3 blockSize(16, 16);
        thrust::transform(input_wv.begin(),input_wv.end(),output_wv.begin(),nulla.begin(),d_bilateral_filter_functor(e_d,radius));
        // d_dest_block.download(dDest,hipMemoryTypeDevice);
        // sync host and stop computation timer
        checkCudaErrors(hipDeviceSynchronize());
        dKernelTime += sdkGetTimerValue(&timer);

        if (iterations > 1)
        {
            // checkCudaErrors(hipMemcpy2D(dTemp, pitch, d_dest_block.data_pointer, sizeof(int)*width,
            //                              sizeof(int)*width, height, hipMemcpyDeviceToDevice));
            // checkCudaErrors(hipBindTexture2D(0, rgbaTex, dTemp, desc, width, height, pitch));
        }

    }

    return ((dKernelTime/1000.)/(double)iterations);
}
