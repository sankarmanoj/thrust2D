#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_vector_types.h>
#include <thrust/window_2d.h>
#include <thrust/window_transform.h>
// includes, cuda
#include <hip/hip_runtime_api.h>

typedef unsigned int uint;
typedef unsigned char uchar;

#include "bicubicTexture_kernel.cuh"

hipArray *d_imageArray = 0;

extern "C"
void initTexture(int imageWidth, int imageHeight, uchar *h_data)
{
    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors(hipMallocArray(&d_imageArray, &channelDesc, imageWidth, imageHeight));
    uint size = imageWidth * imageHeight * sizeof(uchar);
    checkCudaErrors(hipMemcpyToArray(d_imageArray, 0, 0, h_data, size, hipMemcpyHostToDevice));
    free(h_data);

    // set texture parameters
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = false;    // access with integer texture coordinates

    getLastCudaError("initTexture");

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(tex, d_imageArray));

    // bind same array to 2nd texture reference with point sampling
    tex2.addressMode[0] = hipAddressModeClamp;
    tex2.addressMode[1] = hipAddressModeClamp;
    tex2.filterMode = hipFilterModePoint;
    tex2.normalized = false;    // access with integer texture coordinates

    checkCudaErrors(hipBindTextureToArray(tex2, d_imageArray));
}

extern "C"
void freeTexture()
{
    checkCudaErrors(hipFreeArray(d_imageArray));
}


// render image using CUDA
extern "C"
void render(int width, int height, float tx, float ty, float scale, float cx, float cy,
            dim3 blockSize, dim3 gridSize, int filter_mode, uchar4 *output,thrust::block_2d<uchar4> &block_d_output)
{
    // call CUDA kernel, writing results to PBO memory
    thrust::window_vector<uchar4> render_window_vector(&block_d_output,1,1,1,1);
    switch (filter_mode)
    {
        case MODE_NEAREST:
            tex.filterMode = hipFilterModePoint;
            thrust::for_each(render_window_vector.begin(),render_window_vector.end(),d_render_functor(tx,ty,scale,cx,cy));
            d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
            break;

        case MODE_BILINEAR:
            tex.filterMode = hipFilterModeLinear;
            thrust::for_each(render_window_vector.begin(),render_window_vector.end(),d_render_functor(tx,ty,scale,cx,cy));
            d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
            break;

        case MODE_BICUBIC:
            tex.filterMode = hipFilterModePoint;
            thrust::for_each(render_window_vector.begin(),render_window_vector.end(),d_renderBicubic_functor(tx,ty,scale,cx,cy));
            // thrust::for_each(thrust::cuda::shared,render_window_vector.begin(),render_window_vector.end(),d_renderBicubic_functor(tx,ty,scale,cx,cy));
            d_renderBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
            break;

        case MODE_FAST_BICUBIC:
            tex.filterMode = hipFilterModeLinear;
            thrust::for_each(render_window_vector.begin(),render_window_vector.end(),d_renderFastBicubic_functor(tx,ty,scale,cx,cy));
            d_renderFastBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
            break;

        case MODE_CATROM:
            tex.filterMode = hipFilterModePoint;
            thrust::for_each(render_window_vector.begin(),render_window_vector.end(),d_renderCatrom_functor(tx,ty,scale,cx,cy));
            d_renderCatRom<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
            break;
    }

    getLastCudaError("kernel failed");
}

#endif
